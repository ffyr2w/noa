#include <hipblas.h>
#include "noa/core/indexing/Layout.hpp"
#include "noa/gpu/cuda/Blas.hpp"
#include "noa/gpu/cuda/Exception.hpp"

namespace {
    using namespace noa::types;
    using noa::check;

    /// Throws an Exception if the result is not hipSuccess.
    constexpr void check(hipblasStatus_t result, const std::source_location& location = std::source_location::current()) {
        if (result == hipblasStatus_t::HIPBLAS_STATUS_SUCCESS) {
            /*do nothing*/
        } else {
            noa::panic_at_location(location, "cublas failed with error: {}", cublasGetStatusString(result));
        }
    }

    class CuBlasHandle {
    public:
        hipblasHandle_t handle{};
        CuBlasHandle() {
            check(hipblasCreate(&handle));
        }
        ~CuBlasHandle() {
            const hipblasStatus_t err = hipblasDestroy(handle);
            NOA_ASSERT(err == HIPBLAS_STATUS_SUCCESS);
            (void) err;
        }
    };

    std::unique_ptr<CuBlasHandle>& cublas_cache_handle_(int device) {
        constexpr size_t MAX_DEVICES = 16;
        thread_local std::unique_ptr<CuBlasHandle> g_cache[MAX_DEVICES];

        auto& cache = g_cache[device];
        if (not cache)
            cache = std::make_unique<CuBlasHandle>();
        return cache;
    }

    template<typename T>
    void cublas_gemm_(
        bool is_column_major, bool lhs_transpose, bool rhs_transpose,
        Shape3<int> mnk, Vec3<i32> labc, Vec3<i64> sabc, int batches, T alpha, T beta,
        const T* lhs, const T* rhs, T* output, noa::cuda::Stream& stream
    ) {
        // OpenBlas GEMM is slower than its DOT function, so we check for this condition and
        // redirect to dot if necessary. Here, cublas GEMM is about as fast as the dot function,
        // so let it do a matrix-matrix product even if it is a dot product.
        hipblasHandle_t handle = cublas_cache_handle_(stream.device().id())->handle;
        check(hipblasSetStream(handle, stream.id()));
        check(hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST));

        auto lhs_op = lhs_transpose ? HIPBLAS_OP_T : HIPBLAS_OP_N;
        auto rhs_op = rhs_transpose ? HIPBLAS_OP_T : HIPBLAS_OP_N;

        // cublas thinks everything is column-major, so if we are row-major we need to compute B.T @ A.T = C.T
        // https://stackoverflow.com/questions/56043539/cublassgemm-row-major-multiplication
        // https://peterwittek.com/cublas-matrix-c-style.html
        if (not is_column_major) {
            std::swap(lhs_op, rhs_op);
            std::swap(mnk[0], mnk[1]);
            std::swap(lhs, rhs);
            std::swap(labc[0], labc[1]);
            std::swap(sabc[0], sabc[1]);
        }

        if constexpr (std::is_same_v<f32, T>) {
            check(hipblasSgemmStridedBatched(
                handle, lhs_op, rhs_op, mnk[0], mnk[1], mnk[2], &alpha,
                lhs, labc[0], sabc[0],
                rhs, labc[1], sabc[1], &beta,
                output, labc[2], sabc[2], batches));
        } else if constexpr (std::is_same_v<f64, T>) {
            check(hipblasDgemmStridedBatched(
                handle, lhs_op, rhs_op, mnk[0], mnk[1], mnk[2], &alpha,
                lhs, labc[0], sabc[0],
                rhs, labc[1], sabc[1], &beta,
                output, labc[2], sabc[2], batches));
        } else if constexpr (std::is_same_v<c32, T>) {
            check(hipblasCgemmStridedBatched(
                handle, lhs_op, rhs_op, mnk[0], mnk[1], mnk[2],
                reinterpret_cast<const hipComplex*>(&alpha),
                reinterpret_cast<const hipComplex*>(lhs), labc[0], sabc[0],
                reinterpret_cast<const hipComplex*>(rhs), labc[1], sabc[1],
                reinterpret_cast<const hipComplex*>(&beta),
                reinterpret_cast<hipComplex*>(output), labc[2], sabc[2], batches));
        } else if constexpr (std::is_same_v<c64, T>) {
            check(hipblasZgemmStridedBatched(
                handle, lhs_op, rhs_op, mnk[0], mnk[1], mnk[2],
                reinterpret_cast<const hipDoubleComplex*>(&alpha),
                reinterpret_cast<const hipDoubleComplex*>(lhs), labc[0], sabc[0],
                reinterpret_cast<const hipDoubleComplex*>(rhs), labc[1], sabc[1],
                reinterpret_cast<const hipDoubleComplex*>(&beta),
                reinterpret_cast<hipDoubleComplex*>(output), labc[2], sabc[2], batches));
        }
    }
}

namespace noa::cuda {
    void cublas_clear_cache(i32 device) {
        std::unique_ptr<CuBlasHandle>& cached_handle = cublas_cache_handle_(device);
        cached_handle = nullptr;
    }

    template<typename T>
    void matmul(
        const T* lhs, const Strides4<i64>& lhs_strides, const Shape4<i64>& lhs_shape,
        const T* rhs, const Strides4<i64>& rhs_strides, const Shape4<i64>& rhs_shape,
        T alpha, T beta, bool lhs_transpose, bool rhs_transpose,
        T* output, const Strides4<i64>& output_strides, const Shape4<i64>& output_shape,
        Stream& stream
    ) {
        auto [mnk, secondmost_strides, are_column_major] = ni::extract_matmul_layout(
            lhs_strides, lhs_shape, rhs_strides, rhs_shape, output_strides, output_shape,
            lhs_transpose, rhs_transpose);

        const auto labc = secondmost_strides.vec.as_safe<i32>();
        const auto sabc = Vec3<i64>{lhs_strides[0], rhs_strides[0], output_strides[0]};

        cublas_gemm_(are_column_major, lhs_transpose, rhs_transpose,
                     mnk.as_safe<i32>(), labc, sabc, static_cast<i32>(output_shape[0]), alpha, beta,
                     lhs, rhs, output, stream);
    }

    #define NOA_INSTANTIATE_MATMUL_(T)                          \
    template void matmul<T>(                                    \
        const T*, const Strides4<i64>&, const Shape4<i64>&,     \
        const T*, const Strides4<i64>&, const Shape4<i64>&,     \
        T, T, bool, bool,                                       \
        T*, const Strides4<i64>&, const Shape4<i64>&,           \
        Stream&)

    NOA_INSTANTIATE_MATMUL_(f32);
    NOA_INSTANTIATE_MATMUL_(f64);
    NOA_INSTANTIATE_MATMUL_(c32);
    NOA_INSTANTIATE_MATMUL_(c64);
}
