#include "hip/hip_runtime.h"
#include "noa/gpu/cuda/Exception.h"
#include "noa/gpu/cuda/memory/Set.h"

namespace {
    using namespace noa;

    template<typename T>
    __global__ void set_(T* array, size_t elements, T value) {
        #pragma unroll 10
        for (size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
             idx < elements;
             idx += blockDim.x * gridDim.x)
            array[idx] = value;
    }

    constexpr dim3 THREADS(32, 16);

    template<typename T>
    __global__ __launch_bounds__(THREADS.x * THREADS.y)
    void set_(T* array, uint array_pitch, uint3_t shape, T value) {
        uint3_t gid(blockIdx.x * blockDim.x + threadIdx.x,
                    blockIdx.y * blockDim.y + threadIdx.y,
                    blockIdx.z);
        if (gid.y > shape.y) // x is checked later and z cannot be OOB
            return;
        array += (gid.z * shape.y + gid.y) * array_pitch; // offset to current line

        // One wrap per line.
        #pragma unroll 8
        for (uint x = gid.x; x < shape.x; x += THREADS.x)
            array[x] = value;
    }
}

namespace noa::cuda::memory::details {
    template<typename T>
    void set(T* array, size_t elements, T value, Stream& stream) {
        uint threads = 512U;
        uint blocks = math::min(noa::math::divideUp(static_cast<uint>(elements), threads), 8192U);
        set_<<<blocks, threads, 0, stream.id()>>>(array, elements, value);
        NOA_THROW_IF(hipGetLastError());
    }

    template<typename T>
    void set(T* array, size_t array_pitch, size3_t shape, T value, Stream& stream) {
        uint3_t u_shape(shape);
        uint blocks_y = math::divideUp(u_shape.y, THREADS.y);
        dim3 blocks{1, blocks_y, u_shape.z}; // one wrap in X
        set_<<<blocks, THREADS, 0, stream.id()>>>(array, array_pitch, u_shape, value);
        NOA_THROW_IF(hipGetLastError());
    }

    #define NOA_INSTANTIATE_SET_(T) \
    template void set<T>(T*, size_t, T, Stream&); \
    template void set<T>(T*, size_t, size3_t, T, Stream&);

    NOA_INSTANTIATE_SET_(char);
    NOA_INSTANTIATE_SET_(short);
    NOA_INSTANTIATE_SET_(int);
    NOA_INSTANTIATE_SET_(long);
    NOA_INSTANTIATE_SET_(long long);
    NOA_INSTANTIATE_SET_(unsigned char);
    NOA_INSTANTIATE_SET_(unsigned short);
    NOA_INSTANTIATE_SET_(unsigned int);
    NOA_INSTANTIATE_SET_(unsigned long);
    NOA_INSTANTIATE_SET_(unsigned long long);
    NOA_INSTANTIATE_SET_(float);
    NOA_INSTANTIATE_SET_(double);
    NOA_INSTANTIATE_SET_(cfloat_t);
    NOA_INSTANTIATE_SET_(cdouble_t);
}
