#include "hip/hip_runtime.h"
#include "noa/gpu/cuda/Exception.h"
#include "noa/gpu/cuda/memory/Set.h"

namespace {
    using namespace noa;

    template<typename T>
    __global__ void set_(T* array, size_t elements, T value) {
        for (size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
             idx < elements;
             idx += blockDim.x * gridDim.x)
            array[idx] = value;
    }
}

namespace noa::cuda::memory {
    template<typename T>
    void set(T* array, size_t elements, T value, Stream& stream) {
        if (value == 0) {
            NOA_THROW_IF(hipMemsetAsync(array, 0, elements * sizeof(T), stream.id()));
        } else {
            uint threads = 512U;
            uint blocks = math::min((elements + threads - 1) / threads, static_cast<size_t>(8192U));
            set_<<<blocks, threads, 0, stream.id()>>>(array, elements, value);
            NOA_THROW_IF(hipPeekAtLastError());
        }
    }
    #define NOA_INSTANTIATE_SET_(T) \
    template void set<T>(T*, size_t, T, Stream&)

    NOA_INSTANTIATE_SET_(char);
    NOA_INSTANTIATE_SET_(short);
    NOA_INSTANTIATE_SET_(int);
    NOA_INSTANTIATE_SET_(long);
    NOA_INSTANTIATE_SET_(long long);
    NOA_INSTANTIATE_SET_(unsigned char);
    NOA_INSTANTIATE_SET_(unsigned short);
    NOA_INSTANTIATE_SET_(unsigned int);
    NOA_INSTANTIATE_SET_(unsigned long);
    NOA_INSTANTIATE_SET_(unsigned long long);
    NOA_INSTANTIATE_SET_(float);
    NOA_INSTANTIATE_SET_(double);
}
