#include "hip/hip_runtime.h"
#include "noa/common/Assert.h"
#include "noa/common/Math.h"
#include "noa/gpu/cuda/memory/Set.h"
#include "noa/gpu/cuda/util/Block.cuh"
#include "noa/gpu/cuda/util/Pointers.h"

namespace {
    using namespace ::noa;

    constexpr uint ELEMENTS_PER_THREAD = 4;
    constexpr uint BLOCK_SIZE = 256;
    constexpr uint BLOCK_WORK_SIZE = BLOCK_SIZE * ELEMENTS_PER_THREAD;

    constexpr dim3 BLOCK_SIZE_2D(32, BLOCK_SIZE / 32);
    constexpr dim3 ELEMENTS_PER_THREAD_2D(1, 4);
    constexpr dim3 BLOCK_WORK_SIZE_2D(BLOCK_SIZE_2D.x * ELEMENTS_PER_THREAD_2D.x,
                                      BLOCK_SIZE_2D.y * ELEMENTS_PER_THREAD_2D.y);

    template<typename T, int VEC_SIZE>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void set1D_(T* src, uint2_t stride, uint elements_per_batch, T value) {
        const uint base = BLOCK_WORK_SIZE * blockIdx.x;
        src += blockIdx.y * stride[0];

        if constexpr (VEC_SIZE == 1) {
            #pragma unroll
            for (int i = 0; i < ELEMENTS_PER_THREAD; ++i) {
                const uint gid = base + BLOCK_SIZE * i + threadIdx.x;
                if (gid < elements_per_batch)
                    src[gid * stride[1]] = value;
            }
        } else {
            NOA_ASSERT(stride[1] == 1);
            const uint remaining = elements_per_batch - base;
            src += base;
            if (remaining < BLOCK_WORK_SIZE) {
                #pragma unroll
                for (int i = 0; i < ELEMENTS_PER_THREAD; ++i) {
                    const uint gid = BLOCK_SIZE * i + threadIdx.x;
                    if (gid < remaining)
                        src[gid] = value;
                }
            } else {
                T values[ELEMENTS_PER_THREAD];
                #pragma unroll
                for (uint i = 0; i < ELEMENTS_PER_THREAD; ++i)
                    values[i] = value;
                cuda::util::block::vectorizedStore<BLOCK_SIZE, ELEMENTS_PER_THREAD, VEC_SIZE>(values, src, threadIdx.x);
            }
        }
    }

    template<typename T>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void set4D_(T* src, uint4_t src_stride,
                uint2_t shape, T value, uint blocks_x) {
        const uint2_t index = indexing::indexes(blockIdx.x, blocks_x);
        const int4_t gid(blockIdx.z,
                         blockIdx.y,
                         BLOCK_WORK_SIZE_2D.y * index[0] + threadIdx.y,
                         BLOCK_WORK_SIZE_2D.x * index[1] + threadIdx.x);
        src += indexing::at(gid[0], gid[1], src_stride);

        #pragma unroll
        for (int k = 0; k < ELEMENTS_PER_THREAD_2D.y; ++k) {
            #pragma unroll
            for (int l = 0; l < ELEMENTS_PER_THREAD_2D.x; ++l) {
                const uint ik = gid[2] + BLOCK_SIZE_2D.y * k;
                const uint il = gid[3] + BLOCK_SIZE_2D.x * l;
                if (ik < shape[0] && il < shape[1])
                    src[ik * src_stride[2] + il * src_stride[3]] = value;
            }
        }
    }
}

namespace noa::cuda::memory::details {
    template<typename T, typename>
    void set(T* src, size_t elements, T value, Stream& stream) {
        if (!elements)
            return;

        const uint2_t stride{0, 1};
        const auto elements_per_batch = static_cast<uint>(elements);
        const dim3 blocks(noa::math::divideUp(elements_per_batch, BLOCK_WORK_SIZE));
        const int vec_size = noa::cuda::util::maxVectorCount(src);
        if (vec_size == 4) {
            stream.enqueue("memory::set", set1D_<T, 4>,
                           {blocks, BLOCK_SIZE}, src, stride, elements_per_batch, value);
        } else if (vec_size == 2) {
            stream.enqueue("memory::set", set1D_<T, 2>,
                           {blocks, BLOCK_SIZE}, src, stride, elements_per_batch, value);
        } else {
            stream.enqueue("memory::set", set1D_<T, 1>,
                           {blocks, BLOCK_SIZE}, src, stride, elements_per_batch, value);
        }
    }

    template<typename T, typename>
    void set(const shared_t<T[]>& src, size4_t stride, size4_t shape, T value, Stream& stream) {
        if (!shape.elements())
            return;

        const bool4_t is_contiguous = indexing::isContiguous(stride, shape);
        if (is_contiguous[1] && is_contiguous[2]) {
            const uint4_t uint_shape{shape};
            const uint2_t uint_stride{stride[0], stride[3]};
            const uint elements_per_batch = is_contiguous[0] ?
                                            uint_shape.elements() :
                                            uint3_t{uint_shape.get() + 1}.elements();
            const dim3 blocks(noa::math::divideUp(elements_per_batch, BLOCK_WORK_SIZE),
                              is_contiguous[0] ? 1 : shape[0]);

            uint vec_size = is_contiguous[3] ? noa::cuda::util::maxVectorCount(src.get()) : 1;
            if (blocks.y > 1) // make sure the beginning of each batch preserves the alignment
                vec_size = stride[0] % vec_size ? 1 : vec_size;

            if (vec_size == 4) {
                stream.enqueue("memory::set", set1D_<T, 4>,
                               {blocks, BLOCK_SIZE}, src.get(), uint_stride, elements_per_batch, value);
            } else if (vec_size == 2) {
                stream.enqueue("memory::set", set1D_<T, 2>,
                               {blocks, BLOCK_SIZE}, src.get(), uint_stride, elements_per_batch, value);
            } else {
                stream.enqueue("memory::set", set1D_<T, 1>,
                               {blocks, BLOCK_SIZE}, src.get(), uint_stride, elements_per_batch, value);
            }
        } else {
            const uint2_t i_shape{shape.get() + 2};
            const uint blocks_x = noa::math::divideUp(i_shape[1], BLOCK_WORK_SIZE_2D.x);
            const uint blocks_y = noa::math::divideUp(i_shape[0], BLOCK_WORK_SIZE_2D.y);
            const dim3 blocks(blocks_x * blocks_y, shape[1], shape[0]);
            stream.enqueue("memory::set", set4D_<T>, {blocks, BLOCK_SIZE_2D},
                           src.get(), uint4_t{stride}, i_shape, value, blocks_x);
        }
        stream.attach(src);
    }

    #define NOA_INSTANTIATE_SET_(T)                     \
    template void set<T, void>(T*, size_t, T, Stream&); \
    template void set<T, void>(const shared_t<T[]>&, size4_t, size4_t, T, Stream&);

    NOA_INSTANTIATE_SET_(bool);
    NOA_INSTANTIATE_SET_(int8_t);
    NOA_INSTANTIATE_SET_(int16_t);
    NOA_INSTANTIATE_SET_(int32_t);
    NOA_INSTANTIATE_SET_(int64_t);
    NOA_INSTANTIATE_SET_(uint8_t);
    NOA_INSTANTIATE_SET_(uint16_t);
    NOA_INSTANTIATE_SET_(uint32_t);
    NOA_INSTANTIATE_SET_(uint64_t);
    NOA_INSTANTIATE_SET_(half_t);
    NOA_INSTANTIATE_SET_(float);
    NOA_INSTANTIATE_SET_(double);
    NOA_INSTANTIATE_SET_(chalf_t);
    NOA_INSTANTIATE_SET_(cfloat_t);
    NOA_INSTANTIATE_SET_(cdouble_t);
}
