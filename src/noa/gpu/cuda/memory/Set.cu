#include "hip/hip_runtime.h"
#include "noa/gpu/cuda/Exception.h"
#include "noa/gpu/cuda/memory/Set.h"

namespace {
    using namespace noa;

    template<typename T>
    __global__ void set_(T* array, size_t elements, T value) {
        for (size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
             idx < elements;
             idx += blockDim.x * gridDim.x)
            array[idx] = value;
    }
}

namespace noa::cuda::memory {
    template<typename T>
    void set(T* array, size_t elements, T value, Stream& stream) {
        if (value == 0) {
            NOA_THROW_IF(hipMemsetAsync(array, 0, elements * sizeof(T), stream.id()));
        } else {
            uint threads = 512U;
            uint blocks = math::min((elements + threads - 1) / threads, static_cast<size_t>(8192U));
            set_<<<blocks, threads, 0, stream.id()>>>(array, elements, value);
            NOA_THROW_IF(hipPeekAtLastError());
        }
    }
    #define INSTANTIATE_SET(T) \
    template void set<T>(T*, size_t, T, Stream&)

    INSTANTIATE_SET(char);
    INSTANTIATE_SET(short);
    INSTANTIATE_SET(int);
    INSTANTIATE_SET(long);
    INSTANTIATE_SET(long long);
    INSTANTIATE_SET(unsigned char);
    INSTANTIATE_SET(unsigned short);
    INSTANTIATE_SET(unsigned int);
    INSTANTIATE_SET(unsigned long);
    INSTANTIATE_SET(unsigned long long);
    INSTANTIATE_SET(float);
    INSTANTIATE_SET(double);
}
