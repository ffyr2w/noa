#include "hip/hip_runtime.h"
#include "noa/common/Assert.h"
#include "noa/common/Math.h"

#include "noa/gpu/cuda/Exception.h"
#include "noa/gpu/cuda/Types.h"
#include "noa/gpu/cuda/memory/Index.h"
#include "noa/gpu/cuda/memory/PtrDevice.h"
#include "noa/gpu/cuda/util/Pointers.h"

namespace {
    using namespace noa;
    constexpr uint BLOCK_SIZE = 256;
    constexpr dim3 BLOCK_SIZE_2D(32, BLOCK_SIZE / 32);
    constexpr uint ELEMENTS_PER_THREAD = 4;
    constexpr dim3 BLOCK_WORK_SIZE_2D(BLOCK_SIZE_2D.x * ELEMENTS_PER_THREAD, BLOCK_SIZE_2D.y);

    template<typename T>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void extractOrNothing_(const T* __restrict__ input, uint4_t input_strides, int4_t input_shape,
                           T* __restrict__ subregions, uint4_t subregion_strides, int2_t subregion_shape,
                           const int4_t* __restrict__ origins, int4_t order, uint blocks_x) {
        const uint2_t index = indexing::indexes(blockIdx.x, blocks_x);
        const int4_t gid{blockIdx.z,
                         blockIdx.y,
                         BLOCK_WORK_SIZE_2D.y * index[0] + threadIdx.y,
                         BLOCK_WORK_SIZE_2D.x * index[1] + threadIdx.x};
        if (gid[2] >= subregion_shape[0])
            return;

        const int4_t origin = indexing::reorder(origins[gid[0]], order); // TODO constant memory?
        const int ii = origin[0];
        const int ij = origin[1] + gid[1];
        const int ik = origin[2] + gid[2];
        if (ii < 0 || ii >= input_shape[0] ||
            ij < 0 || ij >= input_shape[1] ||
            ik < 0 || ik >= input_shape[2])
            return;

        input += indexing::at(ii, ij, ik, input_strides);
        subregions += indexing::at(gid[0], gid[1], gid[2], subregion_strides);

        for (int i = 0; i < ELEMENTS_PER_THREAD; ++i) {
            const int ol = gid[3] + static_cast<int>(BLOCK_SIZE_2D.x) * i;
            const int il = origin[3] + ol;
            if (ol < subregion_shape[1] && il >= 0 && il < input_shape[3])
                subregions[ol * subregion_strides[3]] = input[il * input_strides[3]];
        }
    }

    template<typename T>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void extractOrValue_(const T* __restrict__ input, uint4_t input_strides, int4_t input_shape,
                         T* __restrict__ subregions, uint4_t subregion_strides, int2_t subregion_shape,
                         const int4_t* __restrict__ origins, T value, int4_t order, uint blocks_x) {
        const uint2_t index = indexing::indexes(blockIdx.x, blocks_x);
        const int4_t gid{blockIdx.z,
                         blockIdx.y,
                         BLOCK_WORK_SIZE_2D.y * index[0] + threadIdx.y,
                         BLOCK_WORK_SIZE_2D.x * index[1] + threadIdx.x};
        if (gid[2] >= subregion_shape[0])
            return;

        const int4_t origin = indexing::reorder(origins[gid[0]], order); // TODO constant memory?
        const int ii = origin[0];
        const int ij = origin[1] + gid[1];
        const int ik = origin[2] + gid[2];
        const bool is_in = ii >= 0 && ii < input_shape[0] &&
                           ij >= 0 && ij < input_shape[1] &&
                           ik >= 0 && ik < input_shape[2];

        subregions += indexing::at(gid[0], gid[1], gid[2], subregion_strides);
        for (int i = 0; i < ELEMENTS_PER_THREAD; ++i) {
            const int ol = gid[3] + static_cast<int>(BLOCK_SIZE_2D.x) * i;
            if (ol >= subregion_shape[1])
                return;

            const int il = origin[3] + ol;
            if (is_in && il >= 0 && il < input_shape[3])
                subregions[ol * subregion_strides[3]] = input[indexing::at(ii, ij, ik, il, input_strides)];
            else
                subregions[ol * subregion_strides[3]] = value;
        }
    }

    template<BorderMode MODE, typename T>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void extract_(const T* __restrict__ input, uint4_t input_strides, int4_t input_shape,
                  T* __restrict__ subregions, uint4_t subregion_strides, int2_t subregion_shape,
                  const int4_t* __restrict__ origins, int4_t order, uint blocks_x) {
        const uint2_t index = indexing::indexes(blockIdx.x, blocks_x);
        const int4_t gid{blockIdx.z,
                         blockIdx.y,
                         BLOCK_WORK_SIZE_2D.y * index[0] + threadIdx.y,
                         BLOCK_WORK_SIZE_2D.x * index[1] + threadIdx.x};
        if (gid[2] >= subregion_shape[0])
            return;

        const int4_t origin = indexing::reorder(origins[gid[0]], order); // TODO constant memory?
        const int ii = indexing::at<MODE>(origin[0], input_shape[0]);
        const int ij = indexing::at<MODE>(origin[1] + gid[1], input_shape[1]);
        const int ik = indexing::at<MODE>(origin[2] + gid[2], input_shape[2]);

        input += indexing::at(ii, ij, ik, input_strides);
        subregions += indexing::at(gid[0], gid[1], gid[2], subregion_strides);

        for (int i = 0; i < ELEMENTS_PER_THREAD; ++i) {
            const int ol = gid[2] + static_cast<int>(BLOCK_SIZE_2D.x) * i;
            const int il = indexing::at<MODE>(origin[3] + ol, input_shape[3]);
            if (ol < subregion_shape[1])
                subregions[ol * subregion_strides[3]] = input[il * input_strides[3]];
        }
    }

    template<typename T>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void insert_(const T* __restrict__ subregions, uint4_t subregion_strides, int2_t subregion_shape,
                 T* __restrict__ output, uint4_t output_strides, int4_t output_shape,
                 const int4_t* __restrict__ origins, int4_t order, uint blocks_x) {
        const uint2_t index = indexing::indexes(blockIdx.x, blocks_x);
        const int4_t gid{blockIdx.z,
                         blockIdx.y,
                         BLOCK_WORK_SIZE_2D.y * index[0] + threadIdx.y,
                         BLOCK_WORK_SIZE_2D.x * index[1] + threadIdx.x};
        if (gid[2] >= subregion_shape[0])
            return;

        const int4_t origin = indexing::reorder(origins[gid[0]], order); // TODO constant memory?
        const int oi = origin[0];
        const int oj = origin[1] + gid[1];
        const int ok = origin[2] + gid[2];
        if (oi < 0 || oi >= output_shape[0] ||
            oj < 0 || oj >= output_shape[1] ||
            ok < 0 || ok >= output_shape[2])
            return;

        output += indexing::at(oi, oj, ok, output_strides);
        subregions += indexing::at(gid[0], gid[1], gid[2], subregion_strides);

        for (int i = 0; i < ELEMENTS_PER_THREAD; ++i) {
            const int il = gid[3] + static_cast<int>(BLOCK_SIZE_2D.x) * i;
            const int ol = origin[3] + il;
            if (il < subregion_shape[1] && ol >= 0 && ol < output_shape[3])
                output[ol * output_strides[3]] = subregions[il * subregion_strides[3]];
        }
    }
}

namespace noa::cuda::memory {
    template<typename T, typename>
    void extract(const shared_t<T[]>& input, size4_t input_strides, size4_t input_shape,
                 const shared_t<T[]>& subregions, size4_t subregion_strides, size4_t subregion_shape,
                 const shared_t<int4_t[]>& origins, BorderMode border_mode, T border_value, Stream& stream) {
        NOA_ASSERT(input != subregions);

        // Reorder the DHW dimensions to the rightmost order.
        // We'll have to reorder the origins similarly later.
        const size3_t order_3d = indexing::order(size3_t(subregion_strides.get(1)),
                                                 size3_t(subregion_shape.get(1))) + 1;
        const int4_t order(0, order_3d[0], order_3d[1], order_3d[2]);
        input_strides = indexing::reorder(input_strides, order);
        input_shape = indexing::reorder(input_shape, order);
        subregion_strides = indexing::reorder(subregion_strides, order);
        subregion_shape = indexing::reorder(subregion_shape, order);

        const shared_t<int4_t[]> d_origins = util::ensureDeviceAccess(origins, stream, subregion_shape[0]);
        const int4_t i_shape(input_shape);
        const int2_t o_shape(subregion_shape.get(2));

        const uint blocks_x = math::divideUp(static_cast<uint>(o_shape[1]), BLOCK_WORK_SIZE_2D.x);
        const uint blocks_y = math::divideUp(static_cast<uint>(o_shape[0]), BLOCK_WORK_SIZE_2D.y);
        const dim3 blocks(blocks_x * blocks_y, subregion_shape[1], subregion_shape[0]);
        switch (border_mode) {
            case BORDER_NOTHING:
                stream.enqueue("memory::extractOrNothing", extractOrNothing_<T>, {blocks, BLOCK_SIZE_2D},
                               input.get(), uint4_t(input_strides), i_shape,
                               subregions.get(), uint4_t(subregion_strides),
                               o_shape, d_origins.get(), order, blocks_x);
                break;
            case BORDER_ZERO:
                stream.enqueue("memory::extractOrValue", extractOrValue_<T>, {blocks, BLOCK_SIZE_2D},
                               input.get(), uint4_t(input_strides), i_shape,
                               subregions.get(), uint4_t(subregion_strides),
                               o_shape, d_origins.get(), static_cast<T>(0), order, blocks_x);
                break;
            case BORDER_VALUE:
                stream.enqueue("memory::extractOrValue", extractOrValue_<T>, {blocks, BLOCK_SIZE_2D},
                               input.get(), uint4_t(input_strides), i_shape,
                               subregions.get(), uint4_t(subregion_strides),
                               o_shape, d_origins.get(), border_value, order, blocks_x);
                break;
            case BORDER_CLAMP:
                stream.enqueue("memory::extract<CLAMP>", extract_<BORDER_CLAMP, T>, {blocks, BLOCK_SIZE_2D},
                               input.get(), uint4_t(input_strides), i_shape,
                               subregions.get(), uint4_t(subregion_strides),
                               o_shape, d_origins.get(), order, blocks_x);
                break;
            case BORDER_MIRROR:
                stream.enqueue("memory::extract<MIRROR>", extract_<BORDER_MIRROR, T>, {blocks, BLOCK_SIZE_2D},
                               input.get(), uint4_t(input_strides), i_shape,
                               subregions.get(), uint4_t(subregion_strides),
                               o_shape, d_origins.get(), order, blocks_x);
                break;
            case BORDER_REFLECT:
                stream.enqueue("memory::extract<REFLECT>", extract_<BORDER_REFLECT, T>, {blocks, BLOCK_SIZE_2D},
                               input.get(), uint4_t(input_strides), i_shape,
                               subregions.get(), uint4_t(subregion_strides),
                               o_shape, d_origins.get(), order, blocks_x);
                break;
            default:
                NOA_THROW("Border mode {} is not supported", border_mode);
        }
        stream.attach(input, subregions, d_origins);
    }

    template<typename T, typename>
    void insert(const shared_t<T[]>& subregions, size4_t subregion_strides, size4_t subregion_shape,
                const shared_t<T[]>& output, size4_t output_strides, size4_t output_shape,
                const shared_t<int4_t[]>& origins, Stream& stream) {
        NOA_ASSERT(subregions != output);

        // Reorder the DHW dimensions to the rightmost order.
        // We'll have to reorder the origins similarly later.
        const size3_t order_3d = indexing::order(size3_t(subregion_strides.get(1)),
                                                 size3_t(subregion_shape.get(1))) + 1;
        const int4_t order(0, order_3d[0], order_3d[1], order_3d[2]);
        output_strides = indexing::reorder(output_strides, order);
        output_shape = indexing::reorder(output_shape, order);
        subregion_strides = indexing::reorder(subregion_strides, order);
        subregion_shape = indexing::reorder(subregion_shape, order);

        const shared_t<int4_t[]> d_origins = util::ensureDeviceAccess(origins, stream, subregion_shape[0]);
        const int2_t i_shape(subregion_shape.get(2));
        const uint blocks_x = math::divideUp(static_cast<uint>(i_shape[1]), BLOCK_WORK_SIZE_2D.x);
        const uint blocks_y = math::divideUp(static_cast<uint>(i_shape[0]), BLOCK_WORK_SIZE_2D.y);
        const dim3 blocks(blocks_x * blocks_y, subregion_shape[1], subregion_shape[0]);
        stream.enqueue("memory::insert", insert_<T>, {blocks, BLOCK_SIZE_2D},
                       subregions.get(), uint4_t(subregion_strides), i_shape,
                       output.get(), uint4_t(output_strides), int4_t{output_shape}, d_origins.get(),
                       order, blocks_x);
        stream.attach(subregions, output, d_origins);
    }

    #define INSTANTIATE_EXTRACT_INSERT_(T)                                                                                                                              \
    template void extract<T, void>(const shared_t<T[]>&, size4_t, size4_t, const shared_t<T[]>&, size4_t, size4_t, const shared_t<int4_t[]>&, BorderMode, T, Stream&);  \
    template void insert<T, void>(const shared_t<T[]>&, size4_t, size4_t, const shared_t<T[]>&, size4_t, size4_t, const shared_t<int4_t[]>&, Stream&)

    INSTANTIATE_EXTRACT_INSERT_(bool);
    INSTANTIATE_EXTRACT_INSERT_(int8_t);
    INSTANTIATE_EXTRACT_INSERT_(int16_t);
    INSTANTIATE_EXTRACT_INSERT_(int32_t);
    INSTANTIATE_EXTRACT_INSERT_(int64_t);
    INSTANTIATE_EXTRACT_INSERT_(uint8_t);
    INSTANTIATE_EXTRACT_INSERT_(uint16_t);
    INSTANTIATE_EXTRACT_INSERT_(uint32_t);
    INSTANTIATE_EXTRACT_INSERT_(uint64_t);
    INSTANTIATE_EXTRACT_INSERT_(half_t);
    INSTANTIATE_EXTRACT_INSERT_(float);
    INSTANTIATE_EXTRACT_INSERT_(double);
    INSTANTIATE_EXTRACT_INSERT_(chalf_t);
    INSTANTIATE_EXTRACT_INSERT_(cfloat_t);
    INSTANTIATE_EXTRACT_INSERT_(cdouble_t);
}
