#include "hip/hip_runtime.h"
#include "noa/common/Math.h"
#include "noa/gpu/cuda/Exception.h"
#include "noa/gpu/cuda/util/Traits.h"
#include "noa/gpu/cuda/memory/Permute.h"

#include "noa/gpu/cuda/util/Block.cuh"

namespace {
    using namespace ::noa;
    using namespace ::noa::cuda;

    // Transpose XZ plane (by chunk of 32x32 tiles) for every Y.
    constexpr uint32_t TILE_DIM = 32;
    constexpr dim3 BLOCK_SIZE(TILE_DIM, 256 / TILE_DIM);

    // Out-of-place.
    // The XZ tile along Y becomes X'Y' (X'=Z, Y'=X) along Z' (Z'=Y)
    template<typename T, bool IS_MULTIPLE_OF_TILE>
    __global__ __launch_bounds__(BLOCK_SIZE.x * BLOCK_SIZE.y)
    void permute0231_(AccessorRestrict<const T, 4, uint32_t> input_swapped,
                      AccessorRestrict<T, 4, uint32_t> output,
                      uint2_t shape /* ZX */, uint32_t blocks_x) {
        using uninit_t = cuda::util::traits::uninitialized_type_t<T>;
        __shared__ uninit_t buffer[TILE_DIM][TILE_DIM + 1];
        T(& tile)[TILE_DIM][TILE_DIM + 1] = *reinterpret_cast<T(*)[TILE_DIM][TILE_DIM + 1]>(&buffer);

        const auto input_swapped_ = input_swapped[blockIdx.z][blockIdx.y];
        const auto output_ = output[blockIdx.z][blockIdx.y];

        // Get the current indexes.
        const uint2_t tid{threadIdx.y, threadIdx.x};
        const uint2_t index = indexing::indexes(blockIdx.x, blocks_x);
        const uint2_t offset = TILE_DIM * index; // ZX

        // Read tile to shared memory.
        const uint2_t old_gid = offset + tid;
        for (uint32_t repeat = 0; repeat < TILE_DIM; repeat += BLOCK_SIZE.y) {
            const uint32_t gz = old_gid[0] + repeat;
            if (IS_MULTIPLE_OF_TILE || (old_gid[1] < shape[1] && gz < shape[0]))
                tile[tid[0] + repeat][tid[1]] = input_swapped_(gz, old_gid[1]);
        }

        util::block::synchronize();

        // Write permuted tile to global memory.
        const uint2_t new_gid = offset.flip() + tid; // ZX.flip() -> XZ -> Y'X'
        for (uint32_t repeat = 0; repeat < TILE_DIM; repeat += BLOCK_SIZE.y) {
            const uint32_t gy = new_gid[0] + repeat;
            if (IS_MULTIPLE_OF_TILE || (new_gid[1] < shape[0] && gy < shape[1]))
                output_(gy, new_gid[1]) = tile[tid[1]][tid[0] + repeat];
        }
    }

    // Since all axes are permuted, in-place permute cannot easily be expressed as a 2D transposition
    // along a COMMON plane. https://www.aldapa.eus/res/cuTranspose/Readme.html has an implementation
    // based on a 3D shared memory array, but since it is unlikely to be used anyway, don't bother for now.
}

namespace noa::cuda::memory::details {
    template<typename T>
    void permute0231(const shared_t<T[]>& input, dim4_t input_strides,
                     const shared_t<T[]>& output, dim4_t output_strides,
                     dim4_t shape, Stream& stream) {
        const auto uint_shape = safe_cast<uint2_t>(dim2_t{shape[1], shape[3]});
        const bool are_multiple_tile = all((uint_shape % TILE_DIM) == 0);

        const uint32_t blocks_x = math::divideUp(uint_shape[1], TILE_DIM);
        const uint32_t blocks_z = math::divideUp(uint_shape[0], TILE_DIM);
        const dim3 blocks(blocks_x * blocks_z, shape[2], shape[0]);

        const AccessorRestrict<const T, 4, uint32_t> input_accessor(input.get(), safe_cast<uint4_t>(input_strides));
        const AccessorRestrict<T, 4, uint32_t> output_accessor(output.get(), safe_cast<uint4_t>(output_strides));
        const auto swapped_input = input_accessor.swap(1, 2); // Y -> Z'

        if (are_multiple_tile) {
            stream.enqueue("memory::permute0231", permute0231_<T, true>, {blocks, BLOCK_SIZE},
                           swapped_input, output_accessor, uint_shape, blocks_x);
        } else {
            stream.enqueue("memory::permute0231", permute0231_<T, false>, {blocks, BLOCK_SIZE},
                           swapped_input, output_accessor, uint_shape, blocks_x);
        }
        stream.attach(input, output);
    }
}

#define NOA_INSTANTIATE_TRANSPOSE_(T) \
template void noa::cuda::memory::details::permute0231<T>(const shared_t<T[]>&, dim4_t, const shared_t<T[]>&, dim4_t, dim4_t, Stream&)

NOA_INSTANTIATE_TRANSPOSE_(bool);
NOA_INSTANTIATE_TRANSPOSE_(int8_t);
NOA_INSTANTIATE_TRANSPOSE_(int16_t);
NOA_INSTANTIATE_TRANSPOSE_(int32_t);
NOA_INSTANTIATE_TRANSPOSE_(int64_t);
NOA_INSTANTIATE_TRANSPOSE_(uint8_t);
NOA_INSTANTIATE_TRANSPOSE_(uint16_t);
NOA_INSTANTIATE_TRANSPOSE_(uint32_t);
NOA_INSTANTIATE_TRANSPOSE_(uint64_t);
NOA_INSTANTIATE_TRANSPOSE_(half_t);
NOA_INSTANTIATE_TRANSPOSE_(float);
NOA_INSTANTIATE_TRANSPOSE_(double);
NOA_INSTANTIATE_TRANSPOSE_(chalf_t);
NOA_INSTANTIATE_TRANSPOSE_(cfloat_t);
NOA_INSTANTIATE_TRANSPOSE_(cdouble_t);
