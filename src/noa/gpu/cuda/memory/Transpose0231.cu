#include "hip/hip_runtime.h"
#include "noa/common/Math.h"
#include "noa/gpu/cuda/Exception.h"
#include "noa/gpu/cuda/util/Traits.h"
#include "noa/gpu/cuda/memory/Transpose.h"

#include "noa/gpu/cuda/util/Block.cuh"

namespace {
    using namespace ::noa;
    using namespace ::noa::cuda;

    // Transpose XZ plane (by chunk of 32x32 tiles) for every Y.
    constexpr uint TILE_DIM = 32;
    constexpr dim3 BLOCK_SIZE(TILE_DIM, 256 / TILE_DIM);

    // Out-of-place.
    // The XZ tile along Y becomes X'Y' (X'=Z, Y'=X) along Z' (Z'=Y)
    template<typename T, bool IS_MULTIPLE_OF_TILE>
    __global__ __launch_bounds__(BLOCK_SIZE.x * BLOCK_SIZE.y)
    void transpose0231_(const T* __restrict__ input, uint4_t input_stride,
                       T* __restrict__ output, uint4_t output_stride,
                       uint2_t shape /* ZX */, uint blocks_x) {
        using uninit_t = cuda::util::traits::uninitialized_type_t<T>;
        __shared__ uninit_t buffer[TILE_DIM][TILE_DIM + 1];
        T(& tile)[TILE_DIM][TILE_DIM + 1] = *reinterpret_cast<T(*)[TILE_DIM][TILE_DIM + 1]>(&buffer);

        input += blockIdx.z * input_stride[0];
        output += blockIdx.z * output_stride[0];
        input += blockIdx.y * input_stride[2]; // Y->Z'
        output += blockIdx.y * output_stride[1];

        // Get the current indexes.
        const uint2_t tid(threadIdx.y, threadIdx.x);
        const uint2_t index = indexing::indexes(blockIdx.x, blocks_x);
        const uint2_t offset = TILE_DIM * index; // ZX

        // Read tile to shared memory.
        const uint2_t old_gid = offset + tid;
        for (uint repeat = 0; repeat < TILE_DIM; repeat += BLOCK_SIZE.y) {
            const uint gz = old_gid[0] + repeat;
            if (IS_MULTIPLE_OF_TILE || (old_gid[1] < shape[1] && gz < shape[0]))
                tile[tid[0] + repeat][tid[1]] = input[gz * input_stride[1] + old_gid[1] * input_stride[3]];
        }

        util::block::synchronize();

        // Write transposed tile to global memory.
        const uint2_t new_gid = offset.flip() + tid; // ZX.flip() -> XZ -> Y'X'
        for (uint repeat = 0; repeat < TILE_DIM; repeat += BLOCK_SIZE.y) {
            const uint gy = new_gid[0] + repeat;
            if (IS_MULTIPLE_OF_TILE || (new_gid[1] < shape[0] && gy < shape[1]))
                output[gy * output_stride[2] + new_gid[1] * output_stride[3]] = tile[tid[1]][tid[0] + repeat];
        }
    }

    // Since all axes are permuted, in-place transpose cannot easily be expressed as a 2D transposition
    // along a COMMON plane. https://www.aldapa.eus/res/cuTranspose/Readme.html has an implementation
    // based on a 3D shared memory array, but since it is unlikely to be used anyway, don't bother for now.
}

namespace noa::cuda::memory::details {
    template<typename T>
    void transpose0231(const shared_t<T[]>& input, size4_t input_stride,
                       const shared_t<T[]>& output, size4_t output_stride,
                       size4_t shape, Stream& stream) {
        const uint2_t uint_shape{shape[1], shape[3]};
        const bool are_multiple_tile = all((uint_shape % TILE_DIM) == 0);

        const uint blocks_x = math::divideUp(uint_shape[1], TILE_DIM);
        const uint blocks_z = math::divideUp(uint_shape[0], TILE_DIM);
        const dim3 blocks(blocks_x * blocks_z, shape[2], shape[0]);
        if (are_multiple_tile) {
            stream.enqueue("memory::transpose0231", transpose0231_<T, true>, {blocks, BLOCK_SIZE},
                           input.get(), uint4_t{input_stride}, output.get(), uint4_t{output_stride},
                           uint_shape, blocks_x);
        } else {
            stream.enqueue("memory::transpose0231", transpose0231_<T, false>, {blocks, BLOCK_SIZE},
                           input.get(), uint4_t{input_stride}, output.get(), uint4_t{output_stride},
                           uint_shape, blocks_x);
        }
        stream.attach(input, output);
    }
}

#define NOA_INSTANTIATE_TRANSPOSE_(T) \
template void noa::cuda::memory::details::transpose0231<T>(const shared_t<T[]>&, size4_t, const shared_t<T[]>&, size4_t, size4_t, Stream&)

NOA_INSTANTIATE_TRANSPOSE_(bool);
NOA_INSTANTIATE_TRANSPOSE_(int8_t);
NOA_INSTANTIATE_TRANSPOSE_(int16_t);
NOA_INSTANTIATE_TRANSPOSE_(int32_t);
NOA_INSTANTIATE_TRANSPOSE_(int64_t);
NOA_INSTANTIATE_TRANSPOSE_(uint8_t);
NOA_INSTANTIATE_TRANSPOSE_(uint16_t);
NOA_INSTANTIATE_TRANSPOSE_(uint32_t);
NOA_INSTANTIATE_TRANSPOSE_(uint64_t);
NOA_INSTANTIATE_TRANSPOSE_(half_t);
NOA_INSTANTIATE_TRANSPOSE_(float);
NOA_INSTANTIATE_TRANSPOSE_(double);
NOA_INSTANTIATE_TRANSPOSE_(chalf_t);
NOA_INSTANTIATE_TRANSPOSE_(cfloat_t);
NOA_INSTANTIATE_TRANSPOSE_(cdouble_t);
