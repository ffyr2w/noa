#include "hip/hip_runtime.h"
#include "noa/common/Assert.h"
#include "noa/common/Math.h"
#include "noa/gpu/cuda/memory/Arange.h"
#include "noa/gpu/cuda/util/Block.cuh"
#include "noa/gpu/cuda/util/Pointers.h"

namespace {
    using namespace ::noa;

    constexpr uint ELEMENTS_PER_THREAD = 4;
    constexpr uint BLOCK_SIZE = 128;
    constexpr uint BLOCK_WORK_SIZE = BLOCK_SIZE * ELEMENTS_PER_THREAD;

    constexpr dim3 BLOCK_SIZE_2D(32, BLOCK_SIZE / 32);
    constexpr dim3 ELEMENTS_PER_THREAD_2D(1, 4);
    constexpr dim3 BLOCK_WORK_SIZE_2D(BLOCK_SIZE_2D.x * ELEMENTS_PER_THREAD_2D.x,
                                      BLOCK_SIZE_2D.y * ELEMENTS_PER_THREAD_2D.y);

    template<typename T, int VEC_SIZE>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void arange1D_(T* src, uint stride, uint elements, T start, T step) {
        const uint base = BLOCK_WORK_SIZE * blockIdx.x;

        if constexpr (VEC_SIZE == 1) {
            #pragma unroll
            for (int i = 0; i < ELEMENTS_PER_THREAD; ++i) {
                const uint gid = base + BLOCK_SIZE * i + threadIdx.x;
                if (gid < elements)
                    src[gid * stride] = start + static_cast<T>(gid) * step;
            }
        } else {
            NOA_ASSERT(stride == 1);
            (void) stride;
            const uint remaining = elements - base;
            src += base;
            if (remaining < BLOCK_WORK_SIZE) {
                #pragma unroll
                for (int i = 0; i < ELEMENTS_PER_THREAD; ++i) {
                    const uint gid = BLOCK_SIZE * i + threadIdx.x;
                    if (gid < remaining)
                        src[gid] = start + static_cast<T>(base + gid) * step;
                }
            } else {
                T values[ELEMENTS_PER_THREAD];
                #pragma unroll
                for (int i = 0; i < ELEMENTS_PER_THREAD; ++i) {
                    const uint gid = base + i + ELEMENTS_PER_THREAD * threadIdx.x;
                    values[i] = start + static_cast<T>(gid) * step;
                }
                cuda::util::block::vectorizedStore<BLOCK_SIZE, ELEMENTS_PER_THREAD, VEC_SIZE>(values, src, threadIdx.x);
            }
        }
    }

    template<typename T>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void arange4D_(T* src, uint4_t stride, uint4_t shape, T start, T step, uint blocks_x) {
        const uint4_t logical_stride = shape.strides();
        const uint2_t index = indexing::indexes(blockIdx.x, blocks_x);
        const int4_t gid(blockIdx.z,
                         blockIdx.y,
                         BLOCK_WORK_SIZE_2D.y * index[0] + threadIdx.y,
                         BLOCK_WORK_SIZE_2D.x * index[1] + threadIdx.x);
        src += indexing::at(gid[0], gid[1], stride);

        #pragma unroll
        for (int k = 0; k < ELEMENTS_PER_THREAD_2D.y; ++k) {
            #pragma unroll
            for (int l = 0; l < ELEMENTS_PER_THREAD_2D.x; ++l) {
                const uint ik = gid[2] + BLOCK_SIZE_2D.y * k;
                const uint il = gid[3] + BLOCK_SIZE_2D.x * l;
                if (ik < shape[2] && il < shape[3]) {
                    const uint offset = indexing::at(gid[0], gid[1], ik, il, logical_stride);
                    src[ik * stride[2] + il * stride[3]] = start + static_cast<T>(offset) * step;
                }
            }
        }
    }
}

namespace noa::cuda::memory {
    template<typename T, typename>
    void arange(const shared_t<T[]>& src, size_t elements, T start, T step, Stream& stream) {
        if (!elements)
            return;

        const auto uint_elements = static_cast<uint>(elements);
        const dim3 blocks(noa::math::divideUp(uint_elements, BLOCK_WORK_SIZE));
        const int vec_size = noa::cuda::util::maxVectorCount(src.get());
        if (vec_size == 4) {
            stream.enqueue("memory::arange", arange1D_<T, 4>,
                           {blocks, BLOCK_SIZE}, src.get(), 1, uint_elements, start, step);
        } else if (vec_size == 2) {
            stream.enqueue("memory::arange", arange1D_<T, 2>,
                           {blocks, BLOCK_SIZE}, src.get(), 1, uint_elements, start, step);
        } else {
            stream.enqueue("memory::arange", arange1D_<T, 1>,
                           {blocks, BLOCK_SIZE}, src.get(), 1, uint_elements, start, step);
        }
        stream.attach(src);
    }

    template<typename T, typename>
    void arange(const shared_t<T[]>& src, size4_t stride, size4_t shape, T start, T step, Stream& stream) {
        if (!shape.elements())
            return;

        const bool4_t is_contiguous = indexing::isContiguous(stride, shape);
        if (is_contiguous[0] && is_contiguous[1] && is_contiguous[2]) {
            const auto uint_elements = static_cast<uint>(shape.elements());
            const dim3 blocks(noa::math::divideUp(uint_elements, BLOCK_WORK_SIZE));
            const uint vec_size = is_contiguous[3] ? noa::cuda::util::maxVectorCount(src.get()) : 1;

            if (vec_size == 4) {
                stream.enqueue("memory::arange", arange1D_<T, 4>,
                               {blocks, BLOCK_SIZE}, src.get(), stride[3], uint_elements, start, step);
            } else if (vec_size == 2) {
                stream.enqueue("memory::arange", arange1D_<T, 2>,
                               {blocks, BLOCK_SIZE}, src.get(), stride[3], uint_elements, start, step);
            } else {
                stream.enqueue("memory::arange", arange1D_<T, 1>,
                               {blocks, BLOCK_SIZE}, src.get(), stride[3], uint_elements, start, step);
            }
        } else {
            const uint4_t uint_shape{shape};
            const uint blocks_x = noa::math::divideUp(uint_shape[3], BLOCK_WORK_SIZE_2D.x);
            const uint blocks_y = noa::math::divideUp(uint_shape[2], BLOCK_WORK_SIZE_2D.y);
            const dim3 blocks(blocks_x * blocks_y, uint_shape[1], uint_shape[0]);
            stream.enqueue("memory::arange", arange4D_<T>, {blocks, BLOCK_SIZE_2D},
                           src.get(), uint4_t{stride}, uint_shape, start, step, blocks_x);
        }
        stream.attach(src);
    }

    #define NOA_INSTANTIATE_ARANGE_(T)                                          \
    template void arange<T, void>(const shared_t<T[]>&, size_t, T, T, Stream&); \
    template void arange<T, void>(const shared_t<T[]>&, size4_t, size4_t, T, T, Stream&)

    NOA_INSTANTIATE_ARANGE_(int8_t);
    NOA_INSTANTIATE_ARANGE_(int16_t);
    NOA_INSTANTIATE_ARANGE_(int32_t);
    NOA_INSTANTIATE_ARANGE_(int64_t);
    NOA_INSTANTIATE_ARANGE_(uint8_t);
    NOA_INSTANTIATE_ARANGE_(uint16_t);
    NOA_INSTANTIATE_ARANGE_(uint32_t);
    NOA_INSTANTIATE_ARANGE_(uint64_t);
    NOA_INSTANTIATE_ARANGE_(half_t);
    NOA_INSTANTIATE_ARANGE_(float);
    NOA_INSTANTIATE_ARANGE_(double);
    NOA_INSTANTIATE_ARANGE_(chalf_t);
    NOA_INSTANTIATE_ARANGE_(cfloat_t);
    NOA_INSTANTIATE_ARANGE_(cdouble_t);
}
