#include "hip/hip_runtime.h"
#include "noa/common/Assert.h"
#include "noa/common/Math.h"
#include "noa/gpu/cuda/memory/Arange.h"
#include "noa/gpu/cuda/util/Block.cuh"
#include "noa/gpu/cuda/util/Pointers.h"

namespace {
    using namespace ::noa;

    constexpr uint32_t ELEMENTS_PER_THREAD = 4;
    constexpr uint32_t BLOCK_SIZE = 128;
    constexpr uint32_t BLOCK_WORK_SIZE = BLOCK_SIZE * ELEMENTS_PER_THREAD;

    constexpr dim3 BLOCK_SIZE_2D(32, BLOCK_SIZE / 32);
    constexpr dim3 ELEMENTS_PER_THREAD_2D(1, 4);
    constexpr dim3 BLOCK_WORK_SIZE_2D(BLOCK_SIZE_2D.x * ELEMENTS_PER_THREAD_2D.x,
                                      BLOCK_SIZE_2D.y * ELEMENTS_PER_THREAD_2D.y);

    template<typename T, int32_t VEC_SIZE>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void arange1D_(T* src, uint32_t strides, uint32_t elements, T start, T step) {
        const uint32_t base = BLOCK_WORK_SIZE * blockIdx.x;

        if constexpr (VEC_SIZE == 1) {
            #pragma unroll
            for (int32_t i = 0; i < ELEMENTS_PER_THREAD; ++i) {
                const uint32_t gid = base + BLOCK_SIZE * i + threadIdx.x;
                if (gid < elements)
                    src[gid * strides] = start + static_cast<T>(gid) * step;
            }
        } else {
            NOA_ASSERT(strides == 1);
            (void) strides;
            const uint32_t remaining = elements - base;
            src += base;
            if (remaining < BLOCK_WORK_SIZE) {
                #pragma unroll
                for (int32_t i = 0; i < ELEMENTS_PER_THREAD; ++i) {
                    const uint32_t gid = BLOCK_SIZE * i + threadIdx.x;
                    if (gid < remaining)
                        src[gid] = start + static_cast<T>(base + gid) * step;
                }
            } else {
                T values[ELEMENTS_PER_THREAD];
                #pragma unroll
                for (int32_t i = 0; i < ELEMENTS_PER_THREAD; ++i) {
                    const uint32_t gid = base + i + ELEMENTS_PER_THREAD * threadIdx.x;
                    values[i] = start + static_cast<T>(gid) * step;
                }
                cuda::util::block::vectorizedStore<BLOCK_SIZE, ELEMENTS_PER_THREAD, VEC_SIZE>(values, src, threadIdx.x);
            }
        }
    }

    template<typename T>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void arange4D_(Accessor<T, 4, uint32_t> src, uint4_t shape, T start, T step, uint32_t blocks_x) {
        const uint4_t logical_strides = shape.strides();
        const uint2_t index = indexing::indexes(blockIdx.x, blocks_x);
        const int4_t gid{blockIdx.z,
                         blockIdx.y,
                         BLOCK_WORK_SIZE_2D.y * index[0] + threadIdx.y,
                         BLOCK_WORK_SIZE_2D.x * index[1] + threadIdx.x};
        const auto src_ = src[gid[0]][gid[1]];

        #pragma unroll
        for (int32_t k = 0; k < ELEMENTS_PER_THREAD_2D.y; ++k) {
            #pragma unroll
            for (int32_t l = 0; l < ELEMENTS_PER_THREAD_2D.x; ++l) {
                const uint32_t ik = gid[2] + BLOCK_SIZE_2D.y * k;
                const uint32_t il = gid[3] + BLOCK_SIZE_2D.x * l;
                if (ik < shape[2] && il < shape[3]) {
                    const uint32_t offset = indexing::at(gid[0], gid[1], ik, il, logical_strides);
                    src_(ik, il) = start + static_cast<T>(offset) * step;
                }
            }
        }
    }
}

namespace noa::cuda::memory {
    template<typename T, typename>
    void arange(const shared_t<T[]>& src, dim_t elements, T start, T step, Stream& stream) {
        if (!elements)
            return;

        NOA_ASSERT_DEVICE_PTR(src.get(), stream.device());
        const auto uint_elements = static_cast<uint32_t>(elements);
        const dim3 blocks(noa::math::divideUp(uint_elements, BLOCK_WORK_SIZE));
        const int32_t vec_size = noa::cuda::util::maxVectorCount(src.get());
        if (vec_size == 4) {
            stream.enqueue("memory::arange", arange1D_<T, 4>,
                           {blocks, BLOCK_SIZE}, src.get(), 1, uint_elements, start, step);
        } else if (vec_size == 2) {
            stream.enqueue("memory::arange", arange1D_<T, 2>,
                           {blocks, BLOCK_SIZE}, src.get(), 1, uint_elements, start, step);
        } else {
            stream.enqueue("memory::arange", arange1D_<T, 1>,
                           {blocks, BLOCK_SIZE}, src.get(), 1, uint_elements, start, step);
        }
        stream.attach(src);
    }

    template<typename T, typename>
    void arange(const shared_t<T[]>& src, dim4_t strides, dim4_t shape, T start, T step, Stream& stream) {
        if (!shape.elements())
            return;

        NOA_ASSERT_DEVICE_PTR(src.get(), stream.device());
        const bool4_t is_contiguous = indexing::isContiguous(strides, shape);
        if (is_contiguous[0] && is_contiguous[1] && is_contiguous[2]) {
            const auto uint_elements = safe_cast<uint32_t>(shape.elements());
            const dim3 blocks(noa::math::divideUp(uint_elements, BLOCK_WORK_SIZE));
            const uint32_t vec_size = is_contiguous[3] ? noa::cuda::util::maxVectorCount(src.get()) : 1;

            if (vec_size == 4) {
                stream.enqueue("memory::arange", arange1D_<T, 4>,
                               {blocks, BLOCK_SIZE}, src.get(), strides[3], uint_elements, start, step);
            } else if (vec_size == 2) {
                stream.enqueue("memory::arange", arange1D_<T, 2>,
                               {blocks, BLOCK_SIZE}, src.get(), strides[3], uint_elements, start, step);
            } else {
                stream.enqueue("memory::arange", arange1D_<T, 1>,
                               {blocks, BLOCK_SIZE}, src.get(), strides[3], uint_elements, start, step);
            }
        } else {
            const uint4_t uint_shape(shape);
            const uint32_t blocks_x = noa::math::divideUp(uint_shape[3], BLOCK_WORK_SIZE_2D.x);
            const uint32_t blocks_y = noa::math::divideUp(uint_shape[2], BLOCK_WORK_SIZE_2D.y);
            const dim3 blocks(blocks_x * blocks_y, uint_shape[1], uint_shape[0]);
            const Accessor<T, 4, uint32_t> src_accessor(src.get(), safe_cast<uint4_t>(strides));
            stream.enqueue("memory::arange", arange4D_<T>, {blocks, BLOCK_SIZE_2D},
                           src_accessor, uint_shape, start, step, blocks_x);
        }
        stream.attach(src);
    }

    #define NOA_INSTANTIATE_ARANGE_(T)                                          \
    template void arange<T, void>(const shared_t<T[]>&, dim_t, T, T, Stream&);  \
    template void arange<T, void>(const shared_t<T[]>&, dim4_t, dim4_t, T, T, Stream&)

    NOA_INSTANTIATE_ARANGE_(int8_t);
    NOA_INSTANTIATE_ARANGE_(int16_t);
    NOA_INSTANTIATE_ARANGE_(int32_t);
    NOA_INSTANTIATE_ARANGE_(int64_t);
    NOA_INSTANTIATE_ARANGE_(uint8_t);
    NOA_INSTANTIATE_ARANGE_(uint16_t);
    NOA_INSTANTIATE_ARANGE_(uint32_t);
    NOA_INSTANTIATE_ARANGE_(uint64_t);
    NOA_INSTANTIATE_ARANGE_(half_t);
    NOA_INSTANTIATE_ARANGE_(float);
    NOA_INSTANTIATE_ARANGE_(double);
    NOA_INSTANTIATE_ARANGE_(chalf_t);
    NOA_INSTANTIATE_ARANGE_(cfloat_t);
    NOA_INSTANTIATE_ARANGE_(cdouble_t);
}
