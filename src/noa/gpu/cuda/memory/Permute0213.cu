#include "hip/hip_runtime.h"
#include "noa/common/Math.h"
#include "noa/gpu/cuda/Exception.h"
#include "noa/gpu/cuda/util/Traits.h"
#include "noa/gpu/cuda/memory/Permute.h"

namespace {
    using namespace ::noa;

    constexpr uint32_t TILE_DIM = 32;
    constexpr dim3 BLOCK_SIZE(TILE_DIM, 256 / TILE_DIM);

    // Out-of-place.
    // Transpose 0213 is a specific case: the innermost dimension is unchanged,
    // which makes everything much simpler. Only the last two dimensions are swapped:
    //  - input_strides[1]->output_strides[2]
    //  - input_strides[2]->output_strides[1]
    template<typename T, bool IS_MULTIPLE_OF_TILE>
    __global__ __launch_bounds__(BLOCK_SIZE.x * BLOCK_SIZE.y)
    void permute0213_(AccessorRestrict<const T, 4, uint32_t> input,
                      AccessorRestrict<T, 4, uint32_t> output_swapped,
                      uint2_t shape /* YX */ , uint32_t blocks_x) {
        const uint2_t tid{threadIdx.y, threadIdx.x};
        const uint2_t index = indexing::indexes(blockIdx.x, blocks_x);
        const uint2_t gid = TILE_DIM * index + tid;
        if (!IS_MULTIPLE_OF_TILE && gid[1] >= shape[1])
            return;

        const auto input_ = input[blockIdx.z][blockIdx.y];
        const auto output_ = output_swapped[blockIdx.z][blockIdx.y];

        for (uint32_t repeat = 0; repeat < TILE_DIM; repeat += BLOCK_SIZE.y) {
            const uint32_t gy = gid[0] + repeat;
            if (IS_MULTIPLE_OF_TILE || gy < shape[0])
                output_(gy, gid[1]) = input_(gy, gid[1]);
        }
    }

    // In-place.
    // This is simply swapping the Y with the X, such as swap(o[z][y][x], o[y][z][x]).
    // Only process one triangle, plus the diagonal. The other blocks are idle...
    // The shared memory simply acts as a per thread buffer.
    template<typename T, bool IS_MULTIPLE_OF_TILE>
    __global__ __launch_bounds__(BLOCK_SIZE.x * BLOCK_SIZE.y)
    void permute0213_inplace_(Accessor<T, 4, uint32_t> output, uint2_t shape, uint32_t blocks_x) {
        using uninit_t = cuda::util::traits::uninitialized_type_t<T>;
        __shared__ uninit_t buffer[BLOCK_SIZE.y][BLOCK_SIZE.x];
        T(& tile)[BLOCK_SIZE.y][BLOCK_SIZE.x] = *reinterpret_cast<T(*)[BLOCK_SIZE.y][BLOCK_SIZE.x]>(&buffer);

        const uint2_t tid{threadIdx.y, threadIdx.x};
        const uint2_t index = indexing::indexes(blockIdx.x, blocks_x);
        const uint4_t gid{blockIdx.z,
                          blockIdx.y,
                          TILE_DIM * index[0] + tid[0],
                          TILE_DIM * index[1] + tid[1]};
        if (gid[3] >= shape[1])
            return;

        const auto output_ = output[gid[0]];
        for (uint32_t repeat = 0; repeat < TILE_DIM; repeat += BLOCK_SIZE.y) {
            const uint32_t gy = gid[2] + repeat;
            if (gid[1] > gy) // process one triangle + diagonal
                continue;

            if (IS_MULTIPLE_OF_TILE || gy < shape[0]) {
                T& src = output_(gid[1], gy, gid[3]);
                T& dst = output_(gy, gid[1], gid[3]); // permutation 1 <-> 2
                tile[tid[0]][tid[1]] = dst;
                dst = src;
                src = tile[tid[0]][tid[1]];
            }
        }
    }
}

namespace noa::cuda::memory::details {
    template<typename T>
    void permute0213(const shared_t<T[]>& input, dim4_t input_strides,
                     const shared_t<T[]>& output, dim4_t output_strides,
                     dim4_t shape, Stream& stream) {
        NOA_ASSERT_DEVICE_PTR(input.get(), stream.device());
        NOA_ASSERT_DEVICE_PTR(output.get(), stream.device());
        const auto uint_shape = safe_cast<uint2_t>(dim2_t(shape.get(2)));
        const bool are_multiple_tile = all((uint_shape % TILE_DIM) == 0);

        const uint32_t blocks_x = math::divideUp(uint_shape[1], TILE_DIM);
        const uint32_t blocks_y = math::divideUp(uint_shape[0], TILE_DIM);
        const dim3 blocks(blocks_x * blocks_y, shape[1], shape[0]);

        const AccessorRestrict<const T, 4, uint32_t> input_accessor(input.get(), safe_cast<uint4_t>(input_strides));
        const AccessorRestrict<T, 4, uint32_t> output_accessor(output.get(), safe_cast<uint4_t>(output_strides));
        const auto swapped_output = output_accessor.swap(1, 2);

        if (are_multiple_tile) {
            stream.enqueue("memory::permute0213", permute0213_<T, true>, {blocks, BLOCK_SIZE},
                           input_accessor, swapped_output, uint_shape, blocks_x);
        } else {
            stream.enqueue("memory::permute0213", permute0213_<T, false>, {blocks, BLOCK_SIZE},
                           input_accessor, swapped_output, uint_shape, blocks_x);
        }
        stream.attach(input, output);
    }
}

namespace noa::cuda::memory::details::inplace {
    template<typename T>
    void permute0213(const shared_t<T[]>& output, dim4_t output_strides, dim4_t shape, Stream& stream) {
        NOA_ASSERT_DEVICE_PTR(output.get(), stream.device());
        if (shape[1] != shape[2])
            NOA_THROW("For a \"0213\" in-place permutation, shape[1] should be equal to shape[2]. Got {}", shape);

        const auto uint_shape = safe_cast<uint2_t>(dim2_t(shape.get(2)));
        const bool are_multiple_tile = (uint_shape[0] % TILE_DIM) == 0;

        const uint32_t blocks_x = math::divideUp(uint_shape[1], TILE_DIM);
        const uint32_t blocks_y = math::divideUp(uint_shape[0], TILE_DIM);
        const dim3 blocks(blocks_x * blocks_y, shape[1], shape[0]);
        const Accessor<T, 4, uint32_t> accessor(output.get(), safe_cast<uint4_t>(output_strides));

        if (are_multiple_tile) {
            stream.enqueue("memory::permute0213_inplace", permute0213_inplace_<T, true>, {blocks, BLOCK_SIZE},
                           accessor, uint_shape, blocks_x);
        } else {
            stream.enqueue("memory::permute0213_inplace", permute0213_inplace_<T, false>, {blocks, BLOCK_SIZE},
                           accessor, uint_shape, blocks_x);
        }
        stream.attach(output);
    }
}

#define NOA_INSTANTIATE_TRANSPOSE_(T)                                                                                                   \
template void noa::cuda::memory::details::permute0213<T>(const shared_t<T[]>&, dim4_t, const shared_t<T[]>&, dim4_t, dim4_t, Stream&);  \
template void noa::cuda::memory::details::inplace::permute0213<T>(const shared_t<T[]>&, dim4_t, dim4_t, Stream&)

NOA_INSTANTIATE_TRANSPOSE_(bool);
NOA_INSTANTIATE_TRANSPOSE_(int8_t);
NOA_INSTANTIATE_TRANSPOSE_(int16_t);
NOA_INSTANTIATE_TRANSPOSE_(int32_t);
NOA_INSTANTIATE_TRANSPOSE_(int64_t);
NOA_INSTANTIATE_TRANSPOSE_(uint8_t);
NOA_INSTANTIATE_TRANSPOSE_(uint16_t);
NOA_INSTANTIATE_TRANSPOSE_(uint32_t);
NOA_INSTANTIATE_TRANSPOSE_(uint64_t);
NOA_INSTANTIATE_TRANSPOSE_(half_t);
NOA_INSTANTIATE_TRANSPOSE_(float);
NOA_INSTANTIATE_TRANSPOSE_(double);
NOA_INSTANTIATE_TRANSPOSE_(chalf_t);
NOA_INSTANTIATE_TRANSPOSE_(cfloat_t);
NOA_INSTANTIATE_TRANSPOSE_(cdouble_t);
