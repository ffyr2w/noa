#include "hip/hip_runtime.h"
#include "noa/gpu/cuda/memory/Resize.h"
#include "noa/Session.h"
#include "Copy.h"

namespace {
    using namespace Noa;

    template<typename T>
    __global__ void resizeWithNothing(const T* inputs, uint input_pitch, uint3_t input_shape, uint input_elements,
                                      T* outputs, uint output_pitch, uint3_t output_shape, uint output_elements,
                                      int3_t crop_left, int3_t pad_left, int3_t pad_right, uint batches) {
        uint o_y = blockIdx.y;
        uint o_z = blockIdx.z;
        int i_y = static_cast<int>(o_y) - pad_left.y + crop_left.y; // negative if withing padding
        int i_z = static_cast<int>(o_z) - pad_left.z + crop_left.z;

        if (o_z < pad_left.z || o_z >= output_shape.z - pad_right.z ||
            o_y < pad_left.y || o_y >= output_shape.y - pad_right.y)
            return;

        outputs += (o_z * output_shape.y + o_y) * output_pitch;
        for (uint o_x = blockIdx.x * blockDim.x + threadIdx.x; o_x < output_shape.x; o_x += blockDim.x * gridDim.x) {
            if (o_x < pad_left.x || o_x >= output_shape.x - pad_right.x)
                break;

            uint i_x = o_x - pad_left.x + crop_left.x; // cannot be negative
            inputs += (i_z * input_shape.y + i_y) * input_pitch + i_x;
            for (uint batch = 0; batch < batches; ++batch)
                outputs[batch * output_elements + o_x] = inputs[batch * input_elements];
        }
    }

    template<typename T>
    NOA_HOST void launchResizeWithNothing(const T* inputs, uint input_pitch, uint3_t input_shape,
                                          T* outputs, uint output_pitch, uint3_t output_shape,
                                          int3_t border_left, int3_t border_right, uint batches, CUDA::Stream& stream) {
        uint input_elements = getRows(input_shape) * input_pitch;
        uint output_elements = getRows(output_shape) * output_pitch;
        int3_t crop_left(Math::min(border_left, 0) * -1);
        int3_t pad_left(Math::max(border_left, 0));
        int3_t pad_right(Math::max(border_right, 0));

        uint threads = Math::min(256U, Math::nextMultipleOf(output_shape.x, 32U));
        dim3 blocks{(output_shape.x + threads - 1) / threads, output_shape.y, output_shape.z};
        NOA_CUDA_LAUNCH(blocks, threads, 0, stream.id(),
                        resizeWithNothing,
                        inputs, input_pitch, input_shape, input_elements,
                        outputs, output_pitch, output_shape, output_elements,
                        crop_left, pad_left, pad_right, batches);
    }

    template<typename T>
    __global__ void resizeWithValue(const T* inputs, uint input_pitch, uint3_t input_shape, uint input_elements,
                                    T* outputs, uint output_pitch, uint3_t output_shape, uint output_elements,
                                    int3_t crop_left, int3_t pad_left, int3_t pad_right, T value, uint batches) {
        uint o_y = blockIdx.y;
        uint o_z = blockIdx.z;
        int i_y = static_cast<int>(o_y) - pad_left.y + crop_left.y;
        int i_z = static_cast<int>(o_z) - pad_left.z + crop_left.z;

        bool is_padding = o_z < pad_left.z || o_z >= output_shape.z - pad_right.z ||
                          o_y < pad_left.y || o_y >= output_shape.y - pad_right.y;

        outputs += (o_z * output_shape.y + o_y) * output_pitch;
        for (uint o_x = blockIdx.x * blockDim.x + threadIdx.x; o_x < output_shape.x; o_x += blockDim.x * gridDim.x) {
            if (is_padding || o_x < pad_left.x || o_x >= output_shape.x - pad_right.x) {
                for (uint batch = 0; batch < batches; ++batch)
                    outputs[batch * output_elements + o_x] = value;
            } else {
                uint i_x = o_x - pad_left.x + crop_left.x;
                inputs += (i_z * input_shape.y + i_y) * input_pitch + i_x;
                for (uint batch = 0; batch < batches; ++batch)
                    outputs[batch * output_elements + o_x] = inputs[batch * input_elements];
            }
        }
    }

    template<typename T>
    NOA_HOST void launchResizeWithValue(const T* inputs, uint input_pitch, uint3_t input_shape,
                                        T* outputs, uint output_pitch, uint3_t output_shape,
                                        int3_t border_left, int3_t border_right, T value,
                                        uint batches, CUDA::Stream& stream) {
        uint input_elements = getRows(input_shape) * input_pitch;
        uint output_elements = getRows(output_shape) * output_pitch;
        int3_t crop_left(Math::min(border_left, 0) * -1);
        int3_t pad_left(Math::max(border_left, 0));
        int3_t pad_right(Math::max(border_right, 0));

        uint threads = Math::min(256U, Math::nextMultipleOf(output_shape.x, 32U));
        dim3 blocks{(output_shape.x + threads - 1) / threads, output_shape.y, output_shape.z};
        NOA_CUDA_LAUNCH(blocks, threads, 0, stream.id(),
                        resizeWithValue,
                        inputs, input_pitch, input_shape, input_elements,
                        outputs, output_pitch, output_shape, output_elements,
                        crop_left, pad_left, pad_right, value, batches);
    }

    template<int MODE>
    NOA_ID int getBorderIndex(int idx, int pad_left, int crop_left, int len) {
        static_assert(MODE == BORDER_CLAMP || MODE == BORDER_PERIODIC || MODE == BORDER_MIRROR);
        int out_idx;
        if constexpr (MODE == BORDER_CLAMP) {
            out_idx = Math::max(0, Math::min(idx - pad_left + crop_left, len - 1));
        } else if constexpr (MODE == BORDER_PERIODIC) {
            int rem = (idx - pad_left + crop_left) % len;
            out_idx = rem < 0 ? rem + len : rem;
        } else if constexpr (MODE == BORDER_MIRROR) {
            out_idx = idx - pad_left + crop_left;
            if (out_idx < 0) {
                int offset = (Math::abs(out_idx) - 1) % len;
                out_idx = offset;
            } else if (out_idx >= len) {
                int offset = Math::abs(out_idx) % len;
                out_idx = len - offset - 1;
            }
        }
        return out_idx;
    }

    template<int MODE, typename T>
    __global__ void resizeWith(const T* inputs, uint input_pitch, uint3_t input_shape, uint input_elements,
                               T* outputs, uint output_pitch, uint3_t output_shape, uint output_elements,
                               int3_t crop_left, int3_t pad_left, uint batches) {
        uint o_y = blockIdx.y;
        uint o_z = blockIdx.z;
        int3_t int_input_shape(input_shape);

        uint i_z = getBorderIndex<MODE>(o_z, pad_left.z, crop_left.z, int_input_shape.z);
        uint i_y = getBorderIndex<MODE>(o_y, pad_left.y, crop_left.y, int_input_shape.y);

        outputs += (o_z * output_shape.y + o_y) * output_pitch;
        inputs += (i_z * input_shape.y + i_y) * input_pitch;
        for (uint o_x = blockIdx.x * blockDim.x + threadIdx.x; o_x < output_shape.x; o_x += blockDim.x * gridDim.x) {
            uint i_x = getBorderIndex<MODE>(o_x, pad_left.x, crop_left.x, int_input_shape.x);
            for (uint batch = 0; batch < batches; ++batch)
                outputs[batch * output_elements + o_x] = inputs[batch * input_elements + i_x];
        }
    }

    template<int MODE, typename T>
    NOA_HOST void launchResizeWith(const T* inputs, uint input_pitch, uint3_t input_shape,
                                   T* outputs, uint output_pitch, uint3_t output_shape,
                                   int3_t border_left, int3_t border_right,
                                   uint batches, CUDA::Stream& stream) {
        uint input_elements = getRows(input_shape) * input_pitch;
        uint output_elements = getRows(output_shape) * output_pitch;
        int3_t crop_left(Math::min(border_left, 0) * -1);
        int3_t pad_left(Math::max(border_left, 0));
        int3_t pad_right(Math::max(border_right, 0));

        if constexpr (MODE == BORDER_MIRROR) {
            int3_t int_input_shape(input_shape);
            if (pad_left > int_input_shape || pad_right > int_input_shape)
                Session::logger.warn("Edge case: BORDER_MIRROR used with padding larger than the original shape. "
                                     "This might not produce the expect result. "
                                     "Got: pad_left={}, pad_right={}, input_shape={}",
                                     pad_left, pad_right, int_input_shape);
        }

        uint threads = Math::min(256U, Math::nextMultipleOf(output_shape.x, 32U));
        dim3 blocks{(output_shape.x + threads - 1) / threads, output_shape.y, output_shape.z};
        NOA_CUDA_LAUNCH(blocks, threads, 0, stream.id(),
                        resizeWith<MODE>,
                        inputs, input_pitch, input_shape, input_elements,
                        outputs, output_pitch, output_shape, output_elements,
                        crop_left, pad_left, batches);
    }
}

namespace Noa::CUDA::Memory {
    template<typename T>
    void resize(const T* inputs, size_t input_pitch, size3_t input_shape,
                T* outputs, size_t output_pitch, size3_t output_shape,
                int3_t border_left, int3_t border_right, BorderMode border_mode, T border_value,
                uint batches, Stream& stream) {
        if (int3_t(input_shape) + border_left + border_right != int3_t(output_shape)) {
            NOA_THROW("Cannot resize an array with shape {} to {} given the borders left:{}, right:{}",
                      input_shape, output_shape, border_left, border_right);
        } else if (inputs == outputs) {
            NOA_THROW("In-place resizing is not allowed");
        } else if (border_left == 0 && border_right == 0) {
            copy(inputs, input_pitch, outputs, output_pitch, {input_shape.x, input_shape.y * input_shape.z, batches});
            return;
        }

        if (border_mode == BORDER_NOTHING)
            launchResizeWithNothing(inputs, input_pitch, uint3_t(input_shape),
                                    outputs, output_pitch, uint3_t(output_shape),
                                    border_left, border_right, batches, stream);
        else if (border_mode == BORDER_ZERO)
            launchResizeWithValue(inputs, input_pitch, uint3_t(input_shape),
                                  outputs, output_pitch, uint3_t(output_shape),
                                  border_left, border_right, T{0}, batches, stream);
        else if (border_mode == BORDER_VALUE)
            launchResizeWithValue(inputs, input_pitch, uint3_t(input_shape),
                                  outputs, output_pitch, uint3_t(output_shape),
                                  border_left, border_right, border_value, batches, stream);
        else if (border_mode == BORDER_CLAMP)
            launchResizeWith<BORDER_CLAMP>(inputs, input_pitch, uint3_t(input_shape),
                                           outputs, output_pitch, uint3_t(output_shape),
                                           border_left, border_right, batches, stream);
        else if (border_mode == BORDER_PERIODIC)
            launchResizeWith<BORDER_PERIODIC>(inputs, input_pitch, uint3_t(input_shape),
                                              outputs, output_pitch, uint3_t(output_shape),
                                              border_left, border_right, batches, stream);
        else if (border_mode == BORDER_MIRROR)
            launchResizeWith<BORDER_MIRROR>(inputs, input_pitch, uint3_t(input_shape),
                                            outputs, output_pitch, uint3_t(output_shape),
                                            border_left, border_right, batches, stream);
        else
            NOA_THROW("BorderMode not recognized. Got: {}", border_mode);
    }

    #define INSTANTIATE_RESIZE(T) \
    template void resize<T>(const T*, size_t, size3_t, T*, size_t, size3_t, int3_t, int3_t, BorderMode, T, uint, Stream&)

    INSTANTIATE_RESIZE(float);
    INSTANTIATE_RESIZE(double);
    INSTANTIATE_RESIZE(bool);
    INSTANTIATE_RESIZE(char);
    INSTANTIATE_RESIZE(short);
    INSTANTIATE_RESIZE(int);
    INSTANTIATE_RESIZE(long);
    INSTANTIATE_RESIZE(long long);
    INSTANTIATE_RESIZE(unsigned char);
    INSTANTIATE_RESIZE(unsigned short);
    INSTANTIATE_RESIZE(unsigned int);
    INSTANTIATE_RESIZE(unsigned long);
    INSTANTIATE_RESIZE(unsigned long long);
}
