#include "hip/hip_runtime.h"
#include "noa/common/Assert.h"
#include "noa/gpu/cuda/Exception.h"
#include "noa/gpu/cuda/memory/Resize.h"
#include "noa/gpu/cuda/memory/Copy.h"

namespace {
    using namespace noa;
    constexpr uint32_t BLOCK_SIZE = 512;
    constexpr dim3 BLOCK_SIZE_2D(32, BLOCK_SIZE / 32);
    constexpr uint32_t ELEMENT_PER_THREAD = 4;
    constexpr dim3 BLOCK_WORK_SIZE_2D(BLOCK_SIZE_2D.x * ELEMENT_PER_THREAD, BLOCK_SIZE_2D.y);

    // Computes two elements per thread.
    template<typename T>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void resizeWithNothing_(AccessorRestrict<const T, 4, uint32_t> input,
                            AccessorRestrict<T, 4, uint32_t> output, uint2_t output_shape /* YX */,
                            int4_t crop_left, int4_t pad_left, int4_t pad_right, uint32_t blocks_x) {
        const uint2_t idx = indexing::indexes(blockIdx.x, blocks_x);
        const int4_t gid{blockIdx.z,
                         blockIdx.y,
                         BLOCK_WORK_SIZE_2D.y * idx[0] + threadIdx.y,
                         BLOCK_WORK_SIZE_2D.x * idx[1] + threadIdx.x};

        // If within the padding, stop.
        if (gid[0] < pad_left[0] || gid[0] >= static_cast<int32_t>(gridDim.z) - pad_right[0] ||
            gid[1] < pad_left[1] || gid[1] >= static_cast<int32_t>(gridDim.y) - pad_right[1] ||
            gid[2] < pad_left[2] || gid[2] >= static_cast<int32_t>(output_shape[0]) - pad_right[2])
            return;

        const int32_t ii = gid[0] - pad_left[0] + crop_left[0]; // cannot be negative
        const int32_t ij = gid[1] - pad_left[1] + crop_left[1];
        const int32_t ik = gid[2] - pad_left[2] + crop_left[2];

        const auto input_row = input[ii][ij][ik];
        const auto output_row = output[gid[0]][gid[1]][gid[2]];

        for (int32_t i = 0; i < ELEMENT_PER_THREAD; ++i) {
            const int32_t ol = gid[3] + static_cast<int32_t>(BLOCK_SIZE_2D.x) * i;
            if (ol >= pad_left[3] && ol < static_cast<int32_t>(output_shape[1]) - pad_right[3]) {
                const int32_t il = ol - pad_left[3] + crop_left[3]; // cannot be negative
                output_row[ol] = input_row[il];
            }
        }
    }

    template<typename T>
    void launchResizeWithNothing_(const shared_t<T[]>& input, uint4_t input_strides,
                                  const shared_t<T[]>& output, uint4_t output_strides, uint4_t output_shape,
                                  int4_t border_left, int4_t border_right, cuda::Stream& stream) {
        const int4_t crop_left(math::min(border_left, 0) * -1);
        const int4_t pad_left(math::max(border_left, 0));
        const int4_t pad_right(math::max(border_right, 0));

        const uint2_t uint_shape(output_shape.get(2));
        const uint32_t blocks_x = math::divideUp(uint_shape[1], BLOCK_WORK_SIZE_2D.x);
        const uint32_t blocks_y = math::divideUp(uint_shape[0], BLOCK_WORK_SIZE_2D.y);
        const dim3 blocks{blocks_x * blocks_y, output_shape[1], output_shape[0]};
        const AccessorRestrict<const T, 4, uint32_t> input_accessor(input.get(), input_strides);
        const AccessorRestrict<T, 4, uint32_t> output_accessor(output.get(), output_strides);

        stream.enqueue("memory::resizeWithNothing", resizeWithNothing_<T>, {blocks, BLOCK_SIZE_2D},
                       input_accessor, output_accessor, uint_shape, crop_left, pad_left, pad_right, blocks_x);
        stream.attach(input, output);
    }

    template<typename T>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void resizeWithValue_(AccessorRestrict<const T, 4, uint32_t> input,
                          AccessorRestrict<T, 4, uint32_t> output, uint2_t output_shape /* YX */,
                          int4_t crop_left, int4_t pad_left, int4_t pad_right, T value, uint32_t blocks_x) {
        const uint2_t idx = indexing::indexes(blockIdx.x, blocks_x);
        const int4_t ogid{blockIdx.z,
                          blockIdx.y,
                          BLOCK_WORK_SIZE_2D.y * idx[0] + threadIdx.y,
                          BLOCK_WORK_SIZE_2D.x * idx[1] + threadIdx.x};
        if (ogid[2] >= output_shape[0])
            return;

        const bool is_valid = ogid[0] >= pad_left[0] && ogid[0] < static_cast<int32_t>(gridDim.z) - pad_right[0] &&
                              ogid[1] >= pad_left[1] && ogid[1] < static_cast<int32_t>(gridDim.y) - pad_right[1] &&
                              ogid[2] >= pad_left[2] && ogid[2] < static_cast<int32_t>(output_shape[0]) - pad_right[2];

        const int32_t ii = ogid[0] - pad_left[0] + crop_left[0]; // can be negative, but is_valid protects against it-
        const int32_t ij = ogid[1] - pad_left[1] + crop_left[1];
        const int32_t ik = ogid[2] - pad_left[2] + crop_left[2];
        // Cast the indexes here, since the accessor asserts against negative indexes and loss of range.
        // In this case, we allow it since we precompute the offset but only use it when the index is valid.
        const auto input_row = input[static_cast<uint32_t>(ii)][static_cast<uint32_t>(ij)][static_cast<uint32_t>(ik)];
        const auto output_row = output[ogid[0]][ogid[1]][ogid[2]];

        for (int32_t i = 0; i < ELEMENT_PER_THREAD; ++i) {
            const int32_t ol = ogid[3] + static_cast<int32_t>(BLOCK_SIZE_2D.x) * i;
            if (ol >= output_shape[1])
                return;

            if (is_valid && ol >= pad_left[3] && ol < static_cast<int32_t>(output_shape[1]) - pad_right[3]) {
                const auto il = static_cast<uint32_t>(ol - pad_left[3] + crop_left[3]); // cannot be negative
                output_row[ol] = input_row[il];
            } else {
                output_row[ol] = value;
            }
        }
    }

    template<typename T>
    void launchResizeWithValue_(const shared_t<T[]>& input, uint4_t input_strides,
                                const shared_t<T[]>& output, uint4_t output_strides, uint4_t output_shape,
                                int4_t border_left, int4_t border_right, T value, cuda::Stream& stream) {
        const int4_t crop_left(math::min(border_left, 0) * -1);
        const int4_t pad_left(math::max(border_left, 0));
        const int4_t pad_right(math::max(border_right, 0));

        const uint2_t uint_shape(output_shape.get(2));
        const uint32_t blocks_x = math::divideUp(uint_shape[1], BLOCK_WORK_SIZE_2D.x);
        const uint32_t blocks_y = math::divideUp(uint_shape[0], BLOCK_WORK_SIZE_2D.y);
        const dim3 blocks{blocks_x * blocks_y, output_shape[1], output_shape[0]};
        const AccessorRestrict<const T, 4, uint32_t> input_accessor(input.get(), input_strides);
        const AccessorRestrict<T, 4, uint32_t> output_accessor(output.get(), output_strides);

        stream.enqueue("memory::resizeWithValue", resizeWithValue_<T>, {blocks, BLOCK_SIZE_2D},
                       input_accessor, output_accessor, uint_shape, crop_left, pad_left, pad_right, value, blocks_x);
        stream.attach(input, output);
    }

    template<BorderMode MODE, typename T>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void resizeWith_(AccessorRestrict<const T, 4, uint32_t> input, uint4_t input_shape,
                     AccessorRestrict<T, 4, uint32_t> output, uint2_t output_shape /* YX */,
                     int4_t crop_left, int4_t pad_left, uint32_t blocks_x) {
        const uint2_t idx = indexing::indexes(blockIdx.x, blocks_x);
        const int4_t ogid{blockIdx.z,
                          blockIdx.y,
                          BLOCK_WORK_SIZE_2D.y * idx[0] + threadIdx.y,
                          BLOCK_WORK_SIZE_2D.x * idx[1] + threadIdx.x};
        if (ogid[2] >= output_shape[0])
            return;

        int3_t igid{ogid[0] - pad_left[0] + crop_left[0],
                    ogid[1] - pad_left[1] + crop_left[1],
                    ogid[2] - pad_left[2] + crop_left[2]};
        igid[0] = indexing::at<MODE>(igid[0], static_cast<int32_t>(input_shape[0]));
        igid[1] = indexing::at<MODE>(igid[1], static_cast<int32_t>(input_shape[1]));
        igid[2] = indexing::at<MODE>(igid[2], static_cast<int32_t>(input_shape[2]));
        const auto input_row = input[igid[0]][igid[1]][igid[2]];
        const auto output_row = output[ogid[0]][ogid[1]][ogid[2]];

        for (int32_t i = 0; i < ELEMENT_PER_THREAD; ++i) {
            const int32_t ol = ogid[3] + static_cast<int32_t>(BLOCK_SIZE_2D.x) * i;
            if (ol >= output_shape[1])
                return;
            int32_t il = ol - pad_left[3] + crop_left[3];
            il = indexing::at<MODE>(il, static_cast<int32_t>(input_shape[3]));
            output_row[ol] = input_row[il];
        }
    }

    template<BorderMode MODE, typename T>
    void launchResizeWith_(const shared_t<T[]>& input, uint4_t input_strides, uint4_t input_shape,
                           const shared_t<T[]>& output, uint4_t output_strides, uint4_t output_shape,
                           int4_t border_left, cuda::Stream& stream) {
        const int4_t crop_left(math::min(border_left, 0) * -1);
        const int4_t pad_left(math::max(border_left, 0));

        const uint2_t uint_shape(output_shape.get(2));
        const uint32_t blocks_x = math::divideUp(uint_shape[1], BLOCK_WORK_SIZE_2D.x);
        const uint32_t blocks_y = math::divideUp(uint_shape[0], BLOCK_WORK_SIZE_2D.y);
        const dim3 blocks{blocks_x * blocks_y, output_shape[1], output_shape[0]};
        const AccessorRestrict<const T, 4, uint32_t> input_accessor(input.get(), input_strides);
        const AccessorRestrict<T, 4, uint32_t> output_accessor(output.get(), output_strides);

        stream.enqueue("memory::resizeWith", resizeWith_<MODE, T>, {blocks, BLOCK_SIZE_2D},
                       input_accessor, input_shape, output_accessor, uint_shape,
                       crop_left, pad_left, blocks_x);
        stream.attach(input, output);
    }
}

namespace noa::cuda::memory {
    template<typename T, typename>
    void resize(const shared_t<T[]>& input, dim4_t input_strides, dim4_t input_shape,
                int4_t border_left, int4_t border_right,
                const shared_t<T[]>& output, dim4_t output_strides,
                BorderMode border_mode, T border_value, Stream& stream) {
        if (all(border_left == 0) && all(border_right == 0))
            return copy(input, input_strides, output, output_strides, input_shape, stream);

        NOA_ASSERT_DEVICE_PTR(input.get(), stream.device());
        NOA_ASSERT_DEVICE_PTR(output.get(), stream.device());
        const auto tmp = safe_cast<int4_t>(input_shape) + border_left + border_right;
        NOA_ASSERT(all(tmp >= 1));
        uint4_t output_shape(tmp);

        // Optimize reads/writes for output:
        auto input_strides_ = safe_cast<uint4_t>(input_strides);
        auto output_strides_ = safe_cast<uint4_t>(output_strides);
        auto input_shape_ = safe_cast<uint4_t>(input_shape);
        const auto order = indexing::order(output_strides_, output_shape);
        if (!all(order == uint4_t{0, 1, 2, 3})) {
            input_strides_ = indexing::reorder(input_strides_, order);
            input_shape_ = indexing::reorder(input_shape_, order);
            border_left = indexing::reorder(border_left, order);
            border_right = indexing::reorder(border_right, order);
            output_strides_ = indexing::reorder(output_strides_, order);
            output_shape = indexing::reorder(output_shape, order);
        }

        switch (border_mode) {
            case BORDER_NOTHING:
                return launchResizeWithNothing_(input, input_strides_,
                                                output, output_strides_, output_shape,
                                                border_left, border_right, stream);
            case BORDER_ZERO:
                return launchResizeWithValue_(input, input_strides_,
                                              output, output_strides_, output_shape,
                                              border_left, border_right, T{0}, stream);
            case BORDER_VALUE:
                return launchResizeWithValue_(input, input_strides_,
                                              output, output_strides_, output_shape,
                                              border_left, border_right, border_value, stream);
            case BORDER_CLAMP:
                return launchResizeWith_<BORDER_CLAMP>(input, input_strides_, input_shape_,
                                                       output, output_strides_, output_shape,
                                                       border_left, stream);
            case BORDER_PERIODIC:
                return launchResizeWith_<BORDER_PERIODIC>(input, input_strides_, input_shape_,
                                                          output, output_strides_, output_shape,
                                                          border_left, stream);
            case BORDER_REFLECT:
                return launchResizeWith_<BORDER_REFLECT>(input, input_strides_, input_shape_,
                                                         output, output_strides_, output_shape,
                                                         border_left, stream);
            case BORDER_MIRROR:
                return launchResizeWith_<BORDER_MIRROR>(input, input_strides_, input_shape_,
                                                        output, output_strides_, output_shape,
                                                        border_left, stream);
            default:
                NOA_THROW("BorderMode not supported. Got: {}", border_mode);
        }
    }

    #define NOA_INSTANTIATE_RESIZE_(T) \
    template void resize<T, void>(const shared_t<T[]>&, dim4_t, dim4_t, int4_t, int4_t, const shared_t<T[]>&, dim4_t, BorderMode, T, Stream&)

    NOA_INSTANTIATE_RESIZE_(bool);
    NOA_INSTANTIATE_RESIZE_(int8_t);
    NOA_INSTANTIATE_RESIZE_(int16_t);
    NOA_INSTANTIATE_RESIZE_(int32_t);
    NOA_INSTANTIATE_RESIZE_(int64_t);
    NOA_INSTANTIATE_RESIZE_(uint8_t);
    NOA_INSTANTIATE_RESIZE_(uint16_t);
    NOA_INSTANTIATE_RESIZE_(uint32_t);
    NOA_INSTANTIATE_RESIZE_(uint64_t);
    NOA_INSTANTIATE_RESIZE_(half_t);
    NOA_INSTANTIATE_RESIZE_(float);
    NOA_INSTANTIATE_RESIZE_(double);
    NOA_INSTANTIATE_RESIZE_(chalf_t);
    NOA_INSTANTIATE_RESIZE_(cfloat_t);
    NOA_INSTANTIATE_RESIZE_(cdouble_t);
}
