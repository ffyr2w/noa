#include "hip/hip_runtime.h"
#include "noa/common/Profiler.h"
#include "noa/common/Math.h"
#include "noa/gpu/cuda/signal/Shape.h"

// TODO Add vectorized loads/stores?
namespace {
    using namespace noa;
    constexpr dim3 BLOCK_SIZE(32, 8);

    template<bool INVERT>
    __forceinline__ __device__ float getSoftMask_(float dst_sqd, float radius, float radius_sqd,
                                                  float radius_taper_sqd, float taper_size) {
        float mask_value;
        constexpr float PI = math::Constants<float>::PI;
        if constexpr (INVERT) {
            if (dst_sqd > radius_taper_sqd) {
                mask_value = 1.f;
            } else if (dst_sqd <= radius_sqd) {
                mask_value = 0.f;
            } else {
                float dst = math::sqrt(dst_sqd);
                mask_value = (1.f - math::cos(PI * (dst - radius) / taper_size)) * 0.5f;
            }
        } else {
            if (dst_sqd > radius_taper_sqd) {
                mask_value = 0.f;
            } else if (dst_sqd <= radius_sqd) {
                mask_value = 1.f;
            } else {
                dst_sqd = math::sqrt(dst_sqd);
                mask_value = (1.f + math::cos(PI * (dst_sqd - radius) / taper_size)) * 0.5f;
            }
        }
        return mask_value;
    }

    template<bool INVERT>
    __forceinline__ __device__ float getHardMask_(float dst_sqd, float radius_sqd) {
        float mask_value;
        if constexpr (INVERT) {
            if (dst_sqd > radius_sqd)
                mask_value = 1;
            else
                mask_value = 0;
        } else {
            if (dst_sqd > radius_sqd)
                mask_value = 0;
            else
                mask_value = 1;
        }
        return mask_value;
    }

    template<bool TAPER, bool INVERT, typename T>
    __global__ __launch_bounds__(BLOCK_SIZE.x * BLOCK_SIZE.y)
    void sphere_(const T* input, uint4_t input_stride, T* output, uint4_t output_stride,
                 uint2_t shape, uint batches,
                 float3_t center, float radius, float taper_size) {
        const uint3_t gid{blockIdx.z,
                          blockIdx.y * BLOCK_SIZE.y + threadIdx.y,
                          blockIdx.x * BLOCK_SIZE.x + threadIdx.x};
        if (gid[1] >= shape[0] || gid[2] >= shape[1])
            return;

        const float radius_sqd = radius * radius;
        const float3_t tmp{float3_t(gid) - center};
        const float dst_sqd = math::dot(tmp, tmp);

        float mask;
        if constexpr (TAPER) {
            float radius_taper_sqd = radius + taper_size;
            radius_taper_sqd *= radius_taper_sqd;
            mask = getSoftMask_<INVERT>(dst_sqd, radius, radius_sqd, radius_taper_sqd, taper_size);
        } else {
            mask = getHardMask_<INVERT>(dst_sqd, radius_sqd);
            (void) taper_size;
        }

        using real_t = traits::value_type_t<T>;
        const uint offset = gid[0] * input_stride[1] + gid[1] * input_stride[2] + gid[2] * input_stride[3];
        output += gid[0] * output_stride[1] + gid[1] * output_stride[2] + gid[2] * output_stride[3];
        for (uint batch = 0; batch < batches; ++batch) {
            output[batch * output_stride[0]] =
                    input ?
                    input[batch * input_stride[0] + offset] * static_cast<real_t>(mask) :
                    static_cast<real_t>(mask);
        }
    }
}

namespace noa::cuda::signal {
    template<bool INVERT, typename T>
    void sphere(const shared_t<T[]>& input, size4_t input_stride,
                const shared_t<T[]>& output, size4_t output_stride, size4_t shape,
                float3_t center, float radius, float taper_size, Stream& stream) {
        NOA_PROFILE_FUNCTION();
        const uint2_t u_shape{shape.get() + 2};
        const bool taper = taper_size > 1e-5f;
        const dim3 blocks(math::divideUp(u_shape[1], BLOCK_SIZE.x),
                          math::divideUp(u_shape[0], BLOCK_SIZE.y),
                          shape[1]);
        const LaunchConfig config{blocks, BLOCK_SIZE};
        stream.enqueue("filter::sphere", taper ? sphere_<true, INVERT, T> : sphere_<false, INVERT, T>, config,
                       input.get(), uint4_t{input_stride}, output.get(), uint4_t{output_stride}, u_shape, shape[0],
                       center, radius, taper_size);
        stream.attach(input, output);
    }

    #define NOA_INSTANTIATE_SPHERE_(T)                                                                                                      \
    template void sphere<true, T>(const shared_t<T[]>&, size4_t, const shared_t<T[]>&, size4_t, size4_t, float3_t, float, float, Stream&);  \
    template void sphere<false, T>(const shared_t<T[]>&, size4_t, const shared_t<T[]>&, size4_t, size4_t, float3_t, float, float, Stream&)

    NOA_INSTANTIATE_SPHERE_(half_t);
    NOA_INSTANTIATE_SPHERE_(float);
    NOA_INSTANTIATE_SPHERE_(double);
    NOA_INSTANTIATE_SPHERE_(chalf_t);
    NOA_INSTANTIATE_SPHERE_(cfloat_t);
    NOA_INSTANTIATE_SPHERE_(cdouble_t);
}
