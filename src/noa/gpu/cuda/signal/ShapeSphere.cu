#include "hip/hip_runtime.h"
#include "noa/common/Math.h"
#include "noa/gpu/cuda/signal/Shape.h"

// TODO Add vectorized loads/stores?
namespace {
    using namespace noa;
    constexpr dim3 BLOCK_SIZE(32, 8);

    template<bool INVERT>
    __forceinline__ __device__ float getSoftMask_(float dst_sqd, float radius, float radius_sqd,
                                                  float radius_taper_sqd, float taper_size) {
        float mask_value;
        constexpr float PI = math::Constants<float>::PI;
        if constexpr (INVERT) {
            if (dst_sqd > radius_taper_sqd) {
                mask_value = 1.f;
            } else if (dst_sqd <= radius_sqd) {
                mask_value = 0.f;
            } else {
                float dst = math::sqrt(dst_sqd);
                mask_value = (1.f - math::cos(PI * (dst - radius) / taper_size)) * 0.5f;
            }
        } else {
            if (dst_sqd > radius_taper_sqd) {
                mask_value = 0.f;
            } else if (dst_sqd <= radius_sqd) {
                mask_value = 1.f;
            } else {
                dst_sqd = math::sqrt(dst_sqd);
                mask_value = (1.f + math::cos(PI * (dst_sqd - radius) / taper_size)) * 0.5f;
            }
        }
        return mask_value;
    }

    template<bool INVERT>
    __forceinline__ __device__ float getHardMask_(float dst_sqd, float radius_sqd) {
        float mask_value;
        if constexpr (INVERT) {
            if (dst_sqd > radius_sqd)
                mask_value = 1;
            else
                mask_value = 0;
        } else {
            if (dst_sqd > radius_sqd)
                mask_value = 0;
            else
                mask_value = 1;
        }
        return mask_value;
    }

    template<bool TAPER, bool INVERT, typename T>
    __global__ __launch_bounds__(BLOCK_SIZE.x * BLOCK_SIZE.y)
    void sphere_(const T* input, uint4_t input_stride, T* output, uint4_t output_stride,
                 uint2_t start, uint2_t end, uint batches,
                 float3_t center, float radius, float taper_size) {
        const uint3_t gid{blockIdx.z,
                          blockIdx.y * BLOCK_SIZE.y + threadIdx.y + start[0],
                          blockIdx.x * BLOCK_SIZE.x + threadIdx.x + start[1]};
        if (gid[1] >= end[0] || gid[2] >= end[1])
            return;

        const float radius_sqd = radius * radius;
        const float3_t tmp{float3_t(gid) - center};
        const float dst_sqd = math::dot(tmp, tmp);

        float mask;
        if constexpr (TAPER) {
            float radius_taper_sqd = radius + taper_size;
            radius_taper_sqd *= radius_taper_sqd;
            mask = getSoftMask_<INVERT>(dst_sqd, radius, radius_sqd, radius_taper_sqd, taper_size);
        } else {
            mask = getHardMask_<INVERT>(dst_sqd, radius_sqd);
            (void) taper_size;
        }

        using real_t = traits::value_type_t<T>;
        const uint offset = gid[0] * input_stride[1] + gid[1] * input_stride[2] + gid[2] * input_stride[3];
        output += gid[0] * output_stride[1] + gid[1] * output_stride[2] + gid[2] * output_stride[3];
        for (uint batch = 0; batch < batches; ++batch) {
            output[batch * output_stride[0]] =
                    input ?
                    input[batch * input_stride[0] + offset] * static_cast<real_t>(mask) :
                    static_cast<real_t>(mask);
        }
    }
}

namespace noa::cuda::signal {
    template<bool INVERT, typename T, typename>
    void sphere(const shared_t<T[]>& input, size4_t input_stride,
                const shared_t<T[]>& output, size4_t output_stride, size4_t shape,
                float3_t center, float radius, float taper_size, Stream& stream) {
        uint3_t start{0}, end{shape.get(1)};
        if (INVERT && input.get() == output.get()) {
            start = uint3_t{noa::math::clamp(int3_t{center - (radius + taper_size)}, int3_t{}, int3_t{end})};
            end = uint3_t{noa::math::clamp(int3_t{center + (radius + taper_size) + 1}, int3_t{}, int3_t{end})};
            if (any(end <= start))
                return;
        }
        const uint3_t shape_{end - start};

        const dim3 blocks(math::divideUp(shape_[2], BLOCK_SIZE.x),
                          math::divideUp(shape_[1], BLOCK_SIZE.y),
                          shape_[0]);
        const LaunchConfig config{blocks, BLOCK_SIZE};
        const bool taper = taper_size > 1e-5f;
        stream.enqueue("signal::sphere", taper ? sphere_<true, INVERT, T> : sphere_<false, INVERT, T>, config,
                       input.get(), uint4_t{input_stride}, output.get(), uint4_t{output_stride},
                       uint2_t{start.get(1)}, uint2_t{end.get(1)}, shape[0],
                       center, radius, taper_size);
        stream.attach(input, output);
    }

    #define NOA_INSTANTIATE_SPHERE_(T)                                                                                                              \
    template void sphere<true, T, void>(const shared_t<T[]>&, size4_t, const shared_t<T[]>&, size4_t, size4_t, float3_t, float, float, Stream&);    \
    template void sphere<false, T, void>(const shared_t<T[]>&, size4_t, const shared_t<T[]>&, size4_t, size4_t, float3_t, float, float, Stream&)

    NOA_INSTANTIATE_SPHERE_(half_t);
    NOA_INSTANTIATE_SPHERE_(float);
    NOA_INSTANTIATE_SPHERE_(double);
    NOA_INSTANTIATE_SPHERE_(chalf_t);
    NOA_INSTANTIATE_SPHERE_(cfloat_t);
    NOA_INSTANTIATE_SPHERE_(cdouble_t);
}
