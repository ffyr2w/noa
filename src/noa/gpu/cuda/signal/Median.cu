#include "hip/hip_runtime.h"
#include "noa/common/Math.h"

#include "noa/gpu/cuda/Types.h"
#include "noa/gpu/cuda/Exception.h"
#include "noa/gpu/cuda/signal/Median.h"
#include "noa/gpu/cuda/memory/Copy.h"
#include "noa/gpu/cuda/util/Block.cuh"

// The current implementations only supports small squared windows. This allows to:
//  1)  Load the windows for all threads in a block in shared memory. This is useful because windows overlap.
//  2)  The exchange search can be done on the per thread registers. Only about half of the window needs to
//      be on the registers at a single time. The rest stays in shared memory. This also requires the indexing
//      to be constant, i.e. the window size should be a template argument.
// TODO Maybe look at other implementations for larger windows (e.g. with textures)?

namespace {
    using namespace ::noa;
    using namespace ::noa::cuda;
    constexpr dim3 BLOCK_SIZE(16, 16);

    // Ensures a < b. If not, swap.
    template<typename T>
    __forceinline__ __device__ void swap_(T& a, T& b) {
        if (b < a) {
            T tmp = a;
            a = b;
            b = tmp;
        }
    }

    // Sets the array v, so that the min is at v[0] and max is at v[length-1].
    // TODO make sure the force inline still allows constant indexing, otherwise replace with a macro.
    template<typename T>
    __forceinline__ __device__ void order_(T* v, int32_t length) {
        for (int32_t i = 0; i < length / 2; i++) // ensure min in first half, max in second half
            swap_(v[i], v[length - 1 - i]);
        for (int32_t i = 1; i < (length + 1) / 2; i++) // move min in first half to first pos
            swap_(v[0], v[i]);
        for (int32_t i = length - 2; i >= length / 2; i--) // move max in second half to last pos
            swap_(v[i], v[length - 1]);
    }

    // Loads the shared memory array according to the BORDER_MODE.
    // input_row:   device memory. Should point to the beginning of the current row.
    // stride:      innermost stride of the input.
    // s_mem:       shared memory. Should point to the current element.
    // shape_x:     number of logical elements in x, x being the dimension of the 1D window.
    // gx:          index of the current element in x. If out of bound, add padding according to the BORDER_MODE.
    template<typename T, int32_t BORDER_MODE, int32_t HALO>
    __device__ void loadToShared1D_(AccessorReference<const T, 1, uint32_t> input_row,
                                    T* s_mem, int32_t shape_x, int32_t gx) {
        static_assert(BORDER_MODE == BORDER_REFLECT || BORDER_MODE == BORDER_ZERO);
        if constexpr (BORDER_MODE == BORDER_REFLECT) {
            if (gx < 0)
                *s_mem = input_row[-gx]; // pad left; requires shape_x >= HALO + 1, since gx >= -HALO
            else if (gx < shape_x)
                *s_mem = input_row[gx]; // in array, no padding
            else if (gx < shape_x + HALO)
                *s_mem = input_row[(2 * (shape_x - 1) - gx)]; // pad right; requires shape_x >= HALO  + 1
            // Otherwise, don't do anything since the *s_mem element will not be used anyway.

        } else if constexpr (BORDER_MODE == BORDER_ZERO) {
            if (gx < 0 || gx >= shape_x)
                *s_mem = static_cast<T>(0);
            else
                *s_mem = input_row[gx];
        }
    }

    template<typename T, int32_t BORDER_MODE, int32_t WINDOW_SIZE>
    __global__ __launch_bounds__(BLOCK_SIZE.x * BLOCK_SIZE.y)
    void medfilt1_(Accessor<const T, 4, uint32_t> input,
                   Accessor<T, 4, uint32_t> output,
                   uint2_t shape, uint32_t blocks_x) {
        static_assert(WINDOW_SIZE % 2); // only support odd windows.
        constexpr int32_t PADDING = WINDOW_SIZE - 1; // assume odd
        constexpr int32_t HALO = PADDING / 2;

        // The shared memory is the shape of the block + the padding in the first dimension.
        using uninit_t = util::traits::uninitialized_type_t<T>;
        constexpr int2_t SHARED_SIZE(BLOCK_SIZE.y, BLOCK_SIZE.x + PADDING);
        __shared__ uninit_t buffer[math::prod(SHARED_SIZE)];
        T* shared_mem = reinterpret_cast<T*>(buffer);

        const uint2_t index = indexing::indexes(blockIdx.x, blocks_x);
        const int2_t tid{threadIdx.y, threadIdx.x};
        const int4_t gid{blockIdx.z,
                         blockIdx.y,
                         BLOCK_SIZE.y * index[0] + tid[0],
                         BLOCK_SIZE.x * index[1] + tid[1]};

        const auto input_ = input[gid[0]][gid[1]][gid[2]];

        // There's no padding in y, so if out of bounds, stop.
        if (gid[2] < shape[0]) {
            // Load shared memory. Loop to take into account padding.
            for (int32_t lx = tid[1], gx = gid[3]; lx < SHARED_SIZE[1]; lx += BLOCK_SIZE.x, gx += BLOCK_SIZE.x) {
                loadToShared1D_<T, BORDER_MODE, HALO>(input_,
                                                      shared_mem + tid[0] * SHARED_SIZE[1] + lx,
                                                      shape[1], gx - HALO);
            }
            util::block::synchronize();

            // Only continue if not out of bound.
            if (gid[3] < shape[1]) {
                // The goal is to reduce register pressure as much as possible, but still use registers
                // to do the exchange sort. The window is divided into two half: the first "activate" half,
                // which is where the sorting happens, and the second half, which is the pool of contestants
                // that will be progressively added.
                constexpr int32_t ACTIVE = WINDOW_SIZE / 2 + 2; // the final size should be at least 3.

                // Load active half to, hopefully, the local registers (otherwise spill to device memory).
                T v[ACTIVE]; // all indexing are known at compile, so registers should be used
                for (int32_t x = 0; x < ACTIVE; ++x)
                    v[x] = shared_mem[tid[0] * SHARED_SIZE[1] + tid[1] + x];

                order_(v, ACTIVE); // ensure min at 0, max at ACTIVE - 1

                // We can replace the min by a new contestant and then reorder.
                // We also don't have to compare the max at ACTIVE - 1, we just need to stack the new max at the end.
                // As such, as we add new contestants, we right-truncate the active half to ignore the previous max.
                int32_t length = ACTIVE;
                for (int32_t k = ACTIVE; k < WINDOW_SIZE; ++k) {
                    v[0] = shared_mem[tid[0] * SHARED_SIZE[1] + tid[1] + k]; // replace min by new contestant
                    --length; // ignore the previous max at the end
                    order_(v, length); // min at 0, max at length - 1
                }

                // Since we know the min and max are at v[0] and v[length-1] respectively, to find the median,
                // we simply need to (exchange) sort the elements from v[1] to v[length-2] included.
                // The median will then be at length/2.
                for (int32_t k = 1; k < length - 1; k++)
                    order_(v + k, length - k);
                output(gid) = v[length / 2];
            }
        }
    }

    // Loads the (gx,gy) element according to the BORDER_MODE.
    // input_slice: device memory. Should point to the beginning of the current slice (z).
    // stride_y/x:  stride in y and x.
    // s_mem:       shared memory. Should point to the current element.
    // shape_y/x:   number of logical elements in y/x.
    // gy/x:        index of the current element in y/x. If out of bound, add padding according to the BORDER_MODE.
    template<typename T, int32_t BORDER_MODE, int32_t HALO>
    __device__ void loadToShared2D_(AccessorReference<const T, 2, uint32_t> input_slice, T* s_mem,
                                    int32_t shape_y, int32_t gy, int32_t shape_x, int32_t gx) {
        static_assert(BORDER_MODE == BORDER_REFLECT || BORDER_MODE == BORDER_ZERO);
        if constexpr (BORDER_MODE == BORDER_REFLECT) {
            if (gx < 0)
                gx *= -1;
            else if (gx >= shape_x) {
                if (gx >= shape_x + HALO)
                    return; // don't do anything; gx is out of the shape + window so *s_mem won't be used anyway
                gx = 2 * (shape_x - 1) - gx;
            }
            if (gy < 0)
                gy *= -1;
            else if (gy >= shape_y) {
                if (gy >= shape_y + HALO)
                    return;
                gy = 2 * (shape_y - 1) - gy;
            }
            *s_mem = input_slice(gy, gx);

        } else if constexpr (BORDER_MODE == BORDER_ZERO) {
            if (gx < 0 || gx >= shape_x || gy < 0 || gy >= shape_y)
                *s_mem = static_cast<T>(0);
            else
                *s_mem = input_slice(gy, gx);
        }
    }

    template<typename T, int32_t BORDER_MODE, int32_t WINDOW_SIZE>
    __global__ __launch_bounds__(BLOCK_SIZE.x * BLOCK_SIZE.y)
    void medfilt2_(Accessor<const T, 4, uint32_t> input,
                   Accessor<T, 4, uint32_t> output,
                   uint2_t shape, uint32_t blocks_x) {
        static_assert(WINDOW_SIZE % 2); // only support odd windows.
        constexpr int32_t TILE_SIZE = WINDOW_SIZE * WINDOW_SIZE;
        constexpr int32_t PADDING = WINDOW_SIZE - 1; // assume odd
        constexpr int32_t HALO = PADDING / 2;

        // The shared memory is the shape of the block + the padding the first and second dimension.
        using uninit_t = util::traits::uninitialized_type_t<T>;
        constexpr int2_t SHARED_SIZE(BLOCK_SIZE.y + PADDING, BLOCK_SIZE.x + PADDING);
        __shared__ uninit_t buffer[math::prod(SHARED_SIZE)];
        T* shared_mem = reinterpret_cast<T*>(buffer);

        const uint2_t index = indexing::indexes(blockIdx.x, blocks_x);
        const int2_t tid{threadIdx.y, threadIdx.x};
        const int4_t gid{blockIdx.z,
                         blockIdx.y,
                         BLOCK_SIZE.y * index[0] + tid[0],
                         BLOCK_SIZE.x * index[1] + tid[1]};

        const auto input_ = input[gid[0]][gid[1]];

        // Load shared memory. Loop to account for the halo.
        for (int32_t ly = tid[0], gy = gid[2]; ly < SHARED_SIZE[0]; ly += BLOCK_SIZE.y, gy += BLOCK_SIZE.y)
            for (int32_t lx = tid[1], gx = gid[3]; lx < SHARED_SIZE[1]; lx += BLOCK_SIZE.x, gx += BLOCK_SIZE.x)
                loadToShared2D_<T, BORDER_MODE, HALO>(input_,
                                                      shared_mem + ly * SHARED_SIZE[1] + lx,
                                                      shape[0], gy - HALO,
                                                      shape[1], gx - HALO);
        util::block::synchronize();

        // Only continue if not out of bound. gid.z cannot be out of bound.
        if (gid[2] < shape[0] && gid[3] < shape[1]) {
            constexpr int32_t ACTIVE = TILE_SIZE / 2 + 2;

            // Load active window from shared memory into this 1D array.
            T v[ACTIVE];
            for (int32_t count = 0, y = 0; y < WINDOW_SIZE; ++y)
                for (int32_t x = 0; count < ACTIVE && x < WINDOW_SIZE; ++x, ++count)
                    v[count] = shared_mem[(tid[0] + y) * SHARED_SIZE[1] + tid[1] + x];

            order_(v, ACTIVE);
            int32_t length = ACTIVE;
            for (int32_t k = ACTIVE; k < TILE_SIZE; ++k) {
                // k is the index, but we want the corresponding (x,y) coordinates in the 2D window.
                // Then offset these coordinates to the current thread.
                const int32_t coord_y = k / WINDOW_SIZE;
                const int32_t coord_x = k - coord_y * WINDOW_SIZE;
                v[0] = shared_mem[(coord_y + tid[0]) * SHARED_SIZE[1] + coord_x + tid[1]];

                --length;
                order_(v, length);
            }

            // Sort the final elements.
            for (int32_t k = 1; k < length - 1; k++)
                order_(v + k, length - k);
            output(gid) = v[length / 2];
        }
    }

    // Loads the (gx,gy,gz) element according to the BORDER_MODE.
    // input:        device memory. Should point to the beginning of the batch.
    // stride_z/y/x: strides
    // s_mem:        shared memory. Should point to the current element.
    // shape_z/y/x:  number of logical elements in z/y/x.
    // gz/y/x:       index of the current element in z/y/x. If out of bound, add padding according to the BORDER_MODE.
    template<typename T, int32_t BORDER_MODE, int32_t HALO>
    __device__ void loadToShared3D_(AccessorReference<const T, 3, uint32_t> input, T* s_mem,
                                    int32_t shape_z, int32_t gz,
                                    int32_t shape_y, int32_t gy,
                                    int32_t shape_x, int32_t gx) {
        static_assert(BORDER_MODE == BORDER_REFLECT || BORDER_MODE == BORDER_ZERO);
        if constexpr (BORDER_MODE == BORDER_REFLECT) {
            if (gx < 0) {
                gx *= -1;
            } else if (gx >= shape_x) {
                if (gx >= shape_x + HALO)
                    return;
                gx = 2 * (shape_x - 1) - gx;
            }
            if (gy < 0) {
                gy *= -1;
            } else if (gy >= shape_y) {
                if (gy >= shape_y + HALO)
                    return;
                gy = 2 * (shape_y - 1) - gy;
            }
            if (gz < 0) {
                gz *= -1;
            } else if (gz >= shape_z) {
                if (gz >= shape_z + HALO)
                    return;
                gz = 2 * (shape_z - 1) - gz;
            }
            *s_mem = input(gz, gy, gx);

        } else if constexpr (BORDER_MODE == BORDER_ZERO) {
            if (gx < 0 || gx >= shape_x ||
                gy < 0 || gy >= shape_y ||
                gz < 0 || gz >= shape_z)
                *s_mem = static_cast<T>(0);
            else
                *s_mem = input(gz, gy, gx);
        }
    }

    // The launch config and block size is like medfilt1_.
    template<typename T, int32_t BORDER_MODE, uint32_t WINDOW_SIZE>
    __global__ __launch_bounds__(BLOCK_SIZE.x * BLOCK_SIZE.y)
    void medfilt3_(Accessor<const T, 4, uint32_t> input,
                   Accessor<T, 4, uint32_t> output,
                   uint3_t shape, uint32_t blocks_x) {
        static_assert(WINDOW_SIZE % 2); // only support odd windows.
        constexpr int32_t TILE_SIZE = WINDOW_SIZE * WINDOW_SIZE * WINDOW_SIZE;
        constexpr int32_t PADDING = WINDOW_SIZE - 1; // assume odd
        constexpr int32_t HALO = PADDING / 2;

        // The shared memory is the shape of the block + the padding all 3 dimensions.
        // For the largest supported window (5), this goes up to 20*20*5=2205 elements.
        using uninit_t = util::traits::uninitialized_type_t<T>;
        constexpr int3_t SHARED_SIZE(WINDOW_SIZE, BLOCK_SIZE.y + PADDING, BLOCK_SIZE.x + PADDING);
        __shared__ uninit_t buffer[math::prod(SHARED_SIZE)];
        T* shared_mem = reinterpret_cast<T*>(buffer);

        const uint2_t index = indexing::indexes(blockIdx.x, blocks_x);
        const int2_t tid{threadIdx.y, threadIdx.x};
        const int4_t gid{blockIdx.z,
                         blockIdx.y,
                         BLOCK_SIZE.y * index[0] + tid[0],
                         BLOCK_SIZE.x * index[1] + tid[1]};

        const auto input_ = input[gid[0]];

        // Load shared memory.
        // Each thread processes at least WINDOW_SIZE elements (the z dimension).
        for (int32_t lz = 0, gz = gid[1]; lz < SHARED_SIZE[0]; ++lz, ++gz)
            for (int32_t ly = tid[0], gy = gid[2]; ly < SHARED_SIZE[1]; ly += BLOCK_SIZE.y, gy += BLOCK_SIZE.y)
                for (int32_t lx = tid[1], gx = gid[3]; lx < SHARED_SIZE[2]; lx += BLOCK_SIZE.x, gx += BLOCK_SIZE.x)
                    loadToShared3D_<T, BORDER_MODE, HALO>(
                            input_,
                            shared_mem + (lz * SHARED_SIZE[1] + ly) * SHARED_SIZE[2] + lx,
                            shape[0], gz - HALO, shape[1], gy - HALO, shape[2], gx - HALO);
        util::block::synchronize();

        // Only continue if not out of bound. gid.z cannot be out of bound.
        if (gid[2] < shape[1] && gid[3] < shape[2]) {
            constexpr int32_t ACTIVE = TILE_SIZE / 2 + 2;

            // Load active window from shared memory into this 1D array.
            T v[ACTIVE];
            for (int32_t count = 0, z = 0; z < WINDOW_SIZE; ++z)
                for (int32_t y = 0; count < ACTIVE && y < WINDOW_SIZE; ++y)
                    for (int32_t x = 0; count < ACTIVE && x < WINDOW_SIZE; ++x, ++count)
                        v[count] = shared_mem[(z * SHARED_SIZE[1] + tid[0] + y) * SHARED_SIZE[2] + tid[1] + x];

            order_(v, ACTIVE);
            int32_t length = ACTIVE;
            for (int32_t k = ACTIVE; k < TILE_SIZE; ++k) {
                // k is the index, but we want the corresponding (x,y,z) coordinates in the 3D window.
                // Then offset these coordinates to the current thread.
                const int32_t coord_z = k / (WINDOW_SIZE * WINDOW_SIZE);
                const int32_t tmp = k - coord_z * WINDOW_SIZE * WINDOW_SIZE;
                const int32_t coord_y = tmp / WINDOW_SIZE;
                const int32_t coord_x = tmp - coord_y * WINDOW_SIZE;
                v[0] = shared_mem[(coord_z * SHARED_SIZE[1] + coord_y + tid[0]) * SHARED_SIZE[2] + coord_x + tid[1]];
                --length;
                order_(v, length);
            }

            // Sort the final elements.
            for (int32_t k = 1; k < length - 1; k++)
                order_(v + k, length - k);
            output(gid) = v[length / 2];
        }
    }
}

namespace noa::cuda::signal {
    template<typename T, typename>
    void median1(const shared_t<T[]>& input, dim4_t input_strides,
                 const shared_t<T[]>& output, dim4_t output_strides,
                 dim4_t shape, BorderMode border_mode, dim_t window_size, Stream& stream) {
        NOA_ASSERT(input != output && all(shape > 0));
        NOA_ASSERT_DEVICE_PTR(input.get(), stream.device());
        NOA_ASSERT_DEVICE_PTR(output.get(), stream.device());
        if (window_size <= 1)
            return memory::copy(input, input_strides, output, output_strides, shape, stream);

        const auto uint_shape = safe_cast<uint2_t>(dim2_t(shape.get(2)));
        const auto uint_input_strides = safe_cast<uint4_t>(input_strides);
        const auto uint_output_strides = safe_cast<uint4_t>(output_strides);
        const uint32_t blocks_x = math::divideUp(uint_shape[1], BLOCK_SIZE.x);
        const uint32_t blocks_y = math::divideUp(uint_shape[0], BLOCK_SIZE.y);
        const dim3 blocks(blocks_x * blocks_y, shape[1], shape[0]);
        const LaunchConfig config{blocks, BLOCK_SIZE};

        const Accessor<const T, 4, uint32_t> input_accessor(input.get(), uint_input_strides);
        const Accessor<T, 4, uint32_t> output_accessor(output.get(), uint_output_strides);

        NOA_ASSERT(border_mode != BORDER_REFLECT || window_size / 2 + 1 <= shape[3]);
        if (border_mode != BORDER_REFLECT && border_mode != BORDER_ZERO) {
            NOA_THROW("BorderMode not supported. Should be {} or {}, got {}",
                      BORDER_REFLECT, BORDER_ZERO, border_mode);
        }

        switch (window_size) {
            case 3:
                stream.enqueue(
                        "signal::medfilt1",
                        border_mode == BORDER_REFLECT ? medfilt1_<T, BORDER_REFLECT, 3> : medfilt1_<T, BORDER_ZERO, 3>,
                        config, input_accessor, output_accessor, uint_shape, blocks_x);
                break;
            case 5:
                stream.enqueue(
                        "signal::medfilt1",
                        border_mode == BORDER_REFLECT ? medfilt1_<T, BORDER_REFLECT, 5> : medfilt1_<T, BORDER_ZERO, 5>,
                        config, input_accessor, output_accessor, uint_shape, blocks_x);
                break;
            case 7:
                stream.enqueue(
                        "signal::medfilt1",
                        border_mode == BORDER_REFLECT ? medfilt1_<T, BORDER_REFLECT, 7> : medfilt1_<T, BORDER_ZERO, 7>,
                        config, input_accessor, output_accessor, uint_shape, blocks_x);
                break;
            case 9:
                stream.enqueue(
                        "signal::medfilt1",
                        border_mode == BORDER_REFLECT ? medfilt1_<T, BORDER_REFLECT, 9> : medfilt1_<T, BORDER_ZERO, 9>,
                        config, input_accessor, output_accessor, uint_shape, blocks_x);
                break;
            case 11:
                stream.enqueue(
                        "signal::medfilt1",
                        border_mode == BORDER_REFLECT ? medfilt1_<T, BORDER_REFLECT, 11> : medfilt1_<T, BORDER_ZERO, 11>,
                        config, input_accessor, output_accessor, uint_shape, blocks_x);
                break;
            case 13:
                stream.enqueue(
                        "signal::medfilt1",
                        border_mode == BORDER_REFLECT ? medfilt1_<T, BORDER_REFLECT, 13> : medfilt1_<T, BORDER_ZERO, 13>,
                        config, input_accessor, output_accessor, uint_shape, blocks_x);
                break;
            case 15:
                stream.enqueue(
                        "signal::medfilt1",
                        border_mode == BORDER_REFLECT ? medfilt1_<T, BORDER_REFLECT, 15> : medfilt1_<T, BORDER_ZERO, 15>,
                        config, input_accessor, output_accessor, uint_shape, blocks_x);
                break;
            case 17:
                stream.enqueue(
                        "signal::medfilt1",
                        border_mode == BORDER_REFLECT ? medfilt1_<T, BORDER_REFLECT, 17> : medfilt1_<T, BORDER_ZERO, 17>,
                        config, input_accessor, output_accessor, uint_shape, blocks_x);
                break;
            case 19:
                stream.enqueue(
                        "signal::medfilt1",
                        border_mode == BORDER_REFLECT ? medfilt1_<T, BORDER_REFLECT, 19> : medfilt1_<T, BORDER_ZERO, 19>,
                        config, input_accessor, output_accessor, uint_shape, blocks_x);
                break;
            case 21:
                stream.enqueue(
                        "signal::medfilt1",
                        border_mode == BORDER_REFLECT ? medfilt1_<T, BORDER_REFLECT, 21> : medfilt1_<T, BORDER_ZERO, 21>,
                        config, input_accessor, output_accessor, uint_shape, blocks_x);
                break;
            default:
                NOA_THROW("Unsupported window size. It should be an odd number from 1 to 21, got {}", window_size);
        }
        stream.attach(input, output);
    }

    template<typename T, typename>
    void median2(const shared_t<T[]>& input, dim4_t input_strides,
                 const shared_t<T[]>& output, dim4_t output_strides,
                 dim4_t shape, BorderMode border_mode, dim_t window_size, Stream& stream) {
        NOA_ASSERT(input != output && all(shape > 0));
        NOA_ASSERT_DEVICE_PTR(input.get(), stream.device());
        NOA_ASSERT_DEVICE_PTR(output.get(), stream.device());
        if (window_size <= 1)
            return memory::copy(input, input_strides, output, output_strides, shape, stream);

        const dim2_t order_2d = indexing::order(dim2_t(output_strides.get(2)), dim2_t(shape.get(2)));
        if (any(order_2d != dim2_t{0, 1})) {
            std::swap(input_strides[2], input_strides[3]);
            std::swap(output_strides[2], output_strides[3]);
            std::swap(shape[2], shape[3]);
        }

        const auto uint_shape = safe_cast<uint2_t>(dim2_t(shape.get(2)));
        const auto uint_input_strides = safe_cast<uint4_t>(input_strides);
        const auto uint_output_strides = safe_cast<uint4_t>(output_strides);
        const uint32_t blocks_x = math::divideUp(uint_shape[1], BLOCK_SIZE.x);
        const uint32_t blocks_y = math::divideUp(uint_shape[0], BLOCK_SIZE.y);
        const dim3 blocks(blocks_x * blocks_y, shape[1], shape[0]);
        const LaunchConfig config{blocks, BLOCK_SIZE};

        const Accessor<const T, 4, uint32_t> input_accessor(input.get(), uint_input_strides);
        const Accessor<T, 4, uint32_t> output_accessor(output.get(), uint_output_strides);

        NOA_ASSERT(border_mode != BORDER_REFLECT || all(window_size / 2 + 1 <= uint_shape));
        if (border_mode != BORDER_REFLECT && border_mode != BORDER_ZERO) {
            NOA_THROW("BorderMode not supported. Should be {} or {}, got {}",
                      BORDER_REFLECT, BORDER_ZERO, border_mode);
        }

        switch (window_size) {
            case 3:
                stream.enqueue(
                        "signal::medfilt2",
                        border_mode == BORDER_REFLECT ? medfilt2_<T, BORDER_REFLECT, 3> : medfilt2_<T, BORDER_ZERO, 3>,
                        config, input_accessor, output_accessor, uint_shape, blocks_x);
                break;
            case 5:
                stream.enqueue(
                        "signal::medfilt2",
                        border_mode == BORDER_REFLECT ? medfilt2_<T, BORDER_REFLECT, 5> : medfilt2_<T, BORDER_ZERO, 5>,
                        config, input_accessor, output_accessor, uint_shape, blocks_x);
                break;
            case 7:
                stream.enqueue(
                        "signal::medfilt2",
                        border_mode == BORDER_REFLECT ? medfilt2_<T, BORDER_REFLECT, 7> : medfilt2_<T, BORDER_ZERO, 7>,
                        config, input_accessor, output_accessor, uint_shape, blocks_x);
                break;
            case 9:
                stream.enqueue(
                        "signal::medfilt2",
                        border_mode == BORDER_REFLECT ? medfilt2_<T, BORDER_REFLECT, 9> : medfilt2_<T, BORDER_ZERO, 9>,
                        config, input_accessor, output_accessor, uint_shape, blocks_x);
                break;
            case 11:
                stream.enqueue(
                        "signal::medfilt2",
                        border_mode == BORDER_REFLECT ? medfilt2_<T, BORDER_REFLECT, 11> : medfilt2_<T, BORDER_ZERO, 11>,
                        config, input_accessor, output_accessor, uint_shape, blocks_x);
                break;
            default:
                NOA_THROW("Unsupported window size. It should be an odd number from 1 to 11, got {}", window_size);
        }
        stream.attach(input, output);
    }

    template<typename T, typename>
    void median3(const shared_t<T[]>& input, dim4_t input_strides,
                 const shared_t<T[]>& output, dim4_t output_strides,
                 dim4_t shape, BorderMode border_mode, dim_t window_size, Stream& stream) {
        NOA_ASSERT(input != output && all(shape > 0));
        NOA_ASSERT_DEVICE_PTR(input.get(), stream.device());
        NOA_ASSERT_DEVICE_PTR(output.get(), stream.device());
        if (window_size <= 1)
            return memory::copy(input, input_strides, output, output_strides, shape, stream);

        const dim3_t order_3d = indexing::order(dim3_t(output_strides.get(1)), dim3_t(shape.get(1))) + 1;
        if (any(order_3d != dim3_t{1, 2, 3})) {
            const dim4_t order{0, order_3d[0], order_3d[1], order_3d[2]};
            input_strides = indexing::reorder(input_strides, order);
            output_strides = indexing::reorder(output_strides, order);
            shape = indexing::reorder(shape, order);
        }

        const auto uint_shape = safe_cast<uint3_t>(dim3_t(shape.get(1)));
        const auto uint_input_strides = safe_cast<uint4_t>(input_strides);
        const auto uint_output_strides = safe_cast<uint4_t>(output_strides);
        const uint32_t blocks_x = math::divideUp(uint_shape[2], BLOCK_SIZE.x);
        const uint32_t blocks_y = math::divideUp(uint_shape[1], BLOCK_SIZE.y);
        const dim3 blocks(blocks_x * blocks_y, uint_shape[0], shape[0]);
        const LaunchConfig config{blocks, BLOCK_SIZE};

        const Accessor<const T, 4, uint32_t> input_accessor(input.get(), uint_input_strides);
        const Accessor<T, 4, uint32_t> output_accessor(output.get(), uint_output_strides);

        NOA_ASSERT(border_mode != BORDER_REFLECT || all(window_size / 2 + 1 <= uint_shape));
        if (border_mode != BORDER_REFLECT && border_mode != BORDER_ZERO) {
            NOA_THROW("BorderMode not supported. Should be {} or {}, got {}",
                      BORDER_REFLECT, BORDER_ZERO, border_mode);
        }

        switch (window_size) {
            case 3:
                stream.enqueue(
                        "signal::medfilt3",
                        border_mode == BORDER_REFLECT ? medfilt3_<T, BORDER_REFLECT, 3> : medfilt3_<T, BORDER_ZERO, 3>,
                        config, input_accessor, output_accessor, uint_shape, blocks_x);
                break;
            case 5:
                stream.enqueue(
                        "signal::medfilt3",
                        border_mode == BORDER_REFLECT ? medfilt3_<T, BORDER_REFLECT, 5> : medfilt3_<T, BORDER_ZERO, 5>,
                        config, input_accessor, output_accessor, uint_shape, blocks_x);
                break;
            default:
                NOA_THROW("Unsupported window size. It should be an odd number from 1 to 11, got {}", window_size);
        }
        stream.attach(input, output);
    }

    #define NOA_INSTANTIATE_MEDFILT_(T)                                                                                             \
    template void median1<T, void>(const shared_t<T[]>&, dim4_t, const shared_t<T[]>&, dim4_t, dim4_t, BorderMode, dim_t, Stream&); \
    template void median2<T, void>(const shared_t<T[]>&, dim4_t, const shared_t<T[]>&, dim4_t, dim4_t, BorderMode, dim_t, Stream&); \
    template void median3<T, void>(const shared_t<T[]>&, dim4_t, const shared_t<T[]>&, dim4_t, dim4_t, BorderMode, dim_t, Stream&)

    NOA_INSTANTIATE_MEDFILT_(half_t);
    NOA_INSTANTIATE_MEDFILT_(float);
    NOA_INSTANTIATE_MEDFILT_(double);
    NOA_INSTANTIATE_MEDFILT_(int32_t);
    NOA_INSTANTIATE_MEDFILT_(int64_t);
    NOA_INSTANTIATE_MEDFILT_(uint32_t);
    NOA_INSTANTIATE_MEDFILT_(uint64_t);
}
