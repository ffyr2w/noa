#include "hip/hip_runtime.h"
#include "noa/core/Math.hpp"
#include "noa/gpu/cuda/memory/Copy.hpp"
#include "noa/gpu/cuda/signal/Convolve.hpp"
#include "noa/gpu/cuda/utils/Block.cuh"

namespace {
    using namespace ::noa;

    constexpr dim3 BLOCK_SIZE(16, 16);
    constexpr i32 MAX_FILTER_SIZE = 11;
    constexpr i32 MAX_FILTER_BYTES = MAX_FILTER_SIZE * MAX_FILTER_SIZE * sizeof(f64);
    __constant__ char cfilter[MAX_FILTER_BYTES];

    template<typename T>
    __global__ __launch_bounds__(BLOCK_SIZE.x * BLOCK_SIZE.y)
    void convolve_2d_(AccessorRestrict<const T, 4, u32> input,
                      AccessorRestrict<T, 4, u32> output,
                      Shape2<i32> shape, Shape2<i32> filter_shape, u32 blocks_x) {

        const auto index = noa::indexing::offset2index(blockIdx.x, blocks_x);
        const auto tid = Vec2<i32>{threadIdx.y, threadIdx.x};
        const auto gid = Vec4<i32>{blockIdx.z,
                                   blockIdx.y,
                                   BLOCK_SIZE.y * index[0] + tid[0],
                                   BLOCK_SIZE.x * index[1] + tid[1]};
        const auto input_2d = input[gid[0]][gid[1]];

        const auto OFFSET = static_cast<i32>(BLOCK_SIZE.x); // block is 16x16 square
        const auto PADDING = Vec2<i32>(filter_shape.vec() - 1);
        const auto HALO = Vec2<i32>(PADDING / 2);
        const auto SHARED_LEN = Vec2<i32>(OFFSET + PADDING);
        T* shared = noa::cuda::utils::block_dynamic_shared_resource<T>();

        // Load to shared memory. Loop to take into account padding.
        for (i32 ly = tid[0], gy = gid[2]; ly < SHARED_LEN[0]; ly += OFFSET, gy += OFFSET) {
            const i32 i_y = gy - HALO[0];
            const bool is_in_y = i_y >= 0 && i_y < shape[0];
            for (i32 lx = tid[1], gx = gid[3]; lx < SHARED_LEN[1]; lx += OFFSET, gx += OFFSET) {
                const i32 i_x = gx - HALO[1];
                const bool is_in_x = i_x >= 0 && i_x < shape[1];
                shared[ly * SHARED_LEN[1] + lx] = is_in_y && is_in_x ? input_2d(i_y, i_x) : T{0};
            }
        }
        noa::cuda::utils::block_synchronize();

        if (gid[2] < shape[0] && gid[3] < shape[1]) {
            // Weighted sum.
            const T* window = reinterpret_cast<T*>(cfilter);
            T result{0};
            for (i32 y = 0; y < filter_shape[0]; ++y)
                for (i32 x = 0; x < filter_shape[1]; ++x)
                    result += shared[(tid[0] + y) * SHARED_LEN[1] + tid[1] + x] * window[y * filter_shape[1] + x];
            output(gid) = result;
        }
    }
}

namespace noa::cuda::signal {
    template<typename T, typename U, typename>
    void convolve_2d(const T* input, const Strides4<i64>& input_strides,
                     T* output, const Strides4<i64>& output_strides, const Shape4<i64>& shape,
                     const U* filter, const Shape2<i64>& filter_shape, Stream& stream) {
        NOA_ASSERT(input != output && noa::all(shape > 0));
        NOA_ASSERT_DEVICE_PTR(input, stream.device());
        NOA_ASSERT_DEVICE_PTR(output, stream.device());
        NOA_ASSERT(filter_shape.elements() * sizeof(T) <= MAX_FILTER_BYTES);
        NOA_ASSERT(noa::all(filter_shape % 2 == 1));

        if (noa::all(filter_shape <= 1))
            return noa::cuda::memory::copy(input, input_strides, output, output_strides, shape, stream);

        NOA_THROW_IF(hipMemcpyToSymbolAsync(HIP_SYMBOL(
                cfilter), filter, filter_shape.elements() * sizeof(T),
                0, hipMemcpyDefault, stream.get()));

        const auto shape_2d = shape.filter(2, 3).as_safe<i32>();
        const u32 blocks_x = noa::math::divide_up(static_cast<u32>(shape_2d[1]), BLOCK_SIZE.x);
        const u32 blocks_y = noa::math::divide_up(static_cast<u32>(shape_2d[0]), BLOCK_SIZE.y);
        const dim3 blocks(blocks_x * blocks_y, shape[1], shape[0]);
        const u32 shared_bytes = (BLOCK_SIZE.x + filter_shape[1] - 1) *
                                 (BLOCK_SIZE.y + filter_shape[0] - 1) * sizeof(T);
        const auto config = LaunchConfig{blocks, BLOCK_SIZE, shared_bytes};
        const auto input_accessor = AccessorRestrict<const T, 4, u32> (input, input_strides.as_safe<u32>());
        const auto output_accessor = AccessorRestrict<T, 4, u32> (output, output_strides.as_safe<u32>());

        stream.enqueue("convolve_2d", convolve_2d_<T>, config,
                       input_accessor, output_accessor, shape_2d,
                       filter_shape.as_safe<i32>(), blocks_x);
    }

    #define NOA_INSTANTIATE_CONV2_(T) \
    template void convolve_2d<T,T,void>(    \
        const T*, const Strides4<i64>&,     \
        T*, const Strides4<i64>&,           \
        const Shape4<i64>&,                 \
        const T*, const Shape2<i64>&, Stream&)

    NOA_INSTANTIATE_CONV2_(f16);
    NOA_INSTANTIATE_CONV2_(f32);
    NOA_INSTANTIATE_CONV2_(f64);
}
