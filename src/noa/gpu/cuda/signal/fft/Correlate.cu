#include "hip/hip_runtime.h"
#include "noa/gpu/cuda/fft/Transforms.h"
#include "noa/gpu/cuda/math/Ewise.h"
#include "noa/gpu/cuda/math/Reduce.h"
#include "noa/gpu/cuda/memory/PtrPinned.h"
#include "noa/gpu/cuda/signal/fft/Correlate.h"
#include "noa/gpu/cuda/signal/fft/Shift.h"

#include "noa/gpu/cuda/utils/EwiseBinary.cuh"
#include "noa/gpu/cuda/utils/ReduceUnary.cuh"
#include "noa/gpu/cuda/utils/ReduceBinary.cuh"

namespace noa::cuda::signal::fft {
    template<Remap REMAP, typename T, typename U>
    void xmap(const shared_t<Complex<T>[]>& lhs, dim4_t lhs_strides,
              const shared_t<Complex<T>[]>& rhs, dim4_t rhs_strides,
              const shared_t<T[]>& output, dim4_t output_strides,
              dim4_t shape, bool normalize, Norm norm, Stream& stream,
              const shared_t<Complex<T>[]>& tmp, dim4_t tmp_strides) {

        const shared_t<Complex<T>[]>& buffer = tmp ? tmp : rhs;
        const dim4_t& buffer_strides = tmp ? tmp_strides : rhs_strides;
        NOA_ASSERT(all(buffer_strides > 0));

        if (normalize) {
            cuda::utils::ewise::binary(
                    "signal::fft::xmap",
                    lhs.get(), lhs_strides, rhs.get(), rhs_strides,
                    buffer.get(), buffer_strides,
                    shape.fft(), true, stream,
                    []__device__(Complex<T> l, Complex<T> r) {
                        const Complex<T> product = l * noa::math::conj(r);
                        const T magnitude = noa::math::abs(product);
                        return product / (magnitude + static_cast<T>(1e-13));
                        // The epsilon could be scaled by the max(abs(rhs)), but this seems to be useful only
                        // for input values close to zero (less than 1e-10). In most cases, this is fine.
                        // Note that the normalization can sharpen the peak considerably.
                    });
        } else {
            cuda::math::ewise(lhs, lhs_strides, rhs, rhs_strides, buffer, buffer_strides,
                              shape.fft(), noa::math::multiply_conj_t{}, stream);
        }

        if constexpr (REMAP == Remap::H2FC) {
            const dim3_t shape_3d(shape.get(1));
            if (shape_3d.ndim() == 3) {
                cuda::signal::fft::shift3D<Remap::H2H>(buffer, buffer_strides, buffer, buffer_strides, shape,
                                                       float3_t(shape_3d / 2), 1, stream);
            } else {
                cuda::signal::fft::shift2D<Remap::H2H>(buffer, buffer_strides, buffer, buffer_strides, shape,
                                                       float2_t{shape_3d[1] / 2, shape_3d[2] / 2}, 1, stream);
            }
        }

        cuda::fft::c2r(buffer, buffer_strides, output, output_strides, shape, norm, stream);
    }


    #define INSTANTIATE_XMAP(R, T)              \
    template void xmap<R, T, void>(             \
        const shared_t<Complex<T>[]>&, dim4_t,  \
        const shared_t<Complex<T>[]>&, dim4_t,  \
        const shared_t<T[]>&, dim4_t, dim4_t,   \
        bool, Norm, Stream&, const shared_t<Complex<T>[]>&, dim4_t)

    #define INSTANTIATE_XMAP_ALL(T)     \
    INSTANTIATE_XMAP(Remap::H2F, T);    \
    INSTANTIATE_XMAP(Remap::H2FC, T)

    INSTANTIATE_XMAP_ALL(float);
    INSTANTIATE_XMAP_ALL(double);
}

namespace noa::cuda::signal::fft::details {
    template<typename T>
    void xcorr(const shared_t<Complex<T>[]>& lhs, dim4_t lhs_stride,
               const shared_t<Complex<T>[]>& rhs, dim4_t rhs_stride,
               dim4_t shape, const shared_t<T[]>& coefficients,
               Stream& stream, bool is_half) {
        const dim_t batches = shape[0];
        const dim4_t shape_fft = is_half ? shape.fft() : shape;

        cuda::memory::PtrPinned<T> buffer(batches * 3);
        auto denominator_lhs = buffer.get() + batches;
        auto denominator_rhs = buffer.get() + batches * 2;

        T* null{};
        cuda::utils::reduce(
                "signal::fft::xcorr", lhs.get(), lhs_stride, shape_fft,
                noa::math::abs_squared_t{}, noa::math::plus_t{}, T{0},
                denominator_lhs, 1, noa::math::copy_t{},
                null, 0, noa::math::copy_t{},
                false, true, stream);
        cuda::utils::reduce(
                "signal::fft::xcorr", rhs.get(), rhs_stride, shape_fft,
                noa::math::abs_squared_t{}, noa::math::plus_t{}, T{0},
                denominator_rhs, 1, noa::math::copy_t{},
                null, 0, noa::math::copy_t{},
                false, true, stream);

        auto combine_op = []__device__(Complex<T> l, Complex<T> r) { return noa::math::real(l * r); };
        cuda::utils::reduce<false>(
                "signal::fft::xcorr",
                lhs.get(), lhs_stride, rhs.get(), rhs_stride, shape_fft,
                noa::math::copy_t{}, noa::math::conj_t{}, combine_op, noa::math::plus_t{}, T{0},
                buffer.get(), 1, noa::math::copy_t{}, null, 1, noa::math::copy_t{}, false, stream);

        stream.synchronize(); // FIXME Add callback
        for (dim_t batch = 0; batch < batches; ++batch) {
            coefficients.get()[batch] =
                    buffer[batch] / noa::math::sqrt(denominator_lhs[batch] * denominator_rhs[batch]);
        }
    }

    template<typename T>
    T xcorr(const shared_t<Complex<T>[]>& lhs, dim4_t lhs_stride,
            const shared_t<Complex<T>[]>& rhs, dim4_t rhs_stride,
            dim4_t shape, Stream& stream, bool is_half) {
        NOA_ASSERT(shape[0] == 1);
        const dim4_t shape_fft = is_half ? shape.fft() : shape;

        T numerator{}, denominator_lhs{}, denominator_rhs{};
        T* null{};
        cuda::utils::reduce(
                "signal::fft::xcorr", lhs.get(), lhs_stride, shape_fft,
                noa::math::abs_squared_t{}, noa::math::plus_t{}, T{0},
                &denominator_lhs, 1, noa::math::copy_t{},
                null, 0, noa::math::copy_t{},
                true, true, stream);
        cuda::utils::reduce(
                "signal::fft::xcorr", rhs.get(), rhs_stride, shape_fft,
                noa::math::abs_squared_t{}, noa::math::plus_t{}, T{0},
                &denominator_rhs, 1, noa::math::copy_t{},
                null, 0, noa::math::copy_t{},
                true, true, stream);

        auto combine_op = []__device__(Complex<T> l, Complex<T> r) { return noa::math::real(l * r); };
        cuda::utils::reduce<false>(
                "signal::fft::xcorr",
                lhs.get(), lhs_stride, rhs.get(), rhs_stride, shape_fft,
                noa::math::copy_t{}, noa::math::conj_t{}, combine_op, noa::math::plus_t{}, T{0},
                &numerator, 1, noa::math::copy_t{}, null, 1, noa::math::copy_t{}, false, stream);

        stream.synchronize();
        const T denominator = noa::math::sqrt(denominator_lhs * denominator_rhs);
        return numerator / denominator;
    }

    #define INSTANTIATE_XCORR(T) \
    template void xcorr<T>(const shared_t<Complex<T>[]>&, dim4_t, const shared_t<Complex<T>[]>&, dim4_t, dim4_t, const shared_t<T[]>&, Stream&, bool); \
    template T xcorr<T>(const shared_t<Complex<T>[]>&, dim4_t, const shared_t<Complex<T>[]>&, dim4_t, dim4_t, Stream&, bool)

    INSTANTIATE_XCORR(float);
    INSTANTIATE_XCORR(double);
}
