#include "hip/hip_runtime.h"
#include "noa/gpu/cuda/fft/Transforms.h"
#include "noa/gpu/cuda/math/Ewise.h"
#include "noa/gpu/cuda/math/Find.h"
#include "noa/gpu/cuda/math/Reduce.h"
#include "noa/gpu/cuda/memory/PtrPinned.h"
#include "noa/gpu/cuda/memory/PtrDevice.h"
#include "noa/gpu/cuda/signal/fft/Correlate.h"
#include "noa/gpu/cuda/signal/fft/Shift.h"

#include "noa/gpu/cuda/util/EwiseBinary.cuh"
#include "noa/gpu/cuda/util/ReduceUnary.cuh"
#include "noa/gpu/cuda/util/ReduceBinary.cuh"
#include "noa/gpu/cuda/util/Warp.cuh"

namespace {
    using namespace ::noa;
    constexpr uint BLOCK_SIZE = cuda::Limits::WARP_SIZE;

    // From the DC-centered frequency to a valid index in the non-centered output.
    // The input frequency should be in-bound, i.e. -n/2 <= frequency <= (n-1)/2
    constexpr NOA_FD int64_t getIndex_(int64_t frequency, int64_t volume_dim) {
        return frequency < 0 ? volume_dim + frequency : frequency;
    }

    // From a valid index to the DC-centered frequency.
    constexpr NOA_FD int3_t getFrequency_(int3_t index, int3_t shape) {
        return {index[0] < (shape[0] + 1) / 2 ? index[0] : index[0] - shape[0],
                index[1] < (shape[1] + 1) / 2 ? index[1] : index[1] - shape[1],
                index[2] < (shape[2] + 1) / 2 ? index[2] : index[2] - shape[2]};
    }

    // From a valid index to the DC-centered frequency.
    constexpr NOA_FD int2_t getFrequency_(int2_t index, int2_t shape) {
        return {index[0] < (shape[0] + 1) / 2 ? index[0] : index[0] - shape[0],
                index[1] < (shape[1] + 1) / 2 ? index[1] : index[1] - shape[1]};
    }

    constexpr NOA_FD int getFrequency_(int index, int shape) {
        return index < (shape + 1) / 2 ? index : index - shape;
    }

    // Given values at three successive positions, y[0], y[1], y[2], where
    // y[1] is the peak value, this fits a parabola to the values and returns the
    // offset (from -0.5 to 0.5) from the center position.
    template<typename T>
    constexpr NOA_FD T getParabolicVertex_(T y0, T y1, T y2) noexcept {
        const T d = 2 * (y0 + y2 - 2 * y1);
        T x = 0;
        // From IMOD/libcfshr/filtxcorr.c::parabolicFitPosition
        if (math::abs(d) > math::abs(static_cast<T>(1e-2) * (y0 - y2)))
            x = (y0 - y2) / d;
        if (x > T{0.5})
            x = T{0.5};
        if (x < T{-0.5})
            x = T{-0.5};
        return x;
    }

    template<bool IS_CENTERED, typename T>
    constexpr NOA_FD T fetchPeack1D_(const T* input, uint stride, int shape, int peak, int tidx, int offset) {
        T value = 0;
        if (tidx < 3) {
            if constexpr (!IS_CENTERED) {
                const int tid = getFrequency_(peak, shape) + offset;
                if (-shape / 2 <= tid && tid <= (shape - 1) / 2) {
                    value = input[getIndex_(tid, shape) * stride];
                }
            } else {
                const int tid = peak + offset;
                if (0 <= tid && tid < shape)
                    value = input[tid * stride];
            }
        }
        return value;
    }

    // Fetch the 3x3 window around the peak
    // No coalescing here I'm afraid.
    template<bool IS_CENTERED, typename T>
    constexpr NOA_FD T fetchPeack2D_(const T* input, uint2_t stride, int2_t shape, int2_t peak,
                                     int tidx, int2_t offset) {
        T value = 0;
        if (tidx < 9) {
            if constexpr (!IS_CENTERED) {
                const int2_t tid = getFrequency_(peak, shape) + offset;
                if (all(-shape / 2 <= tid && tid <= (shape - 1) / 2)) {
                    value = input[indexing::at(getIndex_(tid[0], shape[0]),
                                               getIndex_(tid[1], shape[1]),
                                               stride)];
                }
            } else {
                const int2_t tid = peak + offset;
                if (all(0 <= tid && tid < shape))
                    value = input[indexing::at(tid, stride)];
            }
        }
        return value;
    }

    // Fetch the 3x3x3 window around the peak
    // No coalescing here I'm afraid.
    template<bool IS_CENTERED, typename T>
    constexpr NOA_FD T fetchPeack3D_(const T* input, uint3_t stride, int3_t shape, int3_t peak,
                                     int tidx, int3_t offset) {
        T value = 0;
        if (tidx < 27) {
            if constexpr (!IS_CENTERED) {
                const int3_t tid = getFrequency_(peak, shape) + offset;
                if (all(-shape / 2 <= tid && tid <= (shape - 1) / 2)) {
                    value = input[indexing::at(getIndex_(tid[0], shape[0]),
                                               getIndex_(tid[1], shape[1]),
                                               getIndex_(tid[2], shape[2]),
                                               stride)];
                }
            } else {
                const int3_t tid = peak + offset;
                if (all(0 <= tid && tid < shape))
                    value = input[indexing::at(tid, stride)];
            }
        }
        return value;
    }

    template<bool IS_CENTERED, typename T>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void singlePeak1D_(const T* __restrict__ input, uint stride, int shape,
                       int peak, float* __restrict__ coordinates) {
        using namespace cuda::util;
        const auto tidx = static_cast<int>(threadIdx.x);
        const int offset = tidx - 1;

        __shared__ T square[BLOCK_SIZE];
        square[tidx] = fetchPeack1D_<IS_CENTERED>(input, stride, shape, peak, tidx, offset);
        block::synchronize();

        if (tidx == 0) {
            float refined_peak = IS_CENTERED ? peak : math::FFTShift(peak, shape);
            refined_peak += getParabolicVertex_(square[0], square[1], square[2]);
            *coordinates = refined_peak;
        }
    }

    template<bool IS_CENTERED, typename T>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void singlePeak1DBatched_(const T* __restrict__ input, uint batch_stride,
                              uint stride, int shape,
                              const uint32_t* __restrict__ peaks,
                              float* __restrict__ coordinates) {
        using namespace cuda::util;
        const uint batch = blockIdx.x;
        const auto tidx = static_cast<int>(threadIdx.x);
        const int offset = tidx - 1;

        const auto peak = static_cast<int>(peaks[batch] / stride);
        input += batch_stride * batch;

        __shared__ T square[BLOCK_SIZE];
        square[tidx] = fetchPeack1D_<IS_CENTERED>(input, stride, shape, peak, tidx, offset);
        block::synchronize();

        if (tidx == 0) {
            float refined_peak = IS_CENTERED ? peak : math::FFTShift(peak, shape);
            refined_peak += getParabolicVertex_(square[0], square[1], square[2]);
            coordinates[batch] = refined_peak;
        }
    }

    template<bool IS_CENTERED, typename T>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void singlePeak2D_(const T* __restrict__ input, uint2_t stride, int2_t shape,
                       int2_t peak, float2_t* __restrict__ coordinates) {
        using namespace cuda::util;
        const auto tidx = static_cast<int>(threadIdx.x);
        const int2_t offset = indexing::indexes(tidx, 3) - 1;

        __shared__ T square[BLOCK_SIZE];
        square[tidx] = fetchPeack2D_<IS_CENTERED>(input, stride, shape, peak, tidx, offset);
        block::synchronize();

        if (tidx == 0) {
            float refined_peak0 = IS_CENTERED ? peak[0] : math::FFTShift(peak[0], shape[0]);
            float refined_peak1 = IS_CENTERED ? peak[1] : math::FFTShift(peak[1], shape[1]);
            const T peak_value = square[4];
            refined_peak0 += getParabolicVertex_(square[1], peak_value, square[7]);
            refined_peak1 += getParabolicVertex_(square[3], peak_value, square[5]);
            *coordinates = float2_t{refined_peak0, refined_peak1};
        }
    }

    template<bool IS_CENTERED, typename T>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void singlePeak2DBatched_(const T* __restrict__ input, uint batch_stride,
                              uint2_t stride, int2_t shape, uint pitch_x,
                              const uint32_t* __restrict__ peak_offsets,
                              float2_t* __restrict__ coordinates) {
        using namespace cuda::util;
        const uint batch = blockIdx.x;
        const auto tidx = static_cast<int>(threadIdx.x);
        const int2_t offset = indexing::indexes(tidx, 3) - 1;

        const uint32_t peak_offset = peak_offsets[batch];
        const int2_t peak{indexing::indexes(peak_offset, pitch_x)};
        input += batch_stride * batch;

        __shared__ T square[BLOCK_SIZE];
        square[tidx] = fetchPeack2D_<IS_CENTERED>(input, stride, shape, peak, tidx, offset);
        block::synchronize();

        if (tidx == 0) {
            float refined_peak0 = IS_CENTERED ? peak[0] : math::FFTShift(peak[0], shape[0]);
            float refined_peak1 = IS_CENTERED ? peak[1] : math::FFTShift(peak[1], shape[1]);
            const T peak_value = square[4];
            refined_peak0 += getParabolicVertex_(square[1], peak_value, square[7]);
            refined_peak1 += getParabolicVertex_(square[3], peak_value, square[5]);
            coordinates[batch] = float2_t{refined_peak0, refined_peak1};
        }
    }

    template<bool IS_CENTERED, typename T>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void singlePeak3D_(const T* __restrict__ input, uint3_t stride, int3_t shape,
                       int3_t peak, float3_t* __restrict__ coordinates) {
        using namespace cuda::util;
        const auto tidx = static_cast<int>(threadIdx.x);
        const int3_t offset = indexing::indexes(tidx, 3, 3) - 1;

        __shared__ T square[BLOCK_SIZE];
        square[tidx] = fetchPeack3D_<IS_CENTERED>(input, stride, shape, peak, tidx, offset);
        block::synchronize();

        if (tidx == 0) {
            float refined_peak0 = IS_CENTERED ? peak[0] : math::FFTShift(peak[0], shape[0]);
            float refined_peak1 = IS_CENTERED ? peak[1] : math::FFTShift(peak[1], shape[1]);
            float refined_peak2 = IS_CENTERED ? peak[2] : math::FFTShift(peak[2], shape[2]);
            const T peak_value = square[13];
            refined_peak0 += getParabolicVertex_(square[4], peak_value, square[22]);
            refined_peak1 += getParabolicVertex_(square[10], peak_value, square[16]);
            refined_peak2 += getParabolicVertex_(square[12], peak_value, square[14]);
            *coordinates = float3_t{refined_peak0, refined_peak1, refined_peak2};
        }
    }

    template<bool IS_CENTERED, typename T>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void singlePeak3DBatched_(const T* __restrict__ input, uint batch_stride,
                              uint3_t stride, int3_t shape, uint pitch_y, uint pitch_x,
                              const uint32_t* __restrict__ peak_offsets,
                              float3_t* __restrict__ coordinates) {
        using namespace cuda::util;
        const uint batch = blockIdx.x;
        const auto tidx = static_cast<int>(threadIdx.x);
        const int3_t offset = indexing::indexes(tidx, 3, 3) - 1;

        const uint32_t peak_offset = peak_offsets[batch];
        const int3_t peak{indexing::indexes(peak_offset, pitch_y, pitch_x)};
        input += batch_stride * batch;

        __shared__ T square[BLOCK_SIZE];
        square[tidx] = fetchPeack3D_<IS_CENTERED>(input, stride, shape, peak, tidx, offset);
        block::synchronize();

        if (tidx == 0) {
            float refined_peak0 = IS_CENTERED ? peak[0] : math::FFTShift(peak[0], shape[0]);
            float refined_peak1 = IS_CENTERED ? peak[1] : math::FFTShift(peak[1], shape[1]);
            float refined_peak2 = IS_CENTERED ? peak[2] : math::FFTShift(peak[2], shape[2]);
            const T peak_value = square[13];
            refined_peak0 += getParabolicVertex_(square[4], peak_value, square[22]);
            refined_peak1 += getParabolicVertex_(square[10], peak_value, square[16]);
            refined_peak2 += getParabolicVertex_(square[12], peak_value, square[14]);
            coordinates[batch] = float3_t{refined_peak0, refined_peak1, refined_peak2};
        }
    }
}

namespace noa::cuda::signal::fft {
    template<Remap REMAP, typename T, typename U>
    void xmap(const shared_t<Complex<T>[]>& lhs, size4_t lhs_stride,
              const shared_t<Complex<T>[]>& rhs, size4_t rhs_stride,
              const shared_t<T[]>& output, size4_t output_stride,
              size4_t shape, bool normalize, Norm norm, Stream& stream,
              const shared_t<Complex<T>[]>& tmp, size4_t tmp_stride) {

        const shared_t<Complex<T>[]>& buffer = tmp ? tmp : rhs;
        const size4_t& buffer_stride = tmp ? tmp_stride : rhs_stride;
        NOA_ASSERT(all(buffer_stride > 0));

        if (normalize) {
            cuda::util::ewise::binary(
                    "signal::fft::xmap",
                    lhs.get(), lhs_stride, rhs.get(), rhs_stride, buffer.get(), buffer_stride, shape.fft(), stream,
                    []__device__(Complex<T> l, Complex<T> r) {
                        const Complex<T> product = l * noa::math::conj(r);
                        const T magnitude = noa::math::abs(product);
                        return product / (magnitude + static_cast<T>(1e-13));
                        // The epsilon could be scaled by the max(abs(rhs)), but this seems to be useful only
                        // for input values close to zero (less than 1e-10). In most cases, this is fine.
                        // Note that the normalization can sharpen the peak considerably.
                    });
        } else {
            cuda::math::ewise(lhs, lhs_stride, rhs, rhs_stride, buffer, buffer_stride,
                              shape.fft(), noa::math::multiply_conj_t{}, stream);
        }

        if constexpr (REMAP == Remap::H2FC) {
            const size3_t shape_3d{shape.get() + 1};
            if (shape_3d.ndim() == 3) {
                cuda::signal::fft::shift3D<Remap::H2H>(buffer, buffer_stride, buffer, buffer_stride, shape,
                                                       float3_t{shape_3d / 2}, 1, stream);
            } else {
                cuda::signal::fft::shift2D<Remap::H2H>(buffer, buffer_stride, buffer, buffer_stride, shape,
                                                       float2_t{shape_3d[1] / 2, shape_3d[2] / 2}, 1, stream);
            }
        }

        cuda::fft::c2r(buffer, buffer_stride, output, output_stride, shape, norm, stream);
    }

    template<Remap REMAP, typename T, typename>
    void xpeak1D(const shared_t<T[]>& map, size4_t stride, size4_t shape,
                 const shared_t<float[]>& coordinates, Stream& stream) {
        NOA_ASSERT(shape[1] == 1);
        NOA_ASSERT(stride[2] > 0);
        cuda::memory::PtrDevice<uint32_t> offsets{shape[0], stream};
        cuda::math::find(noa::math::max_t{}, map, stride, shape, offsets.share(), true, stream);

        float* coordinates_ptr = util::devicePointer(coordinates.get(), stream.device());
        memory::PtrDevice<float> buffer;
        if (!coordinates_ptr) {
            buffer = memory::PtrDevice<float>{shape[0], stream};
            coordinates_ptr = buffer.get();
        }
        constexpr bool IS_CENTERED = static_cast<std::underlying_type_t<Remap>>(REMAP) & noa::fft::Layout::DST_CENTERED;
        stream.enqueue("signal::fft::xpeak1D", singlePeak1DBatched_<IS_CENTERED, T>, LaunchConfig{shape[0], BLOCK_SIZE},
                       map.get(), stride[0], stride[3], shape[3], offsets.get(), coordinates_ptr);

        if (!buffer.empty())
            memory::copy(coordinates_ptr, coordinates.get(), shape[0], stream);
        stream.attach(map, coordinates);
    }

    template<Remap REMAP, typename T, typename>
    float xpeak1D(const shared_t<T[]>& xmap, size4_t stride, size4_t shape, Stream& stream) {
        NOA_ASSERT(shape.ndim() == 1);
        NOA_ASSERT(stride[3] > 0);
        const auto peak_offset = cuda::math::find<uint32_t>(noa::math::max_t{}, xmap, stride, shape, stream);
        const uint32_t peak_index = peak_offset / static_cast<uint32_t>(stride[3]);

        cuda::memory::PtrPinned<float> coordinate{1};
        constexpr bool IS_CENTERED = static_cast<std::underlying_type_t<Remap>>(REMAP) & noa::fft::Layout::DST_CENTERED;
        stream.enqueue("signal::fft::xpeak1D", singlePeak1D_<IS_CENTERED, T>, LaunchConfig{1, BLOCK_SIZE},
                       xmap.get(), stride[3], shape[3], peak_index, coordinate.get());
        stream.synchronize();
        return coordinate[0];
    }

    template<Remap REMAP, typename T, typename>
    void xpeak2D(const shared_t<T[]>& map, size4_t stride, size4_t shape,
                 const shared_t<float2_t[]>& coordinates, Stream& stream) {
        NOA_ASSERT(shape[1] == 1);
        NOA_ASSERT(stride[2] > 0);
        cuda::memory::PtrDevice<uint32_t> offsets{shape[0], stream};
        cuda::math::find(noa::math::max_t{}, map, stride, shape, offsets.share(), true, stream);

        float2_t* coordinates_ptr = util::devicePointer(coordinates.get(), stream.device());
        memory::PtrDevice<float2_t> buffer;
        if (!coordinates_ptr) {
            buffer = memory::PtrDevice<float2_t>{shape[0], stream};
            coordinates_ptr = buffer.get();
        }
        constexpr bool IS_CENTERED = static_cast<std::underlying_type_t<Remap>>(REMAP) & noa::fft::Layout::DST_CENTERED;
        stream.enqueue("signal::fft::xpeak2D", singlePeak2DBatched_<IS_CENTERED, T>, LaunchConfig{shape[0], BLOCK_SIZE},
                       map.get(), stride[0], uint2_t{stride.get() + 2}, int2_t{shape.get() + 2},
                       stride[2], offsets.get(), coordinates_ptr);

        if (!buffer.empty())
            memory::copy(coordinates_ptr, coordinates.get(), shape[0], stream);
        stream.attach(map, coordinates);
    }

    template<Remap REMAP, typename T, typename>
    float2_t xpeak2D(const shared_t<T[]>& xmap, size4_t stride, size4_t shape, Stream& stream) {
        NOA_ASSERT(shape.ndim() == 2);
        NOA_ASSERT(stride[2] > 0);
        const auto peak_offset = cuda::math::find<uint32_t>(noa::math::max_t{}, xmap, stride, shape, stream);
        const uint2_t peak_index = indexing::indexes(peak_offset, static_cast<uint32_t>(stride[2]));

        cuda::memory::PtrPinned<float2_t> coordinate{1};
        constexpr bool IS_CENTERED = static_cast<std::underlying_type_t<Remap>>(REMAP) & noa::fft::Layout::DST_CENTERED;
        stream.enqueue("signal::fft::xpeak2D", singlePeak2D_<IS_CENTERED, T>, LaunchConfig{1, BLOCK_SIZE},
                       xmap.get(), uint2_t{stride.get() + 2}, int2_t{shape.get() + 2},
                       int2_t{peak_index}, coordinate.get());
        stream.synchronize();
        return coordinate[0];
    }

    template<Remap REMAP, typename T, typename>
    void xpeak3D(const shared_t<T[]>& map, size4_t stride, size4_t shape,
                 const shared_t<float3_t[]>& coordinates, Stream& stream) {
        NOA_ASSERT(stride[1] > 0 && stride[2] > 0);
        cuda::memory::PtrPinned<uint32_t> offsets{shape[0]};
        cuda::math::find(noa::math::max_t{}, map, stride, shape, offsets.share(), true, stream);

        float3_t* coordinates_ptr = util::devicePointer(coordinates.get(), stream.device());
        memory::PtrDevice<float3_t> buffer;
        if (!coordinates_ptr) {
            buffer = memory::PtrDevice<float3_t>{shape[0], stream};
            coordinates_ptr = buffer.get();
        }
        constexpr bool IS_CENTERED = static_cast<std::underlying_type_t<Remap>>(REMAP) & noa::fft::Layout::DST_CENTERED;
        stream.enqueue("signal::fft::xpeak3D", singlePeak3DBatched_<IS_CENTERED, T>, LaunchConfig{shape[0], BLOCK_SIZE},
                       map.get(), stride[0], uint3_t{stride.get() + 1}, int3_t{shape.get() + 1},
                       stride[1] / stride[2], stride[2], offsets.get(), coordinates_ptr);

        if (!buffer.empty())
            memory::copy(coordinates_ptr, coordinates.get(), shape[0], stream);
        stream.attach(map, offsets.share(), coordinates);
    }

    template<Remap REMAP, typename T, typename>
    float3_t xpeak3D(const shared_t<T[]>& xmap, size4_t stride, size4_t shape, Stream& stream) {
        NOA_ASSERT(shape.ndim() == 3);
        NOA_ASSERT(stride[1] > 0 && stride[2] > 0);
        const auto peak_offset = cuda::math::find<uint32_t>(noa::math::max_t{}, xmap, stride, shape, stream);
        const uint3_t peak_index = indexing::indexes(peak_offset,
                                                     static_cast<uint32_t>(stride[1] / stride[2]),
                                                     static_cast<uint32_t>(stride[2]));

        cuda::memory::PtrPinned<float3_t> coordinate{1};
        constexpr bool IS_CENTERED = static_cast<std::underlying_type_t<Remap>>(REMAP) & noa::fft::Layout::DST_CENTERED;
        stream.enqueue("signal::fft::xpeak3D", singlePeak3D_<IS_CENTERED, T>, LaunchConfig{1, BLOCK_SIZE},
                       xmap.get(), uint3_t{stride.get() + 1}, int3_t{shape.get() + 1},
                       int3_t{peak_index}, coordinate.get());
        stream.synchronize();
        return coordinate[0];
    }

    #define INSTANTIATE_XMAP(T) \
    template void xmap<Remap::H2F, T, void>(const shared_t<Complex<T>[]>&, size4_t, const shared_t<Complex<T>[]>&, size4_t, const shared_t<T[]>&, size4_t, size4_t, bool, Norm, Stream&, const shared_t<Complex<T>[]>&, size4_t);   \
    template void xmap<Remap::H2FC, T, void>(const shared_t<Complex<T>[]>&, size4_t, const shared_t<Complex<T>[]>&, size4_t, const shared_t<T[]>&, size4_t, size4_t, bool, Norm, Stream&, const shared_t<Complex<T>[]>&, size4_t);  \
    template void xpeak1D<Remap::F2F, T, void>(const shared_t<T[]>&, size4_t, size4_t, const shared_t<float[]>&, Stream&);      \
    template void xpeak1D<Remap::FC2FC, T, void>(const shared_t<T[]>&, size4_t, size4_t, const shared_t<float[]>&, Stream&);    \
    template void xpeak2D<Remap::F2F, T, void>(const shared_t<T[]>&, size4_t, size4_t, const shared_t<float2_t[]>&, Stream&);   \
    template void xpeak2D<Remap::FC2FC, T, void>(const shared_t<T[]>&, size4_t, size4_t, const shared_t<float2_t[]>&, Stream&); \
    template void xpeak3D<Remap::F2F, T, void>(const shared_t<T[]>&, size4_t, size4_t, const shared_t<float3_t[]>&, Stream&);   \
    template void xpeak3D<Remap::FC2FC, T, void>(const shared_t<T[]>&, size4_t, size4_t, const shared_t<float3_t[]>&, Stream&); \
    template float xpeak1D<Remap::F2F, T, void>(const shared_t<T[]>&, size4_t, size4_t, Stream&);       \
    template float xpeak1D<Remap::FC2FC, T, void>(const shared_t<T[]>&, size4_t, size4_t, Stream&);     \
    template float2_t xpeak2D<Remap::F2F, T, void>(const shared_t<T[]>&, size4_t, size4_t, Stream&);    \
    template float2_t xpeak2D<Remap::FC2FC, T, void>(const shared_t<T[]>&, size4_t, size4_t, Stream&);  \
    template float3_t xpeak3D<Remap::F2F, T, void>(const shared_t<T[]>&, size4_t, size4_t, Stream&);    \
    template float3_t xpeak3D<Remap::FC2FC, T, void>(const shared_t<T[]>&, size4_t, size4_t, Stream&)

    INSTANTIATE_XMAP(float);
    INSTANTIATE_XMAP(double);
}

namespace noa::cuda::signal::fft::details {
    template<typename T>
    void xcorr(const shared_t<Complex<T>[]>& lhs, size4_t lhs_stride,
               const shared_t<Complex<T>[]>& rhs, size4_t rhs_stride,
               size4_t shape, const shared_t<T[]>& coefficients,
               Stream& stream, bool is_half) {
        const size_t batches = shape[0];
        const size4_t shape_fft = is_half ? shape.fft() : shape;
        const size4_t stride_fft = shape_fft.stride();
        const size4_t reduced_shape{batches, 1, 1, 1};
        const size4_t reduced_stride = reduced_shape.stride();

        cuda::memory::PtrPinned<T> buffer{batches * 3};
        auto denominator_lhs = buffer.get() + batches;
        auto denominator_rhs = buffer.get() + batches * 2;

        T* null{};
        cuda::util::reduce<false>(
                "signal::fft::xcorr", lhs.get(), uint4_t{lhs_stride}, uint4_t{shape_fft},
                noa::math::abs_squared_t{}, noa::math::plus_t{}, T{0},
                denominator_lhs, 1, noa::math::copy_t{},
                null, 0, noa::math::copy_t{},
                stream);
        cuda::util::reduce<false>(
                "signal::fft::xcorr", rhs.get(), uint4_t{rhs_stride}, uint4_t{shape_fft},
                noa::math::abs_squared_t{}, noa::math::plus_t{}, T{0},
                denominator_rhs, 1, noa::math::copy_t{},
                null, 0, noa::math::copy_t{},
                stream);

        auto combine_op = []__device__(Complex<T> l, Complex<T> r) { return noa::math::real(l * r); };
        cuda::util::reduce<false, false>(
                "signal::fft::xcorr",
                lhs.get(), uint4_t{lhs_stride}, rhs.get(), uint4_t{rhs_stride}, uint4_t{shape_fft},
                noa::math::copy_t{}, noa::math::conj_t{}, combine_op, noa::math::plus_t{}, T{0},
                buffer.get(), 1, noa::math::copy_t{}, null, 1, noa::math::copy_t{}, stream);

        stream.synchronize(); // FIXME Add callback
        for (size_t batch = 0; batch < batches; ++batch) {
            coefficients.get()[batch] =
                    buffer[batch] / noa::math::sqrt(denominator_lhs[batch] * denominator_rhs[batch]);
        }
    }

    template<typename T>
    T xcorr(const shared_t<Complex<T>[]>& lhs, size4_t lhs_stride,
            const shared_t<Complex<T>[]>& rhs, size4_t rhs_stride,
            size4_t shape, Stream& stream, bool is_half) {
        NOA_ASSERT(shape[0] == 1);
        const size4_t shape_fft = is_half ? shape.fft() : shape;
        const size4_t stride_fft = shape_fft.stride();

        T numerator{}, denominator_lhs{}, denominator_rhs{};
        T* null{};
        cuda::util::reduce<true>(
                "signal::fft::xcorr", lhs.get(), uint4_t{lhs_stride}, uint4_t{shape_fft},
                noa::math::abs_squared_t{}, noa::math::plus_t{}, T{0},
                &denominator_lhs, 1, noa::math::copy_t{},
                null, 0, noa::math::copy_t{},
                stream);
        cuda::util::reduce<true>(
                "signal::fft::xcorr", rhs.get(), uint4_t{rhs_stride}, uint4_t{shape_fft},
                noa::math::abs_squared_t{}, noa::math::plus_t{}, T{0},
                &denominator_rhs, 1, noa::math::copy_t{},
                null, 0, noa::math::copy_t{},
                stream);

        auto combine_op = []__device__(Complex<T> l, Complex<T> r) { return noa::math::real(l * r); };
        cuda::util::reduce<false, false>(
                "signal::fft::xcorr",
                lhs.get(), uint4_t{lhs_stride}, rhs.get(), uint4_t{rhs_stride}, uint4_t{shape_fft},
                noa::math::copy_t{}, noa::math::conj_t{}, combine_op, noa::math::plus_t{}, T{0},
                &numerator, 1, noa::math::copy_t{}, null, 1, noa::math::copy_t{}, stream);

        stream.synchronize();
        const T denominator = noa::math::sqrt(denominator_lhs * denominator_rhs);
        return numerator / denominator;
    }

    #define INSTANTIATE_XCORR(T) \
    template void xcorr<T>(const shared_t<Complex<T>[]>&, size4_t, const shared_t<Complex<T>[]>&, size4_t, size4_t, const shared_t<T[]>&, Stream&, bool); \
    template T xcorr<T>(const shared_t<Complex<T>[]>&, size4_t, const shared_t<Complex<T>[]>&, size4_t, size4_t, Stream&, bool)

    INSTANTIATE_XCORR(float);
    INSTANTIATE_XCORR(double);
}
