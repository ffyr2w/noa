#include "hip/hip_runtime.h"
#include "noa/gpu/cuda/fft/Transforms.h"
#include "noa/gpu/cuda/math/Ewise.h"
#include "noa/gpu/cuda/math/Find.h"
#include "noa/gpu/cuda/math/Reduce.h"
#include "noa/gpu/cuda/memory/PtrPinned.h"
#include "noa/gpu/cuda/memory/PtrDevice.h"
#include "noa/gpu/cuda/signal/fft/Correlate.h"
#include "noa/gpu/cuda/signal/fft/Shift.h"

#include "noa/gpu/cuda/util/EwiseBinary.cuh"
#include "noa/gpu/cuda/util/ReduceUnary.cuh"
#include "noa/gpu/cuda/util/ReduceBinary.cuh"
#include "noa/gpu/cuda/util/Warp.cuh"

namespace {
    using namespace ::noa;
    constexpr uint BLOCK_SIZE = cuda::Limits::WARP_SIZE;

    // From the DC-centered frequency to a valid index in the non-centered output.
    // The input frequency should be in-bound, i.e. -n/2 <= frequency <= (n-1)/2
    constexpr NOA_FD int64_t getIndex_(int64_t frequency, int64_t volume_dim) {
        return frequency < 0 ? volume_dim + frequency : frequency;
    }

    // From a valid index to the DC-centered frequency.
    constexpr NOA_FD int3_t getFrequency_(int3_t index, int3_t shape) {
        return {index[0] < (shape[0] + 1) / 2 ? index[0] : index[0] - shape[0],
                index[1] < (shape[1] + 1) / 2 ? index[1] : index[1] - shape[1],
                index[2] < (shape[2] + 1) / 2 ? index[2] : index[2] - shape[2]};
    }

    // From a valid index to the DC-centered frequency.
    constexpr NOA_FD int2_t getFrequency_(int2_t index, int2_t shape) {
        return {index[0] < (shape[0] + 1) / 2 ? index[0] : index[0] - shape[0],
                index[1] < (shape[1] + 1) / 2 ? index[1] : index[1] - shape[1]};
    }

    constexpr NOA_FD int getFrequency_(int index, int shape) {
        return index < (shape + 1) / 2 ? index : index - shape;
    }

    // Given values at three successive positions, y[0], y[1], y[2], where
    // y[1] is the peak value, this fits a parabola to the values and returns the
    // offset (from -0.5 to 0.5) from the center position.
    template<typename T>
    constexpr NOA_FD T getParabolicVertex_(T y0, T y1, T y2) noexcept {
        const T d = 2 * (y0 + y2 - 2 * y1);
        T x = 0;
        // From IMOD/libcfshr/filtxcorr.c::parabolicFitPosition
        if (math::abs(d) > math::abs(static_cast<T>(1e-2) * (y0 - y2)))
            x = (y0 - y2) / d;
        if (x > T{0.5})
            x = T{0.5};
        if (x < T{-0.5})
            x = T{-0.5};
        return x;
    }

    template<bool IS_CENTERED, typename T>
    constexpr NOA_FD T fetchPeack1D_(const T* input, uint stride, int shape, int peak, int tidx, int offset) {
        T value = 0;
        if (tidx < 3) {
            if constexpr (!IS_CENTERED) {
                const int tid = getFrequency_(peak, shape) + offset;
                if (-shape / 2 <= tid && tid <= (shape - 1) / 2) {
                    value = input[getIndex_(tid, shape) * stride];
                }
            } else {
                const int tid = peak + offset;
                if (0 <= tid && tid < shape)
                    value = input[tid * stride];
            }
        }
        return value;
    }

    // Fetch the 3x3 window around the peak
    // No coalescing here I'm afraid.
    template<bool IS_CENTERED, typename T>
    constexpr NOA_FD T fetchPeack2D_(const T* input, uint2_t strides, int2_t shape, int2_t peak,
                                     int tidx, int2_t offset) {
        T value = 0;
        if (tidx < 9) {
            if constexpr (!IS_CENTERED) {
                const int2_t tid = getFrequency_(peak, shape) + offset;
                if (all(-shape / 2 <= tid && tid <= (shape - 1) / 2)) {
                    value = input[indexing::at(getIndex_(tid[0], shape[0]),
                                               getIndex_(tid[1], shape[1]),
                                               strides)];
                }
            } else {
                const int2_t tid = peak + offset;
                if (all(0 <= tid && tid < shape))
                    value = input[indexing::at(tid, strides)];
            }
        }
        return value;
    }

    // Fetch the 3x3x3 window around the peak
    // No coalescing here I'm afraid.
    template<bool IS_CENTERED, typename T>
    constexpr NOA_FD T fetchPeack3D_(const T* input, uint3_t strides, int3_t shape, int3_t peak,
                                     int tidx, int3_t offset) {
        T value = 0;
        if (tidx < 27) {
            if constexpr (!IS_CENTERED) {
                const int3_t tid = getFrequency_(peak, shape) + offset;
                if (all(-shape / 2 <= tid && tid <= (shape - 1) / 2)) {
                    value = input[indexing::at(getIndex_(tid[0], shape[0]),
                                               getIndex_(tid[1], shape[1]),
                                               getIndex_(tid[2], shape[2]),
                                               strides)];
                }
            } else {
                const int3_t tid = peak + offset;
                if (all(0 <= tid && tid < shape))
                    value = input[indexing::at(tid, strides)];
            }
        }
        return value;
    }

    template<bool IS_CENTERED, typename T>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void singlePeak1D_(const T* __restrict__ input, uint stride, int shape,
                       int peak, float* __restrict__ coordinates) {
        using namespace cuda::util;
        const auto tidx = static_cast<int>(threadIdx.x);
        const int offset = tidx - 1;

        __shared__ T square[BLOCK_SIZE];
        square[tidx] = fetchPeack1D_<IS_CENTERED>(input, stride, shape, peak, tidx, offset);
        block::synchronize();

        if (tidx == 0) {
            float refined_peak = IS_CENTERED ? peak : math::FFTShift(peak, shape);
            refined_peak += getParabolicVertex_(square[0], square[1], square[2]);
            *coordinates = refined_peak;
        }
    }

    template<bool IS_CENTERED, typename T>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void singlePeak1DBatched_(const T* __restrict__ input, uint batch_stride,
                              uint stride, int shape,
                              const uint32_t* __restrict__ peaks,
                              float* __restrict__ coordinates) {
        using namespace cuda::util;
        const uint batch = blockIdx.x;
        const auto tidx = static_cast<int>(threadIdx.x);
        const int offset = tidx - 1;

        const auto peak = static_cast<int>(peaks[batch] / stride);
        input += batch_stride * batch;

        __shared__ T square[BLOCK_SIZE];
        square[tidx] = fetchPeack1D_<IS_CENTERED>(input, stride, shape, peak, tidx, offset);
        block::synchronize();

        if (tidx == 0) {
            float refined_peak = IS_CENTERED ? peak : math::FFTShift(peak, shape);
            refined_peak += getParabolicVertex_(square[0], square[1], square[2]);
            coordinates[batch] = refined_peak;
        }
    }

    template<bool IS_CENTERED, typename T>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void singlePeak2D_(const T* __restrict__ input, uint2_t strides, int2_t shape,
                       int2_t peak, float2_t* __restrict__ coordinates) {
        using namespace cuda::util;
        const auto tidx = static_cast<int>(threadIdx.x);
        const int2_t offset = indexing::indexes(tidx, 3) - 1;

        __shared__ T square[BLOCK_SIZE];
        square[tidx] = fetchPeack2D_<IS_CENTERED>(input, strides, shape, peak, tidx, offset);
        block::synchronize();

        if (tidx == 0) {
            float refined_peak0 = IS_CENTERED ? peak[0] : math::FFTShift(peak[0], shape[0]);
            float refined_peak1 = IS_CENTERED ? peak[1] : math::FFTShift(peak[1], shape[1]);
            const T peak_value = square[4];
            refined_peak0 += getParabolicVertex_(square[1], peak_value, square[7]);
            refined_peak1 += getParabolicVertex_(square[3], peak_value, square[5]);
            *coordinates = float2_t{refined_peak0, refined_peak1};
        }
    }

    template<bool IS_CENTERED, typename T>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void singlePeak2DBatched_(const T* __restrict__ input, uint batch_stride,
                              uint2_t strides, int2_t shape,
                              const uint32_t* __restrict__ peak_offsets,
                              float2_t* __restrict__ coordinates) {
        using namespace cuda::util;
        const uint batch = blockIdx.x;
        const auto tidx = static_cast<int>(threadIdx.x);
        const int2_t offset = indexing::indexes(tidx, 3) - 1;

        const uint32_t peak_offset = peak_offsets[batch];
        const int2_t peak(indexing::indexes(peak_offset, strides, uint2_t(shape)));
        input += batch_stride * batch;

        __shared__ T square[BLOCK_SIZE];
        square[tidx] = fetchPeack2D_<IS_CENTERED>(input, strides, shape, peak, tidx, offset);
        block::synchronize();

        if (tidx == 0) {
            float refined_peak0 = IS_CENTERED ? peak[0] : math::FFTShift(peak[0], shape[0]);
            float refined_peak1 = IS_CENTERED ? peak[1] : math::FFTShift(peak[1], shape[1]);
            const T peak_value = square[4];
            refined_peak0 += getParabolicVertex_(square[1], peak_value, square[7]);
            refined_peak1 += getParabolicVertex_(square[3], peak_value, square[5]);
            coordinates[batch] = float2_t{refined_peak0, refined_peak1};
        }
    }

    template<bool IS_CENTERED, typename T>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void singlePeak3D_(const T* __restrict__ input, uint3_t strides, int3_t shape,
                       int3_t peak, float3_t* __restrict__ coordinates) {
        using namespace cuda::util;
        const auto tidx = static_cast<int>(threadIdx.x);
        const int3_t offset = indexing::indexes(tidx, 3, 3) - 1;

        __shared__ T square[BLOCK_SIZE];
        square[tidx] = fetchPeack3D_<IS_CENTERED>(input, strides, shape, peak, tidx, offset);
        block::synchronize();

        if (tidx == 0) {
            float refined_peak0 = IS_CENTERED ? peak[0] : math::FFTShift(peak[0], shape[0]);
            float refined_peak1 = IS_CENTERED ? peak[1] : math::FFTShift(peak[1], shape[1]);
            float refined_peak2 = IS_CENTERED ? peak[2] : math::FFTShift(peak[2], shape[2]);
            const T peak_value = square[13];
            refined_peak0 += getParabolicVertex_(square[4], peak_value, square[22]);
            refined_peak1 += getParabolicVertex_(square[10], peak_value, square[16]);
            refined_peak2 += getParabolicVertex_(square[12], peak_value, square[14]);
            *coordinates = float3_t{refined_peak0, refined_peak1, refined_peak2};
        }
    }

    template<bool IS_CENTERED, typename T>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void singlePeak3DBatched_(const T* __restrict__ input, uint batch_stride,
                              uint3_t strides, int3_t shape,
                              const uint32_t* __restrict__ peak_offsets,
                              float3_t* __restrict__ coordinates) {
        using namespace cuda::util;
        const uint batch = blockIdx.x;
        const auto tidx = static_cast<int>(threadIdx.x);
        const int3_t offset = indexing::indexes(tidx, 3, 3) - 1;

        const uint32_t peak_offset = peak_offsets[batch];
        const int3_t peak(indexing::indexes(peak_offset, strides, uint3_t(shape)));
        input += batch_stride * batch;

        __shared__ T square[BLOCK_SIZE];
        square[tidx] = fetchPeack3D_<IS_CENTERED>(input, strides, shape, peak, tidx, offset);
        block::synchronize();

        if (tidx == 0) {
            float refined_peak0 = IS_CENTERED ? peak[0] : math::FFTShift(peak[0], shape[0]);
            float refined_peak1 = IS_CENTERED ? peak[1] : math::FFTShift(peak[1], shape[1]);
            float refined_peak2 = IS_CENTERED ? peak[2] : math::FFTShift(peak[2], shape[2]);
            const T peak_value = square[13];
            refined_peak0 += getParabolicVertex_(square[4], peak_value, square[22]);
            refined_peak1 += getParabolicVertex_(square[10], peak_value, square[16]);
            refined_peak2 += getParabolicVertex_(square[12], peak_value, square[14]);
            coordinates[batch] = float3_t{refined_peak0, refined_peak1, refined_peak2};
        }
    }
}

namespace noa::cuda::signal::fft {
    template<Remap REMAP, typename T, typename U>
    void xmap(const shared_t<Complex<T>[]>& lhs, size4_t lhs_strides,
              const shared_t<Complex<T>[]>& rhs, size4_t rhs_strides,
              const shared_t<T[]>& output, size4_t output_strides,
              size4_t shape, bool normalize, Norm norm, Stream& stream,
              const shared_t<Complex<T>[]>& tmp, size4_t tmp_strides) {

        const shared_t<Complex<T>[]>& buffer = tmp ? tmp : rhs;
        const size4_t& buffer_strides = tmp ? tmp_strides : rhs_strides;
        NOA_ASSERT(all(buffer_strides > 0));

        if (normalize) {
            cuda::util::ewise::binary(
                    "signal::fft::xmap",
                    lhs.get(), lhs_strides, rhs.get(), rhs_strides,
                    buffer.get(), buffer_strides,
                    shape.fft(), true, stream,
                    []__device__(Complex<T> l, Complex<T> r) {
                        const Complex<T> product = l * noa::math::conj(r);
                        const T magnitude = noa::math::abs(product);
                        return product / (magnitude + static_cast<T>(1e-13));
                        // The epsilon could be scaled by the max(abs(rhs)), but this seems to be useful only
                        // for input values close to zero (less than 1e-10). In most cases, this is fine.
                        // Note that the normalization can sharpen the peak considerably.
                    });
        } else {
            cuda::math::ewise(lhs, lhs_strides, rhs, rhs_strides, buffer, buffer_strides,
                              shape.fft(), noa::math::multiply_conj_t{}, stream);
        }

        if constexpr (REMAP == Remap::H2FC) {
            const size3_t shape_3d(shape.get(1));
            if (shape_3d.ndim() == 3) {
                cuda::signal::fft::shift3D<Remap::H2H>(buffer, buffer_strides, buffer, buffer_strides, shape,
                                                       float3_t(shape_3d / 2), 1, stream);
            } else {
                cuda::signal::fft::shift2D<Remap::H2H>(buffer, buffer_strides, buffer, buffer_strides, shape,
                                                       float2_t{shape_3d[1] / 2, shape_3d[2] / 2}, 1, stream);
            }
        }

        cuda::fft::c2r(buffer, buffer_strides, output, output_strides, shape, norm, stream);
    }

    template<Remap REMAP, typename T, typename>
    void xpeak1D(const shared_t<T[]>& map, size4_t strides, size4_t shape,
                 const shared_t<float[]>& coordinates, Stream& stream) {
        NOA_ASSERT(size3_t(shape.get(1)).ndim() == 1);
        cuda::memory::PtrDevice<uint32_t> offsets(shape[0], stream);
        cuda::math::find(noa::math::first_max_t{}, map, strides, shape, offsets.share(), true, true, stream);

        float* coordinates_ptr = util::devicePointer(coordinates.get(), stream.device());
        memory::PtrDevice<float> buffer;
        if (!coordinates_ptr) {
            buffer = memory::PtrDevice<float>(shape[0], stream);
            coordinates_ptr = buffer.get();
        }

        constexpr bool IS_CENTERED = static_cast<std::underlying_type_t<Remap>>(REMAP) & noa::fft::Layout::DST_CENTERED;
        const bool is_column = shape[3] == 1;
        NOA_ASSERT(strides[3 - is_column] > 0);
        stream.enqueue("signal::fft::xpeak1D", singlePeak1DBatched_<IS_CENTERED, T>, LaunchConfig{shape[0], BLOCK_SIZE},
                       map.get(), strides[0], strides[3 - is_column], shape[3 - is_column],
                       offsets.get(), coordinates_ptr);

        if (!buffer.empty())
            memory::copy(coordinates_ptr, coordinates.get(), shape[0], stream);
        stream.attach(map, coordinates);
    }

    template<Remap REMAP, typename T, typename>
    float xpeak1D(const shared_t<T[]>& xmap, size4_t strides, size4_t shape, Stream& stream) {
        const bool is_column = shape[3] == 1;
        NOA_ASSERT(strides[3 - is_column] > 0);
        NOA_ASSERT(shape.ndim() == 1);

        const auto peak_offset = cuda::math::find<uint32_t>(
                noa::math::first_max_t{}, xmap, strides, shape, true, stream);
        const uint32_t peak_index = peak_offset / static_cast<uint32_t>(strides[3 - is_column]);

        cuda::memory::PtrPinned<float> coordinate(1);
        constexpr bool IS_CENTERED = static_cast<std::underlying_type_t<Remap>>(REMAP) & noa::fft::Layout::DST_CENTERED;
        stream.enqueue("signal::fft::xpeak1D", singlePeak1D_<IS_CENTERED, T>, LaunchConfig{1, BLOCK_SIZE},
                       xmap.get(), strides[3 - is_column], shape[3 - is_column], peak_index, coordinate.get());
        stream.synchronize();
        return coordinate[0];
    }

    template<Remap REMAP, typename T, typename>
    void xpeak2D(const shared_t<T[]>& map, size4_t strides, size4_t shape,
                 const shared_t<float2_t[]>& coordinates, Stream& stream) {
        NOA_ASSERT(shape[1] == 1);
        cuda::memory::PtrDevice<uint32_t> offsets(shape[0], stream);
        cuda::math::find(noa::math::first_max_t{}, map, strides, shape, offsets.share(), true, true, stream);

        float2_t* coordinates_ptr = util::devicePointer(coordinates.get(), stream.device());
        memory::PtrDevice<float2_t> buffer;
        if (!coordinates_ptr) {
            buffer = memory::PtrDevice<float2_t>(shape[0], stream);
            coordinates_ptr = buffer.get();
        }
        constexpr bool IS_CENTERED = static_cast<std::underlying_type_t<Remap>>(REMAP) & noa::fft::Layout::DST_CENTERED;
        stream.enqueue("signal::fft::xpeak2D", singlePeak2DBatched_<IS_CENTERED, T>, LaunchConfig{shape[0], BLOCK_SIZE},
                       map.get(), strides[0], uint2_t(strides.get(2)), int2_t(shape.get(2)),
                       offsets.get(), coordinates_ptr);

        if (!buffer.empty())
            memory::copy(coordinates_ptr, coordinates.get(), shape[0], stream);
        stream.attach(map, coordinates);
    }

    template<Remap REMAP, typename T, typename>
    float2_t xpeak2D(const shared_t<T[]>& xmap, size4_t strides, size4_t shape, Stream& stream) {
        NOA_ASSERT(shape.ndim() == 2);
        const auto peak_offset = cuda::math::find<uint32_t>(noa::math::first_max_t{}, xmap, strides, shape, true, stream);
        const uint2_t shape_2d(shape.get(2));
        const uint2_t strides_2d(strides.get(2));
        const int2_t peak_index(indexing::indexes(peak_offset, strides_2d, shape_2d));

        cuda::memory::PtrPinned<float2_t> coordinate(1);
        constexpr bool IS_CENTERED = static_cast<std::underlying_type_t<Remap>>(REMAP) & noa::fft::Layout::DST_CENTERED;
        stream.enqueue("signal::fft::xpeak2D", singlePeak2D_<IS_CENTERED, T>, LaunchConfig{1, BLOCK_SIZE},
                       xmap.get(), strides_2d, int2_t(shape_2d),
                       peak_index, coordinate.get());
        stream.synchronize();
        return coordinate[0];
    }

    template<Remap REMAP, typename T, typename>
    void xpeak3D(const shared_t<T[]>& map, size4_t stride, size4_t shape,
                 const shared_t<float3_t[]>& coordinates, Stream& stream) {
        cuda::memory::PtrPinned<uint32_t> offsets(shape[0]);
        cuda::math::find(noa::math::first_max_t{}, map, stride, shape, offsets.share(), true, true, stream);

        float3_t* coordinates_ptr = util::devicePointer(coordinates.get(), stream.device());
        memory::PtrDevice<float3_t> buffer;
        if (!coordinates_ptr) {
            buffer = memory::PtrDevice<float3_t>(shape[0], stream);
            coordinates_ptr = buffer.get();
        }
        constexpr bool IS_CENTERED = static_cast<std::underlying_type_t<Remap>>(REMAP) & noa::fft::Layout::DST_CENTERED;
        stream.enqueue("signal::fft::xpeak3D", singlePeak3DBatched_<IS_CENTERED, T>, LaunchConfig{shape[0], BLOCK_SIZE},
                       map.get(), stride[0], uint3_t(stride.get(1)), int3_t(shape.get(1)),
                       offsets.get(), coordinates_ptr);

        if (!buffer.empty())
            memory::copy(coordinates_ptr, coordinates.get(), shape[0], stream);
        stream.attach(map, offsets.share(), coordinates);
    }

    template<Remap REMAP, typename T, typename>
    float3_t xpeak3D(const shared_t<T[]>& xmap, size4_t strides, size4_t shape, Stream& stream) {
        NOA_ASSERT(shape.ndim() == 3);
        const auto peak_offset = cuda::math::find<uint32_t>(noa::math::first_max_t{}, xmap, strides, shape, true, stream);
        const uint3_t shape_3d(shape.get(1));
        const uint3_t strides_3d(strides.get(1));
        const int3_t peak_index(indexing::indexes(peak_offset, strides_3d, shape_3d));

        cuda::memory::PtrPinned<float3_t> coordinate(1);
        constexpr bool IS_CENTERED = static_cast<std::underlying_type_t<Remap>>(REMAP) & noa::fft::Layout::DST_CENTERED;
        stream.enqueue("signal::fft::xpeak3D", singlePeak3D_<IS_CENTERED, T>, LaunchConfig{1, BLOCK_SIZE},
                       xmap.get(), strides_3d, int3_t(shape_3d), peak_index, coordinate.get());
        stream.synchronize();
        return coordinate[0];
    }

    #define INSTANTIATE_XMAP(T) \
    template void xmap<Remap::H2F, T, void>(const shared_t<Complex<T>[]>&, size4_t, const shared_t<Complex<T>[]>&, size4_t, const shared_t<T[]>&, size4_t, size4_t, bool, Norm, Stream&, const shared_t<Complex<T>[]>&, size4_t);   \
    template void xmap<Remap::H2FC, T, void>(const shared_t<Complex<T>[]>&, size4_t, const shared_t<Complex<T>[]>&, size4_t, const shared_t<T[]>&, size4_t, size4_t, bool, Norm, Stream&, const shared_t<Complex<T>[]>&, size4_t);  \
    template void xpeak1D<Remap::F2F, T, void>(const shared_t<T[]>&, size4_t, size4_t, const shared_t<float[]>&, Stream&);      \
    template void xpeak1D<Remap::FC2FC, T, void>(const shared_t<T[]>&, size4_t, size4_t, const shared_t<float[]>&, Stream&);    \
    template void xpeak2D<Remap::F2F, T, void>(const shared_t<T[]>&, size4_t, size4_t, const shared_t<float2_t[]>&, Stream&);   \
    template void xpeak2D<Remap::FC2FC, T, void>(const shared_t<T[]>&, size4_t, size4_t, const shared_t<float2_t[]>&, Stream&); \
    template void xpeak3D<Remap::F2F, T, void>(const shared_t<T[]>&, size4_t, size4_t, const shared_t<float3_t[]>&, Stream&);   \
    template void xpeak3D<Remap::FC2FC, T, void>(const shared_t<T[]>&, size4_t, size4_t, const shared_t<float3_t[]>&, Stream&); \
    template float xpeak1D<Remap::F2F, T, void>(const shared_t<T[]>&, size4_t, size4_t, Stream&);       \
    template float xpeak1D<Remap::FC2FC, T, void>(const shared_t<T[]>&, size4_t, size4_t, Stream&);     \
    template float2_t xpeak2D<Remap::F2F, T, void>(const shared_t<T[]>&, size4_t, size4_t, Stream&);    \
    template float2_t xpeak2D<Remap::FC2FC, T, void>(const shared_t<T[]>&, size4_t, size4_t, Stream&);  \
    template float3_t xpeak3D<Remap::F2F, T, void>(const shared_t<T[]>&, size4_t, size4_t, Stream&);    \
    template float3_t xpeak3D<Remap::FC2FC, T, void>(const shared_t<T[]>&, size4_t, size4_t, Stream&)

    INSTANTIATE_XMAP(float);
    INSTANTIATE_XMAP(double);
}

namespace noa::cuda::signal::fft::details {
    template<typename T>
    void xcorr(const shared_t<Complex<T>[]>& lhs, size4_t lhs_stride,
               const shared_t<Complex<T>[]>& rhs, size4_t rhs_stride,
               size4_t shape, const shared_t<T[]>& coefficients,
               Stream& stream, bool is_half) {
        const size_t batches = shape[0];
        const size4_t shape_fft = is_half ? shape.fft() : shape;

        cuda::memory::PtrPinned<T> buffer(batches * 3);
        auto denominator_lhs = buffer.get() + batches;
        auto denominator_rhs = buffer.get() + batches * 2;

        T* null{};
        cuda::util::reduce(
                "signal::fft::xcorr", lhs.get(), uint4_t(lhs_stride), uint4_t(shape_fft),
                noa::math::abs_squared_t{}, noa::math::plus_t{}, T{0},
                denominator_lhs, 1, noa::math::copy_t{},
                null, 0, noa::math::copy_t{},
                false, true, stream);
        cuda::util::reduce(
                "signal::fft::xcorr", rhs.get(), uint4_t(rhs_stride), uint4_t(shape_fft),
                noa::math::abs_squared_t{}, noa::math::plus_t{}, T{0},
                denominator_rhs, 1, noa::math::copy_t{},
                null, 0, noa::math::copy_t{},
                false, true, stream);

        auto combine_op = []__device__(Complex<T> l, Complex<T> r) { return noa::math::real(l * r); };
        cuda::util::reduce<false>(
                "signal::fft::xcorr",
                lhs.get(), uint4_t(lhs_stride), rhs.get(), uint4_t(rhs_stride), uint4_t(shape_fft),
                noa::math::copy_t{}, noa::math::conj_t{}, combine_op, noa::math::plus_t{}, T{0},
                buffer.get(), 1, noa::math::copy_t{}, null, 1, noa::math::copy_t{}, false, stream);

        stream.synchronize(); // FIXME Add callback
        for (size_t batch = 0; batch < batches; ++batch) {
            coefficients.get()[batch] =
                    buffer[batch] / noa::math::sqrt(denominator_lhs[batch] * denominator_rhs[batch]);
        }
    }

    template<typename T>
    T xcorr(const shared_t<Complex<T>[]>& lhs, size4_t lhs_stride,
            const shared_t<Complex<T>[]>& rhs, size4_t rhs_stride,
            size4_t shape, Stream& stream, bool is_half) {
        NOA_ASSERT(shape[0] == 1);
        const size4_t shape_fft = is_half ? shape.fft() : shape;

        T numerator{}, denominator_lhs{}, denominator_rhs{};
        T* null{};
        cuda::util::reduce(
                "signal::fft::xcorr", lhs.get(), uint4_t(lhs_stride), uint4_t(shape_fft),
                noa::math::abs_squared_t{}, noa::math::plus_t{}, T{0},
                &denominator_lhs, 1, noa::math::copy_t{},
                null, 0, noa::math::copy_t{},
                true, true, stream);
        cuda::util::reduce(
                "signal::fft::xcorr", rhs.get(), uint4_t(rhs_stride), uint4_t(shape_fft),
                noa::math::abs_squared_t{}, noa::math::plus_t{}, T{0},
                &denominator_rhs, 1, noa::math::copy_t{},
                null, 0, noa::math::copy_t{},
                true, true, stream);

        auto combine_op = []__device__(Complex<T> l, Complex<T> r) { return noa::math::real(l * r); };
        cuda::util::reduce<false>(
                "signal::fft::xcorr",
                lhs.get(), uint4_t(lhs_stride), rhs.get(), uint4_t(rhs_stride), uint4_t(shape_fft),
                noa::math::copy_t{}, noa::math::conj_t{}, combine_op, noa::math::plus_t{}, T{0},
                &numerator, 1, noa::math::copy_t{}, null, 1, noa::math::copy_t{}, false, stream);

        stream.synchronize();
        const T denominator = noa::math::sqrt(denominator_lhs * denominator_rhs);
        return numerator / denominator;
    }

    #define INSTANTIATE_XCORR(T) \
    template void xcorr<T>(const shared_t<Complex<T>[]>&, size4_t, const shared_t<Complex<T>[]>&, size4_t, size4_t, const shared_t<T[]>&, Stream&, bool); \
    template T xcorr<T>(const shared_t<Complex<T>[]>&, size4_t, const shared_t<Complex<T>[]>&, size4_t, size4_t, Stream&, bool)

    INSTANTIATE_XCORR(float);
    INSTANTIATE_XCORR(double);
}
