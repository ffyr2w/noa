#include "hip/hip_runtime.h"
#include "noa/common/Assert.h"
#include "noa/common/Math.h"
#include "noa/common/Profiler.h"
#include "noa/gpu/cuda/fft/Exception.h"
#include "noa/gpu/cuda/fft/Filters.h"

namespace {
    using namespace noa;
    constexpr dim3 BLOCK_SIZE(32, 8);

    enum class Type {
        LOWPASS,
        HIGHPASS
    };

    template<bool IS_CENTERED>
    __forceinline__ __device__ int getFrequency_(int idx, int dim) {
        if constexpr(IS_CENTERED)
            return idx - dim / 2;
        else
            return idx < (dim + 1) / 2 ? idx : idx - dim;
        return 0;
    }

    template<bool IS_SRC_CENTERED, bool IS_DST_CENTERED>
    __forceinline__ __device__ int getOutputIndex_(int i_idx, [[maybe_unused]] int dim) {
        (void) dim;
        if constexpr (IS_SRC_CENTERED == IS_DST_CENTERED)
            return i_idx;
        else if constexpr (IS_SRC_CENTERED)
            return noa::math::iFFTShift(i_idx, dim);
        else
            return noa::math::FFTShift(i_idx, dim);
        return 0;
    }

    template<bool IS_CENTERED>
    __forceinline__ __device__ float getNormalizedFrequencySqd(int3_t gid, int3_t shape, float3_t norm) {
        float3_t distance_sqd(getFrequency_<IS_CENTERED>(gid[0], shape[0]),
                              getFrequency_<IS_CENTERED>(gid[1], shape[1]),
                              gid[2]);
        distance_sqd *= norm;
        return math::dot(distance_sqd, distance_sqd);
    }
}

namespace {
    template<Type PASS>
    inline __device__ float getSoftWindow_(float freq_cutoff, float freq_width, float freq) {
        constexpr float PI = math::Constants<float>::PI;
        float filter;
        if constexpr (PASS == Type::LOWPASS) {
            if (freq <= freq_cutoff)
                filter = 1;
            else if (freq_cutoff + freq_width <= freq)
                filter = 0;
            else
                filter = (1.f + math::cos(PI * (freq_cutoff - freq) / freq_width)) * 0.5f;
        } else if constexpr (PASS == Type::HIGHPASS) {
            if (freq_cutoff <= freq)
                filter = 1;
            else if (freq <= freq_cutoff - freq_width)
                filter = 0;
            else
                filter = (1.f + math::cos(PI * (freq - freq_cutoff) / freq_width)) * 0.5f;
        }
        return filter;
    }

    template<Type PASS>
    inline __device__ float getHardWindow_(float freq_cutoff_sqd, float freq_sqd) {
        float filter;
        if constexpr (PASS == Type::LOWPASS) {
            if (freq_cutoff_sqd < freq_sqd)
                filter = 0;
            else
                filter = 1;
        } else if constexpr (PASS == Type::HIGHPASS) {
            if (freq_sqd < freq_cutoff_sqd)
                filter = 0;
            else
                filter = 1;
        }
        return filter;
    }

    template<bool IS_SRC_CENTERED, bool IS_DST_CENTERED, Type PASS, bool HAS_WIDTH, typename T>
    __global__ __launch_bounds__(BLOCK_SIZE.x * BLOCK_SIZE.y)
    void singlePass_(const T* input, uint4_t input_stride, T* output, uint4_t output_stride,
                     int3_t shape, float3_t norm, float cutoff, float width, uint blocks_x) {
        using real_t = noa::traits::value_type_t<T>;

        const uint batch = blockIdx.z;
        const uint2_t index = indexing::indexes(blockIdx.x, blocks_x);
        const int3_t gid(blockIdx.y,
                         BLOCK_SIZE.y * index[0] + threadIdx.y,
                         BLOCK_SIZE.x * index[1] + threadIdx.x);
        if (gid[2] >= shape[2] / 2 + 1 || gid[1] >= shape[1])
            return;

        // Get filter for current input frequency.
        real_t filter;
        if constexpr (HAS_WIDTH) {
            const float frequency = math::sqrt(getNormalizedFrequencySqd<IS_SRC_CENTERED>(gid, shape, norm));
            filter = static_cast<real_t>(getSoftWindow_<PASS>(cutoff, width, frequency));
        } else {
            const float frequency_sqd = getNormalizedFrequencySqd<IS_SRC_CENTERED>(gid, shape, norm);
            filter = static_cast<real_t>(getHardWindow_<PASS>(cutoff * cutoff, frequency_sqd));
            (void) width;
        }

        // Save to output.
        const int oz = getOutputIndex_<IS_SRC_CENTERED, IS_DST_CENTERED>(gid[0], shape[0]);
        const int oy = getOutputIndex_<IS_SRC_CENTERED, IS_DST_CENTERED>(gid[1], shape[1]);
        output[indexing::at(batch, oz, oy, gid[2], output_stride)] =
                input ? input[indexing::at(batch, gid[0], gid[1], gid[2], input_stride)] * filter : filter;
    }

    template<bool IS_SRC_CENTERED, bool IS_DST_CENTERED, bool HAS_WIDTH, typename T>
    __global__ __launch_bounds__(BLOCK_SIZE.x * BLOCK_SIZE.y)
    void bandPass_(const T* input, uint4_t input_stride, T* output, uint4_t output_stride,
                   int3_t shape, float3_t norm,
                   float cutoff_1, float cutoff_2,
                   float width_1, float width_2, uint blocks_x) {
        using real_t = noa::traits::value_type_t<T>;

        const uint batch = blockIdx.z;
        const uint2_t index = indexing::indexes(blockIdx.x, blocks_x);
        const int3_t gid(blockIdx.y,
                         BLOCK_SIZE.y * index[0] + threadIdx.y,
                         BLOCK_SIZE.x * index[1] + threadIdx.x);
        if (gid[2] >= shape[2] / 2 + 1 || gid[1] >= shape[1])
            return;

        // Get filter for current input frequency.
        real_t filter;
        if constexpr (HAS_WIDTH) {
            const float frequency = math::sqrt(getNormalizedFrequencySqd<IS_SRC_CENTERED>(gid, shape, norm));
            filter = static_cast<real_t>(getSoftWindow_<Type::HIGHPASS>(cutoff_1, width_1, frequency) *
                                         getSoftWindow_<Type::LOWPASS>(cutoff_2, width_2, frequency));
        } else {
            const float frequency_sqd = getNormalizedFrequencySqd<IS_SRC_CENTERED>(gid, shape, norm);
            filter = static_cast<real_t>(getSoftWindow_<Type::HIGHPASS>(cutoff_1 * cutoff_1, width_1, frequency_sqd) *
                                         getSoftWindow_<Type::LOWPASS>(cutoff_2 * cutoff_2, width_2, frequency_sqd));
            (void) width_1;
            (void) width_2;
        }

        // Save to output.
        const int oz = getOutputIndex_<IS_SRC_CENTERED, IS_DST_CENTERED>(gid[0], shape[0]);
        const int oy = getOutputIndex_<IS_SRC_CENTERED, IS_DST_CENTERED>(gid[1], shape[1]);
        output[indexing::at(batch, oz, oy, gid[2], output_stride)] =
                input ? input[indexing::at(batch, gid[0], gid[1], gid[2], input_stride)] * filter : filter;
    }
}

namespace {
    template<Type PASS, ::noa::fft::Remap REMAP, typename T>
    void launchSinglePass_(const shared_t<T[]>& input, size4_t input_stride,
                           const shared_t<T[]>& output, size4_t output_stride,
                           size4_t shape, float cutoff, float width, cuda::Stream& stream) {
        using Layout = ::noa::fft::Layout;
        constexpr auto REMAP_ = static_cast<uint8_t>(REMAP);
        constexpr bool IS_SRC_CENTERED = REMAP_ & Layout::SRC_CENTERED;
        constexpr bool IS_DST_CENTERED = REMAP_ & Layout::DST_CENTERED;
        if constexpr (REMAP_ & Layout::SRC_FULL || REMAP_ & Layout::DST_FULL)
            static_assert(traits::always_false_v<T>);
        NOA_ASSERT(input != output || IS_SRC_CENTERED == IS_DST_CENTERED);

        const int3_t s_shape{shape.get() + 1};
        float3_t norm{s_shape / 2 * 2 + int3_t{s_shape == 1}};
        norm = 1.f / norm;

        const uint blocks_x = math::divideUp(s_shape[2] / 2 + 1, static_cast<int>(BLOCK_SIZE.x));
        const uint blocks_y = math::divideUp(s_shape[1], static_cast<int>(BLOCK_SIZE.y));
        const dim3 blocks(blocks_x * blocks_y, s_shape[0], shape[0]);
        const cuda::LaunchConfig config{blocks, BLOCK_SIZE};
        stream.enqueue(
                "singlePass_",
                width > 1e-6f ?
                singlePass_<IS_SRC_CENTERED, IS_DST_CENTERED, PASS, true, T> :
                singlePass_<IS_SRC_CENTERED, IS_DST_CENTERED, PASS, false, T>, config,
                input.get(), uint4_t{input_stride}, output.get(), uint4_t{output_stride}, s_shape,
                norm, cutoff, width, blocks_x);
        stream.attach(input, output);
    }
}

namespace noa::cuda::fft {
    template<Remap REMAP, typename T>
    void lowpass(const shared_t<T[]>& input, size4_t input_stride,
                 const shared_t<T[]>& output, size4_t output_stride,
                 size4_t shape, float cutoff, float width, Stream& stream) {
        NOA_PROFILE_FUNCTION();
        launchSinglePass_<Type::LOWPASS, REMAP>(
                input, input_stride, output, output_stride, shape, cutoff, width, stream);
    }

    template<Remap REMAP, typename T>
    void highpass(const shared_t<T[]>& input, size4_t input_stride,
                  const shared_t<T[]>& output, size4_t output_stride,
                  size4_t shape, float cutoff, float width, Stream& stream) {
        NOA_PROFILE_FUNCTION();
        launchSinglePass_<Type::HIGHPASS, REMAP>(
                input, input_stride, output, output_stride, shape, cutoff, width, stream);
    }

    template<Remap REMAP, typename T>
    void bandpass(const shared_t<T[]>& input, size4_t input_stride,
                  const shared_t<T[]>& output, size4_t output_stride, size4_t shape,
                  float cutoff1, float cutoff2, float width1, float width2, Stream& stream) {
        using Layout = ::noa::fft::Layout;
        constexpr auto REMAP_ = static_cast<uint8_t>(REMAP);
        constexpr bool IS_SRC_CENTERED = REMAP_ & Layout::SRC_CENTERED;
        constexpr bool IS_DST_CENTERED = REMAP_ & Layout::DST_CENTERED;
        if constexpr (REMAP_ & Layout::SRC_FULL || REMAP_ & Layout::DST_FULL)
            static_assert(traits::always_false_v<T>);
        NOA_ASSERT(input != output || IS_SRC_CENTERED == IS_DST_CENTERED);
        NOA_PROFILE_FUNCTION();
        const int3_t s_shape(shape.get() + 1);
        float3_t norm(s_shape / 2 * 2 + int3_t{s_shape == 1});
        norm = 1.f / norm;

        const uint blocks_x = math::divideUp(s_shape[2] / 2 + 1, static_cast<int>(BLOCK_SIZE.x));
        const uint blocks_y = math::divideUp(s_shape[1], static_cast<int>(BLOCK_SIZE.y));
        const dim3 blocks(blocks_x * blocks_y, s_shape[0], shape[0]);
        const LaunchConfig config{blocks, BLOCK_SIZE};
        stream.enqueue(
                "bandPass_",
                width1 > 1e-6f || width2 > 1e-6f ?
                bandPass_<IS_SRC_CENTERED, IS_DST_CENTERED, true, T> :
                bandPass_<IS_SRC_CENTERED, IS_DST_CENTERED, false, T>, config,
                input.get(), uint4_t{input_stride}, output.get(), uint4_t{output_stride}, s_shape, norm,
                cutoff1, cutoff2, width1, width2, blocks_x);
        stream.attach(input, output);
    }

    #define NOA_INSTANTIATE_FILTERS_(T)                                                                                                               \
    template void lowpass<Remap::H2H, T>(const shared_t<T[]>&, size4_t, const shared_t<T[]>&, size4_t, size4_t, float, float, Stream&);               \
    template void highpass<Remap::H2H,T>(const shared_t<T[]>&, size4_t, const shared_t<T[]>&, size4_t, size4_t, float, float, Stream&);               \
    template void bandpass<Remap::H2H,T>(const shared_t<T[]>&, size4_t, const shared_t<T[]>&, size4_t, size4_t, float, float, float, float, Stream&); \
    template void lowpass<Remap::H2HC, T>(const shared_t<T[]>&, size4_t, const shared_t<T[]>&, size4_t, size4_t, float, float, Stream&);              \
    template void highpass<Remap::H2HC,T>(const shared_t<T[]>&, size4_t, const shared_t<T[]>&, size4_t, size4_t, float, float, Stream&);              \
    template void bandpass<Remap::H2HC,T>(const shared_t<T[]>&, size4_t, const shared_t<T[]>&, size4_t, size4_t, float, float, float, float, Stream&);\
    template void lowpass<Remap::HC2H, T>(const shared_t<T[]>&, size4_t, const shared_t<T[]>&, size4_t, size4_t, float, float, Stream&);              \
    template void highpass<Remap::HC2H,T>(const shared_t<T[]>&, size4_t, const shared_t<T[]>&, size4_t, size4_t, float, float, Stream&);              \
    template void bandpass<Remap::HC2H,T>(const shared_t<T[]>&, size4_t, const shared_t<T[]>&, size4_t, size4_t, float, float, float, float, Stream&);\
    template void lowpass<Remap::HC2HC, T>(const shared_t<T[]>&, size4_t, const shared_t<T[]>&, size4_t, size4_t, float, float, Stream&);             \
    template void highpass<Remap::HC2HC,T>(const shared_t<T[]>&, size4_t, const shared_t<T[]>&, size4_t, size4_t, float, float, Stream&);             \
    template void bandpass<Remap::HC2HC,T>(const shared_t<T[]>&, size4_t, const shared_t<T[]>&, size4_t, size4_t, float, float, float, float, Stream&)

    NOA_INSTANTIATE_FILTERS_(half_t);
    NOA_INSTANTIATE_FILTERS_(float);
    NOA_INSTANTIATE_FILTERS_(double);
    NOA_INSTANTIATE_FILTERS_(chalf_t);
    NOA_INSTANTIATE_FILTERS_(cfloat_t);
    NOA_INSTANTIATE_FILTERS_(cdouble_t);
}
