#include "hip/hip_runtime.h"
#include "noa/common/Math.h"
#include "noa/common/Profiler.h"
#include "noa/gpu/cuda/fft/Exception.h"
#include "noa/gpu/cuda/fft/Filters.h"
#include "noa/gpu/cuda/memory/Copy.h"

// Commons:
namespace {
    using namespace noa;
    constexpr dim3 THREADS(32, 8);

    enum class Type {
        LOWPASS,
        HIGHPASS
    };

    __forceinline__ __device__ float getDistance_(uint idx, uint half, uint dimension) {
        return idx >= half ? static_cast<float>(idx) - static_cast<float>(dimension) : static_cast<float>(idx);
    }

    __forceinline__ __device__ float getNormalizedFrequencySqd(uint3_t gid, uint3_t shape, uint3_t half) {
        float3_t distance_sqd(gid.x,
                              getDistance_(gid.y, half.y, shape.y),
                              getDistance_(gid.z, half.z, shape.z));
        distance_sqd /= float3_t(shape);
        return math::dot(distance_sqd, distance_sqd);
    }
}

// Soft edges (Hann window):
namespace {
    template<Type PASS>
    inline __device__ float getSoftWindow_(float freq_cutoff, float freq_width, float freq) {
        constexpr float PI = math::Constants<float>::PI;
        float filter;
        if constexpr (PASS == Type::LOWPASS) {
            if (freq <= freq_cutoff)
                filter = 1;
            else if (freq_cutoff + freq_width <= freq)
                filter = 0;
            else
                filter = (1.f + math::cos(PI * (freq_cutoff - freq) / freq_width)) * 0.5f;
        } else if constexpr (PASS == Type::HIGHPASS) {
            if (freq_cutoff <= freq)
                filter = 1;
            else if (freq <= freq_cutoff - freq_width)
                filter = 0;
            else
                filter = (1.f + math::cos(PI * (freq - freq_cutoff) / freq_width)) * 0.5f;
        }
        return filter;
    }

    template<Type PASS, typename T>
    __global__ __launch_bounds__(THREADS.x * THREADS.y)
    void singlePassSoft_(const T* inputs, uint inputs_pitch, T* outputs, uint outputs_pitch,
                         uint3_t shape, uint3_t half, float freq_cutoff, float freq_width, uint blocks_x) {
        using real_t = noa::traits::value_type_t<T>;

        // Get the current indexes.
        const uint2_t idx = coordinates(blockIdx.x, blocks_x);
        const uint3_t gid(THREADS.x * idx.x + threadIdx.x,
                          THREADS.y * idx.y + threadIdx.y,
                          blockIdx.y);
        if (gid.x >= half.x || gid.y >= shape.y)
            return;

        // Offset to current batch.
        inputs += blockIdx.z * rows(shape) * inputs_pitch;
        outputs += blockIdx.z * rows(shape) * outputs_pitch;

        // Apply filter.
        float frequency = math::sqrt(getNormalizedFrequencySqd(gid, shape, half));
        auto filter = static_cast<real_t>(getSoftWindow_<PASS>(freq_cutoff, freq_width, frequency));
        outputs[(gid.z * shape.y + gid.y) * outputs_pitch + gid.x] =
                inputs[(gid.z * shape.y + gid.y) * inputs_pitch + gid.x] * filter;
    }

    template<Type PASS, typename T>
    __global__ __launch_bounds__(THREADS.x * THREADS.y)
    void singlePassSoft_(T* output_filter, uint output_filter_pitch,
                         uint3_t shape, uint3_t half, float freq_cutoff, float freq_width) {
        using real_t = noa::traits::value_type_t<T>;

        const uint3_t gid(THREADS.x * blockIdx.x + threadIdx.x,
                          THREADS.y * blockIdx.y + threadIdx.y,
                          blockIdx.z);
        if (gid.x >= half.x || gid.y >= shape.y)
            return;

        float frequency = math::sqrt(getNormalizedFrequencySqd(gid, shape, half));
        auto filter = static_cast<real_t>(getSoftWindow_<PASS>(freq_cutoff, freq_width, frequency));
        output_filter[(gid.z * shape.y + gid.y) * output_filter_pitch + gid.x] = filter;
    }

    template<typename T>
    __global__ __launch_bounds__(THREADS.x * THREADS.y)
    void bandPassSoft_(const T* inputs, uint inputs_pitch, T* outputs, uint outputs_pitch,
                       uint3_t shape, uint3_t half, float freq_cutoff_1, float freq_cutoff_2,
                       float freq_width_1, float freq_width_2, uint blocks_x) {
        using real_t = noa::traits::value_type_t<T>;

        const uint2_t idx = coordinates(blockIdx.x, blocks_x);
        const uint3_t gid(THREADS.x * idx.x + threadIdx.x,
                          THREADS.y * idx.y + threadIdx.y,
                          blockIdx.y);
        if (gid.x >= half.x || gid.y >= shape.y)
            return;

        inputs += blockIdx.z * rows(shape) * inputs_pitch;
        outputs += blockIdx.z * rows(shape) * outputs_pitch;

        float frequency = math::sqrt(getNormalizedFrequencySqd(gid, shape, half));
        float filter = getSoftWindow_<Type::HIGHPASS>(freq_cutoff_1, freq_width_1, frequency);
        filter *= getSoftWindow_<Type::LOWPASS>(freq_cutoff_2, freq_width_2, frequency);
        outputs[(gid.z * shape.y + gid.y) * outputs_pitch + gid.x] =
                inputs[(gid.z * shape.y + gid.y) * inputs_pitch + gid.x] * static_cast<real_t>(filter);
    }

    template<typename T>
    __global__ __launch_bounds__(THREADS.x * THREADS.y)
    void bandPassSoft_(T* output_filter, uint output_filter_pitch, uint3_t shape, uint3_t half,
                       float freq_cutoff_1, float freq_cutoff_2, float freq_width_1, float freq_width_2) {
        using real_t = noa::traits::value_type_t<T>;

        const uint3_t gid(THREADS.x * blockIdx.x + threadIdx.x,
                          THREADS.y * blockIdx.y + threadIdx.y,
                          blockIdx.z);
        if (gid.x >= half.x || gid.y >= shape.y)
            return;

        float frequency = math::sqrt(getNormalizedFrequencySqd(gid, shape, half));
        float filter = getSoftWindow_<Type::HIGHPASS>(freq_cutoff_1, freq_width_1, frequency);
        filter *= getSoftWindow_<Type::LOWPASS>(freq_cutoff_2, freq_width_2, frequency);
        output_filter[(gid.z * shape.y + gid.y) * output_filter_pitch + gid.x] = static_cast<real_t>(filter);
    }
}

// Hard edges:
namespace {
    template<Type PASS>
    inline __device__ float getHardWindow_(float freq_cutoff_sqd, float freq_sqd) {
        float filter;
        if constexpr (PASS == Type::LOWPASS) {
            if (freq_cutoff_sqd < freq_sqd)
                filter = 0;
            else
                filter = 1;
        } else if constexpr (PASS == Type::HIGHPASS) {
            if (freq_sqd < freq_cutoff_sqd)
                filter = 0;
            else
                filter = 1;
        }
        return filter;
    }

    template<Type PASS, typename T>
    __global__ __launch_bounds__(THREADS.x * THREADS.y)
    void singlePassHard_(const T* inputs, uint inputs_pitch, T* outputs, uint outputs_pitch,
                         uint3_t shape, uint3_t half, float freq_cutoff, uint blocks_x) {
        using real_t = noa::traits::value_type_t<T>;

        // Get the current indexes.
        const uint2_t idx = coordinates(blockIdx.x, blocks_x);
        const uint3_t gid(THREADS.x * idx.x + threadIdx.x,
                          THREADS.y * idx.y + threadIdx.y,
                          blockIdx.y);
        if (gid.x >= half.x || gid.y >= shape.y)
            return;

        // Offset to current batch.
        inputs += blockIdx.z * rows(shape) * inputs_pitch;
        outputs += blockIdx.z * rows(shape) * outputs_pitch;

        // Apply filter.
        float freq_cutoff_sqd = freq_cutoff * freq_cutoff;
        float frequency_sqd = getNormalizedFrequencySqd(gid, shape, half);
        auto filter = static_cast<real_t>(getHardWindow_<PASS>(freq_cutoff_sqd, frequency_sqd));
        outputs[(gid.z * shape.y + gid.y) * outputs_pitch + gid.x] =
                inputs[(gid.z * shape.y + gid.y) * inputs_pitch + gid.x] * filter;
    }

    template<Type PASS, typename T>
    __global__ __launch_bounds__(THREADS.x * THREADS.y)
    void singlePassHard_(T* output_filter, uint output_filter_pitch,
                         uint3_t shape, uint3_t half, float freq_cutoff) {
        using real_t = noa::traits::value_type_t<T>;

        const uint3_t gid(THREADS.x * blockIdx.x + threadIdx.x,
                          THREADS.y * blockIdx.y + threadIdx.y,
                          blockIdx.z);
        if (gid.x >= half.x || gid.y >= shape.y)
            return;

        float freq_cutoff_sqd = freq_cutoff * freq_cutoff;
        float frequency_sqd = getNormalizedFrequencySqd(gid, shape, half);
        auto filter = static_cast<real_t>(getHardWindow_<PASS>(freq_cutoff_sqd, frequency_sqd));
        output_filter[(gid.z * shape.y + gid.y) * output_filter_pitch + gid.x] = filter;
    }

    template<typename T>
    __global__ __launch_bounds__(THREADS.x * THREADS.y)
    void bandPassHard_(const T* inputs, uint inputs_pitch, T* outputs, uint outputs_pitch,
                       uint3_t shape, uint3_t half, float freq_cutoff_1, float freq_cutoff_2,
                       uint blocks_x) {
        using real_t = noa::traits::value_type_t<T>;

        const uint2_t idx = coordinates(blockIdx.x, blocks_x);
        const uint3_t gid(THREADS.x * idx.x + threadIdx.x,
                          THREADS.y * idx.y + threadIdx.y,
                          blockIdx.y);
        if (gid.x >= half.x || gid.y >= shape.y)
            return;

        inputs += blockIdx.z * rows(shape) * inputs_pitch;
        outputs += blockIdx.z * rows(shape) * outputs_pitch;

        float freq_cutoff_sqd_1 = freq_cutoff_1 * freq_cutoff_1;
        float freq_cutoff_sqd_2 = freq_cutoff_2 * freq_cutoff_2;
        float frequency_sqd = getNormalizedFrequencySqd(gid, shape, half);
        float filter = getHardWindow_<Type::HIGHPASS>(freq_cutoff_sqd_1, frequency_sqd);
        filter *= getHardWindow_<Type::LOWPASS>(freq_cutoff_sqd_2, frequency_sqd);
        outputs[(gid.z * shape.y + gid.y) * outputs_pitch + gid.x] =
                inputs[(gid.z * shape.y + gid.y) * inputs_pitch + gid.x] * static_cast<real_t>(filter);
    }

    template<typename T>
    __global__ __launch_bounds__(THREADS.x * THREADS.y)
    void bandPassHard_(T* output_filter, uint output_filter_pitch, uint3_t shape, uint3_t half,
                       float freq_cutoff_1, float freq_cutoff_2) {
        using real_t = noa::traits::value_type_t<T>;

        const uint3_t gid(THREADS.x * blockIdx.x + threadIdx.x,
                          THREADS.y * blockIdx.y + threadIdx.y,
                          blockIdx.z);
        if (gid.x >= half.x || gid.y >= shape.y)
            return;

        float freq_cutoff_sqd_1 = freq_cutoff_1 * freq_cutoff_1;
        float freq_cutoff_sqd_2 = freq_cutoff_2 * freq_cutoff_2;
        float frequency_sqd = getNormalizedFrequencySqd(gid, shape, half);
        float filter = getHardWindow_<Type::HIGHPASS>(freq_cutoff_sqd_1, frequency_sqd);
        filter *= getHardWindow_<Type::LOWPASS>(freq_cutoff_sqd_2, frequency_sqd);
        output_filter[(gid.z * shape.y + gid.y) * output_filter_pitch + gid.x] = static_cast<real_t>(filter);
    }

    template<Type PASS, typename T>
    void singlePass(const T* inputs, size_t inputs_pitch, T* outputs, size_t outputs_pitch,
                    size3_t shape, size_t batches,
                    float freq_cutoff, float freq_width, cuda::Stream& stream) {
        NOA_PROFILE_FUNCTION();
        uint3_t u_shape(shape);
        uint3_t half(u_shape / 2U + 1U);

        uint blocks_x = math::divideUp(half.x, THREADS.x);
        uint blocks_y = math::divideUp(u_shape.y, THREADS.y);

        if (inputs) {
            dim3 blocks(blocks_x * blocks_y, u_shape.z, batches);
            if (freq_width > 1e-6f) {
                singlePassSoft_<PASS><<<blocks, THREADS, 0, stream.id()>>>(
                        inputs, inputs_pitch, outputs, outputs_pitch, u_shape, half, freq_cutoff, freq_width, blocks_x);
            } else {
                singlePassHard_<PASS><<<blocks, THREADS, 0, stream.id()>>>(
                        inputs, inputs_pitch, outputs, outputs_pitch, u_shape, half, freq_cutoff, blocks_x);
            }
        } else {
            if constexpr(!traits::is_complex_v<T>) {
                dim3 blocks(blocks_x, blocks_y, u_shape.z);
                if (freq_width > 1e-6f) {
                    singlePassSoft_<PASS><<<blocks, THREADS, 0, stream.id()>>>(
                            outputs, outputs_pitch, u_shape, half, freq_cutoff, freq_width);
                } else {
                    singlePassHard_<PASS><<<blocks, THREADS, 0, stream.id()>>>(
                            outputs, outputs_pitch, u_shape, half, freq_cutoff);
                }
                const size_t elements = outputs_pitch * rows(shape);
                for (size_t batch = 1; batch < batches; ++batch)
                    cuda::memory::copy(outputs, outputs + elements * batch, elements, stream);
            } else {
                NOA_THROW_FUNC("(low|high)pass", "Cannot compute a filter of complex type");
            }
        }
        NOA_THROW_IF(hipGetLastError());
    }
}

namespace noa::cuda::fft {
    template<typename T>
    void lowpass(const T* inputs, size_t inputs_pitch, T* outputs, size_t outputs_pitch,
                 size3_t shape, size_t batches,
                 float freq_cutoff, float freq_width, Stream& stream) {
        NOA_PROFILE_FUNCTION();
        singlePass<Type::LOWPASS>(inputs, inputs_pitch, outputs, outputs_pitch,
                                  shape, batches, freq_cutoff, freq_width, stream);
    }

    template<typename T>
    void highpass(const T* inputs, size_t inputs_pitch, T* outputs, size_t outputs_pitch,
                  size3_t shape, size_t batches,
                  float freq_cutoff, float freq_width, Stream& stream) {
        NOA_PROFILE_FUNCTION();
        singlePass<Type::HIGHPASS>(inputs, inputs_pitch, outputs, outputs_pitch,
                                   shape, batches, freq_cutoff, freq_width, stream);
    }

    template<typename T>
    void bandpass(const T* inputs, size_t inputs_pitch, T* outputs, size_t outputs_pitch,
                  size3_t shape, size_t batches,
                  float freq_cutoff_1, float freq_cutoff_2, float freq_width_1, float freq_width_2, Stream& stream) {
        NOA_PROFILE_FUNCTION();
        uint3_t u_shape(shape);
        uint3_t half(u_shape / 2U + 1U);

        uint blocks_x = math::divideUp(half.x, THREADS.x);
        uint blocks_y = math::divideUp(u_shape.y, THREADS.y);

        if (inputs) {
            dim3 blocks(blocks_x * blocks_y, u_shape.z, batches);
            if (freq_width_1 > 1e-6f || freq_width_2 > 1e-6f) {
                bandPassSoft_<<<blocks, THREADS, 0, stream.id()>>>(
                        inputs, inputs_pitch, outputs, outputs_pitch, u_shape, half,
                        freq_cutoff_1, freq_cutoff_2, freq_width_1, freq_width_2, blocks_x);
            } else {
                bandPassHard_<<<blocks, THREADS, 0, stream.id()>>>(
                        inputs, inputs_pitch, outputs, outputs_pitch, u_shape, half,
                        freq_cutoff_1, freq_cutoff_2, blocks_x);
            }
        } else {
            if constexpr(!traits::is_complex_v<T>) {
                dim3 blocks(blocks_x, blocks_y, u_shape.z);
                if (freq_width_1 > 1e-6f || freq_width_2 > 1e-6f) {
                    bandPassSoft_<<<blocks, THREADS, 0, stream.id()>>>(
                            outputs, outputs_pitch, u_shape, half,
                            freq_cutoff_1, freq_cutoff_2, freq_width_1, freq_width_2);
                } else {
                    bandPassHard_<<<blocks, THREADS, 0, stream.id()>>>(
                            outputs, outputs_pitch, u_shape, half,
                            freq_cutoff_1, freq_cutoff_2);
                }
                const size_t elements = outputs_pitch * rows(shape);
                for (size_t batch = 1; batch < batches; ++batch)
                    cuda::memory::copy(outputs, outputs + elements * batch, elements, stream);
            } else {
                NOA_THROW_FUNC("(low|high)pass", "Cannot compute a filter of complex type");
            }
        }

        NOA_THROW_IF(hipGetLastError());
    }

    #define NOA_INSTANTIATE_FILTERS_(T)                                                                 \
    template void lowpass<T>(const T*, size_t, T*, size_t, size3_t, size_t, float, float, Stream&);     \
    template void highpass<T>(const T*, size_t, T*, size_t, size3_t, size_t, float, float, Stream&);    \
    template void bandpass<T>(const T*, size_t, T*, size_t, size3_t, size_t, float, float, float, float, Stream&)

    NOA_INSTANTIATE_FILTERS_(float);
    NOA_INSTANTIATE_FILTERS_(double);
    NOA_INSTANTIATE_FILTERS_(cfloat_t);
    NOA_INSTANTIATE_FILTERS_(cdouble_t);
}
