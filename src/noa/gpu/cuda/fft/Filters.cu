#include "hip/hip_runtime.h"
#include "noa/common/Math.h"
#include "noa/common/Profiler.h"
#include "noa/gpu/cuda/fft/Exception.h"
#include "noa/gpu/cuda/fft/Filters.h"
#include "noa/gpu/cuda/memory/Copy.h"

// Commons:
namespace {
    using namespace noa;
    constexpr dim3 THREADS(32, 8);

    enum class Type {
        LOWPASS,
        HIGHPASS
    };

    template<bool IS_CENTERED>
    __forceinline__ __device__ int getFrequency_(int idx, int dim) {
        if constexpr(IS_CENTERED)
            return idx - dim / 2;
        else
            return idx < (dim + 1) / 2 ? idx : idx - dim;
        return 0;
    }

    template<bool IS_SRC_CENTERED, bool IS_DST_CENTERED>
    __forceinline__ __device__ int getOutputIndex_(int i_idx, [[maybe_unused]] int dim) {
        (void) dim;
        if constexpr (IS_SRC_CENTERED == IS_DST_CENTERED)
            return i_idx;
        else if constexpr (IS_SRC_CENTERED)
            return noa::math::iFFTShift(i_idx, dim);
        else
            return noa::math::FFTShift(i_idx, dim);
        return 0;
    }

    template<bool IS_CENTERED>
    __forceinline__ __device__ float getNormalizedFrequencySqd(int3_t gid, int3_t shape, float3_t norm) {
        float3_t distance_sqd(gid.x,
                              getFrequency_<IS_CENTERED>(gid.y, shape.y),
                              getFrequency_<IS_CENTERED>(gid.z, shape.z));
        distance_sqd *= norm;
        return math::dot(distance_sqd, distance_sqd);
    }
}

namespace {
    template<Type PASS>
    inline __device__ float getSoftWindow_(float freq_cutoff, float freq_width, float freq) {
        constexpr float PI = math::Constants<float>::PI;
        float filter;
        if constexpr (PASS == Type::LOWPASS) {
            if (freq <= freq_cutoff)
                filter = 1;
            else if (freq_cutoff + freq_width <= freq)
                filter = 0;
            else
                filter = (1.f + math::cos(PI * (freq_cutoff - freq) / freq_width)) * 0.5f;
        } else if constexpr (PASS == Type::HIGHPASS) {
            if (freq_cutoff <= freq)
                filter = 1;
            else if (freq <= freq_cutoff - freq_width)
                filter = 0;
            else
                filter = (1.f + math::cos(PI * (freq - freq_cutoff) / freq_width)) * 0.5f;
        }
        return filter;
    }

    template<Type PASS>
    inline __device__ float getHardWindow_(float freq_cutoff_sqd, float freq_sqd) {
        float filter;
        if constexpr (PASS == Type::LOWPASS) {
            if (freq_cutoff_sqd < freq_sqd)
                filter = 0;
            else
                filter = 1;
        } else if constexpr (PASS == Type::HIGHPASS) {
            if (freq_sqd < freq_cutoff_sqd)
                filter = 0;
            else
                filter = 1;
        }
        return filter;
    }

    template<bool IS_SRC_CENTERED, bool IS_DST_CENTERED, Type PASS, bool HAS_WIDTH, typename T>
    __global__ __launch_bounds__(THREADS.x * THREADS.y)
    void singlePass_(const T* inputs, uint3_t input_pitch, T* outputs, uint3_t output_pitch,
                     int3_t shape, float3_t norm, float cutoff, [[maybe_unused]] float width, uint blocks_x) {
        using real_t = noa::traits::value_type_t<T>;

        // Get the current indexes within the input.
        const uint2_t idx = coordinates(blockIdx.x, blocks_x);
        const int3_t gid(THREADS.x * idx.x + threadIdx.x,
                         THREADS.y * idx.y + threadIdx.y,
                         blockIdx.y);
        if (gid.x >= shape.x / 2 + 1 || gid.y >= shape.y)
            return;

        // Offset to current batch.
        inputs += blockIdx.z * elements(input_pitch);
        outputs += blockIdx.z * elements(output_pitch);

        // Offset to current output frequency.
        const int64_t oy = getOutputIndex_<IS_SRC_CENTERED, IS_DST_CENTERED>(gid.y, shape.y);
        const int64_t oz = getOutputIndex_<IS_SRC_CENTERED, IS_DST_CENTERED>(gid.z, shape.z);
        outputs += (oz * output_pitch.y + oy) * output_pitch.x + gid.x;

        // Get filter for current input frequency.
        real_t filter;
        if constexpr (HAS_WIDTH) {
            const float frequency = math::sqrt(getNormalizedFrequencySqd<IS_SRC_CENTERED>(gid, shape, norm));
            filter = static_cast<real_t>(getSoftWindow_<PASS>(cutoff, width, frequency));
        } else {
            const float frequency_sqd = getNormalizedFrequencySqd<IS_SRC_CENTERED>(gid, shape, norm);
            filter = static_cast<real_t>(getHardWindow_<PASS>(cutoff * cutoff, frequency_sqd));
            (void) width;
        }

        // Save to output.
        *outputs = inputs ? inputs[(gid.z * input_pitch.y + gid.y) * input_pitch.x + gid.x] * filter : filter;
    }

    template<bool IS_SRC_CENTERED, bool IS_DST_CENTERED, bool HAS_WIDTH, typename T>
    __global__ __launch_bounds__(THREADS.x * THREADS.y)
    void bandPass_(const T* inputs, uint3_t input_pitch, T* outputs, uint3_t output_pitch,
                   int3_t shape, float3_t norm,
                   float cutoff_1, float cutoff_2,
                   float width_1, float width_2, uint blocks_x) {
        using real_t = noa::traits::value_type_t<T>;

        // Get the current indexes within the input.
        const uint2_t idx = coordinates(blockIdx.x, blocks_x);
        const int3_t gid(THREADS.x * idx.x + threadIdx.x,
                         THREADS.y * idx.y + threadIdx.y,
                         blockIdx.y);
        if (gid.x >= shape.x / 2 + 1 || gid.y >= shape.y)
            return;

        // Offset to current batch.
        inputs += blockIdx.z * elements(input_pitch);
        outputs += blockIdx.z * elements(output_pitch);

        // Offset to current output frequency.
        const int64_t oy = getOutputIndex_<IS_SRC_CENTERED, IS_DST_CENTERED>(gid.y, shape.y);
        const int64_t oz = getOutputIndex_<IS_SRC_CENTERED, IS_DST_CENTERED>(gid.z, shape.z);
        outputs += (oz * output_pitch.y + oy) * output_pitch.x + gid.x;

        // Get filter for current input frequency.
        real_t filter;
        if constexpr (HAS_WIDTH) {
            const float frequency = math::sqrt(getNormalizedFrequencySqd<IS_SRC_CENTERED>(gid, shape, norm));
            filter = static_cast<real_t>(getSoftWindow_<Type::HIGHPASS>(cutoff_1, width_1, frequency) *
                                         getSoftWindow_<Type::LOWPASS>(cutoff_2, width_2, frequency));
        } else {
            const float frequency_sqd = getNormalizedFrequencySqd<IS_SRC_CENTERED>(gid, shape, norm);
            filter = static_cast<real_t>(getSoftWindow_<Type::HIGHPASS>(cutoff_1 * cutoff_1, width_1, frequency_sqd) *
                                         getSoftWindow_<Type::LOWPASS>(cutoff_2 * cutoff_2, width_2, frequency_sqd));
            (void) width_1;
            (void) width_2;
        }

        // Save to output.
        *outputs = inputs ? inputs[(gid.z * input_pitch.y + gid.y) * input_pitch.x + gid.x] * filter : filter;
    }
}

namespace {
    template<Type PASS, ::noa::fft::Remap REMAP, typename T>
    void launchSinglePass_(const T* inputs, size3_t input_pitch, T* outputs, size3_t output_pitch,
                           size3_t shape, size_t batches, float cutoff, float width, cuda::Stream& stream) {
        using Layout = ::noa::fft::Layout;
        constexpr auto REMAP_ = static_cast<uint8_t>(REMAP);
        constexpr bool IS_SRC_CENTERED = REMAP_ & Layout::SRC_CENTERED;
        constexpr bool IS_DST_CENTERED = REMAP_ & Layout::DST_CENTERED;
        if constexpr (REMAP_ & Layout::SRC_FULL || REMAP_ & Layout::DST_FULL) {
            static_assert(traits::always_false_v<T>);
        }
        NOA_ASSERT(inputs != outputs || IS_SRC_CENTERED == IS_DST_CENTERED);

        const int3_t s_shape(shape);
        float3_t norm(shape / 2 * 2 + size3_t{shape == 1});
        norm = 1.f / norm;

        const uint blocks_x = math::divideUp(s_shape.x / 2 + 1, static_cast<int>(THREADS.x));
        const uint blocks_y = math::divideUp(s_shape.y, static_cast<int>(THREADS.y));
        const dim3 blocks(blocks_x * blocks_y, s_shape.z, batches);
        if (width > 1e-6f) {
            singlePass_<IS_SRC_CENTERED, IS_DST_CENTERED, PASS, true><<<blocks, THREADS, 0, stream.id()>>>(
                    inputs, uint3_t{input_pitch}, outputs, uint3_t{output_pitch},
                    s_shape, norm, cutoff, width, blocks_x);
        } else {
            singlePass_<IS_SRC_CENTERED, IS_DST_CENTERED, PASS, false><<<blocks, THREADS, 0, stream.id()>>>(
                    inputs, uint3_t{input_pitch}, outputs, uint3_t{output_pitch},
                    s_shape, norm, cutoff, width, blocks_x);
        }
        NOA_THROW_IF(hipGetLastError());
    }
}

namespace noa::cuda::fft {
    template<Remap REMAP, typename T>
    void lowpass(const T* inputs, size3_t input_pitch, T* outputs, size3_t output_pitch,
                 size3_t shape, size_t batches, float cutoff, float width, Stream& stream) {
        NOA_PROFILE_FUNCTION();
        launchSinglePass_<Type::LOWPASS, REMAP>(
                inputs, input_pitch, outputs, output_pitch, shape, batches, cutoff, width, stream);
    }

    template<Remap REMAP, typename T>
    void highpass(const T* inputs, size3_t input_pitch, T* outputs, size3_t output_pitch,
                  size3_t shape, size_t batches, float cutoff, float width, Stream& stream) {
        NOA_PROFILE_FUNCTION();
        launchSinglePass_<Type::HIGHPASS, REMAP>(
                inputs, input_pitch, outputs, output_pitch, shape, batches, cutoff, width, stream);
    }

    template<Remap REMAP, typename T>
    NOA_HOST void bandpass(const T* inputs, size3_t input_pitch, T* outputs, size3_t output_pitch, size3_t shape,
                           size_t batches, float cutoff1, float cutoff2, float width1, float width2, Stream& stream) {
        using Layout = ::noa::fft::Layout;
        constexpr auto REMAP_ = static_cast<uint8_t>(REMAP);
        constexpr bool IS_SRC_CENTERED = REMAP_ & Layout::SRC_CENTERED;
        constexpr bool IS_DST_CENTERED = REMAP_ & Layout::DST_CENTERED;
        if constexpr (REMAP_ & Layout::SRC_FULL || REMAP_ & Layout::DST_FULL) {
            static_assert(traits::always_false_v<T>);
        }
        NOA_ASSERT(inputs != outputs || IS_SRC_CENTERED == IS_DST_CENTERED);
        NOA_PROFILE_FUNCTION();
        const int3_t s_shape(shape);
        float3_t norm(shape / 2 * 2 + size3_t{shape == 1});
        norm = 1.f / norm;

        const uint blocks_x = math::divideUp(s_shape.x / 2 + 1, static_cast<int>(THREADS.x));
        const uint blocks_y = math::divideUp(s_shape.y, static_cast<int>(THREADS.y));
        const dim3 blocks(blocks_x * blocks_y, s_shape.z, batches);
        if (width1 > 1e-6f || width2 > 1e-6f) {
            bandPass_<IS_SRC_CENTERED, IS_DST_CENTERED, true><<<blocks, THREADS, 0, stream.id()>>>(
                    inputs, uint3_t{input_pitch}, outputs, uint3_t{output_pitch}, s_shape, norm,
                    cutoff1, cutoff2, width1, width2, blocks_x);
        } else {
            bandPass_<IS_SRC_CENTERED, IS_DST_CENTERED, false><<<blocks, THREADS, 0, stream.id()>>>(
                    inputs, uint3_t{input_pitch}, outputs, uint3_t{output_pitch}, s_shape, norm,
                    cutoff1, cutoff2, width1, width2, blocks_x);
        }
        NOA_THROW_IF(hipGetLastError());
    }

    #define NOA_INSTANTIATE_FILTERS_(T)                                                                                         \
    template void lowpass<Remap::H2H, T>(const T*, size3_t, T*, size3_t, size3_t, size_t, float, float, Stream&);               \
    template void highpass<Remap::H2H,T>(const T*, size3_t, T*, size3_t, size3_t, size_t, float, float, Stream&);               \
    template void bandpass<Remap::H2H,T>(const T*, size3_t, T*, size3_t, size3_t, size_t, float, float, float, float, Stream&); \
    template void lowpass<Remap::H2HC, T>(const T*, size3_t, T*, size3_t, size3_t, size_t, float, float, Stream&);              \
    template void highpass<Remap::H2HC,T>(const T*, size3_t, T*, size3_t, size3_t, size_t, float, float, Stream&);              \
    template void bandpass<Remap::H2HC,T>(const T*, size3_t, T*, size3_t, size3_t, size_t, float, float, float, float, Stream&);\
    template void lowpass<Remap::HC2H, T>(const T*, size3_t, T*, size3_t, size3_t, size_t, float, float, Stream&);              \
    template void highpass<Remap::HC2H,T>(const T*, size3_t, T*, size3_t, size3_t, size_t, float, float, Stream&);              \
    template void bandpass<Remap::HC2H,T>(const T*, size3_t, T*, size3_t, size3_t, size_t, float, float, float, float, Stream&);\
    template void lowpass<Remap::HC2HC, T>(const T*, size3_t, T*, size3_t, size3_t, size_t, float, float, Stream&);             \
    template void highpass<Remap::HC2HC,T>(const T*, size3_t, T*, size3_t, size3_t, size_t, float, float, Stream&);             \
    template void bandpass<Remap::HC2HC,T>(const T*, size3_t, T*, size3_t, size3_t, size_t, float, float, float, float, Stream&)

    NOA_INSTANTIATE_FILTERS_(half_t);
    NOA_INSTANTIATE_FILTERS_(float);
    NOA_INSTANTIATE_FILTERS_(double);
    NOA_INSTANTIATE_FILTERS_(chalf_t);
    NOA_INSTANTIATE_FILTERS_(cfloat_t);
    NOA_INSTANTIATE_FILTERS_(cdouble_t);
}
