#include "hip/hip_runtime.h"
#include "noa/common/Math.h"
#include "noa/common/Profiler.h"
#include "noa/gpu/cuda/fft/Exception.h"
#include "noa/gpu/cuda/fft/Filters.h"
#include "noa/gpu/cuda/memory/Copy.h"

// Commons:
namespace {
    using namespace noa;
    constexpr dim3 THREADS(32, 8);

    enum class Type {
        LOWPASS,
        HIGHPASS
    };

    template<bool IS_CENTERED>
    __forceinline__ __device__ int getFrequency_(int idx, int dim) {
        if constexpr(IS_CENTERED)
            return idx - dim / 2;
        else
            return idx < (dim + 1) / 2 ? idx : idx - dim;
        return 0; // false warning: missing return statement at end of non-void function
    }

    __forceinline__ __device__ float getNormalizedFrequencySqd(int3_t gid, int3_t shape, float3_t norm) {
        float3_t distance_sqd(gid.x,
                              getFrequency_<false>(gid.y, shape.y),
                              getFrequency_<false>(gid.z, shape.z));
        distance_sqd *= norm;
        return math::dot(distance_sqd, distance_sqd);
    }
}

// Soft edges (Hann window):
namespace {
    template<Type PASS>
    inline __device__ float getSoftWindow_(float freq_cutoff, float freq_width, float freq) {
        constexpr float PI = math::Constants<float>::PI;
        float filter;
        if constexpr (PASS == Type::LOWPASS) {
            if (freq <= freq_cutoff)
                filter = 1;
            else if (freq_cutoff + freq_width <= freq)
                filter = 0;
            else
                filter = (1.f + math::cos(PI * (freq_cutoff - freq) / freq_width)) * 0.5f;
        } else if constexpr (PASS == Type::HIGHPASS) {
            if (freq_cutoff <= freq)
                filter = 1;
            else if (freq <= freq_cutoff - freq_width)
                filter = 0;
            else
                filter = (1.f + math::cos(PI * (freq - freq_cutoff) / freq_width)) * 0.5f;
        }
        return filter;
    }

    template<Type PASS, typename T>
    __global__ __launch_bounds__(THREADS.x * THREADS.y)
    void singlePassSoft_(const T* inputs, uint inputs_pitch, T* outputs, uint outputs_pitch,
                         int3_t shape, float3_t norm, float freq_cutoff, float freq_width, uint blocks_x) {
        using real_t = noa::traits::value_type_t<T>;

        // Get the current indexes.
        const uint2_t idx = coordinates(blockIdx.x, blocks_x);
        const int3_t gid(THREADS.x * idx.x + threadIdx.x,
                         THREADS.y * idx.y + threadIdx.y,
                         blockIdx.y);
        if (gid.x >= shape.x / 2 + 1 || gid.y >= shape.y)
            return;

        // Offset to current batch.
        inputs += blockIdx.z * rows(shape) * inputs_pitch;
        outputs += blockIdx.z * rows(shape) * outputs_pitch;

        // Apply filter.
        const float frequency = math::sqrt(getNormalizedFrequencySqd(gid, shape, norm));
        const auto filter = static_cast<real_t>(getSoftWindow_<PASS>(freq_cutoff, freq_width, frequency));
        outputs[(gid.z * shape.y + gid.y) * outputs_pitch + gid.x] =
                inputs[(gid.z * shape.y + gid.y) * inputs_pitch + gid.x] * filter;
    }

    template<Type PASS, typename T>
    __global__ __launch_bounds__(THREADS.x * THREADS.y)
    void singlePassSoft_(T* output_filter, uint output_filter_pitch,
                         int3_t shape, float3_t norm, float freq_cutoff, float freq_width) {
        const int3_t gid(THREADS.x * blockIdx.x + threadIdx.x,
                         THREADS.y * blockIdx.y + threadIdx.y,
                         blockIdx.z);
        if (gid.x >= shape.x / 2 + 1 || gid.y >= shape.y)
            return;

        const float frequency = math::sqrt(getNormalizedFrequencySqd(gid, shape, norm));
        const auto filter = static_cast<T>(getSoftWindow_<PASS>(freq_cutoff, freq_width, frequency));
        output_filter[(gid.z * shape.y + gid.y) * output_filter_pitch + gid.x] = filter;
    }

    template<typename T>
    __global__ __launch_bounds__(THREADS.x * THREADS.y)
    void bandPassSoft_(const T* inputs, uint inputs_pitch, T* outputs, uint outputs_pitch,
                       int3_t shape, float3_t norm,
                       float freq_cutoff_1, float freq_cutoff_2,
                       float freq_width_1, float freq_width_2, uint blocks_x) {
        using real_t = noa::traits::value_type_t<T>;

        const uint2_t idx = coordinates(blockIdx.x, blocks_x);
        const int3_t gid(THREADS.x * idx.x + threadIdx.x,
                         THREADS.y * idx.y + threadIdx.y,
                         blockIdx.y);
        if (gid.x >= shape.x / 2 + 1 || gid.y >= shape.y)
            return;

        inputs += blockIdx.z * rows(shape) * inputs_pitch;
        outputs += blockIdx.z * rows(shape) * outputs_pitch;

        const float frequency = math::sqrt(getNormalizedFrequencySqd(gid, shape, norm));
        const float filter = getSoftWindow_<Type::HIGHPASS>(freq_cutoff_1, freq_width_1, frequency) *
                             getSoftWindow_<Type::LOWPASS>(freq_cutoff_2, freq_width_2, frequency);
        outputs[(gid.z * shape.y + gid.y) * outputs_pitch + gid.x] =
                inputs[(gid.z * shape.y + gid.y) * inputs_pitch + gid.x] * static_cast<real_t>(filter);
    }

    template<typename T>
    __global__ __launch_bounds__(THREADS.x * THREADS.y)
    void bandPassSoft_(T* output_filter, uint output_filter_pitch, int3_t shape, float3_t norm,
                       float freq_cutoff_1, float freq_cutoff_2, float freq_width_1, float freq_width_2) {
        using real_t = noa::traits::value_type_t<T>;

        const int3_t gid(THREADS.x * blockIdx.x + threadIdx.x,
                         THREADS.y * blockIdx.y + threadIdx.y,
                         blockIdx.z);
        if (gid.x >= shape.x / 2 + 1 || gid.y >= shape.y)
            return;

        const float frequency = math::sqrt(getNormalizedFrequencySqd(gid, shape, norm));
        const float filter = getSoftWindow_<Type::HIGHPASS>(freq_cutoff_1, freq_width_1, frequency) *
                             getSoftWindow_<Type::LOWPASS>(freq_cutoff_2, freq_width_2, frequency);
        output_filter[(gid.z * shape.y + gid.y) * output_filter_pitch + gid.x] = static_cast<real_t>(filter);
    }
}

// Hard edges:
namespace {
    template<Type PASS>
    inline __device__ float getHardWindow_(float freq_cutoff_sqd, float freq_sqd) {
        float filter;
        if constexpr (PASS == Type::LOWPASS) {
            if (freq_cutoff_sqd < freq_sqd)
                filter = 0;
            else
                filter = 1;
        } else if constexpr (PASS == Type::HIGHPASS) {
            if (freq_sqd < freq_cutoff_sqd)
                filter = 0;
            else
                filter = 1;
        }
        return filter;
    }

    template<Type PASS, typename T>
    __global__ __launch_bounds__(THREADS.x * THREADS.y)
    void singlePassHard_(const T* inputs, uint inputs_pitch, T* outputs, uint outputs_pitch,
                         int3_t shape, float3_t norm, float freq_cutoff_sqd, uint blocks_x) {
        using real_t = noa::traits::value_type_t<T>;

        // Get the current indexes.
        const uint2_t idx = coordinates(blockIdx.x, blocks_x);
        const int3_t gid(THREADS.x * idx.x + threadIdx.x,
                         THREADS.y * idx.y + threadIdx.y,
                         blockIdx.y);
        if (gid.x >= shape.x / 2 + 1 || gid.y >= shape.y)
            return;

        // Offset to current batch.
        inputs += blockIdx.z * rows(shape) * inputs_pitch;
        outputs += blockIdx.z * rows(shape) * outputs_pitch;

        // Apply filter.
        const float frequency_sqd = getNormalizedFrequencySqd(gid, shape, norm);
        const auto filter = static_cast<real_t>(getHardWindow_<PASS>(freq_cutoff_sqd, frequency_sqd));
        outputs[(gid.z * shape.y + gid.y) * outputs_pitch + gid.x] =
                inputs[(gid.z * shape.y + gid.y) * inputs_pitch + gid.x] * filter;
    }

    template<Type PASS, typename T>
    __global__ __launch_bounds__(THREADS.x * THREADS.y)
    void singlePassHard_(T* output_filter, uint output_filter_pitch,
                         int3_t shape, float3_t norm, float freq_cutoff_sqd) {
        using real_t = noa::traits::value_type_t<T>;

        const int3_t gid(THREADS.x * blockIdx.x + threadIdx.x,
                         THREADS.y * blockIdx.y + threadIdx.y,
                         blockIdx.z);
        if (gid.x >= shape.x / 2 + 1 || gid.y >= shape.y)
            return;

        const float frequency_sqd = getNormalizedFrequencySqd(gid, shape, norm);
        const auto filter = static_cast<real_t>(getHardWindow_<PASS>(freq_cutoff_sqd, frequency_sqd));
        output_filter[(gid.z * shape.y + gid.y) * output_filter_pitch + gid.x] = filter;
    }

    template<typename T>
    __global__ __launch_bounds__(THREADS.x * THREADS.y)
    void bandPassHard_(const T* inputs, uint inputs_pitch, T* outputs, uint outputs_pitch,
                       int3_t shape, float3_t norm, float freq_cutoff_sqd_1, float freq_cutoff_sqd_2,
                       uint blocks_x) {
        using real_t = noa::traits::value_type_t<T>;

        const uint2_t idx = coordinates(blockIdx.x, blocks_x);
        const int3_t gid(THREADS.x * idx.x + threadIdx.x,
                         THREADS.y * idx.y + threadIdx.y,
                         blockIdx.y);
        if (gid.x >= shape.x / 2 + 1 || gid.y >= shape.y)
            return;

        inputs += blockIdx.z * rows(shape) * inputs_pitch;
        outputs += blockIdx.z * rows(shape) * outputs_pitch;

        const float frequency_sqd = getNormalizedFrequencySqd(gid, shape, norm);
        const float filter = getHardWindow_<Type::HIGHPASS>(freq_cutoff_sqd_1, frequency_sqd) *
                             getHardWindow_<Type::LOWPASS>(freq_cutoff_sqd_2, frequency_sqd);
        outputs[(gid.z * shape.y + gid.y) * outputs_pitch + gid.x] =
                inputs[(gid.z * shape.y + gid.y) * inputs_pitch + gid.x] * static_cast<real_t>(filter);
    }

    template<typename T>
    __global__ __launch_bounds__(THREADS.x * THREADS.y)
    void bandPassHard_(T* output_filter, uint output_filter_pitch, int3_t shape, float3_t norm,
                       float freq_cutoff_sqd_1, float freq_cutoff_sqd_2) {
        using real_t = noa::traits::value_type_t<T>;

        const int3_t gid(THREADS.x * blockIdx.x + threadIdx.x,
                         THREADS.y * blockIdx.y + threadIdx.y,
                         blockIdx.z);
        if (gid.x >= shape.x / 2 + 1 || gid.y >= shape.y)
            return;

        const float frequency_sqd = getNormalizedFrequencySqd(gid, shape, norm);
        const float filter = getHardWindow_<Type::HIGHPASS>(freq_cutoff_sqd_1, frequency_sqd) *
                             getHardWindow_<Type::LOWPASS>(freq_cutoff_sqd_2, frequency_sqd);
        output_filter[(gid.z * shape.y + gid.y) * output_filter_pitch + gid.x] = static_cast<real_t>(filter);
    }

    template<Type PASS, typename T>
    void singlePass(const T* inputs, size_t inputs_pitch, T* outputs, size_t outputs_pitch,
                    size3_t shape, size_t batches,
                    float freq_cutoff, float freq_width, cuda::Stream& stream) {
        NOA_PROFILE_FUNCTION();
        int3_t s_shape(shape);
        float3_t norm(shape.x / 2 * 2,
                      shape.y > 1 ? shape.y / 2 * 2 : 1,
                      shape.z > 1 ? shape.z / 2 * 2 : 1);
        norm = 1.f / norm;

        const float freq_cutoff_sqd = freq_cutoff * freq_cutoff;
        uint blocks_x = math::divideUp(s_shape.x / 2 + 1, static_cast<int>(THREADS.x));
        uint blocks_y = math::divideUp(s_shape.y, static_cast<int>(THREADS.y));
        if (inputs) {
            dim3 blocks(blocks_x * blocks_y, s_shape.z, batches);
            if (freq_width > 1e-6f) {
                singlePassSoft_<PASS><<<blocks, THREADS, 0, stream.id()>>>(
                        inputs, inputs_pitch, outputs, outputs_pitch, s_shape, norm, freq_cutoff, freq_width, blocks_x);
            } else {
                singlePassHard_<PASS><<<blocks, THREADS, 0, stream.id()>>>(
                        inputs, inputs_pitch, outputs, outputs_pitch, s_shape, norm, freq_cutoff_sqd, blocks_x);
            }
        } else {
            if constexpr(!traits::is_complex_v<T>) {
                dim3 blocks(blocks_x, blocks_y, s_shape.z);
                if (freq_width > 1e-6f) {
                    singlePassSoft_<PASS><<<blocks, THREADS, 0, stream.id()>>>(
                            outputs, outputs_pitch, s_shape, norm, freq_cutoff, freq_width);
                } else {
                    singlePassHard_<PASS><<<blocks, THREADS, 0, stream.id()>>>(
                            outputs, outputs_pitch, s_shape, norm, freq_cutoff_sqd);
                }
                const size_t elements = outputs_pitch * rows(shape);
                for (size_t batch = 1; batch < batches; ++batch)
                    cuda::memory::copy(outputs, outputs + elements * batch, elements, stream);
            } else {
                NOA_THROW_FUNC("(low|high)pass", "Cannot compute a filter of complex type");
            }
        }
        NOA_THROW_IF(hipGetLastError());
    }
}

namespace noa::cuda::fft {
    template<typename T>
    void lowpass(const T* inputs, size_t inputs_pitch, T* outputs, size_t outputs_pitch,
                 size3_t shape, size_t batches,
                 float freq_cutoff, float freq_width, Stream& stream) {
        NOA_PROFILE_FUNCTION();
        singlePass<Type::LOWPASS>(inputs, inputs_pitch, outputs, outputs_pitch,
                                  shape, batches, freq_cutoff, freq_width, stream);
    }

    template<typename T>
    void highpass(const T* inputs, size_t inputs_pitch, T* outputs, size_t outputs_pitch,
                  size3_t shape, size_t batches,
                  float freq_cutoff, float freq_width, Stream& stream) {
        NOA_PROFILE_FUNCTION();
        singlePass<Type::HIGHPASS>(inputs, inputs_pitch, outputs, outputs_pitch,
                                   shape, batches, freq_cutoff, freq_width, stream);
    }

    template<typename T>
    void bandpass(const T* inputs, size_t inputs_pitch, T* outputs, size_t outputs_pitch,
                  size3_t shape, size_t batches,
                  float freq_cutoff_1, float freq_cutoff_2, float freq_width_1, float freq_width_2, Stream& stream) {
        NOA_PROFILE_FUNCTION();
        int3_t s_shape(shape);
        float3_t norm(shape.x / 2 * 2,
                      shape.y > 1 ? shape.y / 2 * 2 : 1,
                      shape.z > 1 ? shape.z / 2 * 2 : 1);
        norm = 1.f / norm;

        const float freq_cutoff_sqd_1 = freq_cutoff_1 * freq_cutoff_1;
        const float freq_cutoff_sqd_2 = freq_cutoff_2 * freq_cutoff_2;
        uint blocks_x = math::divideUp(s_shape.x / 2 + 1, static_cast<int>(THREADS.x));
        uint blocks_y = math::divideUp(s_shape.y, static_cast<int>(THREADS.y));
        if (inputs) {
            dim3 blocks(blocks_x * blocks_y, s_shape.z, batches);
            if (freq_width_1 > 1e-6f || freq_width_2 > 1e-6f) {
                bandPassSoft_<<<blocks, THREADS, 0, stream.id()>>>(
                        inputs, inputs_pitch, outputs, outputs_pitch, s_shape, norm,
                        freq_cutoff_1, freq_cutoff_2, freq_width_1, freq_width_2, blocks_x);
            } else {
                bandPassHard_<<<blocks, THREADS, 0, stream.id()>>>(
                        inputs, inputs_pitch, outputs, outputs_pitch, s_shape, norm,
                        freq_cutoff_sqd_1, freq_cutoff_sqd_2, blocks_x);
            }
        } else {
            if constexpr(!traits::is_complex_v<T>) {
                dim3 blocks(blocks_x, blocks_y, s_shape.z);
                if (freq_width_1 > 1e-6f || freq_width_2 > 1e-6f) {
                    bandPassSoft_<<<blocks, THREADS, 0, stream.id()>>>(
                            outputs, outputs_pitch, s_shape, norm,
                            freq_cutoff_1, freq_cutoff_2, freq_width_1, freq_width_2);
                } else {
                    bandPassHard_<<<blocks, THREADS, 0, stream.id()>>>(
                            outputs, outputs_pitch, s_shape, norm,
                            freq_cutoff_sqd_1, freq_cutoff_sqd_2);
                }
                const size_t elements = outputs_pitch * rows(shape);
                for (size_t batch = 1; batch < batches; ++batch)
                    cuda::memory::copy(outputs, outputs + elements * batch, elements, stream);
            } else {
                NOA_THROW_FUNC("(low|high)pass", "Cannot compute a filter of complex type");
            }
        }

        NOA_THROW_IF(hipGetLastError());
    }

    #define NOA_INSTANTIATE_FILTERS_(T)                                                                 \
    template void lowpass<T>(const T*, size_t, T*, size_t, size3_t, size_t, float, float, Stream&);     \
    template void highpass<T>(const T*, size_t, T*, size_t, size3_t, size_t, float, float, Stream&);    \
    template void bandpass<T>(const T*, size_t, T*, size_t, size3_t, size_t, float, float, float, float, Stream&)

    NOA_INSTANTIATE_FILTERS_(float);
    NOA_INSTANTIATE_FILTERS_(double);
    NOA_INSTANTIATE_FILTERS_(cfloat_t);
    NOA_INSTANTIATE_FILTERS_(cdouble_t);
}
