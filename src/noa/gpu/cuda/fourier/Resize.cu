#include "hip/hip_runtime.h"
#include "noa/gpu/cuda/fourier/Resize.h"
#include "noa/Math.h"
#include "noa/gpu/cuda/Memory.h"

// Forward declare kernels.
namespace Noa::CUDA::Fourier::Kernels {
    template<class T>
    static __global__ void crop(const T* in, uint3_t shape_in, uint pitch_in,
                                T* out, uint3_t shape_out, uint pitch_out);

    template<class T>
    static __global__ void cropFull(const T* in, uint3_t shape_in, uint pitch_in,
                                    T* out, uint3_t shape_out, uint pitch_out);

    template<class T>
    static __global__ void pad(const T* in, uint3_t shape_in, uint pitch_in,
                               T* out, uint3_t shape_out, uint pitch_out);

    template<class T>
    static __global__ void padFull(const T* in, uint3_t shape_in, uint pitch_in,
                                   T* out, uint3_t shape_out, uint pitch_out);
}

// Host functions.
namespace Noa::CUDA::Fourier {
    template<typename T>
    void crop(const T* in, size3_t shape_in, size_t pitch_in, T* out, size3_t shape_out, size_t pitch_out,
              uint batches, Stream& stream) {
        if (shape_in == shape_out) {
            Memory::copy(in, pitch_in * sizeof(T), out, pitch_out * sizeof(T), getShapeFFT(shape_in), stream);
            return;
        }
        uint3_t old_shape(shape_in), new_shape(shape_out);
        uint workers_per_row = Math::min(256U, getNextMultipleOf(new_shape.x / 2U + 1, Limits::warp_size));
        dim3 rows_to_process{new_shape.y, new_shape.z, batches};
        Kernels::crop<<<rows_to_process, workers_per_row, 0, stream.get()>>>(
                in, old_shape, static_cast<uint>(pitch_in), out, new_shape, static_cast<uint>(pitch_out));
        NOA_THROW_IF(hipPeekAtLastError());
    }

    template<typename T>
    void cropFull(const T* in, size3_t shape_in, size_t pitch_in, T* out, size3_t shape_out, size_t pitch_out,
                  uint batches, Stream& stream) {
        if (shape_in == shape_out) {
            Memory::copy(in, pitch_in * sizeof(T), out, pitch_out * sizeof(T), shape_in, stream);
            return;
        }
        uint3_t old_shape(shape_in), new_shape(shape_out);
        uint workers_per_row = Math::min(256U, getNextMultipleOf(new_shape.x, Limits::warp_size));
        dim3 rows_to_process{new_shape.y, new_shape.z, batches};
        Kernels::cropFull<<<rows_to_process, workers_per_row, 0, stream.get()>>>(
                in, old_shape, static_cast<uint>(pitch_in), out, new_shape, static_cast<uint>(pitch_out));
        NOA_THROW_IF(hipPeekAtLastError());
    }

    // TODO: not a priority, but maybe replace memset with a single kernel that loops through output.
    template<typename T>
    void pad(const T* in, size3_t shape_in, size_t pitch_in, T* out, size3_t shape_out, size_t pitch_out,
             uint batches, Stream& stream) {
        if (shape_in == shape_out) {
            Memory::copy(in, pitch_in * sizeof(T), out, pitch_out * sizeof(T), getShapeFFT(shape_in), stream);
            return;
        }
        NOA_THROW_IF(hipMemsetAsync(out, 0, pitch_out * shape_out.y * shape_out.z * sizeof(T), stream.get()));

        uint3_t old_shape(shape_in), new_shape(shape_out);
        uint workers_per_row = Math::min(256U, getNextMultipleOf(old_shape.x / 2U + 1U, Limits::warp_size));
        dim3 rows_to_process{old_shape.y, old_shape.z, batches};
        Kernels::pad<<<rows_to_process, workers_per_row, 0, stream.get()>>>(
                in, old_shape, static_cast<uint>(pitch_in), out, new_shape, static_cast<uint>(pitch_out));
        NOA_THROW_IF(hipPeekAtLastError());
    }

    // TODO: not a priority, but maybe replace memset with kernel that loops through output.
    template<typename T>
    void padFull(const T* in, size3_t shape_in, size_t pitch_in, T* out, size3_t shape_out, size_t pitch_out,
                 uint batches, Stream& stream) {
        if (shape_in == shape_out) {
            Memory::copy(in, pitch_in * sizeof(T), out, pitch_out * sizeof(T), shape_in, stream);
            return;
        }
        NOA_THROW_IF(hipMemsetAsync(out, 0, pitch_out * shape_out.y * shape_out.z * sizeof(T), stream.get()));

        uint3_t old_shape(shape_in), new_shape(shape_out);
        uint workers_per_row = Math::min(256U, getNextMultipleOf(old_shape.x, Limits::warp_size));
        dim3 rows_to_process{old_shape.y, old_shape.z, batches};
        Kernels::padFull<<<rows_to_process, workers_per_row, 0, stream.get()>>>(
                in, old_shape, static_cast<uint>(pitch_in), out, new_shape, static_cast<uint>(pitch_out));
        NOA_THROW_IF(hipPeekAtLastError());
    }
}

// Kernel definitions.
namespace Noa::CUDA::Fourier::Kernels {
    template<class T>
    __global__ void crop(const T* in, uint3_t shape_in, uint pitch_in, T* out, uint3_t shape_out, uint pitch_out) {
        // Rebase to the current batch.
        in += pitch_in * shape_in.y * shape_in.z * blockIdx.z;
        out += pitch_out * shape_out.y * shape_out.z * blockIdx.z;

        // Rebase to the current row.
        uint out_y = blockIdx.x, out_z = blockIdx.y;
        uint in_y = out_y < (shape_out.y + 1) / 2 ? out_y : out_y + shape_in.y - shape_out.y;
        uint in_z = out_z < (shape_out.z + 1) / 2 ? out_z : out_z + shape_in.z - shape_out.z;

        in += (in_z * shape_in.y + in_y) * pitch_in;
        out += (out_z * shape_out.y + out_y) * pitch_out;

        // Copy the row.
        for (uint x = threadIdx.x; x < shape_out.x / 2 + 1; x += blockDim.x)
            out[x] = in[x];
    }

    template<class T>
    __global__ void cropFull(const T* in, uint3_t shape_in, uint pitch_in, T* out, uint3_t shape_out, uint pitch_out) {
        // Rebase to the current batch.
        in += pitch_in * shape_in.y * shape_in.z * blockIdx.z;
        out += pitch_out * shape_out.y * shape_out.z * blockIdx.z;

        // Rebase to the current row.
        uint out_y = blockIdx.x, out_z = blockIdx.y;
        uint in_y = out_y < (shape_out.y + 1) / 2 ? out_y : out_y + shape_in.y - shape_out.y;
        uint in_z = out_z < (shape_out.z + 1) / 2 ? out_z : out_z + shape_in.z - shape_out.z;
        in += (in_z * shape_in.y + in_y) * pitch_in;
        out += (out_z * shape_out.y + out_y) * pitch_out;

        // Similarly to the other dimension, if half in new x is passed, add offset to skip cropped elements.
        for (uint out_x = threadIdx.x; out_x < shape_out.x; out_x += blockDim.x) {
            uint in_x = out_x < (shape_out.x + 1) / 2 ? out_x : out_x + shape_in.x - shape_out.x;
            out[out_x] = in[in_x];
        }
    }

    template<class T>
    __global__ void pad(const T* in, uint3_t shape_in, uint pitch_in, T* out, uint3_t shape_out, uint pitch_out) {
        // Rebase to the current batch.
        in += pitch_in * shape_in.y * shape_in.z * blockIdx.z;
        out += pitch_out * shape_out.y * shape_out.z * blockIdx.z;

        // Rebase to the current row.
        uint in_y = blockIdx.x, in_z = blockIdx.y;
        uint out_y = in_y < (shape_in.y + 1) / 2 ? in_y : in_y + shape_out.y - shape_in.y;
        uint out_z = in_z < (shape_in.z + 1) / 2 ? in_z : in_z + shape_out.z - shape_in.z;
        in += (in_z * shape_in.y + in_y) * pitch_in;
        out += (out_z * shape_out.y + out_y) * pitch_out;

        // Copy the row.
        for (uint x = threadIdx.x; x < shape_in.x / 2 + 1; x += blockDim.x)
            out[x] = in[x];
    }

    template<class T>
    __global__ void padFull(const T* in, uint3_t shape_in, uint pitch_in, T* out, uint3_t shape_out, uint pitch_out) {
        // Rebase to the current batch.
        in += pitch_in * shape_in.y * shape_in.z * blockIdx.z;
        out += pitch_out * shape_out.y * shape_out.z * blockIdx.z;

        // Rebase to the current row.
        uint in_y = blockIdx.x, in_z = blockIdx.y;
        uint out_y = in_y < (shape_in.y + 1) / 2 ? in_y : in_y + shape_out.y - shape_in.y;
        uint out_z = in_z < (shape_in.z + 1) / 2 ? in_z : in_z + shape_out.z - shape_in.z;
        in += (in_z * shape_in.y + in_y) * pitch_in;
        out += (out_z * shape_out.y + out_y) * pitch_out;

        // Similarly to the other dimension, if half in new x is passed, add offset to skip padded elements.
        for (uint in_x = threadIdx.x; in_x < shape_in.x; in_x += blockDim.x) {
            uint out_x = in_x < (shape_in.x + 1) / 2 ? in_x : in_x + shape_out.x - shape_in.x;
            out[out_x] = in[in_x];
        }
    }
}

// Instantiate supported types.
namespace Noa::CUDA::Fourier {
    #define INSTANTIATE_CROP(T) \
    template void crop<T>(const T*, size3_t, size_t, T*, size3_t, size_t, uint, Stream&);       \
    template void cropFull<T>(const T*, size3_t, size_t, T*, size3_t, size_t, uint, Stream&);   \
    template void pad<T>(const T*, size3_t, size_t, T*, size3_t, size_t, uint, Stream&);        \
    template void padFull<T>(const T*, size3_t, size_t, T*, size3_t, size_t, uint, Stream&)

    INSTANTIATE_CROP(cfloat_t);
    INSTANTIATE_CROP(float);
}
