#include "hip/hip_runtime.h"
#include "noa/Math.h"
#include "noa/gpu/cuda/fourier/Exception.h"
#include "noa/gpu/cuda/fourier/Filters.h"

// TODO Test 2D block to reduce idle threads / divergence, e.g. BLOCK_SIZE(32, 8).

// Commons:
namespace {
    enum class Type { LOWPASS, HIGHPASS };

    inline __device__ float getDistanceSquared_(size_t dimension, uint half, size_t idx) {
        float dist = idx >= half ? static_cast<float>(idx) - static_cast<float>(dimension) : static_cast<float>(idx);
        dist /= static_cast<float>(dimension);
        dist *= dist;
        return dist;
    }
}

// Soft edges (Hann window):
namespace {
    using namespace Noa;

    template<Type PASS>
    inline __device__ float getSoftWindow_(float freq_cutoff, float freq_width, float freq) {
        constexpr float PI = Math::Constants<float>::PI;
        float filter;
        if constexpr (PASS == Type::LOWPASS) {
            if (freq <= freq_cutoff)
                filter = 1;
            else if (freq_cutoff + freq_width <= freq)
                filter = 0;
            else
                filter = (1.f + Math::cos(PI * (freq_cutoff - freq) / freq_width)) * 0.5f;
        } else if constexpr (PASS == Type::HIGHPASS) {
            if (freq_cutoff <= freq)
                filter = 1;
            else if (freq <= freq_cutoff - freq_width)
                filter = 0;
            else
                filter = (1.f + Math::cos(PI * (freq - freq_cutoff) / freq_width)) * 0.5f;
        }
        return filter;
    }

    template<Type PASS, typename T>
    __global__ void singlePassSoft_(T* inputs, uint pitch_inputs, T* outputs, uint pitch_outputs,
                                    uint3_t shape, uint3_t half, float freq_cutoff, float freq_width, uint batches) {
        using real_t = Noa::Traits::value_type_t<T>;
        uint y = blockIdx.x, z = blockIdx.y;
        inputs += (z * shape.y + y) * pitch_inputs;
        outputs += (z * shape.y + y) * pitch_outputs;

        uint elements_inputs = 0, elements_outputs = 0;
        if (batches) {
            uint rows = getRows(shape);
            elements_inputs = pitch_inputs * rows;
            elements_outputs = pitch_outputs * rows;
        }

        float3_t distance_sqd;
        distance_sqd.z = getDistanceSquared_(shape.z, half.z, z);
        distance_sqd.y = getDistanceSquared_(shape.y, half.y, y);

        float frequency;
        real_t filter;
        for (uint x = threadIdx.x; x < half.x; x += blockDim.x) {
            distance_sqd.x = static_cast<float>(x) / static_cast<float>(shape.x);
            distance_sqd.x *= distance_sqd.x;
            frequency = Math::sqrt(Math::sum(distance_sqd)); // from 0 to 0.5
            filter = static_cast<real_t>(getSoftWindow_<PASS>(freq_cutoff, freq_width, frequency));
            for (uint batch = 0; batch < batches; ++batch)
                outputs[batch * elements_outputs + x] = inputs[batch * elements_inputs + x] * filter;
        }
    }

    template<Type PASS, typename T>
    __global__ void singlePassSoft_(T* output_filter, uint pitch_output_filter, uint3_t shape, uint3_t half,
                                    float freq_cutoff, float freq_width) {
        using real_t = Noa::Traits::value_type_t<T>;
        uint y = blockIdx.x, z = blockIdx.y;
        output_filter += (z * shape.y + y) * pitch_output_filter;

        float3_t distance_sqd;
        distance_sqd.z = getDistanceSquared_(shape.z, half.z, z);
        distance_sqd.y = getDistanceSquared_(shape.y, half.y, y);

        float frequency, filter;
        for (uint x = threadIdx.x; x < half.x; x += blockDim.x) {
            distance_sqd.x = static_cast<float>(x) / static_cast<float>(shape.x);
            distance_sqd.x *= distance_sqd.x;
            frequency = Math::sqrt(Math::sum(distance_sqd)); // from 0 to 0.5
            filter = getSoftWindow_<PASS>(freq_cutoff, freq_width, frequency);
            output_filter[x] = static_cast<real_t>(filter);
        }
    }

    template<typename T>
    __global__ void bandPassSoft_(T* inputs, uint pitch_inputs, T* outputs, uint pitch_outputs,
                                  uint3_t shape, uint3_t half, float freq_cutoff_1, float freq_cutoff_2,
                                  float freq_width_1, float freq_width_2, uint batches) {
        using real_t = Noa::Traits::value_type_t<T>;
        uint y = blockIdx.x, z = blockIdx.y;
        inputs += (z * shape.y + y) * pitch_inputs;
        outputs += (z * shape.y + y) * pitch_outputs;

        uint elements_inputs = 0, elements_outputs = 0;
        if (batches) {
            uint rows = getRows(shape);
            elements_inputs = pitch_inputs * rows;
            elements_outputs = pitch_outputs * rows;
        }

        float3_t distance_sqd;
        distance_sqd.z = getDistanceSquared_(shape.z, half.z, z);
        distance_sqd.y = getDistanceSquared_(shape.y, half.y, y);

        float frequency, filter;
        for (uint x = threadIdx.x; x < half.x; x += blockDim.x) {
            distance_sqd.x = static_cast<float>(x) / static_cast<float>(shape.x);
            distance_sqd.x *= distance_sqd.x;
            frequency = Math::sqrt(Math::sum(distance_sqd)); // from 0 to 0.5
            filter = getSoftWindow_<Type::HIGHPASS>(freq_cutoff_1, freq_width_1, frequency);
            filter *= getSoftWindow_<Type::LOWPASS>(freq_cutoff_2, freq_width_2, frequency);
            for (uint batch = 0; batch < batches; ++batch)
                outputs[batch * elements_outputs + x] =
                        inputs[batch * elements_inputs + x] * static_cast<real_t>(filter);
        }
    }

    template<typename T>
    __global__ void bandPassSoft_(T* output_filter, uint pitch_output_filter, uint3_t shape, uint3_t half,
                                  float freq_cutoff_1, float freq_cutoff_2, float freq_width_1, float freq_width_2) {
        using real_t = Noa::Traits::value_type_t<T>;
        uint y = blockIdx.x, z = blockIdx.y;
        output_filter += (z * shape.y + y) * pitch_output_filter;

        float3_t distance_sqd;
        distance_sqd.z = getDistanceSquared_(shape.z, half.z, z);
        distance_sqd.y = getDistanceSquared_(shape.y, half.y, y);

        float frequency, filter;
        for (uint x = threadIdx.x; x < half.x; x += blockDim.x) {
            distance_sqd.x = static_cast<float>(x) / static_cast<float>(shape.x);
            distance_sqd.x *= distance_sqd.x;
            frequency = Math::sqrt(Math::sum(distance_sqd)); // from 0 to 0.5
            filter = getSoftWindow_<Type::HIGHPASS>(freq_cutoff_1, freq_width_1, frequency);
            filter *= getSoftWindow_<Type::LOWPASS>(freq_cutoff_2, freq_width_2, frequency);
            output_filter[x] = static_cast<real_t>(filter);
        }
    }
}

// Hard edges:
namespace {
    using namespace Noa;

    template<Type PASS>
    inline __device__ float getHardWindow_(float freq_cutoff_sqd, float freq_sqd) {
        float filter;
        if constexpr (PASS == Type::LOWPASS) {
            if (freq_cutoff_sqd < freq_sqd)
                filter = 0;
            else
                filter = 1;
        } else if constexpr (PASS == Type::HIGHPASS) {
            if (freq_sqd < freq_cutoff_sqd)
                filter = 0;
            else
                filter = 1;
        }
        return filter;
    }

    template<Type PASS, typename T>
    __global__ void singlePassHard_(T* inputs, uint pitch_inputs, T* outputs, uint pitch_outputs,
                                    uint3_t shape, uint3_t half, float freq_cutoff, uint batches) {
        using real_t = Noa::Traits::value_type_t<T>;
        uint y = blockIdx.x, z = blockIdx.y;
        inputs += (z * shape.y + y) * pitch_inputs;
        outputs += (z * shape.y + y) * pitch_outputs;

        uint elements_inputs = 0, elements_outputs = 0;
        if (batches) {
            uint rows = getRows(shape);
            elements_inputs = pitch_inputs * rows;
            elements_outputs = pitch_outputs * rows;
        }

        float3_t distance_sqd;
        distance_sqd.z = getDistanceSquared_(shape.z, half.z, z);
        distance_sqd.y = getDistanceSquared_(shape.y, half.y, y);

        float freq_cutoff_sqd = freq_cutoff * freq_cutoff;
        float frequency_sqd;
        real_t filter;
        for (uint x = threadIdx.x; x < half.x; x += blockDim.x) {
            distance_sqd.x = static_cast<float>(x) / static_cast<float>(shape.x);
            distance_sqd.x *= distance_sqd.x;
            frequency_sqd = Math::sum(distance_sqd); // from 0 to 0.25
            filter = static_cast<real_t>(getHardWindow_<PASS>(freq_cutoff_sqd, frequency_sqd));
            for (uint batch = 0; batch < batches; ++batch)
                outputs[batch * elements_outputs + x] = inputs[batch * elements_inputs + x] * filter;
        }
    }

    template<Type PASS, typename T>
    __global__ void singlePassHard_(T* output_filter, uint pitch_output_filter,
                                    uint3_t shape, uint3_t half, float freq_cutoff) {
        using real_t = Noa::Traits::value_type_t<T>;
        uint y = blockIdx.x, z = blockIdx.y;
        output_filter += (z * shape.y + y) * pitch_output_filter;

        float3_t distance_sqd;
        distance_sqd.z = getDistanceSquared_(shape.z, half.z, z);
        distance_sqd.y = getDistanceSquared_(shape.y, half.y, y);

        float freq_cutoff_sqd = freq_cutoff * freq_cutoff;
        float frequency_sqd, filter;
        for (uint x = threadIdx.x; x < half.x; x += blockDim.x) {
            distance_sqd.x = static_cast<float>(x) / static_cast<float>(shape.x);
            distance_sqd.x *= distance_sqd.x;
            frequency_sqd = Math::sum(distance_sqd); // from 0 to 0.25
            filter = getHardWindow_<PASS>(freq_cutoff_sqd, frequency_sqd);
            output_filter[x] = static_cast<real_t>(filter);
        }
    }

    template<typename T>
    __global__ void bandPassHard_(T* inputs, uint pitch_inputs, T* outputs, uint pitch_outputs,
                                  uint3_t shape, uint3_t half, float freq_cutoff_1, float freq_cutoff_2, uint batches) {
        using real_t = Noa::Traits::value_type_t<T>;
        uint y = blockIdx.x, z = blockIdx.y;
        inputs += (z * shape.y + y) * pitch_inputs;
        outputs += (z * shape.y + y) * pitch_outputs;

        uint elements_inputs = 0, elements_outputs = 0;
        if (batches) {
            uint rows = getRows(shape);
            elements_inputs = pitch_inputs * rows;
            elements_outputs = pitch_outputs * rows;
        }

        float3_t distance_sqd;
        distance_sqd.z = getDistanceSquared_(shape.z, half.z, z);
        distance_sqd.y = getDistanceSquared_(shape.y, half.y, y);

        float freq_cutoff_sqd_1 = freq_cutoff_1 * freq_cutoff_1;
        float freq_cutoff_sqd_2 = freq_cutoff_2 * freq_cutoff_2;
        float frequency_sqd, filter;
        for (uint x = threadIdx.x; x < half.x; x += blockDim.x) {
            distance_sqd.x = static_cast<float>(x) / static_cast<float>(shape.x);
            distance_sqd.x *= distance_sqd.x;
            frequency_sqd = Math::sum(distance_sqd); // from 0 to 0.25
            filter = getHardWindow_<Type::HIGHPASS>(freq_cutoff_sqd_1, frequency_sqd);
            filter *= getHardWindow_<Type::LOWPASS>(freq_cutoff_sqd_2, frequency_sqd);
            for (uint batch = 0; batch < batches; ++batch)
                outputs[batch * elements_outputs + x] =
                        inputs[batch * elements_inputs + x] * static_cast<real_t>(filter);
        }
    }

    template<typename T>
    __global__ void bandPassHard_(T* output_filter, uint pitch_output_filter, uint3_t shape, uint3_t half,
                                  float freq_cutoff_1, float freq_cutoff_2) {
        using real_t = Noa::Traits::value_type_t<T>;
        uint y = blockIdx.x, z = blockIdx.y;
        output_filter += (z * shape.y + y) * pitch_output_filter;

        float3_t distance_sqd;
        distance_sqd.z = getDistanceSquared_(shape.z, half.z, z);
        distance_sqd.y = getDistanceSquared_(shape.y, half.y, y);

        float freq_cutoff_sqd_1 = freq_cutoff_1 * freq_cutoff_1;
        float freq_cutoff_sqd_2 = freq_cutoff_2 * freq_cutoff_2;
        float frequency_sqd, filter;
        for (uint x = threadIdx.x; x < half.x; x += blockDim.x) {
            distance_sqd.x = static_cast<float>(x) / static_cast<float>(shape.x);
            distance_sqd.x *= distance_sqd.x;
            frequency_sqd = Math::sum(distance_sqd); // from 0 to 0.25
            filter = getHardWindow_<Type::HIGHPASS>(freq_cutoff_sqd_1, frequency_sqd);
            filter *= getHardWindow_<Type::LOWPASS>(freq_cutoff_sqd_2, frequency_sqd);
            output_filter[x] = static_cast<real_t>(filter);
        }
    }
}

namespace Noa::CUDA::Fourier {
    template<typename T>
    void lowpass(T* inputs, size_t pitch_inputs, T* outputs, size_t pitch_outputs, size3_t shape,
                 float freq_cutoff, float freq_width, uint batches, Stream& stream) {
        uint3_t tmp_shape(shape);
        uint3_t half(tmp_shape / 2U + 1U);

        uint threads = Math::min(128U, Math::nextMultipleOf(half.x, 32U));
        dim3 blocks(tmp_shape.y, tmp_shape.z);
        if (freq_width > 1e-8f) {
            NOA_CUDA_LAUNCH(blocks, threads, 0, stream.id(),
                            singlePassSoft_<Type::LOWPASS>,
                            inputs, pitch_inputs, outputs, pitch_outputs, tmp_shape, half,
                            freq_cutoff, freq_width, batches);
        } else {
            NOA_CUDA_LAUNCH(blocks, threads, 0, stream.id(),
                            singlePassHard_<Type::LOWPASS>,
                            inputs, pitch_inputs, outputs, pitch_outputs, tmp_shape, half,
                            freq_cutoff, batches);
        }
    }

    template<typename T>
    void lowpass(T* output_lowpass, size_t pitch_output_lowpass, size3_t shape,
                 float freq_cutoff, float freq_width, Stream& stream) {
        uint3_t tmp_shape(shape);
        uint3_t half(tmp_shape / 2U + 1U);

        uint threads = Math::min(128U, Math::nextMultipleOf(half.x, 32U));
        dim3 blocks(tmp_shape.y, tmp_shape.z);
        if (freq_width > 1e-8f) {
            NOA_CUDA_LAUNCH(blocks, threads, 0, stream.id(),
                            singlePassSoft_<Type::LOWPASS>,
                            output_lowpass, pitch_output_lowpass, tmp_shape, half, freq_cutoff, freq_width);
        } else {
            NOA_CUDA_LAUNCH(blocks, threads, 0, stream.id(),
                            singlePassHard_<Type::LOWPASS>,
                            output_lowpass, pitch_output_lowpass, tmp_shape, half, freq_cutoff);
        }
    }

    template<typename T>
    void highpass(T* inputs, size_t pitch_inputs, T* outputs, size_t pitch_outputs, size3_t shape,
                  float freq_cutoff, float freq_width, uint batches, Stream& stream) {
        uint3_t tmp_shape(shape);
        uint3_t half(tmp_shape / 2U + 1U);

        uint threads = Math::min(128U, Math::nextMultipleOf(half.x, 32U));
        dim3 blocks(tmp_shape.y, tmp_shape.z);
        if (freq_width > 1e-8f) {
            NOA_CUDA_LAUNCH(blocks, threads, 0, stream.id(),
                            singlePassSoft_<Type::HIGHPASS>,
                            inputs, pitch_inputs, outputs, pitch_outputs, tmp_shape, half,
                            freq_cutoff, freq_width, batches);
        } else {
            NOA_CUDA_LAUNCH(blocks, threads, 0, stream.id(),
                            singlePassHard_<Type::HIGHPASS>,
                            inputs, pitch_inputs, outputs, pitch_outputs, tmp_shape, half,
                            freq_cutoff, batches);
        }
    }

    template<typename T>
    void highpass(T* output_highpass, size_t pitch_output_highpass, size3_t shape,
                  float freq_cutoff, float freq_width, Stream& stream) {
        uint3_t tmp_shape(shape);
        uint3_t half(tmp_shape / 2U + 1U);

        uint threads = Math::min(128U, Math::nextMultipleOf(half.x, 32U));
        dim3 blocks(tmp_shape.y, tmp_shape.z);
        if (freq_width > 1e-8f) {
            NOA_CUDA_LAUNCH(blocks, threads, 0, stream.id(),
                            singlePassSoft_<Type::HIGHPASS>,
                            output_highpass, pitch_output_highpass, tmp_shape, half,
                            freq_cutoff, freq_width);
        } else {
            NOA_CUDA_LAUNCH(blocks, threads, 0, stream.id(),
                            singlePassHard_<Type::HIGHPASS>,
                            output_highpass, pitch_output_highpass, tmp_shape, half, freq_cutoff);
        }
    }

    template<typename T>
    void bandpass(T* inputs, size_t pitch_inputs, T* outputs, size_t pitch_outputs, size3_t shape,
                  float freq_cutoff_1, float freq_cutoff_2, float freq_width_1, float freq_width_2,
                  uint batches, Stream& stream) {
        uint3_t tmp_shape(shape);
        uint3_t half(tmp_shape / 2U + 1U);

        uint threads = Math::min(128U, Math::nextMultipleOf(half.x, 32U));
        dim3 blocks(tmp_shape.y, tmp_shape.z);
        if (freq_width_1 > 1e-8f || freq_width_2 > 1e-8f) {
            NOA_CUDA_LAUNCH(blocks, threads, 0, stream.id(),
                            bandPassSoft_,
                            inputs, pitch_inputs, outputs, pitch_outputs, tmp_shape, half,
                            freq_cutoff_1, freq_cutoff_2, freq_width_1, freq_width_2, batches);
        } else {
            NOA_CUDA_LAUNCH(blocks, threads, 0, stream.id(),
                            bandPassHard_,
                            inputs, pitch_inputs, outputs, pitch_outputs, tmp_shape, half,
                            freq_cutoff_1, freq_cutoff_2, batches);
        }
    }

    template<typename T>
    void bandpass(T* output_bandpass, size_t pitch_output_bandpass, size3_t shape,
                  float freq_cutoff_1, float freq_cutoff_2, float freq_width_1, float freq_width_2, Stream& stream) {
        uint3_t tmp_shape(shape);
        uint3_t half(tmp_shape / 2U + 1U);

        uint threads = Math::min(128U, Math::nextMultipleOf(half.x, 32U));
        dim3 blocks(tmp_shape.y, tmp_shape.z);
        if (freq_width_1 > 1e-8f || freq_width_2 > 1e-8f) {
            NOA_CUDA_LAUNCH(blocks, threads, 0, stream.id(),
                            bandPassSoft_,
                            output_bandpass, pitch_output_bandpass, tmp_shape, half,
                            freq_cutoff_1, freq_cutoff_2, freq_width_1, freq_width_2);
        } else {
            NOA_CUDA_LAUNCH(blocks, threads, 0, stream.id(),
                            bandPassHard_,
                            output_bandpass, pitch_output_bandpass, tmp_shape, half,
                            freq_cutoff_1, freq_cutoff_2);
        }
    }

    #define INSTANTIATE_FILTERS(REAL, CPLX)                                                                         \
    template void lowpass<CPLX>(CPLX*, size_t, CPLX*, size_t, size3_t, float, float, uint, Stream&);                \
    template void lowpass<REAL>(REAL*, size_t, REAL*, size_t, size3_t, float, float, uint, Stream&);                \
    template void lowpass<REAL>(REAL*, size_t, size3_t, float, float, Stream&);                                     \
    template void highpass<CPLX>(CPLX*, size_t, CPLX*, size_t, size3_t, float, float, uint, Stream&);               \
    template void highpass<REAL>(REAL*, size_t, REAL*, size_t, size3_t, float, float, uint, Stream&);               \
    template void highpass<REAL>(REAL*, size_t, size3_t, float, float, Stream&);                                    \
    template void bandpass<CPLX>(CPLX*, size_t, CPLX*, size_t, size3_t, float, float, float, float, uint, Stream&); \
    template void bandpass<REAL>(REAL*, size_t, REAL*, size_t, size3_t, float, float, float, float, uint, Stream&); \
    template void bandpass<REAL>(REAL*, size_t, size3_t, float, float, float, float, Stream&)

    INSTANTIATE_FILTERS(float, cfloat_t);
    INSTANTIATE_FILTERS(double, cdouble_t);
}
