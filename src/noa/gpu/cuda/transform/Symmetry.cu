#include "hip/hip_runtime.h"
#include "noa/common/Profiler.h"
#include "noa/gpu/cuda/Exception.h"
#include "noa/gpu/cuda/memory/Copy.h"
#include "noa/gpu/cuda/memory/PtrArray.h"
#include "noa/gpu/cuda/memory/PtrDevice.h"
#include "noa/gpu/cuda/memory/PtrTexture.h"
#include "noa/gpu/cuda/transform/Interpolate.h"
#include "noa/gpu/cuda/transform/Symmetry.h"

// TODO(TF) Is it faster to replace the first copy by a texture fetching? Or, take the input array directly and
//          perform the copy in the kernel...

namespace {
    using namespace ::noa;
    constexpr dim3 THREADS(16, 16);

    template<typename T, InterpMode INTERP>
    __global__ void __launch_bounds__(THREADS.x * THREADS.y)
    symmetrize_(hipTextureObject_t texture, T* output, size_t output_pitch, size2_t shape,
                const float33_t* matrix, uint count, float scaling, float2_t center) {

        const uint2_t gid(blockIdx.x * blockDim.x + threadIdx.x,
                          blockIdx.y * blockDim.y + threadIdx.y);
        if (gid.x >= shape.x || gid.y >= shape.y)
            return;

        float2_t coordinates(gid.x, gid.y);
        coordinates -= center;

        T value = 0;
        for (uint i = 0; i < count; ++i) {
            float2_t i_coordinates(float22_t(matrix[i]) * coordinates);
            value += cuda::transform::tex2D<T, INTERP>(texture, i_coordinates + center + 0.5f);
        }

        output += gid.y * output_pitch + gid.x;
        *output += value;
        *output *= scaling;
    }

    template<typename T, InterpMode INTERP>
    __global__ void __launch_bounds__(THREADS.x * THREADS.y)
    symmetrize_(hipTextureObject_t texture, T* output, size_t output_pitch, size3_t shape,
                const float33_t* matrix, uint count, float scaling, float3_t center) {

        const uint3_t gid(blockIdx.x * blockDim.x + threadIdx.x,
                          blockIdx.y * blockDim.y + threadIdx.y,
                          blockIdx.z);
        if (gid.x >= shape.x || gid.y >= shape.y) // z cannot be out
            return;

        float3_t coordinates(gid.x, gid.y, gid.z);
        coordinates -= center;

        T value = 0;
        for (uint i = 0; i < count; ++i) {
            float3_t i_coordinates(matrix[i] * coordinates);
            value += cuda::transform::tex3D<T, INTERP>(texture, i_coordinates + center + 0.5f);
        }

        output += (gid.z * shape.y + gid.y) * output_pitch + gid.x;
        *output += value;
        *output *= scaling;
    }

    template<typename T, typename SHAPE, typename CENTER>
    void launch_(hipTextureObject_t texture, InterpMode texture_interp,
                 T* output, size_t output_pitch, SHAPE shape,
                 const float33_t* matrix, uint count, float scaling, CENTER center,
                 dim3 blocks, cuda::Stream& stream) {
        switch (texture_interp) {
            case INTERP_NEAREST:
                symmetrize_<T, INTERP_NEAREST><<<blocks, THREADS, 0, stream.id()>>>(
                        texture, output, output_pitch, shape, matrix, count, scaling, center);
                break;
            case INTERP_LINEAR:
                symmetrize_<T, INTERP_LINEAR><<<blocks, THREADS, 0, stream.id()>>>(
                        texture, output, output_pitch, shape, matrix, count, scaling, center);
                break;
            case INTERP_COSINE:
                symmetrize_<T, INTERP_COSINE><<<blocks, THREADS, 0, stream.id()>>>(
                        texture, output, output_pitch, shape, matrix, count, scaling, center);
                break;
            case INTERP_CUBIC:
                symmetrize_<T, INTERP_CUBIC><<<blocks, THREADS, 0, stream.id()>>>(
                        texture, output, output_pitch, shape, matrix, count, scaling, center);
                break;
            case INTERP_CUBIC_BSPLINE:
                symmetrize_<T, INTERP_CUBIC_BSPLINE><<<blocks, THREADS, 0, stream.id()>>>(
                        texture, output, output_pitch, shape, matrix, count, scaling, center);
                break;
            case INTERP_LINEAR_FAST:
                symmetrize_<T, INTERP_LINEAR_FAST><<<blocks, THREADS, 0, stream.id()>>>(
                        texture, output, output_pitch, shape, matrix, count, scaling, center);
                break;
            case INTERP_COSINE_FAST:
                symmetrize_<T, INTERP_COSINE_FAST><<<blocks, THREADS, 0, stream.id()>>>(
                        texture, output, output_pitch, shape, matrix, count, scaling, center);
                break;
            case INTERP_CUBIC_BSPLINE_FAST:
                symmetrize_<T, INTERP_CUBIC_BSPLINE_FAST><<<blocks, THREADS, 0, stream.id()>>>(
                        texture, output, output_pitch, shape, matrix, count, scaling, center);
                break;
            default:
                NOA_THROW_FUNC("symmetrize(2|3)D", "{} is not supported", texture_interp);
        }
        NOA_THROW_IF(hipGetLastError());
    }
}

// -- Using textures -- //
namespace noa::cuda::transform {
    template<typename T>
    void symmetrize2D(hipTextureObject_t texture, InterpMode texture_interp_mode,
                      T* output, size_t output_pitch, size2_t shape, const float33_t* symmetry_matrices,
                      size_t symmetry_count, float2_t symmetry_center, Stream& stream) {
        NOA_PROFILE_FUNCTION();
        size3_t shape_3d(shape.x, shape.y, 1);
        hipResourceDesc resource = memory::PtrTexture<T>::getResource(texture);
        memory::copy(resource.res.array.array, output, output_pitch, shape_3d, stream);

        if (symmetry_count == 0)
            return;

        uint2_t tmp(shape.x, shape.y);
        const dim3 blocks(noa::math::divideUp(tmp.x, THREADS.x),
                          noa::math::divideUp(tmp.y, THREADS.y));

        float scaling = 1 / static_cast<float>(symmetry_count + 1);
        launch_(texture, texture_interp_mode, output, output_pitch, shape,
                symmetry_matrices, symmetry_count, scaling, symmetry_center, blocks, stream);
    }

    template<typename T>
    void symmetrize3D(hipTextureObject_t texture, InterpMode texture_interp_mode,
                      T* output, size_t output_pitch, size3_t shape, const float33_t* symmetry_matrices,
                      size_t symmetry_count, float3_t symmetry_center, Stream& stream) {
        NOA_PROFILE_FUNCTION();
        hipResourceDesc resource = memory::PtrTexture<T>::getResource(texture);
        memory::copy(resource.res.array.array, output, output_pitch, shape, stream);

        if (symmetry_count == 0)
            return;

        uint3_t tmp(shape);
        const dim3 blocks(noa::math::divideUp(tmp.x, THREADS.x),
                          noa::math::divideUp(tmp.y, THREADS.y),
                          tmp.z);

        float scaling = 1 / static_cast<float>(symmetry_count + 1);
        launch_(texture, texture_interp_mode, output, output_pitch, shape,
                symmetry_matrices, symmetry_count, scaling, symmetry_center, blocks, stream);
    }
}

// -- Using arrays -- //
namespace noa::cuda::transform {
    template<bool PREFILTER, typename T>
    void symmetrize2D(const T* inputs, size_t input_pitch, T* outputs, size_t output_pitch,
                      size2_t shape, size_t batches, const Symmetry& symmetry, float2_t symmetry_center,
                      InterpMode interp_mode, Stream& stream) {
        NOA_PROFILE_FUNCTION();
        const size3_t shape_3d(shape.x, shape.y, 1);
        const size_t count = symmetry.count();
        if (!count) {
            memory::copy(inputs, input_pitch, outputs, output_pitch, shape_3d, batches, stream);
            stream.synchronize(); // be consistent
            return;
        }

        const float33_t* matrices = symmetry.matrices();
        memory::PtrDevice<float33_t> d_matrices(count);
        memory::copy(matrices, d_matrices.get(), count, stream);

        const T* tmp;
        size_t pitch;
        if (PREFILTER && (interp_mode == INTERP_CUBIC_BSPLINE || interp_mode == INTERP_CUBIC_BSPLINE_FAST)) {
            transform::bspline::prefilter2D(inputs, input_pitch, outputs, output_pitch, shape, batches, stream);
            tmp = outputs;
            pitch = output_pitch;
        } else {
            tmp = inputs;
            pitch = input_pitch;
        }

        memory::PtrArray<T> buffer(shape_3d);
        memory::PtrTexture<T> texture(buffer.get(), interp_mode, BORDER_ZERO);
        for (uint batch = 0; batch < batches; ++batch) {
            size_t offset = batch * pitch * shape.y;
            T* output = outputs + batch * output_pitch * shape.y;
            memory::copy(tmp + offset, pitch, buffer.get(), shape_3d, stream);
            symmetrize2D(texture.get(), interp_mode, output, output_pitch, shape,
                         d_matrices.get(), count, symmetry_center, stream);
        }
        stream.synchronize();
    }

    template<bool PREFILTER, typename T>
    void symmetrize3D(const T* inputs, size_t input_pitch, T* outputs, size_t output_pitch,
                      size3_t shape, size_t batches, const Symmetry& symmetry, float3_t symmetry_center,
                      InterpMode interp_mode, Stream& stream) {
        NOA_PROFILE_FUNCTION();
        const size_t count = symmetry.count();
        if (!count) {
            memory::copy(inputs, input_pitch, outputs, output_pitch, shape, batches, stream);
            stream.synchronize();
            return;
        }

        const float33_t* matrices = symmetry.matrices();
        memory::PtrDevice<float33_t> d_matrices(count);
        memory::copy(matrices, d_matrices.get(), count, stream);

        const T* tmp;
        size_t pitch;
        if (PREFILTER && (interp_mode == INTERP_CUBIC_BSPLINE || interp_mode == INTERP_CUBIC_BSPLINE_FAST)) {
            transform::bspline::prefilter3D(inputs, input_pitch, outputs, output_pitch, shape, batches, stream);
            tmp = outputs;
            pitch = output_pitch;
        } else {
            tmp = inputs;
            pitch = input_pitch;
        }

        memory::PtrArray<T> buffer(shape);
        memory::PtrTexture<T> texture(buffer.get(), interp_mode, BORDER_ZERO);
        for (size_t batch = 0; batch < batches; ++batch) {
            size_t offset = batch * pitch * shape.y * shape.z;
            T* output = outputs + batch * output_pitch * shape.y * shape.z;
            memory::copy(tmp + offset, pitch, buffer.get(), shape, stream);
            symmetrize3D(texture.get(), interp_mode, output, output_pitch, shape,
                         d_matrices.get(), count, symmetry_center, stream);
        }
        stream.synchronize();
    }
}

namespace noa::cuda::transform {
    #define NOA_INSTANTIATE_SYMMETRIZE_(T)                                                                                                  \
    template void symmetrize2D<true, T>(const T*, size_t, T*, size_t, size2_t, size_t, const Symmetry&, float2_t, InterpMode, Stream&);     \
    template void symmetrize3D<true, T>(const T*, size_t, T*, size_t, size3_t, size_t, const Symmetry&, float3_t, InterpMode, Stream&);     \
    template void symmetrize2D<false, T>(const T*, size_t, T*, size_t, size2_t, size_t, const Symmetry&, float2_t, InterpMode, Stream&);    \
    template void symmetrize3D<false, T>(const T*, size_t, T*, size_t, size3_t, size_t, const Symmetry&, float3_t, InterpMode, Stream&)

    NOA_INSTANTIATE_SYMMETRIZE_(float);
    NOA_INSTANTIATE_SYMMETRIZE_(cfloat_t);
}
