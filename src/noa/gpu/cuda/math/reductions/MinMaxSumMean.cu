#include "hip/hip_runtime.h"
// Implementation for Math::minMaxSumMean() for contiguous and padded layouts.

#include "noa/gpu/cuda/math/Reductions.h"
#include "noa/gpu/cuda/Exception.h"
#include "noa/Math.h"
#include "noa/gpu/cuda/memory/PtrDevice.h"

namespace {
    using namespace Noa;

    template<typename T>
    __device__ void warpSumReduce_(volatile T* s_data_tid) {
        T t = *s_data_tid;
        t = t + s_data_tid[32];
        *s_data_tid = t;
        t = t + s_data_tid[16];
        *s_data_tid = t;
        t = t + s_data_tid[8];
        *s_data_tid = t;
        t = t + s_data_tid[4];
        *s_data_tid = t;
        t = t + s_data_tid[2];
        *s_data_tid = t;
        t = t + s_data_tid[1];
        *s_data_tid = t;
    }

    template<typename T>
    __device__ void warpMinReduce_(volatile T* s_data_tid) {
        if (s_data_tid[32] < *s_data_tid) *s_data_tid = s_data_tid[32];
        if (s_data_tid[16] < *s_data_tid) *s_data_tid = s_data_tid[16];
        if (s_data_tid[8] < *s_data_tid) *s_data_tid = s_data_tid[8];
        if (s_data_tid[4] < *s_data_tid) *s_data_tid = s_data_tid[4];
        if (s_data_tid[2] < *s_data_tid) *s_data_tid = s_data_tid[2];
        if (s_data_tid[1] < *s_data_tid) *s_data_tid = s_data_tid[1];
    }

    template<typename T>
    __device__ void warpMaxReduce_(volatile T* s_data_tid) {
        if (*s_data_tid < s_data_tid[32]) *s_data_tid = s_data_tid[32];
        if (*s_data_tid < s_data_tid[16]) *s_data_tid = s_data_tid[16];
        if (*s_data_tid < s_data_tid[8]) *s_data_tid = s_data_tid[8];
        if (*s_data_tid < s_data_tid[4]) *s_data_tid = s_data_tid[4];
        if (*s_data_tid < s_data_tid[2]) *s_data_tid = s_data_tid[2];
        if (*s_data_tid < s_data_tid[1]) *s_data_tid = s_data_tid[1];
    }

    template<typename T>
    inline __device__ void inPlaceMinMaxSum_(T* current_min, T* current_max, T* current_sum, T candidate) {
        *current_sum += candidate;
        if (candidate < *current_min) *current_min = candidate;
        if (*current_max < candidate) *current_max = candidate;
    }

    template<typename T>
    inline __device__ void reduceSharedData_(int tid, T* s_mins, T* s_maxs, T* s_sums,
                                             T* output_min, T* output_max, T* output_sum) {
        if (tid < 256) {
            s_sums[tid] += s_sums[tid + 256];
            if (s_mins[tid + 256] < s_mins[tid]) s_mins[tid] = s_mins[tid + 256];
            if (s_maxs[tid] < s_maxs[tid + 256]) s_maxs[tid] = s_maxs[tid + 256];
        }
        __syncthreads();
        if (tid < 128) {
            s_sums[tid] += s_sums[tid + 128];
            if (s_mins[tid + 128] < s_mins[tid]) s_mins[tid] = s_mins[tid + 128];
            if (s_maxs[tid] < s_maxs[tid + 128]) s_maxs[tid] = s_maxs[tid + 128];
        }
        __syncthreads();
        if (tid < 64) {
            s_sums[tid] += s_sums[tid + 64];
            if (s_mins[tid + 64] < s_mins[tid]) s_mins[tid] = s_mins[tid + 64];
            if (s_maxs[tid] < s_maxs[tid + 64]) s_maxs[tid] = s_maxs[tid + 64];
        }
        __syncthreads();
        if (tid < 32) {
            warpSumReduce_(s_sums + tid);
            warpMinReduce_(s_mins + tid);
            warpMaxReduce_(s_maxs + tid);
        }
        if (tid == 0) {
            *output_sum = *s_sums;
            *output_min = *s_mins;
            *output_max = *s_maxs;
        }
    }

    template<int BLOCK_SIZE, typename T>
    inline __device__ void reduceSharedData_(int tid, T* s_mins, T* s_maxs, T* s_sums,
                                             T* out_min, T* out_max, T* out_sum, T* out_mean, T scale) {
        if constexpr (BLOCK_SIZE >= 256) {
            if (tid < 128) {
                s_sums[tid] += s_sums[tid + 128];
                if (s_mins[tid + 128] < s_mins[tid]) s_mins[tid] = s_mins[tid + 128];
                if (s_maxs[tid] < s_maxs[tid + 128]) s_maxs[tid] = s_maxs[tid + 128];
            }
            __syncthreads();

        }
        if constexpr (BLOCK_SIZE >= 128) {
            if (tid < 64) {
                s_sums[tid] += s_sums[tid + 64];
                if (s_mins[tid + 64] < s_mins[tid]) s_mins[tid] = s_mins[tid + 64];
                if (s_maxs[tid] < s_maxs[tid + 64]) s_maxs[tid] = s_maxs[tid + 64];
            }
            __syncthreads();
        }

        if constexpr (BLOCK_SIZE >= 64) {
            if (tid < 32) {
                warpSumReduce_(s_sums + tid);
                warpMinReduce_(s_mins + tid);
                warpMaxReduce_(s_maxs + tid);
            }
        }

        if (tid == 0) {
            if constexpr (BLOCK_SIZE == 32) {
                for (int i = 1; i < 32; ++i) {
                    *s_sums += s_sums[i];
                    if (s_mins[i] < *s_mins) *s_mins = s_mins[i];
                    if (*s_maxs < s_maxs[i]) *s_maxs = s_maxs[i];
                }
            }
            *out_min = *s_mins;
            *out_max = *s_maxs;
            T final_sum = *s_sums;
            if (out_sum)
                *out_sum = final_sum;
            if (out_mean)
                *out_mean = final_sum / scale;
        }
    }

    // Intermediary kernel to reduce large contiguous arrays to max 512 elements.
    namespace Contiguous_ {
        constexpr uint BLOCK_SIZE = 512U;

        template<bool TWO_BY_TWO, typename T>
        __global__ void kernel_(T* input, T* tmp_mins, T* tmp_maxs, T* tmp_sums, uint elements) {
            __shared__ T s_sums[BLOCK_SIZE];
            __shared__ T s_mins[BLOCK_SIZE];
            __shared__ T s_maxs[BLOCK_SIZE];

            T sum = 0, min = *input, max = *input;
            for (uint idx = blockIdx.x * BLOCK_SIZE * 2 + threadIdx.x; idx < elements;
                 idx += BLOCK_SIZE * 2 * gridDim.x) {
                inPlaceMinMaxSum_(&min, &max, &sum, input[idx]);
                if constexpr (TWO_BY_TWO) {
                    inPlaceMinMaxSum_(&min, &max, &sum, input[idx + BLOCK_SIZE]);
                } else {
                    if (idx + BLOCK_SIZE < elements)
                        inPlaceMinMaxSum_(&min, &max, &sum, input[idx + BLOCK_SIZE]);
                }
            }
            s_sums[threadIdx.x] = sum;
            s_mins[threadIdx.x] = min;
            s_maxs[threadIdx.x] = max;
            __syncthreads();

            reduceSharedData_(threadIdx.x, s_mins, s_maxs, s_sums,
                              tmp_mins + blockIdx.x, tmp_maxs + blockIdx.x, tmp_sums + blockIdx.x);
        }

        uint getBlocks_(size_t elements) {
            constexpr uint MAX_BLOCKS = 512U;
            uint blocks = (elements + (BLOCK_SIZE * 2 - 1)) / (BLOCK_SIZE * 2);
            return Noa::Math::min(MAX_BLOCKS, blocks);
        }

        template<typename T>
        void launch_(T* input, T* tmp_mins, T* tmp_maxs, T* tmp_sums,
                     uint elements, uint blocks, hipStream_t stream) {
            bool two_by_two = !(elements % (BLOCK_SIZE * 2));
            if (two_by_two) {
                kernel_<true><<<blocks, BLOCK_SIZE, 0, stream>>>(input, tmp_mins, tmp_maxs, tmp_sums, elements);
            } else {
                kernel_<false><<<blocks, BLOCK_SIZE, 0, stream>>>(input, tmp_mins, tmp_maxs, tmp_sums, elements);
            }
            NOA_THROW_IF(hipPeekAtLastError());
        }
    }

    // Intermediary kernel to reduce large padded arrays to 1-512 elements.
    namespace Padded_ {
        constexpr uint2_t BLOCK_SIZE(32, 16);
        constexpr uint THREADS = BLOCK_SIZE.x * BLOCK_SIZE.y;

        template<bool TWO_BY_TWO, typename T>
        __global__ void kernel_(T* input, uint pitch, T* tmp_mins, T* tmp_maxs, T* tmp_sums, uint2_t shape) {
            uint tid = threadIdx.y * BLOCK_SIZE.x + threadIdx.x;
            __shared__ T s_sums[THREADS];
            __shared__ T s_mins[THREADS];
            __shared__ T s_maxs[THREADS];

            T min = *input, max = *input, sum = 0;
            uint offset;
            for (uint row = BLOCK_SIZE.y * blockIdx.x + threadIdx.y; row < shape.y; row += gridDim.x * BLOCK_SIZE.y) {
                offset = row * pitch;
                if constexpr (TWO_BY_TWO) {
                    for (uint idx = threadIdx.x; idx < shape.x; idx += BLOCK_SIZE.x * 2) {
                        inPlaceMinMaxSum_(&min, &max, &sum, input[offset + idx]);
                        inPlaceMinMaxSum_(&min, &max, &sum, input[offset + idx + BLOCK_SIZE.x]);
                    }
                } else {
                    for (uint idx = threadIdx.x; idx < shape.x; idx += BLOCK_SIZE.x)
                        inPlaceMinMaxSum_(&min, &max, &sum, input[offset + idx]);
                }
            }
            s_sums[tid] = sum;
            s_mins[tid] = min;
            s_maxs[tid] = max;
            __syncthreads();

            reduceSharedData_(tid, s_mins, s_maxs, s_sums,
                              tmp_mins + blockIdx.x, tmp_maxs + blockIdx.x, tmp_sums + blockIdx.x);
        }

        uint getBlocks_(uint rows) {
            constexpr uint MAX_BLOCKS = 512;
            constexpr uint WARPS = BLOCK_SIZE.y;
            uint blocks = (rows + (WARPS - 1)) / WARPS;
            return Noa::Math::min(blocks, MAX_BLOCKS);
        }

        template<typename T>
        void launch_(T* input, uint pitch, T* tmp_mins, T* tmp_maxs, T* tmp_sums,
                     uint2_t shape, uint blocks, hipStream_t stream) {
            dim3 threads(BLOCK_SIZE.x, BLOCK_SIZE.y);
            bool two_by_two = !(shape.x % (BLOCK_SIZE.x * 2));
            if (two_by_two) {
                kernel_<true><<<blocks, threads, 0, stream>>>(input, pitch, tmp_mins, tmp_maxs, tmp_sums, shape);
            } else {
                kernel_<false><<<blocks, threads, 0, stream>>>(input, pitch, tmp_mins, tmp_maxs, tmp_sums, shape);
            }
            NOA_THROW_IF(hipPeekAtLastError());
        }
    }

    namespace Final_ {
        // Kernel to reduce small arrays (one array per block). Computes 4 (2 optional) values per batch.
        uint getThreads_(size_t elements) {
            uint threads = Noa::Math::nextPowerOf2((elements + 1) / 2); // compute at least 2 elements.
            return Noa::Math::clamp(threads, 32U, 256U);
        }

        template<int BLOCK_SIZE, bool TWO_BY_TWO, typename T>
        __global__ void kernel_(T* inputs, uint elements,
                                T* output_mins, T* output_maxs, T* output_sums, T* output_means, T scale) {
            static_assert(BLOCK_SIZE >= 32 && BLOCK_SIZE <= 256);
            __shared__ T s_sums[BLOCK_SIZE];
            __shared__ T s_mins[BLOCK_SIZE];
            __shared__ T s_maxs[BLOCK_SIZE];

            uint batch = blockIdx.x;
            inputs += elements * batch;

            T sum = 0, min = *inputs, max = min;
            for (uint idx = threadIdx.x; idx < elements; idx += BLOCK_SIZE * 2) {
                inPlaceMinMaxSum_(&min, &max, &sum, inputs[idx]);

                if constexpr (TWO_BY_TWO) {
                    inPlaceMinMaxSum_(&min, &max, &sum, inputs[idx + BLOCK_SIZE]);
                } else {
                    if (idx + BLOCK_SIZE < elements)
                        inPlaceMinMaxSum_(&min, &max, &sum, inputs[idx + BLOCK_SIZE]);
                }
            }
            s_sums[threadIdx.x] = sum;
            s_mins[threadIdx.x] = min;
            s_maxs[threadIdx.x] = max;
            __syncthreads();

            reduceSharedData_<BLOCK_SIZE>(threadIdx.x, s_mins, s_maxs, s_sums,
                                          output_mins + batch, output_maxs + batch,
                                          output_sums + batch, output_means + batch, scale);
        }

        template<typename T>
        void launch_(T* inputs, T* output_mins, T* output_maxs, T* output_sums, T* output_means,
                     size_t elements, T scale, uint batches, uint threads, hipStream_t stream) {
            bool two_by_two = !(elements % (threads * 2));
            if (two_by_two) {
                switch (threads) {
                    case 256:
                        kernel_<256, true><<<batches, 256, 0, stream>>>(inputs, elements,
                                                                        output_mins, output_maxs,
                                                                        output_sums, output_means, scale);
                        break;
                    case 128:
                        kernel_<128, true><<<batches, 128, 0, stream>>>(inputs, elements,
                                                                        output_mins, output_maxs,
                                                                        output_sums, output_means, scale);
                        break;
                    case 64:
                        kernel_<64, true><<<batches, 64, 0, stream>>>(inputs, elements,
                                                                      output_mins, output_maxs,
                                                                      output_sums, output_means, scale);
                        break;
                    case 32:
                        kernel_<32, true><<<batches, 32, 0, stream>>>(inputs, elements,
                                                                      output_mins, output_maxs,
                                                                      output_sums, output_means, scale);
                        break;
                    default:
                        NOA_THROW("DEV: block size should be 32, 64, 128 or 256, "
                                  "got threads:{}, with elements:{}", threads, elements);
                }
            } else {
                switch (threads) {
                    case 256:
                        kernel_<256, false><<<batches, 256, 0, stream>>>(inputs, elements,
                                                                         output_mins, output_maxs,
                                                                         output_sums, output_means, scale);
                        break;
                    case 128:
                        kernel_<128, false><<<batches, 128, 0, stream>>>(inputs, elements,
                                                                         output_mins, output_maxs,
                                                                         output_sums, output_means, scale);
                        break;
                    case 64:
                        kernel_<64, false><<<batches, 64, 0, stream>>>(inputs, elements,
                                                                       output_mins, output_maxs,
                                                                       output_sums, output_means, scale);
                        break;
                    case 32:
                        kernel_<32, false><<<batches, 32, 0, stream>>>(inputs, elements,
                                                                       output_mins, output_maxs,
                                                                       output_sums, output_means, scale);
                        break;
                    default:
                        NOA_THROW("DEV: block size should be 32, 64, 128 or 256, "
                                  "got threads:{}, with elements:{}", threads, elements);
                }
            }
            NOA_THROW_IF(hipPeekAtLastError());
        }

        // Kernel to reduce the intermediary results (3 input arrays, per block).
        // Computes 4 (2 optional) values per batch.
        template<int BLOCK_SIZE, bool TWO_BY_TWO, typename T>
        __global__ void kernel_(T* tmp_mins, T* tmp_maxs, T* tmp_sums, uint tmps,
                                T* output_mins, T* output_maxs, T* output_sums, T* output_means, T scale) {
            static_assert(BLOCK_SIZE >= 32 && BLOCK_SIZE <= 256);
            __shared__ T s_sums[BLOCK_SIZE];
            __shared__ T s_mins[BLOCK_SIZE];
            __shared__ T s_maxs[BLOCK_SIZE];

            uint batch = blockIdx.x;
            uint offset = tmps * batch;
            tmp_mins += offset, tmp_maxs += offset, tmp_sums += offset;

            T sum = 0, min = *tmp_mins, max = *tmp_maxs;
            for (uint idx = threadIdx.x; idx < tmps; idx += BLOCK_SIZE * 2) {
                sum += tmp_sums[idx];
                if (tmp_mins[idx] < min) min = tmp_mins[idx];
                if (max < tmp_maxs[idx]) max = tmp_maxs[idx];
                if constexpr (TWO_BY_TWO) {
                    sum += tmp_sums[idx + BLOCK_SIZE];
                    if (tmp_mins[idx + BLOCK_SIZE] < min) min = tmp_mins[idx + BLOCK_SIZE];
                    if (max < tmp_maxs[idx + BLOCK_SIZE]) max = tmp_maxs[idx + BLOCK_SIZE];
                } else {
                    if (idx + BLOCK_SIZE < tmps) {
                        sum += tmp_sums[idx + BLOCK_SIZE];
                        if (tmp_mins[idx + BLOCK_SIZE] < min) min = tmp_mins[idx + BLOCK_SIZE];
                        if (max < tmp_maxs[idx + BLOCK_SIZE]) max = tmp_maxs[idx + BLOCK_SIZE];
                    }
                }
            }
            s_sums[threadIdx.x] = sum;
            s_mins[threadIdx.x] = min;
            s_maxs[threadIdx.x] = max;
            __syncthreads();

            reduceSharedData_<BLOCK_SIZE>(threadIdx.x, s_mins, s_maxs, s_sums,
                                          output_mins + batch, output_maxs + batch,
                                          output_sums + batch, output_means + batch, scale);
        }

        template<typename T>
        void launch_(T* tmp_mins, T* tmp_maxs, T* tmp_sums,
                     T* output_mins, T* output_maxs, T* output_sums, T* output_means,
                     size_t tmps, T scale, uint batches, uint threads, hipStream_t stream) {
            bool two_by_two = !(tmps % (threads * 2));
            if (two_by_two) {
                switch (threads) {
                    case 256:
                        kernel_<256, true><<<batches, 256, 0, stream>>>(tmp_mins, tmp_maxs, tmp_sums, tmps,
                                                                        output_mins, output_maxs,
                                                                        output_sums, output_means, scale);
                        break;
                    case 128:
                        kernel_<128, true><<<batches, 128, 0, stream>>>(tmp_mins, tmp_maxs, tmp_sums, tmps,
                                                                        output_mins, output_maxs,
                                                                        output_sums, output_means, scale);
                        break;
                    case 64:
                        kernel_<64, true><<<batches, 64, 0, stream>>>(tmp_mins, tmp_maxs, tmp_sums, tmps,
                                                                      output_mins, output_maxs,
                                                                      output_sums, output_means, scale);
                        break;
                    case 32:
                        kernel_<32, true><<<batches, 32, 0, stream>>>(tmp_mins, tmp_maxs, tmp_sums, tmps,
                                                                      output_mins, output_maxs,
                                                                      output_sums, output_means, scale);
                        break;
                    default:
                        NOA_THROW("DEV: block size should be 32, 64, 128 or 256, "
                                  "got threads:{}, with tmps:{}", threads, tmps);
                }
            } else {
                switch (threads) {
                    case 256:
                        kernel_<256, false><<<batches, 256, 0, stream>>>(tmp_mins, tmp_maxs, tmp_sums, tmps,
                                                                         output_mins, output_maxs,
                                                                         output_sums, output_means, scale);
                        break;
                    case 128:
                        kernel_<128, false><<<batches, 128, 0, stream>>>(tmp_mins, tmp_maxs, tmp_sums, tmps,
                                                                         output_mins, output_maxs,
                                                                         output_sums, output_means, scale);
                        break;
                    case 64:
                        kernel_<64, false><<<batches, 64, 0, stream>>>(tmp_mins, tmp_maxs, tmp_sums, tmps,
                                                                       output_mins, output_maxs,
                                                                       output_sums, output_means, scale);
                        break;
                    case 32:
                        kernel_<32, false><<<batches, 32, 0, stream>>>(tmp_mins, tmp_maxs, tmp_sums, tmps,
                                                                       output_mins, output_maxs,
                                                                       output_sums, output_means, scale);
                        break;
                    default:
                        NOA_THROW("DEV: block size should be 32, 64, 128 or 256, "
                                  "got threads:{}, with tmps:{}", threads, tmps);
                }
            }
            NOA_THROW_IF(hipPeekAtLastError());
        }
    }
}

namespace Noa::CUDA::Math {
    template<typename T>
    void minMaxSumMean(T* inputs, T* output_mins, T* output_maxs, T* output_sums, T* output_means,
                       size_t elements, uint batches, Stream& stream) {
        if (elements <= 32768 || batches > 16) {
            if (elements) {
                uint threads = Final_::getThreads_(elements);
                auto scale = static_cast<T>(elements);
                for (int batch = 0; batch < batches; batch += 32768U) {
                    T* input = inputs + batch * elements;
                    T* mins = output_mins + batch;
                    T* maxs = output_maxs + batch;
                    T* sums = output_sums == nullptr ? output_sums : output_sums + batch;
                    T* means = output_means == nullptr ? output_means : output_means + batch;
                    uint blocks = Noa::Math::min(batches - batch, 32768U);
                    Final_::launch_(input, mins, maxs, sums, means,
                                    elements, scale, blocks, threads, stream.id());
                }
            }
            Stream::synchronize(stream);

        } else {
            uint blocks = Contiguous_::getBlocks_(elements);
            Memory::PtrDevice<T> tmp(blocks * 3 * batches); // all mins, then all maxs, then all sums.
            T* mins, * maxs, * sums;
            for (uint batch = 0; batch < batches; ++batch) {
                T* input = inputs + batch * elements;
                mins = tmp.get() + batch * blocks;
                maxs = mins + batches * blocks;
                sums = maxs + batches * blocks;
                Contiguous_::launch_(input, mins, maxs, sums, elements, blocks, stream.get());
            }
            uint threads = Final_::getThreads_(blocks);
            auto scale = static_cast<T>(elements);
            mins = tmp.get();
            maxs = mins + batches * blocks;
            sums = maxs + batches * blocks;
            Final_::launch_(mins, maxs, sums, output_mins, output_maxs, output_sums, output_means,
                            blocks, scale, batches, threads, stream.id());
            Stream::synchronize(stream);
        }
    }

    template<typename T>
    void minMaxSumMean(T* inputs, size_t pitch_inputs,
                       T* output_mins, T* output_maxs, T* output_sums, T* output_means,
                       size3_t shape, uint batches, Stream& stream) {
        size_t elements = getElements(shape);
        if (!elements) {
            Stream::synchronize(stream);
            return;
        }

        uint2_t shape_2d(shape.x, getRows(shape));
        uint blocks = Padded_::getBlocks_(shape_2d.y);
        Memory::PtrDevice<T> tmp(blocks * 3 * batches); // all mins, then all maxs, then all sums.
        T* mins, * maxs, * sums;
        for (uint batch = 0; batch < batches; ++batch) {
            T* input = inputs + pitch_inputs * shape_2d.y * batch;
            mins = tmp.get() + batch * blocks;
            maxs = mins + batches * blocks;
            sums = maxs + batches * blocks;
            Padded_::launch_(input, pitch_inputs, mins, maxs, sums, shape_2d, blocks, stream.get());
        }
        uint threads = Final_::getThreads_(blocks);
        auto scale = static_cast<T>(elements);
        mins = tmp.get();
        maxs = mins + batches * blocks;
        sums = maxs + batches * blocks;
        Final_::launch_(mins, maxs, sums, output_mins, output_maxs, output_sums, output_means,
                        blocks, scale, batches, threads, stream.id());
        Stream::synchronize(stream);
    }

    #define INSTANTIATE_MIN_OR_MAX(T)                                                   \
    template void minMaxSumMean<T>(T*, T*, T*, T*, T*, size_t, uint, Stream&);          \
    template void minMaxSumMean<T>(T*, size_t, T*, T*, T*, T*, size3_t, uint, Stream&)

    INSTANTIATE_MIN_OR_MAX(float);
    INSTANTIATE_MIN_OR_MAX(double);
    INSTANTIATE_MIN_OR_MAX(int);
    INSTANTIATE_MIN_OR_MAX(uint);
    INSTANTIATE_MIN_OR_MAX(char);
    INSTANTIATE_MIN_OR_MAX(unsigned char);
}
