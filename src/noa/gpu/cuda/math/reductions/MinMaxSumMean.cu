#include "hip/hip_runtime.h"
// Implementation for math::minMaxSumMean() for contiguous and padded layouts.

#include "noa/common/Math.h"
#include "noa/common/Profiler.h"
#include "noa/gpu/cuda/Exception.h"
#include "noa/gpu/cuda/math/Reductions.h"
#include "noa/gpu/cuda/memory/PtrDevice.h"

namespace {
    using namespace noa;

    template<typename T>
    __device__ void warpSumReduce_(volatile T* s_data_tid) {
        T t = *s_data_tid;
        t = t + s_data_tid[32];
        *s_data_tid = t;
        t = t + s_data_tid[16];
        *s_data_tid = t;
        t = t + s_data_tid[8];
        *s_data_tid = t;
        t = t + s_data_tid[4];
        *s_data_tid = t;
        t = t + s_data_tid[2];
        *s_data_tid = t;
        t = t + s_data_tid[1];
        *s_data_tid = t;
    }

    template<typename T>
    __device__ void warpMinReduce_(volatile T* s_data_tid) {
        if (s_data_tid[32] < *s_data_tid) *s_data_tid = s_data_tid[32];
        if (s_data_tid[16] < *s_data_tid) *s_data_tid = s_data_tid[16];
        if (s_data_tid[8] < *s_data_tid) *s_data_tid = s_data_tid[8];
        if (s_data_tid[4] < *s_data_tid) *s_data_tid = s_data_tid[4];
        if (s_data_tid[2] < *s_data_tid) *s_data_tid = s_data_tid[2];
        if (s_data_tid[1] < *s_data_tid) *s_data_tid = s_data_tid[1];
    }

    template<typename T>
    __device__ void warpMaxReduce_(volatile T* s_data_tid) {
        if (*s_data_tid < s_data_tid[32]) *s_data_tid = s_data_tid[32];
        if (*s_data_tid < s_data_tid[16]) *s_data_tid = s_data_tid[16];
        if (*s_data_tid < s_data_tid[8]) *s_data_tid = s_data_tid[8];
        if (*s_data_tid < s_data_tid[4]) *s_data_tid = s_data_tid[4];
        if (*s_data_tid < s_data_tid[2]) *s_data_tid = s_data_tid[2];
        if (*s_data_tid < s_data_tid[1]) *s_data_tid = s_data_tid[1];
    }

    template<typename T>
    inline __device__ void inPlaceMinMaxSum_(T* __restrict__ current_min,
                                             T* __restrict__ current_max,
                                             T* __restrict__ current_sum, T candidate) {
        *current_sum += candidate;
        if (candidate < *current_min) *current_min = candidate;
        if (*current_max < candidate) *current_max = candidate;
    }

    template<typename T>
    inline __device__ void reduceSharedData_(int tid,
                                             T* __restrict__ s_mins, T* __restrict__ s_maxs, T* __restrict__ s_sums,
                                             T* __restrict__ output_min,
                                             T* __restrict__ output_max,
                                             T* __restrict__ output_sum) {
        if (tid < 256) {
            s_sums[tid] += s_sums[tid + 256];
            if (s_mins[tid + 256] < s_mins[tid]) s_mins[tid] = s_mins[tid + 256];
            if (s_maxs[tid] < s_maxs[tid + 256]) s_maxs[tid] = s_maxs[tid + 256];
        }
        __syncthreads();
        if (tid < 128) {
            s_sums[tid] += s_sums[tid + 128];
            if (s_mins[tid + 128] < s_mins[tid]) s_mins[tid] = s_mins[tid + 128];
            if (s_maxs[tid] < s_maxs[tid + 128]) s_maxs[tid] = s_maxs[tid + 128];
        }
        __syncthreads();
        if (tid < 64) {
            s_sums[tid] += s_sums[tid + 64];
            if (s_mins[tid + 64] < s_mins[tid]) s_mins[tid] = s_mins[tid + 64];
            if (s_maxs[tid] < s_maxs[tid + 64]) s_maxs[tid] = s_maxs[tid + 64];
        }
        __syncthreads();
        if (tid < 32) {
            warpSumReduce_(s_sums + tid);
            warpMinReduce_(s_mins + tid);
            warpMaxReduce_(s_maxs + tid);
        }
        if (tid == 0) {
            *output_sum = *s_sums;
            *output_min = *s_mins;
            *output_max = *s_maxs;
        }
    }

    template<int THREADS, typename T>
    inline __device__ void reduceSharedData_(int tid,
                                             T* __restrict__ s_mins, T* __restrict__ s_maxs, T* __restrict__ s_sums,
                                             T* __restrict__ out_min, T* __restrict__ out_max,
                                             T* __restrict__ out_sum, T* __restrict__ out_mean, T scale) {
        if constexpr (THREADS >= 256) {
            if (tid < 128) {
                s_sums[tid] += s_sums[tid + 128];
                if (s_mins[tid + 128] < s_mins[tid]) s_mins[tid] = s_mins[tid + 128];
                if (s_maxs[tid] < s_maxs[tid + 128]) s_maxs[tid] = s_maxs[tid + 128];
            }
            __syncthreads();

        }
        if constexpr (THREADS >= 128) {
            if (tid < 64) {
                s_sums[tid] += s_sums[tid + 64];
                if (s_mins[tid + 64] < s_mins[tid]) s_mins[tid] = s_mins[tid + 64];
                if (s_maxs[tid] < s_maxs[tid + 64]) s_maxs[tid] = s_maxs[tid + 64];
            }
            __syncthreads();
        }

        if constexpr (THREADS >= 64) {
            if (tid < 32) {
                warpSumReduce_(s_sums + tid);
                warpMinReduce_(s_mins + tid);
                warpMaxReduce_(s_maxs + tid);
            }
        }

        if (tid == 0) {
            if constexpr (THREADS == 32) {
                for (int i = 1; i < 32; ++i) {
                    *s_sums += s_sums[i];
                    if (s_mins[i] < *s_mins) *s_mins = s_mins[i];
                    if (*s_maxs < s_maxs[i]) *s_maxs = s_maxs[i];
                }
            }
            *out_min = *s_mins;
            *out_max = *s_maxs;
            T final_sum = *s_sums;
            if (out_sum)
                *out_sum = final_sum;
            if (out_mean)
                *out_mean = final_sum / scale;
        }
    }

    // Intermediary kernel to reduce large contiguous arrays to max 512 elements.
    namespace contiguous_ {
        constexpr uint THREADS = 512U;

        template<bool TWO_BY_TWO, typename T>
        __global__ __launch_bounds__(THREADS)
        void kernel_(const T* __restrict__ input,
                     T* __restrict__ tmp_mins,
                     T* __restrict__ tmp_maxs,
                     T* __restrict__ tmp_sums, uint elements) {
            __shared__ T s_sums[THREADS];
            __shared__ T s_mins[THREADS];
            __shared__ T s_maxs[THREADS];

            T sum = 0, min = *input, max = *input;
            for (uint idx = blockIdx.x * THREADS * 2 + threadIdx.x; idx < elements;
                 idx += THREADS * 2 * gridDim.x) {
                inPlaceMinMaxSum_(&min, &max, &sum, input[idx]);
                if constexpr (TWO_BY_TWO) {
                    inPlaceMinMaxSum_(&min, &max, &sum, input[idx + THREADS]);
                } else {
                    if (idx + THREADS < elements)
                        inPlaceMinMaxSum_(&min, &max, &sum, input[idx + THREADS]);
                }
            }
            s_sums[threadIdx.x] = sum;
            s_mins[threadIdx.x] = min;
            s_maxs[threadIdx.x] = max;
            __syncthreads();

            reduceSharedData_(threadIdx.x, s_mins, s_maxs, s_sums,
                              tmp_mins + blockIdx.x, tmp_maxs + blockIdx.x, tmp_sums + blockIdx.x);
        }

        uint getBlocks_(uint elements) {
            constexpr uint MAX_BLOCKS = 512U;
            uint blocks = noa::math::divideUp(elements, THREADS * 2);
            return noa::math::min(MAX_BLOCKS, blocks);
        }

        template<typename T>
        void launch_(const T* input, T* tmp_mins, T* tmp_maxs, T* tmp_sums,
                     uint elements, uint blocks, hipStream_t stream) {
            bool two_by_two = !(elements % (THREADS * 2));
            if (two_by_two) {
                kernel_<true><<<blocks, THREADS, 0, stream>>>(input, tmp_mins, tmp_maxs, tmp_sums, elements);
            } else {
                kernel_<false><<<blocks, THREADS, 0, stream>>>(input, tmp_mins, tmp_maxs, tmp_sums, elements);
            }
            NOA_THROW_IF(hipPeekAtLastError());
        }
    }

    // Intermediary kernel to reduce large padded arrays to 1-512 elements.
    namespace padded_ {
        constexpr uint2_t THREADS(32, 16);

        template<bool TWO_BY_TWO, typename T>
        __global__ __launch_bounds__(THREADS.x * THREADS.y)
        void kernel_(const T* __restrict__ input, uint pitch,
                     T* __restrict__ tmp_mins, T* __restrict__ tmp_maxs, T* __restrict__ tmp_sums, uint2_t shape) {
            uint tid = threadIdx.y * THREADS.x + threadIdx.x;
            __shared__ T s_sums[THREADS.x * THREADS.y];
            __shared__ T s_mins[THREADS.x * THREADS.y];
            __shared__ T s_maxs[THREADS.x * THREADS.y];

            T min = *input, max = *input, sum = 0;
            uint offset;
            for (uint row = THREADS.y * blockIdx.x + threadIdx.y; row < shape.y; row += gridDim.x * THREADS.y) {
                offset = row * pitch;
                if constexpr (TWO_BY_TWO) {
                    for (uint idx = threadIdx.x; idx < shape.x; idx += THREADS.x * 2) {
                        inPlaceMinMaxSum_(&min, &max, &sum, input[offset + idx]);
                        inPlaceMinMaxSum_(&min, &max, &sum, input[offset + idx + THREADS.x]);
                    }
                } else {
                    for (uint idx = threadIdx.x; idx < shape.x; idx += THREADS.x)
                        inPlaceMinMaxSum_(&min, &max, &sum, input[offset + idx]);
                }
            }
            s_sums[tid] = sum;
            s_mins[tid] = min;
            s_maxs[tid] = max;
            __syncthreads();

            reduceSharedData_(tid, s_mins, s_maxs, s_sums,
                              tmp_mins + blockIdx.x, tmp_maxs + blockIdx.x, tmp_sums + blockIdx.x);
        }

        uint getBlocks_(uint rows) {
            constexpr uint MAX_BLOCKS = 512;
            uint blocks = noa::math::divideUp(rows, THREADS.y);
            return noa::math::min(blocks, MAX_BLOCKS);
        }

        template<typename T>
        void launch_(const T* input, uint pitch, T* tmp_mins, T* tmp_maxs, T* tmp_sums,
                     uint2_t shape, uint blocks, hipStream_t stream) {
            dim3 threads(THREADS.x, THREADS.y);
            bool two_by_two = !(shape.x % (THREADS.x * 2));
            if (two_by_two) {
                kernel_<true><<<blocks, threads, 0, stream>>>(input, pitch, tmp_mins, tmp_maxs, tmp_sums, shape);
            } else {
                kernel_<false><<<blocks, threads, 0, stream>>>(input, pitch, tmp_mins, tmp_maxs, tmp_sums, shape);
            }
            NOA_THROW_IF(hipPeekAtLastError());
        }
    }

    namespace final_ {
        // Kernel to reduce small arrays (one array per block). Computes 4 (2 optional) values per batch.
        uint getThreads_(size_t elements) {
            uint threads = noa::math::nextPowerOf2((elements + 1) / 2); // compute at least 2 elements.
            return noa::math::clamp(threads, 32U, 256U);
        }

        template<int THREADS, bool TWO_BY_TWO, typename T>
        __global__ __launch_bounds__(256)
        void kernel_(const T* __restrict__ inputs, uint elements,
                     T* __restrict__ output_mins, T* __restrict__ output_maxs,
                     T* __restrict__ output_sums, T* __restrict__ output_means, T scale) {
            static_assert(THREADS >= 32 && THREADS <= 256);
            __shared__ T s_sums[THREADS];
            __shared__ T s_mins[THREADS];
            __shared__ T s_maxs[THREADS];

            uint batch = blockIdx.x;
            inputs += elements * batch;

            T sum = 0, min = *inputs, max = min;
            for (uint idx = threadIdx.x; idx < elements; idx += THREADS * 2) {
                inPlaceMinMaxSum_(&min, &max, &sum, inputs[idx]);

                if constexpr (TWO_BY_TWO) {
                    inPlaceMinMaxSum_(&min, &max, &sum, inputs[idx + THREADS]);
                } else {
                    if (idx + THREADS < elements)
                        inPlaceMinMaxSum_(&min, &max, &sum, inputs[idx + THREADS]);
                }
            }
            s_sums[threadIdx.x] = sum;
            s_mins[threadIdx.x] = min;
            s_maxs[threadIdx.x] = max;
            __syncthreads();

            reduceSharedData_<THREADS>(threadIdx.x, s_mins, s_maxs, s_sums,
                                       output_mins + batch, output_maxs + batch,
                                       output_sums + batch, output_means + batch, scale);
        }

        template<typename T>
        void launch_(const T* inputs, T* output_mins, T* output_maxs, T* output_sums, T* output_means,
                     uint elements, T scale, uint batches, uint threads, hipStream_t stream) {
            bool two_by_two = !(elements % (threads * 2));
            if (two_by_two) {
                switch (threads) {
                    case 256:
                        kernel_<256, true><<<batches, 256, 0, stream>>>(
                                inputs, elements, output_mins, output_maxs, output_sums, output_means, scale);
                        break;
                    case 128:
                        kernel_<128, true><<<batches, 128, 0, stream>>>(
                                inputs, elements, output_mins, output_maxs, output_sums, output_means, scale);
                        break;
                    case 64:
                        kernel_<64, true><<<batches, 64, 0, stream>>>(
                                inputs, elements, output_mins, output_maxs, output_sums, output_means, scale);
                        break;
                    case 32:
                        kernel_<32, true><<<batches, 32, 0, stream>>>(
                                inputs, elements, output_mins, output_maxs, output_sums, output_means, scale);
                        break;
                    default:
                        NOA_THROW("DEV: block size should be 32, 64, 128 or 256, "
                                  "got threads:{}, with elements:{}", threads, elements);
                }
            } else {
                switch (threads) {
                    case 256:
                        kernel_<256, false><<<batches, 256, 0, stream>>>(
                                inputs, elements, output_mins, output_maxs, output_sums, output_means, scale);
                        break;
                    case 128:
                        kernel_<128, false><<<batches, 128, 0, stream>>>(
                                inputs, elements, output_mins, output_maxs, output_sums, output_means, scale);
                        break;
                    case 64:
                        kernel_<64, false><<<batches, 64, 0, stream>>>(
                                inputs, elements, output_mins, output_maxs, output_sums, output_means, scale);
                        break;
                    case 32:
                        kernel_<32, false><<<batches, 32, 0, stream>>>(
                                inputs, elements, output_mins, output_maxs, output_sums, output_means, scale);
                        break;
                    default:
                        NOA_THROW("DEV: block size should be 32, 64, 128 or 256, "
                                  "got threads:{}, with elements:{}", threads, elements);
                }
            }
            NOA_THROW_IF(hipPeekAtLastError());
        }

        // Kernel to reduce the intermediary results (3 input arrays, per block).
        // Computes 4 (2 optional) values per batch.
        template<int THREADS, bool TWO_BY_TWO, typename T>
        __global__ __launch_bounds__(256)
        void kernel_(const T* __restrict__ tmp_mins, const T* __restrict__ tmp_maxs,
                     const T* __restrict__ tmp_sums, uint tmp_elements,
                     T* __restrict__ output_mins, T* __restrict__ output_maxs,
                     T* __restrict__ output_sums, T* __restrict__ output_means, T scale) {
            static_assert(THREADS >= 32 && THREADS <= 256);
            __shared__ T s_sums[THREADS];
            __shared__ T s_mins[THREADS];
            __shared__ T s_maxs[THREADS];

            uint batch = blockIdx.x;
            uint offset = tmp_elements * batch;
            tmp_mins += offset, tmp_maxs += offset, tmp_sums += offset;

            T sum = 0, min = *tmp_mins, max = *tmp_maxs;
            for (uint idx = threadIdx.x; idx < tmp_elements; idx += THREADS * 2) {
                sum += tmp_sums[idx];
                if (tmp_mins[idx] < min) min = tmp_mins[idx];
                if (max < tmp_maxs[idx]) max = tmp_maxs[idx];
                if constexpr (TWO_BY_TWO) {
                    sum += tmp_sums[idx + THREADS];
                    if (tmp_mins[idx + THREADS] < min) min = tmp_mins[idx + THREADS];
                    if (max < tmp_maxs[idx + THREADS]) max = tmp_maxs[idx + THREADS];
                } else {
                    if (idx + THREADS < tmp_elements) {
                        sum += tmp_sums[idx + THREADS];
                        if (tmp_mins[idx + THREADS] < min) min = tmp_mins[idx + THREADS];
                        if (max < tmp_maxs[idx + THREADS]) max = tmp_maxs[idx + THREADS];
                    }
                }
            }
            s_sums[threadIdx.x] = sum;
            s_mins[threadIdx.x] = min;
            s_maxs[threadIdx.x] = max;
            __syncthreads();

            reduceSharedData_<THREADS>(threadIdx.x, s_mins, s_maxs, s_sums,
                                       output_mins + batch, output_maxs + batch,
                                       output_sums + batch, output_means + batch, scale);
        }

        template<typename T>
        void launch_(const T* tmp_mins, const T* tmp_maxs, const T* tmp_sums,
                     T* output_mins, T* output_maxs, T* output_sums, T* output_means,
                     uint tmp_elements, T scale, uint batches, uint threads, hipStream_t stream) {
            bool two_by_two = !(tmp_elements % (threads * 2));
            if (two_by_two) {
                switch (threads) {
                    case 256:
                        kernel_<256, true><<<batches, 256, 0, stream>>>(tmp_mins, tmp_maxs, tmp_sums, tmp_elements,
                                                                        output_mins, output_maxs,
                                                                        output_sums, output_means, scale);
                        break;
                    case 128:
                        kernel_<128, true><<<batches, 128, 0, stream>>>(tmp_mins, tmp_maxs, tmp_sums, tmp_elements,
                                                                        output_mins, output_maxs,
                                                                        output_sums, output_means, scale);
                        break;
                    case 64:
                        kernel_<64, true><<<batches, 64, 0, stream>>>(tmp_mins, tmp_maxs, tmp_sums, tmp_elements,
                                                                      output_mins, output_maxs,
                                                                      output_sums, output_means, scale);
                        break;
                    case 32:
                        kernel_<32, true><<<batches, 32, 0, stream>>>(tmp_mins, tmp_maxs, tmp_sums, tmp_elements,
                                                                      output_mins, output_maxs,
                                                                      output_sums, output_means, scale);
                        break;
                    default:
                        NOA_THROW("DEV: block size should be 32, 64, 128 or 256, "
                                  "got threads:{}, with tmp_elements:{}", threads, tmp_elements);
                }
            } else {
                switch (threads) {
                    case 256:
                        kernel_<256, false><<<batches, 256, 0, stream>>>(tmp_mins, tmp_maxs, tmp_sums, tmp_elements,
                                                                         output_mins, output_maxs,
                                                                         output_sums, output_means, scale);
                        break;
                    case 128:
                        kernel_<128, false><<<batches, 128, 0, stream>>>(tmp_mins, tmp_maxs, tmp_sums, tmp_elements,
                                                                         output_mins, output_maxs,
                                                                         output_sums, output_means, scale);
                        break;
                    case 64:
                        kernel_<64, false><<<batches, 64, 0, stream>>>(tmp_mins, tmp_maxs, tmp_sums, tmp_elements,
                                                                       output_mins, output_maxs,
                                                                       output_sums, output_means, scale);
                        break;
                    case 32:
                        kernel_<32, false><<<batches, 32, 0, stream>>>(tmp_mins, tmp_maxs, tmp_sums, tmp_elements,
                                                                       output_mins, output_maxs,
                                                                       output_sums, output_means, scale);
                        break;
                    default:
                        NOA_THROW("DEV: block size should be 32, 64, 128 or 256, "
                                  "got threads:{}, with tmp_elements:{}", threads, tmp_elements);
                }
            }
            NOA_THROW_IF(hipPeekAtLastError());
        }
    }
}

namespace noa::cuda::math {
    template<typename T>
    void minMaxSumMean(const T* inputs, T* output_mins, T* output_maxs, T* output_sums, T* output_means,
                       size_t elements, size_t batches, Stream& stream) {
        NOA_PROFILE_FUNCTION();
        if (elements <= 32768 || batches > 16) {
            if (elements) {
                uint threads = final_::getThreads_(elements);
                auto scale = static_cast<T>(elements);
                for (size_t batch = 0; batch < batches; batch += 32768U) {
                    const T* input = inputs + batch * elements;
                    T* o_mins = output_mins + batch;
                    T* o_maxs = output_maxs + batch;
                    T* o_sums = output_sums == nullptr ? output_sums : output_sums + batch;
                    T* o_means = output_means == nullptr ? output_means : output_means + batch;
                    uint blocks = noa::math::min(batches - batch, size_t{32768U});
                    final_::launch_(input, o_mins, o_maxs, o_sums, o_means,
                                    elements, scale, blocks, threads, stream.id());
                }
            }
            Stream::synchronize(stream);

        } else {
            uint blocks = contiguous_::getBlocks_(elements);
            memory::PtrDevice<T> tmp(blocks * 3 * batches); // all mins, then all maxs, then all sums.
            T* tmp_mins, * tmp_maxs, * tmp_sums;
            for (size_t batch = 0; batch < batches; ++batch) {
                const T* input = inputs + batch * elements;
                tmp_mins = tmp.get() + batch * blocks;
                tmp_maxs = tmp_mins + batches * blocks;
                tmp_sums = tmp_maxs + batches * blocks;
                contiguous_::launch_(input, tmp_mins, tmp_maxs, tmp_sums, elements, blocks, stream.get());
            }
            uint threads = final_::getThreads_(blocks);
            auto scale = static_cast<T>(elements);
            tmp_mins = tmp.get();
            tmp_maxs = tmp_mins + batches * blocks;
            tmp_sums = tmp_maxs + batches * blocks;
            final_::launch_(tmp_mins, tmp_maxs, tmp_sums, output_mins, output_maxs, output_sums, output_means,
                            blocks, scale, batches, threads, stream.id());
            Stream::synchronize(stream);
        }
    }

    template<typename T>
    void minMaxSumMean(const T* inputs, size_t inputs_pitch,
                       T* output_mins, T* output_maxs, T* output_sums, T* output_means,
                       size3_t shape, size_t batches, Stream& stream) {
        NOA_PROFILE_FUNCTION();
        size_t elements = noa::elements(shape);
        if (!elements)
            return Stream::synchronize(stream);

        uint2_t shape_2d(shape.x, rows(shape));
        uint blocks = padded_::getBlocks_(shape_2d.y);
        memory::PtrDevice<T> tmp(blocks * 3 * batches); // all mins, then all maxs, then all sums.
        T* tmp_mins, * tmp_maxs, * tmp_sums;
        for (size_t batch = 0; batch < batches; ++batch) {
            const T* input = inputs + inputs_pitch * shape_2d.y * batch;
            tmp_mins = tmp.get() + batch * blocks;
            tmp_maxs = tmp_mins + batches * blocks;
            tmp_sums = tmp_maxs + batches * blocks;
            padded_::launch_(input, inputs_pitch, tmp_mins, tmp_maxs, tmp_sums, shape_2d, blocks, stream.get());
        }
        uint threads = final_::getThreads_(blocks);
        auto scale = static_cast<T>(elements);
        tmp_mins = tmp.get();
        tmp_maxs = tmp_mins + batches * blocks;
        tmp_sums = tmp_maxs + batches * blocks;
        final_::launch_(tmp_mins, tmp_maxs, tmp_sums, output_mins, output_maxs, output_sums, output_means,
                        blocks, scale, batches, threads, stream.id());
        Stream::synchronize(stream);
    }

    #define NOA_INSTANTIATE_MIN_OR_MAX_(T)                                              \
    template void minMaxSumMean<T>(const T*, T*, T*, T*, T*, size_t, size_t, Stream&);  \
    template void minMaxSumMean<T>(const T*, size_t, T*, T*, T*, T*, size3_t, size_t, Stream&)

    NOA_INSTANTIATE_MIN_OR_MAX_(float);
    NOA_INSTANTIATE_MIN_OR_MAX_(double);
    NOA_INSTANTIATE_MIN_OR_MAX_(char);
    NOA_INSTANTIATE_MIN_OR_MAX_(short);
    NOA_INSTANTIATE_MIN_OR_MAX_(int);
    NOA_INSTANTIATE_MIN_OR_MAX_(long);
    NOA_INSTANTIATE_MIN_OR_MAX_(long long);
    NOA_INSTANTIATE_MIN_OR_MAX_(unsigned char);
    NOA_INSTANTIATE_MIN_OR_MAX_(unsigned short);
    NOA_INSTANTIATE_MIN_OR_MAX_(unsigned int);
    NOA_INSTANTIATE_MIN_OR_MAX_(unsigned long);
    NOA_INSTANTIATE_MIN_OR_MAX_(unsigned long long);
}
