#include "hip/hip_runtime.h"
// Use cub to do the actual sorting. Thrust seems to have its own merge sort, but its main API does not allow you to
// choose between radix and merge sort. However, it seems that it always selects radix if the type and comparison
// operator is supported by the radix sort, i.e. base types and < or > comparison. In our case, that's all we need,
// so use the radix sort for everything...
#include <cub/device/device_radix_sort.cuh>

#include "noa/gpu/cuda/math/Sort.h"
#include "noa/gpu/cuda/memory/PtrDevice.h"
#include "noa/gpu/cuda/memory/Copy.h"
#include "noa/gpu/cuda/memory/Iota.h"

namespace {
    using namespace ::noa;

    template<typename T>
    hipError_t cubRadixSortKeys_(void* temp_storage, size_t& temp_storage_bytes, hipcub::DoubleBuffer<T>& keys,
                                int32_t size, bool ascending, cuda::Stream& stream) {
        // half_t can be safely reinterpreted to CUDA's __half.
        using cubT = std::conditional_t<std::is_same_v<T, half_t>, __half, T>;

        if (ascending) {
            return hipcub::DeviceRadixSort::SortKeys(
                    temp_storage, temp_storage_bytes,
                    reinterpret_cast<hipcub::DoubleBuffer<cubT>&>(keys),
                    size, 0, static_cast<int>(sizeof(T) * 8),
                    stream.id());
        } else {
            return hipcub::DeviceRadixSort::SortKeysDescending(
                    temp_storage, temp_storage_bytes,
                    reinterpret_cast<hipcub::DoubleBuffer<cubT>&>(keys),
                    size, 0, static_cast<int>(sizeof(T) * 8),
                    stream.id());
        }
    }

    template<typename T, typename U>
    hipError_t cubRadixSortPairs_(void* temp_storage, size_t& temp_storage_bytes,
                                 hipcub::DoubleBuffer<T>& keys, hipcub::DoubleBuffer<U>& values,
                                 int32_t size, bool ascending, cuda::Stream& stream) {
        // half_t can be safely reinterpreted to CUDA's __half.
        using cubT = std::conditional_t<std::is_same_v<T, half_t>, __half, T>;
        using cubU = std::conditional_t<std::is_same_v<U, half_t>, __half, U>;

        if (ascending) {
            return hipcub::DeviceRadixSort::SortPairs(
                    temp_storage, temp_storage_bytes,
                    reinterpret_cast<hipcub::DoubleBuffer<cubT>&>(keys),
                    reinterpret_cast<hipcub::DoubleBuffer<cubU>&>(values),
                    size, 0, static_cast<int>(sizeof(T) * 8),
                    stream.id());
        } else {
            return hipcub::DeviceRadixSort::SortPairsDescending(
                    temp_storage, temp_storage_bytes,
                    reinterpret_cast<hipcub::DoubleBuffer<cubT>&>(keys),
                    reinterpret_cast<hipcub::DoubleBuffer<cubU>&>(values),
                    size, 0, static_cast<int>(sizeof(T) * 8),
                    stream.id());
        }
    }

    // Sorts the third dimension of "values" using cub radix sort.
    // Works with non-contiguous strides. If dim is non-contiguous, allocates one row.
    // If there's a lot of rows to sort, sortBatched_ should be faster.
    template<typename T>
    void sortIterative_(const shared_t<T[]>& values, dim4_t strides, dim4_t shape,
                        int32_t dim, bool ascending, cuda::Stream& stream) {
        NOA_ASSERT(strides[dim] > 0); // nothing to sort if dim is broadcast

        const bool dim_is_contiguous = strides[dim] == 1;
        const auto dim_size = safe_cast<int32_t>(shape[dim]);
        const dim4_t dim_shape{1, 1, 1, shape[dim]};
        const dim_t dim_stride = strides[dim];
        const dim4_t dim_strides(dim_stride);

        // TODO Do one single allocation for buffer(s) and tmp storage. Problem is the alignment?

        // Prepare the alternate buffer.
        using unique_ptr_t = typename cuda::memory::PtrDevice<T>::alloc_unique_t;
        unique_ptr_t key_buffer;
        unique_ptr_t key_buffer_alt;
        if (dim_is_contiguous) {
            key_buffer = nullptr;
            key_buffer_alt = cuda::memory::PtrDevice<T>::alloc(dim_size, stream);
        } else {
            key_buffer = cuda::memory::PtrDevice<T>::alloc(dim_size, stream);
            key_buffer_alt = cuda::memory::PtrDevice<T>::alloc(dim_size, stream);
        }
        hipcub::DoubleBuffer<T> keys(key_buffer.get(), key_buffer_alt.get());

        // Allocates for the small tmp storage.
        size_t temp_storage_bytes;
        NOA_THROW_IF(cubRadixSortKeys_<T>(nullptr, temp_storage_bytes, keys, dim_size, ascending, stream));
        const auto temp_storage = cuda::memory::PtrDevice<byte_t>::alloc(temp_storage_bytes, stream);

        // Prepare the iterations.
        dim3_t iter_shape_;
        dim3_t iter_strides_;
        int32_t count = 0;
        for (int32_t i = 0; i < 4; ++i) {
            if (i != dim) {
                iter_shape_[count] = shape[i];
                iter_strides_[count] = strides[i];
                ++count;
            }
        }

        // Sort the axis.
        for (dim_t i = 0; i < iter_shape_[0]; ++i) {
            for (dim_t j = 0; j < iter_shape_[1]; ++j) {
                for (dim_t k = 0; k < iter_shape_[2]; ++k) {

                    const dim_t offset = indexing::at(i, j, k, iter_strides_);
                    T* values_iter = values.get() + offset;

                    // (Re)set the buffers.
                    keys.selector = 0;
                    if (dim_is_contiguous) {
                        keys.d_buffers[0] = values_iter;
                    } else {
                        cuda::memory::copy(values_iter, dim_strides,
                                           key_buffer.get(), dim_shape.strides(),
                                           dim_shape, stream);
                    }

                    NOA_THROW_IF(cubRadixSortKeys_<T>(temp_storage.get(), temp_storage_bytes, keys,
                                                      dim_size, ascending, stream));

                    if (dim_is_contiguous) {
                        if (keys.selector != 0) {
                            // Unfortunately, the results are in the alternate buffer,
                            // so copy it back to the original array.
                            cuda::memory::copy(key_buffer_alt.get(), dim_shape.strides(),
                                               values_iter /* or key_buffer */, dim_strides,
                                               dim_shape, stream);
                        }
                    } else {
                        cuda::memory::copy(keys.selector == 0 ? key_buffer.get() : key_buffer_alt.get(),
                                           dim_shape.strides(),
                                           values_iter, dim_strides,
                                           dim_shape, stream);
                    }
                }
            }
        }
        stream.attach(values);
    }

    // Sort any dimension [0..3] of the input array, in-place.
    // The array can have non-contiguous strides in any dimension.
    // Basically allocates x2 the shape...
    template<typename T>
    void sortBatched_(const shared_t<T[]>& values, dim4_t strides, dim4_t shape,
                      int32_t dim, bool ascending, cuda::Stream& stream) {
        const bool contiguous = indexing::areContiguous(strides, shape);
        const auto elements = safe_cast<int32_t>(shape.elements());

        // Prepare the keys.
        dim4_t tile = shape;
        tile[dim] = 1; // mark elements with their original axis.
        shared_t<uint[]> key_buffer = cuda::memory::PtrDevice<uint>::alloc(elements, stream);
        const auto key_buffer_alt = cuda::memory::PtrDevice<uint>::alloc(elements, stream);
        cuda::memory::iota(key_buffer, shape.strides(), shape, tile, stream);

        // Prepare the values.
        using unique_ptr_t = typename cuda::memory::PtrDevice<T>::alloc_unique_t;
        shared_t<T[]> val_buffer;
        unique_ptr_t val_buffer_alt;
        if (contiguous) {
            val_buffer = values;
            val_buffer_alt = cuda::memory::PtrDevice<T>::alloc(elements, stream);
        } else {
            val_buffer = cuda::memory::PtrDevice<T>::alloc(elements, stream);
            val_buffer_alt = cuda::memory::PtrDevice<T>::alloc(elements, stream);
            cuda::memory::copy(values.get(), strides, val_buffer.get(), shape.strides(), shape, stream);
        }

        // Gather them in the cub interface.
        hipcub::DoubleBuffer<uint> cub_keys(key_buffer.get(), key_buffer_alt.get());
        hipcub::DoubleBuffer<T> cub_vals(val_buffer.get(), val_buffer_alt.get());

        // Allocates for the small tmp storage.
        // The documentation says this should be a small value and is relative to the input size.
        size_t tmp_bytes0, tmp_bytes1;
        const hipError_t err0 = cubRadixSortPairs_<uint, T>(
                nullptr, tmp_bytes0, cub_keys, cub_vals, elements, ascending, stream);
        const hipError_t err1 = cubRadixSortPairs_<T, uint>(
                nullptr, tmp_bytes1, cub_vals, cub_keys, elements, ascending, stream);
        if (err0 != hipSuccess || err1 != hipSuccess) {
            NOA_THROW("Could not find temporary allocation size. 0:{}, 1:{}",
                      hipGetErrorString(err0), hipGetErrorString(err1));
        }

        tmp_bytes0 = std::max(tmp_bytes0, tmp_bytes1);
        const auto tmp = cuda::memory::PtrDevice<byte_t>::alloc(tmp_bytes0, stream);

        // Sort the entire array based on the values, but updates the original indexes.
        // It is important that the second sort is stable, which is the case with radix sort.
        NOA_THROW_IF((cubRadixSortPairs_<T, uint>(tmp.get(), tmp_bytes0, cub_vals, cub_keys, elements, ascending, stream)));
        NOA_THROW_IF((cubRadixSortPairs_<uint, T>(tmp.get(), tmp_bytes0, cub_keys, cub_vals, elements, true, stream)));

        // Then permute it back to the original order.
        // Find the permutation from "key_val" to "values":
        dim4_t input_shape(shape[dim]);
        int4_t permutation(3);
        int32_t count = 0;
        for (int32_t i = 0; i < 4; ++i) {
            if (i != dim) {
                input_shape[count] = shape[i];
                permutation[i] = count;
                ++count;
            }
        }

        // Permutation. Since we do 2 sorts, it seems that the result is always at the position 0. If "values" was
        // contiguous, it means the result is already in "values" but with a possible permutation. In this case,
        // we have to permute in the alternate buffer and then copy the result back to "values"...
        const dim4_t current_strides_permuted = indexing::reorder(input_shape.strides(), permutation);
        if (values.get() == cub_vals.Current() && !all(permutation == int4_t{0, 1, 2, 3})) {
            cuda::memory::copy(values.get(), current_strides_permuted,
                               val_buffer_alt.get(), shape.strides(),
                               shape, stream);
            cuda::memory::copy(val_buffer_alt.get(), shape.strides(), values.get(), strides, shape, stream);
        } else {
            cuda::memory::copy(cub_vals.selector == 0 ? val_buffer.get() : val_buffer_alt.get(),
                               current_strides_permuted,
                               values.get(), strides, shape, stream);
        }
        stream.attach(values);
    }
}

namespace noa::cuda::math {
    template<typename T, typename>
    void sort(const shared_t<T[]>& array, dim4_t strides, dim4_t shape, bool ascending, int32_t dim, Stream& stream) {
        NOA_ASSERT(array.get() != nullptr && all(shape > 0));

        // Allow dim = -1 to specify the first non-empty dimension in the rightmost order.
        if (dim == -1)
            dim = shape[3] > 1 ? 3 : shape[2] > 1 ? 2 : shape[1] > 1 ? 1 : 0;
        NOA_ASSERT(dim >= 0 && dim <= 3);

        if (strides[dim] == 0)
            return; // there's one value in the dimension to sort...

        // If there's not a lot of axes to sort, use the iterative version which uses less memory
        // and does a single sort per axis. Otherwise, use the batched version which uses more memory
        // but uses 2 sorts (1 being a stable sort), and a possible permutation, for the entire array.
        dim4_t shape_ = shape;
        shape_[dim] = 1;
        const dim_t iterations = shape_.elements();
        if (iterations < 10)
            sortIterative_(array, strides, shape, dim, ascending, stream);
        else
            sortBatched_(array, strides, shape, dim, ascending, stream);
    }

    #define NOA_INSTANTIATE_SORT_(T) \
    template void sort<T,void>(const shared_t<T[]>&, dim4_t, dim4_t, bool, int32_t, Stream&)

    NOA_INSTANTIATE_SORT_(int16_t);
    NOA_INSTANTIATE_SORT_(int32_t);
    NOA_INSTANTIATE_SORT_(int64_t);
    NOA_INSTANTIATE_SORT_(uint16_t);
    NOA_INSTANTIATE_SORT_(uint32_t);
    NOA_INSTANTIATE_SORT_(uint64_t);
    NOA_INSTANTIATE_SORT_(half_t);
    NOA_INSTANTIATE_SORT_(float);
    NOA_INSTANTIATE_SORT_(double);
}
