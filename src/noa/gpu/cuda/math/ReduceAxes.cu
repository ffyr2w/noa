#include "hip/hip_runtime.h"
#include "noa/gpu/cuda/Exception.hpp"
#include "noa/gpu/cuda/math/Reduce.hpp"
#include "noa/gpu/cuda/memory/Copy.hpp"
#include "noa/gpu/cuda/utils/Pointers.hpp"
#include "noa/gpu/cuda/utils/ReduceUnary.cuh"

namespace {
    using namespace ::noa;
    using namespace ::noa::cuda;

    constexpr u32 ELEMENTS_PER_THREAD = 4;
    constexpr u32 BLOCK_SIZE = 512;

    // Reduce rows, one Block.X per row.
    // Since we need the entire block for the reduction, do not return prematurely.
    template<typename Input, typename Reduced, typename Output, typename Index,
             typename PreProcessOp, typename ReduceOp, typename PostProcess,
             u32 BLOCK_DIM_X, u32 VECTOR_SIZE>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void reduce_width_(AccessorRestrict<const Input, 4, Index> input,
                       Shape2<Index> shape_hw, Reduced initial_reduce,
                       AccessorRestrict<Output, 3, Index> output,
                       PreProcessOp preprocess_op, ReduceOp reduce_op,
                       PostProcess post_process_op) {

        NOA_ASSERT(BLOCK_DIM_X == blockDim.x);
        constexpr Index EPT = noa::math::max(ELEMENTS_PER_THREAD, VECTOR_SIZE);
        constexpr Index BLOCK_WORK_SIZE = EPT * BLOCK_DIM_X;

        const Vec2<Index> thread_index{threadIdx.y, threadIdx.x};
        const Vec4<Index> gid{blockIdx.z,
                              blockIdx.y,
                              blockIdx.x * blockDim.y + thread_index[0],
                              thread_index[1]};
        const bool is_valid_row = gid[2] < shape_hw[0];
        const auto input_row = input[gid[0]][gid[1]][gid[2]];

        // Initial reduction. Loop until the end of the row is reached.
        Reduced reduced = initial_reduce;
        for (Index cid = 0; cid < shape_hw[1] && is_valid_row; cid += BLOCK_WORK_SIZE) {
            const Index remaining = shape_hw[1] - cid;
            const Index stride = input_row.template stride<0>();
            const Index offset = cid * stride;
            utils::block_reduce_global_unary<BLOCK_DIM_X, EPT, VECTOR_SIZE>(
                    input_row.get() + offset, stride, remaining,
                    preprocess_op, reduce_op, &reduced, thread_index[1], offset);
        }

        // Share the threads' initial reduction with the rest of the block.
        const Index tid = thread_index[0] * BLOCK_DIM_X + thread_index[1];
        Reduced* s_data = utils::block_dynamic_shared_resource<Reduced>(); // BLOCK_SIZE elements.
        s_data[tid] = reduced;
        utils::block_synchronize();

        // Reduce shared data to one element.
        const Reduced final = utils::block_reduce_shared<BLOCK_DIM_X>(
                s_data + BLOCK_DIM_X * thread_index[0], thread_index[1], reduce_op);
        if (gid[3] == 0 && is_valid_row)
            output(gid[0], gid[1], gid[2]) = post_process_op(final);
    }

    // Keep X to one warp to have memory coalescing, even though a half-warp should be OK as well.
    // The Y dimension of the block is where the reduction happens.
    constexpr dim3 BLOCK_SIZE_2D(32, BLOCK_SIZE / 32);

    // The second-most dimension is reduced, i.e. shape[0] and strides[2].
    // Grid.X Blocks per row.
    // Grid.Z/Y: blocks to reduce the two outermost
    template<typename Input, typename Reduced, typename Output, typename Index,
             typename PreProcessor, typename ReduceOp, typename PostProcess>
    __global__ __launch_bounds__(BLOCK_SIZE)
    void reduce_height_(const Input* __restrict__ input, Strides4<Index> input_strides, Shape2<Index> shape,
                        Reduced initial_reduce, Output* __restrict__ output, Strides4<Index> output_strides,
                        PreProcessor pre_process_op, ReduceOp reduce_op, PostProcess post_process_op) {

        const Vec4<Index> gid{blockIdx.z,
                              blockIdx.y,
                              threadIdx.y, // one block in the dimension to reduce
                              blockIdx.x * BLOCK_SIZE_2D.x + threadIdx.x};
        const bool is_valid_column = gid[3] < shape[1];
        input += noa::indexing::at(gid[0], gid[1], input_strides) + gid[3] * input_strides[3];

        // Initial reduction. Loop until the end of Y is reached.
        Reduced reduced = initial_reduce;
        for (Index tidy = gid[2]; tidy < shape[0] && is_valid_column; tidy += BLOCK_SIZE_2D.y) { // compute entire row
            const auto offset = tidy * input_strides[2];
            if constexpr (noa::traits::is_detected_v<noa::traits::has_binary_operator, PreProcessor, Input, Index>)
                reduced = reduce_op(reduced, pre_process_op(input[offset], offset));
            else
                reduced = reduce_op(reduced, pre_process_op(input[offset]));
        }

        // Share the threads' initial reduction with the rest of the block.
        const Index tid = gid[2] * blockDim.x + threadIdx.x;
        Reduced* s_data = utils::block_dynamic_shared_resource<Reduced>(); // BLOCK_SIZE elements.
        Reduced* s_data_tid = s_data + tid;
        *s_data_tid = reduced;
        utils::block_synchronize();

        // Reduce along Y:
        #pragma unroll
        for (u32 SIZE = BLOCK_SIZE_2D.y; SIZE >= 2; SIZE /= 2) {
            if (gid[2] < SIZE / 2)
                *s_data_tid = reduce_op(*s_data_tid, s_data_tid[BLOCK_SIZE_2D.x * SIZE / 2]);
            utils::block_synchronize();
        }

        if (gid[2] == 0 && is_valid_column) {
            const auto offset = noa::indexing::at(gid[0], gid[1], output_strides) + gid[3] * output_strides[3];
            output[offset] = post_process_op(*s_data_tid);
        }
    }

    template<typename Input, typename Reduced, typename Output,
             typename ReduceOp, typename PreProcessOp, typename PostProcess>
    inline void reduce_axis_(const char* name,
                            const Input* input, const Strides4<i64>& input_strides, const Shape4<i64>& input_shape,
                            Output* output, const Strides4<i64>& output_strides, const Shape4<i64>& output_shape,
                            const Vec4<bool>& mask, Reduced initial_reduce,
                            PreProcessOp pre_process_op, ReduceOp reduce_op,  PostProcess post_process_op,
                            Stream& stream) {
        NOA_ASSERT_DEVICE_PTR(input, stream.device());
        NOA_ASSERT_DEVICE_PTR(output, stream.device());
        NOA_ASSERT(noa::all(input_shape > 0) && noa::all(output_shape > 0));

        if (noa::math::sum(mask.as<i32>()) > 1) {
            NOA_THROW_FUNC(name,
                           "Reducing more than one axis at a time is only supported if the reduction results in "
                           "one value per batch, i.e. the 3 innermost dimensions are shape=1 after reduction. "
                           "Got input:{}, output:{}, reduce:{}", input_shape, output_shape, mask);
        }

        const auto u_input_strides = input_strides.as_safe<u32>();
        const auto u_input_shape = input_shape.as_safe<u32>();
        const auto u_output_strides = output_strides.as_safe<u32>();

        if (mask[3]) {
            const u32 block_dim_x = u_input_shape[3] > 512 ? 256 : 64;
            const dim3 threads(block_dim_x, BLOCK_SIZE / block_dim_x);
            const u32 blocks_y = noa::math::divide_up(u_input_shape[2], threads.y);
            const dim3 blocks(blocks_y, u_input_shape[1], u_input_shape[0]);
            const LaunchConfig config{blocks, threads, BLOCK_SIZE * sizeof(Reduced)};

            // Try to vectorize the loads within a row.
            // Check that the beginning of each row is at the same alignment. This is true for pitch2D arrays.
            u32 vector_size = u_input_strides[3] == 1 ? std::min(utils::max_vector_count(input), i64{4}) : 1;
            for (; vector_size >= 2; vector_size /= 2) {
                if ((!(u_input_strides[2] % vector_size) || u_input_shape[2] == 1) &&
                    (!(u_input_strides[1] % vector_size) || u_input_shape[1] == 1) &&
                    (!(u_input_strides[0] % vector_size) || u_input_shape[0] == 1))
                    break;
            }

            const auto input_accessor = AccessorRestrict<const Input, 4, u32>(input, u_input_strides);
            const auto output_accessor = AccessorRestrict<Output, 3, u32>(output, u_output_strides.pop_back());
            if (threads.x == 256) {
                stream.enqueue(name,
                               vector_size == 4 ? reduce_width_<Input, Reduced, Output, u32, PreProcessOp, ReduceOp, PostProcess, 256, 4> :
                               vector_size == 2 ? reduce_width_<Input, Reduced, Output, u32, PreProcessOp, ReduceOp, PostProcess, 256, 2> :
                                                  reduce_width_<Input, Reduced, Output, u32, PreProcessOp, ReduceOp, PostProcess, 256, 1>,
                               config, input_accessor, u_input_shape.pop_front<2>(), initial_reduce,
                               output_accessor, pre_process_op, reduce_op, post_process_op);
            } else {
                stream.enqueue(name,
                               vector_size == 4 ? reduce_width_<Input, Reduced, Output, u32, PreProcessOp, ReduceOp, PostProcess, 64, 4> :
                               vector_size == 2 ? reduce_width_<Input, Reduced, Output, u32, PreProcessOp, ReduceOp, PostProcess, 64, 2> :
                                                  reduce_width_<Input, Reduced, Output, u32, PreProcessOp, ReduceOp, PostProcess, 64, 1>,
                               config, input_accessor, u_input_shape.pop_front<2>(), initial_reduce,
                               output_accessor, pre_process_op, reduce_op, post_process_op);
            }
        } else if (mask[2]) {
            const u32 blocks_x = noa::math::divide_up(u_input_shape[3], BLOCK_SIZE_2D.x);
            const dim3 blocks(blocks_x, u_input_shape[1], u_input_shape[0]);
            const LaunchConfig config{blocks, BLOCK_SIZE_2D, BLOCK_SIZE * sizeof(Reduced)};
            stream.enqueue(name, reduce_height_<Input, Reduced, Output, u32, PreProcessOp, ReduceOp, PostProcess>, config,
                           input, u_input_strides, u_input_shape.pop_front<2>(), initial_reduce,
                           output, u_output_strides, pre_process_op, reduce_op, post_process_op);

        } else if (mask[1]) {
            const u32 blocks_x = noa::math::divide_up(u_input_shape[3], BLOCK_SIZE_2D.x);
            const dim3 blocks(blocks_x, u_input_shape[2], u_input_shape[0]);
            const LaunchConfig config{blocks, BLOCK_SIZE_2D, BLOCK_SIZE * sizeof(Reduced)};
            stream.enqueue(name, reduce_height_<Input, Reduced, Output, u32, PreProcessOp, ReduceOp, PostProcess>, config,
                           input, u_input_strides.filter(0, 2, 1, 3), u_input_shape.filter(1, 3), initial_reduce,
                           output, u_output_strides.filter(0, 2, 1, 3), pre_process_op, reduce_op, post_process_op);

        } else if (mask[0]) {
            const u32 blocks_x = noa::math::divide_up(u_input_shape[3], BLOCK_SIZE_2D.x);
            const dim3 blocks(blocks_x, u_input_shape[2], u_input_shape[1]);
            const LaunchConfig config{blocks, BLOCK_SIZE_2D, BLOCK_SIZE * sizeof(Reduced)};
            stream.enqueue(name, reduce_height_<Input, Reduced, Output, u32, PreProcessOp, ReduceOp, PostProcess>, config,
                           input, u_input_strides.filter(1, 2, 0, 3), u_input_shape.filter(0, 3), initial_reduce,
                           output, u_output_strides.filter(1, 2, 0, 3), pre_process_op, reduce_op, post_process_op);
        }
    }

    Vec4<bool> get_mask_(const char* func, const Shape4<i64>& input_shape, const Shape4<i64>& output_shape) {
        const Vec4<bool> mask{input_shape != output_shape};
        if (noa::any(mask && (output_shape != 1))) {
            NOA_THROW_FUNC(func,
                           "Dimensions should match the input shape, or be 1, indicating the dimension should be "
                           "reduced to one element. Got input:{}, output:{}", input_shape, output_shape);
        }
        return mask;
    }
}

namespace noa::cuda::math {
    template<typename Value, typename>
    void min(const Value* input, const Strides4<i64>& input_strides, const Shape4<i64>& input_shape,
             Value* output, const Strides4<i64>& output_strides, const Shape4<i64>& output_shape,
             Stream& stream) {
        const char* name = "math::min";
        const auto mask = get_mask_(name, input_shape, output_shape);
        const auto is_or_should_reduce = output_shape == 1 || mask;

        if (!any(mask))
            return cuda::memory::copy(input, input_strides, output, output_strides, output_shape, stream);

        if (is_or_should_reduce[1] && is_or_should_reduce[2] && is_or_should_reduce[3]) {
            utils::reduce_unary(name, input, input_strides, input_shape,
                                output, output_strides.filter(0), noa::math::Limits<Value>::max(),
                                noa::copy_t{}, noa::min_t{}, noa::copy_t{},
                                is_or_should_reduce[0], true, stream);
        } else {
            reduce_axis_(name,
                         input, input_strides, input_shape,
                         output, output_strides, output_shape,
                         mask, noa::math::Limits<Value>::max(),
                         noa::copy_t{}, noa::min_t{}, noa::copy_t{}, stream);
        }
    }

    template<typename Value, typename>
    void max(const Value* input, const Strides4<i64>& input_strides, const Shape4<i64>& input_shape,
             Value* output, const Strides4<i64>& output_strides, const Shape4<i64>& output_shape,
             Stream& stream) {
        const char* name = "math::max";
        const auto mask = get_mask_(name, input_shape, output_shape);
        const auto is_or_should_reduce = output_shape == 1 || mask;

        if (!any(mask))
            return cuda::memory::copy(input, input_strides, output, output_strides, output_shape, stream);

        if (is_or_should_reduce[1] && is_or_should_reduce[2] && is_or_should_reduce[3]) {
            utils::reduce_unary(name, input, input_strides, input_shape,
                                output, output_strides.filter(0), noa::math::Limits<Value>::lowest(),
                                noa::copy_t{}, noa::max_t{}, noa::copy_t{},
                                is_or_should_reduce[0], true, stream);
        } else {
            reduce_axis_(name,
                         input, input_strides, input_shape,
                         output, output_strides, output_shape,
                         mask, noa::math::Limits<Value>::lowest(),
                         noa::copy_t{}, noa::max_t{}, noa::copy_t{}, stream);
        }
    }

    template<typename Value, typename _>
    void sum(const Value* input, const Strides4<i64>& input_strides, const Shape4<i64>& input_shape,
             Value* output, const Strides4<i64>& output_strides, const Shape4<i64>& output_shape,
             Stream& stream) {
        const char* name = "math::sum";
        const auto mask = get_mask_(name, input_shape, output_shape);
        const auto is_or_should_reduce = output_shape == 1 || mask;

        if (!any(mask))
            return cuda::memory::copy(input, input_strides, output, output_strides, output_shape, stream);

        if (is_or_should_reduce[1] && is_or_should_reduce[2] && is_or_should_reduce[3]) {
            utils::reduce_unary(name, input, input_strides, input_shape,
                                output, output_strides.filter(0), Value{0},
                                noa::copy_t{}, noa::plus_t{}, noa::copy_t{},
                                is_or_should_reduce[0], true, stream);
        } else {
            using reduce_t = std::conditional_t<noa::traits::is_complex_v<Value>, c64,
                             std::conditional_t<noa::traits::is_real_v<Value>, f64, Value>>;
            const auto pre_process_op = []__device__(const Value& value) { return static_cast<reduce_t>(value); };
            const auto post_process_op = []__device__(const reduce_t& value) { return static_cast<Value>(value); };
            reduce_axis_(name,
                         input, input_strides, input_shape,
                         output, output_strides, output_shape,
                         mask, reduce_t{0},
                         pre_process_op, noa::plus_t{}, post_process_op, stream);
        }
    }

    template<typename Value, typename _>
    void mean(const Value* input, const Strides4<i64>& input_strides, const Shape4<i64>& input_shape,
              Value* output, const Strides4<i64>& output_strides, const Shape4<i64>& output_shape,
              Stream& stream) {
        const char* name = "math::mean";
        const auto mask = get_mask_(name, input_shape, output_shape);
        const auto is_or_should_reduce = output_shape == 1 || mask;

        if (!any(mask))
            return cuda::memory::copy(input, input_strides, output, output_strides, output_shape, stream);

        if (is_or_should_reduce[1] && is_or_should_reduce[2] && is_or_should_reduce[3]) {
            const auto element_per_batch =
                    input_shape[1] * input_shape[2] * input_shape[3] *
                    (is_or_should_reduce[0] ? input_shape[0] : 1);

            using real_t = noa::traits::value_type_t<Value>;
            const auto count = static_cast<real_t>(element_per_batch);
            auto sum_to_mean_op = [count]__device__(Value v) -> Value {
                if constexpr (noa::traits::is_int_v<real_t>) {
                    return static_cast<real_t>(noa::math::round(static_cast<f64>(v) / static_cast<f64>(count)));
                } else {
                    return v / count;
                }
            };

            utils::reduce_unary(name, input, input_strides, input_shape,
                                output, output_strides.filter(0), Value{0},
                                noa::copy_t{}, noa::plus_t{}, sum_to_mean_op,
                                is_or_should_reduce[0], true, stream);
        } else {
            // Since there's parallelism here, use double precision to preserve good accuracy.
            using reduce_t = std::conditional_t<noa::traits::is_complex_v<Value>, c64,
                             std::conditional_t<noa::traits::is_real_v<Value>, f64, Value>>;
            using reduce_real_t = noa::traits::value_type_t<reduce_t>;
            const auto count = static_cast<reduce_real_t>(noa::math::sum(input_shape * Shape4<i64>(mask)));

            const auto pre_process_op = []__device__(const Value& value) { return static_cast<reduce_t>(value); };
            auto sum_to_mean_op = [count]__device__(reduce_t value) -> Value {
                if constexpr (noa::traits::is_int_v<Value>) {
                    return static_cast<Value>(noa::math::round(static_cast<f64>(value) / static_cast<f64>(count)));
                } else {
                    return static_cast<Value>(value / count);
                }
            };
            reduce_axis_(name,
                         input, input_strides, input_shape,
                         output, output_strides, output_shape,
                         mask, reduce_t{0},
                         pre_process_op, noa::plus_t{}, sum_to_mean_op, stream);
        }
    }

    #define NOA_INSTANTIATE_REDUCE_(T)                          \
    template void min<T, void>(                                 \
        const T*, const Strides4<i64>&, const Shape4<i64>&,     \
        T*, const Strides4<i64>&, const Shape4<i64>&, Stream&); \
    template void max<T, void>(                                 \
        const T*, const Strides4<i64>&, const Shape4<i64>&,     \
        T*, const Strides4<i64>&, const Shape4<i64>&, Stream&); \
    template void sum<T, void>(                                 \
        const T*, const Strides4<i64>&, const Shape4<i64>&,     \
        T*, const Strides4<i64>&, const Shape4<i64>&, Stream&); \
    template void mean<T, void>(                                \
        const T*, const Strides4<i64>&, const Shape4<i64>&,     \
        T*, const Strides4<i64>&, const Shape4<i64>&, Stream&)

    NOA_INSTANTIATE_REDUCE_(f32);
    NOA_INSTANTIATE_REDUCE_(f64);
    NOA_INSTANTIATE_REDUCE_(u32);
    NOA_INSTANTIATE_REDUCE_(u64);
    NOA_INSTANTIATE_REDUCE_(i32);
    NOA_INSTANTIATE_REDUCE_(i64);

    #define NOA_INSTANTIATE_REDUCE_COMPLEX_(T)                  \
    template void sum<T, void>(                                 \
        const T*, const Strides4<i64>&, const Shape4<i64>&,     \
        T*, const Strides4<i64>&, const Shape4<i64>&, Stream&); \
    template void mean<T, void>(                                \
        const T*, const Strides4<i64>&, const Shape4<i64>&,     \
        T*, const Strides4<i64>&, const Shape4<i64>&, Stream&)

    NOA_INSTANTIATE_REDUCE_COMPLEX_(c32);
    NOA_INSTANTIATE_REDUCE_COMPLEX_(c64);
}
