#include "hip/hip_runtime.h"
#include "noa/common/Math.h"
#include "noa/gpu/cuda/math/Reduce.h"
#include "noa/gpu/cuda/math/Sort.h"
#include "noa/gpu/cuda/util/ReduceUnary.cuh"

namespace noa::cuda::math {
    template<typename T, typename>
    T min(const shared_t<T[]>& input, size4_t strides, size4_t shape, Stream& stream) {
        T output{};
        T* null{};
        util::reduce("math::min",
                     input.get(), uint4_t(strides), uint4_t(shape),
                     noa::math::copy_t{}, noa::math::min_t{}, noa::math::Limits<T>::max(),
                     &output, 1, noa::math::copy_t{}, null, 0, noa::math::copy_t{}, true, true, stream);
        stream.synchronize();
        return output;
    }

    template<typename T, typename>
    T max(const shared_t<T[]>& input, size4_t strides, size4_t shape, Stream& stream) {
        T output{};
        T* null{};
        util::reduce("math::max",
                     input.get(), uint4_t(strides), uint4_t(shape),
                     noa::math::copy_t{}, noa::math::max_t{}, noa::math::Limits<T>::lowest(),
                     &output, 1, noa::math::copy_t{}, null, 0, noa::math::copy_t{}, true, true, stream);
        stream.synchronize();
        return output;
    }

    template<typename T, typename>
    T median(const shared_t<T[]>& input, size4_t strides, size4_t shape,
             bool overwrite, Stream& stream) {
        // Make it in rightmost order.
        const size4_t order = indexing::order(strides, shape);
        strides = indexing::reorder(strides, order);
        shape = indexing::reorder(shape, order);

        const size_t elements = shape.elements();
        const shared_t<T[]>* to_sort;
        shared_t<T[]> buffer;
        if (overwrite && indexing::areContiguous(strides, shape)) {
            to_sort = &input;
        } else {
            buffer = memory::PtrDevice<T>::alloc(elements, stream);
            memory::copy(input, strides, buffer, shape.strides(), shape, stream);
            to_sort = &buffer;
        }

        // Sort the entire contiguous array.
        const size4_t shape_1d{1, 1, 1, elements};
        sort(*to_sort, shape_1d.strides(), shape_1d, true, -1, stream);

        // Retrieve the median.
        const bool is_even = !(elements % 2);
        T out[2];
        memory::copy(to_sort->get() + (elements - is_even) / 2, out, 1 + is_even, stream);
        stream.synchronize();

        if (is_even)
            return (out[0] + out[1]) / T{2};
        else
            return out[0];
    }

    template<typename T, typename>
    T sum(const shared_t<T[]>& input, size4_t strides, size4_t shape, Stream& stream) {
        T output{};
        T* null{};
        util::reduce("math::sum",
                     input.get(), uint4_t(strides), uint4_t(shape),
                     noa::math::copy_t{}, noa::math::plus_t{}, T(0),
                     &output, 1, noa::math::copy_t{}, null, 0, noa::math::copy_t{}, true, true, stream);
        stream.synchronize();
        return output;
    }

    template<typename T, typename U>
    T mean(const shared_t<T[]>& input, size4_t strides, size4_t shape, Stream& stream) {
        using real_t = noa::traits::value_type_t<T>;
        T output{};
        T* null{};
        const auto inv_count = static_cast<real_t>(shape.elements());
        auto sum_to_mean_op = [inv_count]__device__(T v) -> T { return v / inv_count; };
        util::reduce("math::mean",
                     input.get(), uint4_t(strides), uint4_t(shape),
                     noa::math::copy_t{}, noa::math::plus_t{}, T(0),
                     &output, 1, sum_to_mean_op, null, 0, noa::math::copy_t{}, true, true, stream);
        stream.synchronize();
        return output;
    }

    template<typename T, typename U, typename>
    U var(const shared_t<T[]>& input, size4_t strides, size4_t shape, int ddof, Stream& stream) {
        U output;
        util::reduceVar<false>("math::var", input.get(), uint4_t(strides), uint4_t(shape), &output, 1,
                               ddof, true, true, stream);
        stream.synchronize();
        return output;
    }

    template<typename T, typename U, typename>
    U std(const shared_t<T[]>& input, size4_t strides, size4_t shape, int ddof, Stream& stream) {
        U output;
        util::reduceVar<true>("math::std", input.get(), uint4_t(strides), uint4_t(shape), &output, 1,
                              ddof, true, true, stream);
        stream.synchronize();
        return output;
    }

    template<typename T, typename U, typename V>
    std::tuple<T, T, U, U> statistics(const shared_t<T[]>& input, size4_t strides, size4_t shape,
                                      int ddof, Stream& stream) {
        // Get the sum and mean:
        T output_sum, output_mean;
        const U inv_count = U(1) / static_cast<U>(shape.elements());
        auto sum_to_mean_op = [inv_count]__device__(T v) -> T { return v * inv_count; };
        util::reduce("math::statistics",
                     input.get(), uint4_t(strides), uint4_t(shape),
                     noa::math::copy_t{}, noa::math::plus_t{}, T{0},
                     &output_sum, 1, noa::math::copy_t{}, &output_mean, 0, sum_to_mean_op,
                     true, true, stream);

        stream.synchronize();
        T mean = output_sum / static_cast<U>(shape.elements() - ddof);

        // Get the variance and stddev:
        auto transform_op = [mean]__device__(T value) -> U {
            if constexpr (noa::traits::is_complex_v<T>) {
                const U distance = noa::math::abs(value - mean);
                return distance * distance;
            } else {
                const U distance = value - mean;
                return distance * distance;
            }
            return U(0); // unreachable
        };
        auto dist2_to_var = [inv_count]__device__(U v) -> U { return v * inv_count; };
        auto var_to_std = []__device__(U v) -> U { return noa::math::sqrt(v); };

        U output_var, output_std;
        util::reduce("math::statistics",
                     input.get(), uint4_t(strides), uint4_t(shape),
                     transform_op, noa::math::plus_t{}, U{0},
                     &output_var, 1, dist2_to_var, &output_std, 0, var_to_std,
                     true, true, stream);
        stream.synchronize();
        return {output_sum, output_mean, output_var, output_std};
    }

    #define NOA_INSTANTIATE_REDUCE_MIN_MAX_(T)                                  \
    template T min<T,void>(const shared_t<T[]>&, size4_t, size4_t, Stream&);    \
    template T max<T,void>(const shared_t<T[]>&, size4_t, size4_t, Stream&);    \
    template T median<T,void>(const shared_t<T[]>&, size4_t, size4_t, bool, Stream&)

    NOA_INSTANTIATE_REDUCE_MIN_MAX_(half_t);
    NOA_INSTANTIATE_REDUCE_MIN_MAX_(float);
    NOA_INSTANTIATE_REDUCE_MIN_MAX_(double);
    NOA_INSTANTIATE_REDUCE_MIN_MAX_(uint16_t);
    NOA_INSTANTIATE_REDUCE_MIN_MAX_(uint32_t);
    NOA_INSTANTIATE_REDUCE_MIN_MAX_(uint64_t);
    NOA_INSTANTIATE_REDUCE_MIN_MAX_(int16_t);
    NOA_INSTANTIATE_REDUCE_MIN_MAX_(int32_t);
    NOA_INSTANTIATE_REDUCE_MIN_MAX_(int64_t);

    #define NOA_INSTANTIATE_REDUCE_SUM_MEAN_(T)                                 \
    template T sum<T,void>(const shared_t<T[]>&, size4_t, size4_t, Stream&);    \
    template T mean<T,void>(const shared_t<T[]>&, size4_t, size4_t, Stream&)

    NOA_INSTANTIATE_REDUCE_SUM_MEAN_(float);
    NOA_INSTANTIATE_REDUCE_SUM_MEAN_(double);
    NOA_INSTANTIATE_REDUCE_SUM_MEAN_(uint32_t);
    NOA_INSTANTIATE_REDUCE_SUM_MEAN_(uint64_t);
    NOA_INSTANTIATE_REDUCE_SUM_MEAN_(int32_t);
    NOA_INSTANTIATE_REDUCE_SUM_MEAN_(int64_t);

    #define NOA_INSTANTIATE_REDUCE_COMPLEX(T)                                   \
    template T sum<T,void>(const shared_t<T[]>&, size4_t, size4_t, Stream&);    \
    template T mean<T,void>(const shared_t<T[]>&, size4_t, size4_t, Stream&)

    NOA_INSTANTIATE_REDUCE_COMPLEX(cfloat_t);
    NOA_INSTANTIATE_REDUCE_COMPLEX(cdouble_t);

    #define NOA_INSTANTIATE_VAR_(T,U)                                               \
    template U var<T,U,void>(const shared_t<T[]>&, size4_t, size4_t, int, Stream&); \
    template U std<T,U,void>(const shared_t<T[]>&, size4_t, size4_t, int, Stream&); \
    template std::tuple<T, T, U, U> statistics<T,U,void>(const shared_t<T[]>&, size4_t, size4_t, int, Stream&)

    NOA_INSTANTIATE_VAR_(float, float);
    NOA_INSTANTIATE_VAR_(double, double);
    NOA_INSTANTIATE_VAR_(cfloat_t, float);
    NOA_INSTANTIATE_VAR_(cdouble_t, double);
}
