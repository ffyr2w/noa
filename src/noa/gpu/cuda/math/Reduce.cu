#include "hip/hip_runtime.h"
#include "noa/common/Math.h"
#include "noa/gpu/cuda/math/Reduce.h"
#include "noa/gpu/cuda/util/ReduceUnary.cuh"

namespace noa::cuda::math {
    template<typename T, typename>
    T min(const shared_t<T[]>& input, size4_t stride, size4_t shape, Stream& stream) {
        T output{};
        T* null{};
        util::reduce<true>("math::min",
                           input.get(), uint4_t{stride}, uint4_t{shape},
                           noa::math::copy_t{}, noa::math::min_t{}, noa::math::Limits<T>::max(),
                           &output, 1, noa::math::copy_t{}, null, 0, noa::math::copy_t{}, stream);
        stream.synchronize();
        return output;
    }

    template<typename T, typename>
    T max(const shared_t<T[]>& input, size4_t stride, size4_t shape, Stream& stream) {
        T output{};
        T* null{};
        util::reduce<true>("math::max",
                           input.get(), uint4_t{stride}, uint4_t{shape},
                           noa::math::copy_t{}, noa::math::max_t{}, noa::math::Limits<T>::lowest(),
                           &output, 1, noa::math::copy_t{}, null, 0, noa::math::copy_t{}, stream);
        stream.synchronize();
        return output;
    }

    template<typename T, typename>
    T sum(const shared_t<T[]>& input, size4_t stride, size4_t shape, Stream& stream) {
        T output{};
        T* null{};
        util::reduce<true>("math::sum",
                          input.get(), uint4_t{stride}, uint4_t{shape},
                          noa::math::copy_t{}, noa::math::plus_t{}, T(0),
                          &output, 1, noa::math::copy_t{}, null, 0, noa::math::copy_t{}, stream);
        stream.synchronize();
        return output;
    }

    template<typename T, typename U>
    T mean(const shared_t<T[]>& input, size4_t stride, size4_t shape, Stream& stream) {
        using real_t = noa::traits::value_type_t<T>;
        T output{};
        T* null{};
        const auto inv_count = static_cast<real_t>(shape.elements());
        auto sum_to_mean_op = [inv_count]__device__(T v) -> T { return v / inv_count; };
        util::reduce<true>("math::mean",
                           input.get(), uint4_t{stride}, uint4_t{shape},
                           noa::math::copy_t{}, noa::math::plus_t{}, T(0),
                           &output, 1, sum_to_mean_op, null, 0, noa::math::copy_t{}, stream);
        stream.synchronize();
        return output;
    }

    template<int DDOF, typename T, typename U, typename>
    U var(const shared_t<T[]>& input, size4_t stride, size4_t shape, Stream& stream) {
        U output;
        util::reduceVar<DDOF, true, false>(
                "math::var", input.get(), uint4_t{stride}, uint4_t{shape}, &output, 1, stream);
        stream.synchronize();
        return output;
    }

    template<int DDOF, typename T, typename U, typename>
    U std(const shared_t<T[]>& input, size4_t stride, size4_t shape, Stream& stream) {
        U output;
        util::reduceVar<DDOF, true, true>(
                "math::std", input.get(), uint4_t{stride}, uint4_t{shape}, &output, 1, stream);
        stream.synchronize();
        return output;
    }

    template<int DDOF, typename T, typename U, typename V>
    std::tuple<T, T, U, U> statistics(const shared_t<T[]>& input, size4_t stride, size4_t shape, Stream& stream) {
        // Get the sum and mean:
        T output_sum, output_mean;
        const U inv_count = U(1) / static_cast<U>(shape.elements());
        auto sum_to_mean_op = [inv_count]__device__(T v) -> T { return v * inv_count; };
        util::reduce<true>("math::statistics",
                           input.get(), uint4_t{stride}, uint4_t{shape},
                           noa::math::copy_t{}, noa::math::plus_t{}, T{0},
                           &output_sum, 1, noa::math::copy_t{}, &output_mean, 0, sum_to_mean_op, stream);

        stream.synchronize();
        T mean = output_sum / static_cast<U>(shape.elements() - DDOF);

        // Get the variance and stddev:
        auto transform_op = [mean]__device__(T value) -> U {
            if constexpr (noa::traits::is_complex_v<T>) {
                const U distance = noa::math::abs(value - mean);
                return distance * distance;
            } else {
                const U distance = value - mean;
                return distance * distance;
            }
            return U(0); // unreachable
        };
        auto dist2_to_var = [inv_count]__device__(U v) -> U { return v * inv_count; };
        auto var_to_std = []__device__(U v) -> U { return noa::math::sqrt(v); };

        U output_var, output_std;
        util::reduce<true>("math::statistics",
                           input.get(), uint4_t{stride}, uint4_t{shape},
                           transform_op, noa::math::plus_t{}, U{0},
                           &output_var, 1, dist2_to_var, &output_std, 0, var_to_std, stream);
        stream.synchronize();
        return {output_sum, output_mean, output_var, output_std};
    }

    #define NOA_INSTANTIATE_REDUCE_MIN_MAX_(T)                                  \
    template T min<T,void>(const shared_t<T[]>&, size4_t, size4_t, Stream&);    \
    template T max<T,void>(const shared_t<T[]>&, size4_t, size4_t, Stream&)

    NOA_INSTANTIATE_REDUCE_MIN_MAX_(half_t);
    NOA_INSTANTIATE_REDUCE_MIN_MAX_(float);
    NOA_INSTANTIATE_REDUCE_MIN_MAX_(double);
    NOA_INSTANTIATE_REDUCE_MIN_MAX_(uint16_t);
    NOA_INSTANTIATE_REDUCE_MIN_MAX_(uint32_t);
    NOA_INSTANTIATE_REDUCE_MIN_MAX_(uint64_t);
    NOA_INSTANTIATE_REDUCE_MIN_MAX_(int16_t);
    NOA_INSTANTIATE_REDUCE_MIN_MAX_(int32_t);
    NOA_INSTANTIATE_REDUCE_MIN_MAX_(int64_t);

    #define NOA_INSTANTIATE_REDUCE_SUM_MEAN_(T)                                 \
    template T sum<T,void>(const shared_t<T[]>&, size4_t, size4_t, Stream&);    \
    template T mean<T,void>(const shared_t<T[]>&, size4_t, size4_t, Stream&)

    NOA_INSTANTIATE_REDUCE_SUM_MEAN_(float);
    NOA_INSTANTIATE_REDUCE_SUM_MEAN_(double);
    NOA_INSTANTIATE_REDUCE_SUM_MEAN_(uint32_t);
    NOA_INSTANTIATE_REDUCE_SUM_MEAN_(uint64_t);
    NOA_INSTANTIATE_REDUCE_SUM_MEAN_(int32_t);
    NOA_INSTANTIATE_REDUCE_SUM_MEAN_(int64_t);

    #define NOA_INSTANTIATE_REDUCE_COMPLEX(T)                                   \
    template T sum<T,void>(const shared_t<T[]>&, size4_t, size4_t, Stream&);    \
    template T mean<T,void>(const shared_t<T[]>&, size4_t, size4_t, Stream&)

    NOA_INSTANTIATE_REDUCE_COMPLEX(cfloat_t);
    NOA_INSTANTIATE_REDUCE_COMPLEX(cdouble_t);

    #define NOA_INSTANTIATE_VAR_(T,U,DDOF)                                          \
    template U var<DDOF,T,U,void>(const shared_t<T[]>&, size4_t, size4_t, Stream&); \
    template U std<DDOF,T,U,void>(const shared_t<T[]>&, size4_t, size4_t, Stream&); \
    template std::tuple<T, T, U, U> statistics<DDOF,T,U,void>(const shared_t<T[]>&, size4_t, size4_t, Stream&)

    NOA_INSTANTIATE_VAR_(float, float, 0);
    NOA_INSTANTIATE_VAR_(double, double, 0);
    NOA_INSTANTIATE_VAR_(float, float, 1);
    NOA_INSTANTIATE_VAR_(double, double, 1);

    NOA_INSTANTIATE_VAR_(cfloat_t, float, 0);
    NOA_INSTANTIATE_VAR_(cdouble_t, double, 0);
    NOA_INSTANTIATE_VAR_(cfloat_t, float, 1);
    NOA_INSTANTIATE_VAR_(cdouble_t, double, 1);
}
