#include "hip/hip_runtime.h"
#include "noa/gpu/cuda/math/ArithmeticsComposite.h"
#include "noa/gpu/cuda/Exception.h"
#include "noa/Math.h"

namespace {
    using namespace Noa;

    namespace Contiguous_ {
        constexpr uint BLOCK_SIZE = 256;

        // One block computes its elements and go to the corresponding elements
        // in next grid, until the end, for each batch.
        uint getBlocks_(uint elements) {
            constexpr uint MAX_GRIDS = 32768;
            return Noa::Math::min((elements + BLOCK_SIZE - 1) / BLOCK_SIZE, MAX_GRIDS);
        }

        template<typename T>
        __global__ void multiplyAddArray_(T* inputs, T* multipliers, T* addends, T* outputs, uint elements) {
            T* tmp_in = inputs + blockIdx.y * elements;
            T* tmp_out = outputs + blockIdx.y * elements;
            for (uint idx = blockIdx.x * BLOCK_SIZE + threadIdx.x; idx < elements; idx += BLOCK_SIZE * gridDim.x)
                tmp_out[idx] = tmp_in[idx] * multipliers[idx] + addends[idx];
        }

        template<typename T>
        __global__ void squaredDistanceFromValue_(T* inputs, T* values, T* outputs, size_t elements) {
            T* tmp_in = inputs + blockIdx.y * elements;
            T* tmp_out = outputs + blockIdx.y * elements;
            T value = values[blockIdx.y];
            T distance;
            for (uint idx = blockIdx.x * BLOCK_SIZE + threadIdx.x; idx < elements; idx += BLOCK_SIZE * gridDim.x) {
                distance = tmp_in[idx] - value;
                tmp_out[idx] = distance * distance;
            }
        }

        template<typename T>
        __global__ void squaredDistanceFromArray_(T* inputs, T* array, T* outputs, size_t elements) {
            T* tmp_in = inputs + blockIdx.y * elements;
            T* tmp_out = outputs + blockIdx.y * elements;
            T distance;
            for (uint idx = blockIdx.x * BLOCK_SIZE + threadIdx.x; idx < elements; idx += BLOCK_SIZE * gridDim.x) {
                distance = tmp_in[idx] - array[idx];
                tmp_out[idx] = distance * distance;
            }
        }
    }

    namespace Padded_ {
        constexpr dim3 BLOCK_SIZE(32, 8);

        uint getBlocks_(uint2_t shape_2d) {
            constexpr uint MAX_BLOCKS = 1024; // the smaller, the more work per warp.
            constexpr uint WARPS = BLOCK_SIZE.y; // warps per block; every warp processes at least one row.
            return Noa::Math::min((shape_2d.y + (WARPS - 1)) / WARPS, MAX_BLOCKS);
        }

        template<typename T>
        __global__ void multiplyAddArray_(T* inputs, uint pitch_inputs,
                                          T* multipliers, uint pitch_multipliers,
                                          T* addends, uint pitch_addends,
                                          T* outputs, uint pitch_outputs,
                                          uint2_t shape) {
            inputs += blockIdx.y * pitch_inputs * shape.y;
            outputs += blockIdx.y * pitch_outputs * shape.y;
            for (uint row = BLOCK_SIZE.y * blockIdx.x + threadIdx.y; row < shape.y; row += gridDim.x * BLOCK_SIZE.y) {
                for (uint idx = threadIdx.x; idx < shape.x; idx += BLOCK_SIZE.x)
                    outputs[row * pitch_outputs + idx] = inputs[row * pitch_inputs + idx] *
                                                         multipliers[row * pitch_multipliers + idx] +
                                                         addends[row * pitch_addends + idx];
            }
        }

        template<typename T>
        __global__ void squaredDistanceFromValue_(T* inputs, uint pitch_inputs, T* values,
                                                  T* outputs, uint pitch_outputs,
                                                  uint2_t shape) {
            inputs += blockIdx.y * pitch_inputs * shape.y;
            outputs += blockIdx.y * pitch_outputs * shape.y;
            T value = values[blockIdx.y];
            for (uint row = BLOCK_SIZE.y * blockIdx.x + threadIdx.y; row < shape.y; row += gridDim.x * BLOCK_SIZE.y) {
                for (uint idx = threadIdx.x; idx < shape.x; idx += BLOCK_SIZE.x) {
                    T distance = inputs[row * pitch_inputs + idx] - value;
                    outputs[row * pitch_outputs + idx] = distance * distance;
                }
            }
        }

        template<typename T>
        __global__ void squaredDistanceFromArray_(T* inputs, uint pitch_inputs,
                                                  T* array, uint pitch_array,
                                                  T* outputs, uint pitch_outputs,
                                                  uint2_t shape) {
            inputs += blockIdx.y * pitch_inputs * shape.y;
            outputs += blockIdx.y * pitch_outputs * shape.y;
            for (uint row = BLOCK_SIZE.y * blockIdx.x + threadIdx.y; row < shape.y; row += gridDim.x * BLOCK_SIZE.y) {
                for (uint idx = threadIdx.x; idx < shape.x; idx += BLOCK_SIZE.x) {
                    T distance = inputs[row * pitch_inputs + idx] - array[row * pitch_array + idx];
                    outputs[row * pitch_outputs + idx] = distance * distance;
                }
            }
        }
    }
}

namespace Noa::CUDA::Math {
    /* ------------------------ */
    /* --- multiplyAddArray --- */
    /* ------------------------ */

    template<typename T>
    void multiplyAddArray(T* inputs, T* multipliers, T* addends, T* outputs,
                          size_t elements, uint batches, Stream& stream) {
        uint blocks = Contiguous_::getBlocks_(elements);
        NOA_CUDA_LAUNCH(dim3(blocks, batches), Contiguous_::BLOCK_SIZE, 0, stream.get(),
                        Contiguous_::multiplyAddArray_,
                        inputs, multipliers, addends, outputs, elements);
    }

    template<typename T>
    void multiplyAddArray(T* inputs, size_t pitch_inputs,
                          T* multipliers, size_t pitch_multipliers,
                          T* addends, size_t pitch_addends,
                          T* outputs, size_t pitch_outputs,
                          size3_t shape, uint batches, Stream& stream) {
        uint2_t shape_2d(shape.x, getRows(shape));
        uint blocks = Padded_::getBlocks_(shape_2d);
        NOA_CUDA_LAUNCH(dim3(blocks, batches), Padded_::BLOCK_SIZE, 0, stream.get(),
                        Padded_::multiplyAddArray_,
                        inputs, pitch_inputs, multipliers, pitch_multipliers, addends, pitch_addends,
                        outputs, pitch_outputs, shape_2d);
    }

    /* ------------------------ */
    /* --- Squared distance --- */
    /* ------------------------ */

    template<typename T>
    void squaredDistanceFromValue(T* inputs, T* values, T* outputs,
                                  size_t elements, uint batches, Stream& stream) {
        uint blocks = Contiguous_::getBlocks_(elements);
        NOA_CUDA_LAUNCH(dim3(blocks, batches), Contiguous_::BLOCK_SIZE, 0, stream.get(),
                        Contiguous_::squaredDistanceFromValue_,
                        inputs, values, outputs, elements);
    }

    template<typename T>
    void squaredDistanceFromValue(T* inputs, size_t pitch_inputs, T* values,
                                  T* outputs, size_t pitch_outputs,
                                  size3_t shape, uint batches, Stream& stream) {
        uint2_t shape_2d(shape.x, getRows(shape));
        uint blocks = Padded_::getBlocks_(shape_2d);
        NOA_CUDA_LAUNCH(dim3(blocks, batches), Padded_::BLOCK_SIZE, 0, stream.get(),
                        Padded_::squaredDistanceFromValue_,
                        inputs, pitch_inputs, values, outputs, pitch_outputs, shape_2d);
    }

    template<typename T>
    void squaredDistanceFromArray(T* inputs, T* array, T* outputs,
                                  size_t elements, uint batches, Stream& stream) {
        uint blocks = Contiguous_::getBlocks_(elements);
        NOA_CUDA_LAUNCH(dim3(blocks, batches), Contiguous_::BLOCK_SIZE, 0, stream.get(),
                        Contiguous_::squaredDistanceFromArray_,
                        inputs, array, outputs, elements);
    }

    template<typename T>
    void squaredDistanceFromArray(T* inputs, size_t pitch_inputs,
                                  T* array, size_t pitch_array,
                                  T* outputs, size_t pitch_outputs,
                                  size3_t shape, uint batches, Stream& stream) {
        uint2_t shape_2d(shape.x, getRows(shape));
        uint blocks = Padded_::getBlocks_(shape_2d);
        NOA_CUDA_LAUNCH(dim3(blocks, batches), Padded_::BLOCK_SIZE, 0, stream.get(),
                        Padded_::squaredDistanceFromArray_,
                        inputs, pitch_inputs, array, pitch_array, outputs, pitch_outputs, shape_2d);
    }

    #define INSTANTIATE_COMPOSITES(T, U)                                                                        \
    template void multiplyAddArray<T>(T*, T*, T*, T*, size_t, uint, Stream&);                                   \
    template void multiplyAddArray<T>(T*, size_t, T*, size_t, T*, size_t, T*, size_t, size3_t, uint, Stream&);  \
    template void squaredDistanceFromValue<T>(T*, T*, T*, size_t, uint, Stream&);                               \
    template void squaredDistanceFromValue<T>(T*, size_t, T*, T*, size_t, size3_t, uint, Stream&);              \
    template void squaredDistanceFromArray<T>(T*, T*, T*, size_t, uint, Stream&);                               \
    template void squaredDistanceFromArray<T>(T*, size_t, T* array, size_t, T*, size_t, size3_t, uint, Stream&)

    INSTANTIATE_COMPOSITES(float, float);
    INSTANTIATE_COMPOSITES(double, double);
    INSTANTIATE_COMPOSITES(int, int);
    INSTANTIATE_COMPOSITES(uint, uint);
}
