#include "hip/hip_runtime.h"
#include "noa/common/Math.h"
#include "noa/gpu/cuda/Exception.h"
#include "noa/gpu/cuda/math/ArithmeticsComposite.h"

namespace {
    using namespace noa;

    namespace contiguous_ {
        constexpr uint BLOCK_SIZE = 256;

        // One block computes its elements and go to the corresponding elements
        // in next grid, until the end, for each batch.
        uint getBlocks_(uint elements) {
            constexpr uint MAX_GRIDS = 32768;
            return noa::math::min((elements + BLOCK_SIZE - 1) / BLOCK_SIZE, MAX_GRIDS);
        }

        template<typename T>
        __global__ void multiplyAddArray_(const T* inputs, const T* multipliers, const T* addends,
                                          T* outputs, uint elements) {
            const T* tmp_in = inputs + blockIdx.y * elements;
            T* tmp_out = outputs + blockIdx.y * elements;
            for (uint idx = blockIdx.x * BLOCK_SIZE + threadIdx.x; idx < elements; idx += BLOCK_SIZE * gridDim.x)
                tmp_out[idx] = tmp_in[idx] * multipliers[idx] + addends[idx];
        }

        template<typename T>
        __global__ void squaredDistanceFromValue_(const T* inputs, const T* values, T* outputs, size_t elements) {
            const T* tmp_in = inputs + blockIdx.y * elements;
            T* tmp_out = outputs + blockIdx.y * elements;
            T value = values[blockIdx.y];
            T distance;
            for (uint idx = blockIdx.x * BLOCK_SIZE + threadIdx.x; idx < elements; idx += BLOCK_SIZE * gridDim.x) {
                distance = tmp_in[idx] - value;
                tmp_out[idx] = distance * distance;
            }
        }

        template<typename T>
        __global__ void squaredDistanceFromValue_(const T* inputs, T value, T* outputs, size_t elements) {
            const T* tmp_in = inputs + blockIdx.y * elements;
            T* tmp_out = outputs + blockIdx.y * elements;
            T distance;
            for (uint idx = blockIdx.x * BLOCK_SIZE + threadIdx.x; idx < elements; idx += BLOCK_SIZE * gridDim.x) {
                distance = tmp_in[idx] - value;
                tmp_out[idx] = distance * distance;
            }
        }

        template<typename T>
        __global__ void squaredDistanceFromArray_(const T* inputs, const T* array, T* outputs, size_t elements) {
            const T* tmp_in = inputs + blockIdx.y * elements;
            T* tmp_out = outputs + blockIdx.y * elements;
            T distance;
            for (uint idx = blockIdx.x * BLOCK_SIZE + threadIdx.x; idx < elements; idx += BLOCK_SIZE * gridDim.x) {
                distance = tmp_in[idx] - array[idx];
                tmp_out[idx] = distance * distance;
            }
        }
    }

    namespace padded_ {
        constexpr dim3 BLOCK_SIZE(32, 8);

        uint getBlocks_(uint2_t shape_2d) {
            constexpr uint MAX_BLOCKS = 1024; // the smaller, the more work per warp.
            constexpr uint WARPS = BLOCK_SIZE.y; // warps per block; every warp processes at least one row.
            return noa::math::min((shape_2d.y + (WARPS - 1)) / WARPS, MAX_BLOCKS);
        }

        template<typename T>
        __global__ void multiplyAddArray_(const T* inputs, uint inputs_pitch,
                                          const T* multipliers, uint multipliers_pitch,
                                          const T* addends, uint addends_pitch,
                                          T* outputs, uint outputs_pitch,
                                          uint2_t shape) {
            inputs += blockIdx.y * inputs_pitch * shape.y;
            outputs += blockIdx.y * outputs_pitch * shape.y;
            for (uint row = BLOCK_SIZE.y * blockIdx.x + threadIdx.y; row < shape.y; row += gridDim.x * BLOCK_SIZE.y)
                for (uint idx = threadIdx.x; idx < shape.x; idx += BLOCK_SIZE.x)
                    outputs[row * outputs_pitch + idx] = inputs[row * inputs_pitch + idx] *
                                                         multipliers[row * multipliers_pitch + idx] +
                                                         addends[row * addends_pitch + idx];
        }

        template<typename T>
        __global__ void squaredDistanceFromValue_(const T* inputs, uint inputs_pitch, const T* values,
                                                  T* outputs, uint outputs_pitch,
                                                  uint2_t shape) {
            inputs += blockIdx.y * inputs_pitch * shape.y;
            outputs += blockIdx.y * outputs_pitch * shape.y;
            T value = values[blockIdx.y];
            for (uint row = BLOCK_SIZE.y * blockIdx.x + threadIdx.y; row < shape.y; row += gridDim.x * BLOCK_SIZE.y) {
                for (uint idx = threadIdx.x; idx < shape.x; idx += BLOCK_SIZE.x) {
                    T distance = inputs[row * inputs_pitch + idx] - value;
                    outputs[row * outputs_pitch + idx] = distance * distance;
                }
            }
        }

        template<typename T>
        __global__ void squaredDistanceFromValue_(const T* inputs, uint inputs_pitch, T value,
                                                  T* outputs, uint outputs_pitch,
                                                  uint2_t shape) {
            inputs += blockIdx.y * inputs_pitch * shape.y;
            outputs += blockIdx.y * outputs_pitch * shape.y;
            for (uint row = BLOCK_SIZE.y * blockIdx.x + threadIdx.y; row < shape.y; row += gridDim.x * BLOCK_SIZE.y) {
                for (uint idx = threadIdx.x; idx < shape.x; idx += BLOCK_SIZE.x) {
                    T distance = inputs[row * inputs_pitch + idx] - value;
                    outputs[row * outputs_pitch + idx] = distance * distance;
                }
            }
        }

        template<typename T>
        __global__ void squaredDistanceFromArray_(const T* inputs, uint inputs_pitch,
                                                  const T* array, uint array_pitch,
                                                  T* outputs, uint outputs_pitch,
                                                  uint2_t shape) {
            inputs += blockIdx.y * inputs_pitch * shape.y;
            outputs += blockIdx.y * outputs_pitch * shape.y;
            for (uint row = BLOCK_SIZE.y * blockIdx.x + threadIdx.y; row < shape.y; row += gridDim.x * BLOCK_SIZE.y) {
                for (uint idx = threadIdx.x; idx < shape.x; idx += BLOCK_SIZE.x) {
                    T distance = inputs[row * inputs_pitch + idx] - array[row * array_pitch + idx];
                    outputs[row * outputs_pitch + idx] = distance * distance;
                }
            }
        }
    }
}

namespace noa::cuda::math {
    template<typename T>
    void multiplyAddArray(const T* inputs, const T* multipliers, const T* addends, T* outputs,
                          size_t elements, uint batches, Stream& stream) {
        uint blocks = contiguous_::getBlocks_(elements);
        contiguous_::multiplyAddArray_<<<dim3(blocks, batches), contiguous_::BLOCK_SIZE, 0, stream.get()>>>(
                inputs, multipliers, addends, outputs, elements);
        NOA_THROW_IF(hipPeekAtLastError());
    }

    template<typename T>
    void multiplyAddArray(const T* inputs, size_t inputs_pitch,
                          const T* multipliers, size_t multipliers_pitch,
                          const T* addends, size_t addends_pitch,
                          T* outputs, size_t outputs_pitch,
                          size3_t shape, uint batches, Stream& stream) {
        uint2_t shape_2d(shape.x, getRows(shape));
        dim3 blocks(padded_::getBlocks_(shape_2d), batches);
        padded_::multiplyAddArray_<<<blocks, padded_::BLOCK_SIZE, 0, stream.get()>>>(
                inputs, inputs_pitch, multipliers, multipliers_pitch, addends,
                        addends_pitch, outputs, outputs_pitch, shape_2d);
        NOA_THROW_IF(hipPeekAtLastError());
    }

    template<typename T>
    void squaredDistanceFromValue(const T* inputs, const T* values, T* outputs,
                                  size_t elements, uint batches, Stream& stream) {
        uint blocks = contiguous_::getBlocks_(elements);
        contiguous_::squaredDistanceFromValue_<<<dim3(blocks, batches), contiguous_::BLOCK_SIZE, 0, stream.get()>>>(
                inputs, values, outputs, elements);
        NOA_THROW_IF(hipPeekAtLastError());
    }

    template<typename T>
    void squaredDistanceFromValue(const T* inputs, size_t inputs_pitch, const T* values,
                                  T* outputs, size_t outputs_pitch,
                                  size3_t shape, uint batches, Stream& stream) {
        uint2_t shape_2d(shape.x, getRows(shape));
        uint blocks = padded_::getBlocks_(shape_2d);
        padded_::squaredDistanceFromValue_<<<dim3(blocks, batches), padded_::BLOCK_SIZE, 0, stream.get()>>>(
                inputs, inputs_pitch, values, outputs, outputs_pitch, shape_2d);
        NOA_THROW_IF(hipPeekAtLastError());
    }

    template<typename T>
    void squaredDistanceFromValue(const T* inputs, T value, T* outputs,
                                  size_t elements, uint batches, Stream& stream) {
        uint blocks = contiguous_::getBlocks_(elements);
        contiguous_::squaredDistanceFromValue_<<<dim3(blocks, batches), contiguous_::BLOCK_SIZE, 0, stream.get()>>>(
                inputs, value, outputs, elements);
        NOA_THROW_IF(hipPeekAtLastError());
    }

    template<typename T>
    void squaredDistanceFromValue(const T* inputs, size_t inputs_pitch, T value,
                                  T* outputs, size_t outputs_pitch,
                                  size3_t shape, uint batches, Stream& stream) {
        uint2_t shape_2d(shape.x, getRows(shape));
        uint blocks = padded_::getBlocks_(shape_2d);
        padded_::squaredDistanceFromValue_<<<dim3(blocks, batches), padded_::BLOCK_SIZE, 0, stream.get()>>>(
                inputs, inputs_pitch, value, outputs, outputs_pitch, shape_2d);
        NOA_THROW_IF(hipPeekAtLastError());
    }

    template<typename T>
    void squaredDistanceFromArray(const T* inputs, const T* array, T* outputs,
                                  size_t elements, uint batches, Stream& stream) {
        uint blocks = contiguous_::getBlocks_(elements);
        contiguous_::squaredDistanceFromArray_<<<dim3(blocks, batches), contiguous_::BLOCK_SIZE, 0, stream.get()>>>(
                inputs, array, outputs, elements);
        NOA_THROW_IF(hipPeekAtLastError());
    }

    template<typename T>
    void squaredDistanceFromArray(const T* inputs, size_t inputs_pitch,
                                  const T* array, size_t array_pitch,
                                  T* outputs, size_t outputs_pitch,
                                  size3_t shape, uint batches, Stream& stream) {
        uint2_t shape_2d(shape.x, getRows(shape));
        uint blocks = padded_::getBlocks_(shape_2d);
        padded_::squaredDistanceFromArray_<<<dim3(blocks, batches), padded_::BLOCK_SIZE, 0, stream.get()>>>(
                inputs, inputs_pitch, array, array_pitch, outputs, outputs_pitch, shape_2d);
        NOA_THROW_IF(hipPeekAtLastError());
    }

    #define NOA_INSTANTIATE_ARITHMETICS_COMPOSITE_(T)                                                                               \
    template void multiplyAddArray<T>(const T*, const T*, const T*, T*, size_t, uint, Stream&);                                     \
    template void multiplyAddArray<T>(const T*, size_t, const T*, size_t, const T*, size_t, T*, size_t, size3_t, uint, Stream&);    \
    template void squaredDistanceFromValue<T>(const T*, const T*, T*, size_t, uint, Stream&);                                       \
    template void squaredDistanceFromValue<T>(const T*, size_t, const T*, T*, size_t, size3_t, uint, Stream&);                      \
    template void squaredDistanceFromValue<T>(const T*, T, T*, size_t, uint, Stream&);                                              \
    template void squaredDistanceFromValue<T>(const T*, size_t, T, T*, size_t, size3_t, uint, Stream&);                             \
    template void squaredDistanceFromArray<T>(const T*, const T*, T*, size_t, uint, Stream&);                                       \
    template void squaredDistanceFromArray<T>(const T*, size_t, const T*, size_t, T*, size_t, size3_t, uint, Stream&)

    NOA_INSTANTIATE_ARITHMETICS_COMPOSITE_(int);
    NOA_INSTANTIATE_ARITHMETICS_COMPOSITE_(long);
    NOA_INSTANTIATE_ARITHMETICS_COMPOSITE_(long long);
    NOA_INSTANTIATE_ARITHMETICS_COMPOSITE_(unsigned int);
    NOA_INSTANTIATE_ARITHMETICS_COMPOSITE_(unsigned long);
    NOA_INSTANTIATE_ARITHMETICS_COMPOSITE_(unsigned long long);
    NOA_INSTANTIATE_ARITHMETICS_COMPOSITE_(float);
    NOA_INSTANTIATE_ARITHMETICS_COMPOSITE_(double);
}
