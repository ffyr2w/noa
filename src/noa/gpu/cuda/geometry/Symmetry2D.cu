#include "hip/hip_runtime.h"
#include "noa/common/Assert.h"
#include "noa/gpu/cuda/Exception.h"

#include "noa/gpu/cuda/memory/Copy.h"
#include "noa/gpu/cuda/memory/PtrArray.h"
#include "noa/gpu/cuda/memory/PtrDevice.h"
#include "noa/gpu/cuda/memory/PtrTexture.h"

#include "noa/gpu/cuda/geometry/Interpolate.h"
#include "noa/gpu/cuda/geometry/Prefilter.h"
#include "noa/gpu/cuda/geometry/Symmetry.h"

namespace {
    using namespace ::noa;
    constexpr dim3 THREADS(16, 16);

    template<typename T, InterpMode INTERP>
    __global__ void __launch_bounds__(THREADS.x * THREADS.y)
    symmetrize2D_(hipTextureObject_t texture, T* output, uint3_t output_stride, uint2_t shape,
                  const float33_t* symmetry_matrices, uint symmetry_count, float2_t center, float scaling) {
        const uint3_t gid{blockIdx.z,
                          blockIdx.y * THREADS.y + threadIdx.y,
                          blockIdx.x * THREADS.x + threadIdx.x};
        if (gid[1] >= shape[0] || gid[2] >= shape[1])
            return;

        float2_t coordinates{gid[1], gid[2]};
        T value = cuda::geometry::tex2D<T, INTERP>(texture, coordinates + 0.5f);
        coordinates -= center;
        for (uint i = 0; i < symmetry_count; ++i) {
            const float33_t& m = symmetry_matrices[i];
            float22_t sym_matrix{m[1][1], m[1][2],
                                 m[2][1], m[2][2]};
            float2_t i_coordinates{sym_matrix * coordinates};
            value += cuda::geometry::tex2D<T, INTERP>(texture, i_coordinates + center + 0.5f);
        }

        output[indexing::at(gid, output_stride)] = value * scaling;
    }
}

namespace noa::cuda::geometry {
    template<bool PREFILTER, typename T, typename>
    void symmetrize2D(const shared_t<T[]>& input, size4_t input_stride,
                      const shared_t<T[]>& output, size4_t output_stride,
                      size4_t shape, const Symmetry& symmetry, float2_t center,
                      InterpMode interp_mode, bool normalize, Stream& stream) {
        NOA_ASSERT(shape[1] == 1);

        if (!symmetry.count()) {
            if (input != output)
                memory::copy(input, input_stride, output, output_stride, shape, stream);
            return;
        }

        size4_t input_shape{shape};
        if (input_stride[0] == 0)
            input_shape[0] = 1;

        // Prepare the input array:
        const T* buffer_ptr;
        size_t buffer_pitch;
        size_t buffer_offset;
        if (PREFILTER && (interp_mode == INTERP_CUBIC_BSPLINE || interp_mode == INTERP_CUBIC_BSPLINE_FAST)) {
            NOA_ASSERT(indexing::isContiguous(output_stride, shape)[3]);
            // Whether input is batched or not, since we copy to the CUDA array, we can use the output as buffer.
            cuda::geometry::bspline::prefilter(input, input_stride, output, output_stride, input_shape, stream);
            buffer_ptr = output.get();
            buffer_pitch = output_stride[2];
            buffer_offset = output_stride[0];
        } else {
            NOA_ASSERT(indexing::isContiguous(input_stride, input_shape)[3]);
            buffer_ptr = input.get();
            buffer_pitch = input_stride[2];
            buffer_offset = input_stride[0];
        }

        // Broadcast input if it is not batched:
        const size4_t o_shape{input_shape[0] > 1 ? 1 : shape[0],
                              shape[1], shape[2], shape[3]};

        // Copy to texture and launch (per input batch):
        const size3_t shape_3d{1, input_shape[2], input_shape[3]};
        cuda::memory::PtrArray<T> array{shape_3d};
        cuda::memory::PtrTexture texture{array.get(), interp_mode, BORDER_ZERO};
        for (size_t i = 0; i < input_shape[0]; ++i) {
            cuda::memory::copy(buffer_ptr + i * buffer_offset, buffer_pitch, array.get(), shape_3d, stream);
            cuda::geometry::symmetrize2D(
                    texture.get(), interp_mode, output.get() + i * output_stride[0], output_stride, o_shape,
                    symmetry, center, normalize, stream);
        }
        stream.attach(input, output, array.share(), texture.share());
    }

    template<typename T, typename>
    void symmetrize2D(hipTextureObject_t texture, InterpMode texture_interp_mode,
                      T* output, size4_t output_stride, size4_t output_shape,
                      const Symmetry& symmetry, float2_t center, bool normalize, Stream& stream) {
        NOA_ASSERT(!memory::PtrTexture::hasNormalizedCoordinates(texture));

        // TODO Move symmetry matrices to constant memory?
        const size_t count = symmetry.count();
        const float33_t* symmetry_matrices = symmetry.matrices();
        memory::PtrDevice<float33_t> d_matrices{count, stream};
        memory::copy(symmetry_matrices, d_matrices.get(), count, stream);
        const float scaling = normalize ? 1 / static_cast<float>(count + 1) : 1;

        const uint2_t o_shape{output_shape.get() + 2};
        const uint3_t o_stride{output_stride[0], output_stride[2], output_stride[3]};
        const dim3 blocks(math::divideUp(o_shape[1], THREADS.x),
                          math::divideUp(o_shape[0], THREADS.y),
                          output_shape[0]);
        const LaunchConfig config{blocks, THREADS};

        switch (texture_interp_mode) {
            case INTERP_NEAREST:
                return stream.enqueue("geometry::symmetrize2D", symmetrize2D_<T, INTERP_NEAREST>,
                                      config, texture, output, o_stride, o_shape,
                                      d_matrices.get(), count, center, scaling);
            case INTERP_LINEAR:
                return stream.enqueue("geometry::symmetrize2D", symmetrize2D_<T, INTERP_LINEAR>,
                                      config, texture, output, o_stride, o_shape,
                                      d_matrices.get(), count, center, scaling);
            case INTERP_COSINE:
                return stream.enqueue("geometry::symmetrize2D", symmetrize2D_<T, INTERP_COSINE>,
                                      config, texture, output, o_stride, o_shape,
                                      d_matrices.get(), count, center, scaling);
            case INTERP_CUBIC:
                return stream.enqueue("geometry::symmetrize2D", symmetrize2D_<T, INTERP_CUBIC>,
                                      config, texture, output, o_stride, o_shape,
                                      d_matrices.get(), count, center, scaling);
            case INTERP_CUBIC_BSPLINE:
                return stream.enqueue("geometry::symmetrize2D", symmetrize2D_<T, INTERP_CUBIC_BSPLINE>,
                                      config, texture, output, o_stride, o_shape,
                                      d_matrices.get(), count, center, scaling);
            case INTERP_LINEAR_FAST:
                return stream.enqueue("geometry::symmetrize2D", symmetrize2D_<T, INTERP_LINEAR_FAST>,
                                      config, texture, output, o_stride, o_shape,
                                      d_matrices.get(), count, center, scaling);
            case INTERP_COSINE_FAST:
                return stream.enqueue("geometry::symmetrize2D", symmetrize2D_<T, INTERP_COSINE_FAST>,
                                      config, texture, output, o_stride, o_shape,
                                      d_matrices.get(), count, center, scaling);
            case INTERP_CUBIC_BSPLINE_FAST:
                return stream.enqueue("geometry::symmetrize2D", symmetrize2D_<T, INTERP_CUBIC_BSPLINE_FAST>,
                                      config, texture, output, o_stride, o_shape,
                                      d_matrices.get(), count, center, scaling);
            default:
                NOA_THROW("{} is not supported", texture_interp_mode);
        }
    }

    #define NOA_INSTANTIATE_TRANSFORM_SYM_(T)                                                                                                                               \
    template void symmetrize2D<true, T, void>(const shared_t<T[]>&, size4_t, const shared_t<T[]>&, size4_t, size4_t, const Symmetry&, float2_t, InterpMode, bool, Stream&); \
    template void symmetrize2D<false, T, void>(const shared_t<T[]>&, size4_t, const shared_t<T[]>&, size4_t, size4_t, const Symmetry&, float2_t, InterpMode, bool, Stream&);

    NOA_INSTANTIATE_TRANSFORM_SYM_(float);
    NOA_INSTANTIATE_TRANSFORM_SYM_(cfloat_t);
}
