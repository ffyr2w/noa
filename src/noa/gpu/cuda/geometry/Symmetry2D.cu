#include "hip/hip_runtime.h"
#include "noa/common/Assert.h"
#include "noa/gpu/cuda/Exception.h"

#include "noa/gpu/cuda/memory/Copy.h"
#include "noa/gpu/cuda/memory/PtrArray.h"
#include "noa/gpu/cuda/memory/PtrDevice.h"
#include "noa/gpu/cuda/memory/PtrTexture.h"

#include "noa/gpu/cuda/geometry/Interpolate.h"
#include "noa/gpu/cuda/geometry/Prefilter.h"
#include "noa/gpu/cuda/geometry/Symmetry.h"

namespace {
    using namespace ::noa;
    constexpr dim3 THREADS(16, 16);

    template<typename T, InterpMode INTERP>
    __global__ void __launch_bounds__(THREADS.x * THREADS.y)
    symmetrize2D_(hipTextureObject_t texture, T* output, uint3_t output_strides, uint2_t shape,
                  const float33_t* symmetry_matrices, uint symmetry_count, float2_t center, float scaling) {
        const uint3_t gid{blockIdx.z,
                          blockIdx.y * THREADS.y + threadIdx.y,
                          blockIdx.x * THREADS.x + threadIdx.x};
        if (gid[1] >= shape[0] || gid[2] >= shape[1])
            return;

        float2_t coordinates{gid[1], gid[2]};
        T value = cuda::geometry::tex2D<T, INTERP>(texture, coordinates + 0.5f);
        coordinates -= center;
        for (uint i = 0; i < symmetry_count; ++i) {
            const float33_t& m = symmetry_matrices[i];
            float22_t sym_matrix{m[1][1], m[1][2],
                                 m[2][1], m[2][2]};
            float2_t i_coordinates(sym_matrix * coordinates);
            value += cuda::geometry::tex2D<T, INTERP>(texture, i_coordinates + center + 0.5f);
        }

        output[indexing::at(gid, output_strides)] = value * scaling;
    }


    template<typename T>
    void launchSymmetrize2D_(hipTextureObject_t texture, InterpMode texture_interp_mode,
                             T* output, size4_t output_strides, size4_t output_shape,
                             const geometry::Symmetry& symmetry, float2_t center, bool normalize,
                             cuda::Stream& stream) {
        NOA_ASSERT(!cuda::memory::PtrTexture::hasNormalizedCoordinates(texture));

        // TODO Move symmetry matrices to constant memory?
        const size_t count = symmetry.count();
        const float33_t* symmetry_matrices = symmetry.matrices();
        cuda::memory::PtrDevice<float33_t> d_matrices(count, stream);
        cuda::memory::copy(symmetry_matrices, d_matrices.get(), count, stream);
        const float scaling = normalize ? 1 / static_cast<float>(count + 1) : 1;

        const uint2_t o_shape(output_shape.get(2));
        const uint3_t o_strides{output_strides[0], output_strides[2], output_strides[3]};
        const dim3 blocks(math::divideUp(o_shape[1], THREADS.x),
                          math::divideUp(o_shape[0], THREADS.y),
                          output_shape[0]);
        const cuda::LaunchConfig config{blocks, THREADS};

        switch (texture_interp_mode) {
            case INTERP_NEAREST:
                return stream.enqueue("geometry::symmetrize2D", symmetrize2D_<T, INTERP_NEAREST>,
                                      config, texture, output, o_strides, o_shape,
                                      d_matrices.get(), count, center, scaling);
            case INTERP_LINEAR:
                return stream.enqueue("geometry::symmetrize2D", symmetrize2D_<T, INTERP_LINEAR>,
                                      config, texture, output, o_strides, o_shape,
                                      d_matrices.get(), count, center, scaling);
            case INTERP_COSINE:
                return stream.enqueue("geometry::symmetrize2D", symmetrize2D_<T, INTERP_COSINE>,
                                      config, texture, output, o_strides, o_shape,
                                      d_matrices.get(), count, center, scaling);
            case INTERP_CUBIC:
                return stream.enqueue("geometry::symmetrize2D", symmetrize2D_<T, INTERP_CUBIC>,
                                      config, texture, output, o_strides, o_shape,
                                      d_matrices.get(), count, center, scaling);
            case INTERP_CUBIC_BSPLINE:
                return stream.enqueue("geometry::symmetrize2D", symmetrize2D_<T, INTERP_CUBIC_BSPLINE>,
                                      config, texture, output, o_strides, o_shape,
                                      d_matrices.get(), count, center, scaling);
            case INTERP_LINEAR_FAST:
                return stream.enqueue("geometry::symmetrize2D", symmetrize2D_<T, INTERP_LINEAR_FAST>,
                                      config, texture, output, o_strides, o_shape,
                                      d_matrices.get(), count, center, scaling);
            case INTERP_COSINE_FAST:
                return stream.enqueue("geometry::symmetrize2D", symmetrize2D_<T, INTERP_COSINE_FAST>,
                                      config, texture, output, o_strides, o_shape,
                                      d_matrices.get(), count, center, scaling);
            case INTERP_CUBIC_BSPLINE_FAST:
                return stream.enqueue("geometry::symmetrize2D", symmetrize2D_<T, INTERP_CUBIC_BSPLINE_FAST>,
                                      config, texture, output, o_strides, o_shape,
                                      d_matrices.get(), count, center, scaling);
            default:
                NOA_THROW("{} is not supported", texture_interp_mode);
        }
    }
}

namespace noa::cuda::geometry {
    template<typename T, typename>
    void symmetrize2D(const shared_t<T[]>& input, size4_t input_strides,
                      const shared_t<T[]>& output, size4_t output_strides,
                      size4_t shape, const Symmetry& symmetry, float2_t center,
                      InterpMode interp_mode, bool prefilter, bool normalize, Stream& stream) {
        NOA_ASSERT(shape[1] == 1);

        if (!symmetry.count()) {
            if (input != output)
                memory::copy(input, input_strides, output, output_strides, shape, stream);
            return;
        }

        size4_t input_shape(shape);
        if (input_strides[0] == 0)
            input_shape[0] = 1;

        // Prepare the input array:
        const T* buffer_ptr;
        size_t buffer_pitch;
        size_t buffer_offset;
        if (prefilter && (interp_mode == INTERP_CUBIC_BSPLINE || interp_mode == INTERP_CUBIC_BSPLINE_FAST)) {
            NOA_ASSERT(indexing::isContiguous(output_strides, shape)[3]);
            // Whether input is batched or not, since we copy to the CUDA array, we can use the output as buffer.
            cuda::geometry::bspline::prefilter(input, input_strides, output, output_strides, input_shape, stream);
            buffer_ptr = output.get();
            buffer_pitch = output_strides[2];
            buffer_offset = output_strides[0];
        } else {
            NOA_ASSERT(indexing::isContiguous(input_strides, input_shape)[3]);
            buffer_ptr = input.get();
            buffer_pitch = input_strides[2];
            buffer_offset = input_strides[0];
        }

        // Broadcast input if it is not batched:
        const size4_t o_shape{input_shape[0] > 1 ? 1 : shape[0],
                              shape[1], shape[2], shape[3]};

        // Copy to texture and launch (per input batch):
        const size3_t shape_3d{1, input_shape[2], input_shape[3]};
        cuda::memory::PtrArray<T> array(shape_3d);
        cuda::memory::PtrTexture texture(array.get(), interp_mode, BORDER_ZERO);
        for (size_t i = 0; i < input_shape[0]; ++i) {
            cuda::memory::copy(buffer_ptr + i * buffer_offset, buffer_pitch, array.get(), shape_3d, stream);
            launchSymmetrize2D_(
                    texture.get(), interp_mode, output.get() + i * output_strides[0], output_strides, o_shape,
                    symmetry, center, normalize, stream);
        }
        stream.attach(input, output, array.share(), texture.share());
    }

    template<typename T, typename>
    void symmetrize2D(const shared_t<hipArray>& array,
                      const shared_t<hipTextureObject_t>& texture, InterpMode texture_interp_mode,
                      const shared_t<T[]>& output, size4_t output_strides, size4_t output_shape,
                      const Symmetry& symmetry, float2_t center, bool normalize, Stream& stream) {
        launchSymmetrize2D_(*texture, texture_interp_mode, output.get(), output_strides, output_shape,
                            symmetry, center, normalize, stream);
        stream.attach(array, texture, output);
    }

    #define NOA_INSTANTIATE_TRANSFORM_SYM_(T)                                                                                                                               \
    template void symmetrize2D<T, void>(const shared_t<T[]>&, size4_t, const shared_t<T[]>&, size4_t, size4_t, const Symmetry&, float2_t, InterpMode, bool, bool, Stream&); \
    template void symmetrize2D<T, void>(const shared_t<hipArray>&, const shared_t<hipTextureObject_t>&, InterpMode, const shared_t<T[]>&, size4_t, size4_t, const Symmetry&, float2_t, bool, Stream&)

    NOA_INSTANTIATE_TRANSFORM_SYM_(float);
    NOA_INSTANTIATE_TRANSFORM_SYM_(cfloat_t);
}
