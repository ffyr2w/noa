#include "noa/common/Assert.h"
#include "noa/common/geometry/details/LinearTransform3DFourier.h"

#include "noa/gpu/cuda/Exception.h"
#include "noa/gpu/cuda/utils/Pointers.h"
#include "noa/gpu/cuda/utils/Iwise.cuh"
#include "noa/gpu/cuda/memory/Copy.h"
#include "noa/gpu/cuda/memory/PtrArray.h"
#include "noa/gpu/cuda/memory/PtrDevice.h"
#include "noa/gpu/cuda/memory/PtrTexture.h"
#include "noa/gpu/cuda/geometry/Interpolator.h"
#include "noa/gpu/cuda/geometry/fft/Transform.h"

namespace {
    using namespace ::noa;

    template<bool IS_OPTIONAL, typename Wrapper, typename Value>
    auto matrixOrShiftOrRawConstPtrOnDevice_(Wrapper wrapper, size_t count,
                                             cuda::memory::PtrDevice<Value>& buffer,
                                             cuda::Stream& stream) {
        using output_t = std::conditional_t<traits::is_floatXX_v<Wrapper> || traits::is_floatX_v<Wrapper>,
                                            traits::remove_ref_cv_t<Wrapper>,
                                            const traits::element_type_t<Wrapper>*>;
        if constexpr (traits::is_floatXX_v<Wrapper> || traits::is_floatX_v<Wrapper>) {
            return output_t(wrapper);
        } else {
            if (IS_OPTIONAL && wrapper.get() == nullptr)
                return output_t{};
            return output_t(cuda::utils::ensureDeviceAccess(wrapper.get(), stream, buffer, count));
        }
    }

    template<fft::Remap REMAP, typename Value, typename Matrix, typename ShiftOrEmpty>
    void linearTransform3D_(hipTextureObject_t texture, InterpMode texture_interp_mode,
                            Value* output, dim4_t output_strides, dim4_t shape,
                            Matrix inv_matrix, ShiftOrEmpty shift, float cutoff,
                            cuda::Stream& stream) {
        const auto iwise_shape = safe_cast<int4_t>(shape).fft();
        const auto output_accessor = AccessorRestrict<Value, 4, uint32_t>(output, safe_cast<uint4_t>(output_strides));

        switch (texture_interp_mode) {
            case INTERP_NEAREST: {
                using interpolator_t = cuda::geometry::Interpolator3D<INTERP_NEAREST, Value>;
                const auto kernel = noa::geometry::fft::details::transform3D<REMAP, int32_t>(
                        interpolator_t(texture), output_accessor, shape, inv_matrix, shift, cutoff);
                return cuda::utils::iwise4D("geometry::fft::transform3D", iwise_shape, kernel, stream);
            }
            case INTERP_LINEAR: {
                using interpolator_t = cuda::geometry::Interpolator3D<INTERP_LINEAR, Value>;
                const auto kernel = noa::geometry::fft::details::transform3D<REMAP, int32_t>(
                        interpolator_t(texture), output_accessor, shape, inv_matrix, shift, cutoff);
                return cuda::utils::iwise4D("geometry::fft::transform3D", iwise_shape, kernel, stream);
            }
            case INTERP_COSINE: {
                using interpolator_t = cuda::geometry::Interpolator3D<INTERP_COSINE, Value>;
                const auto kernel = noa::geometry::fft::details::transform3D<REMAP, int32_t>(
                        interpolator_t(texture), output_accessor, shape, inv_matrix, shift, cutoff);
                return cuda::utils::iwise4D("geometry::fft::transform3D", iwise_shape, kernel, stream);
            }
            case INTERP_LINEAR_FAST: {
                using interpolator_t = cuda::geometry::Interpolator3D<INTERP_LINEAR_FAST, Value>;
                const auto kernel = noa::geometry::fft::details::transform3D<REMAP, int32_t>(
                        interpolator_t(texture), output_accessor, shape, inv_matrix, shift, cutoff);
                return cuda::utils::iwise4D("geometry::fft::transform3D", iwise_shape, kernel, stream);
            }
            case INTERP_COSINE_FAST: {
                using interpolator_t = cuda::geometry::Interpolator3D<INTERP_COSINE_FAST, Value>;
                const auto kernel = noa::geometry::fft::details::transform3D<REMAP, int32_t>(
                        interpolator_t(texture), output_accessor, shape, inv_matrix, shift, cutoff);
                return cuda::utils::iwise4D("geometry::fft::transform3D", iwise_shape, kernel, stream);
            }
            default:
                NOA_THROW_FUNC("transform3D", "{} is not supported", texture_interp_mode);
        }
    }

    template<fft::Remap REMAP, typename Value, typename Matrix, typename Shift>
    void launchLinearTransform3D_(hipTextureObject_t texture, InterpMode texture_interp_mode,
                                  Value* output, dim4_t output_strides, dim4_t shape,
                                  Matrix inv_matrix, Shift shift, float cutoff,
                                  cuda::Stream& stream) {
        const bool do_shift = noa::any(shift != Shift{});
        if (do_shift) {
            linearTransform3D_<REMAP>(
                    texture, texture_interp_mode,
                    output, output_strides, shape,
                    inv_matrix, shift, cutoff, stream);
        } else {
            linearTransform3D_<REMAP>(
                    texture, texture_interp_mode,
                    output, output_strides, shape,
                    inv_matrix, empty_t{}, cutoff, stream);
        }
    }


    template<fft::Remap REMAP, typename Value, typename MatrixOrEmpty, typename ShiftOrEmpty>
    void linearTransformSymmetry3D_(hipTextureObject_t texture, InterpMode texture_interp_mode,
                                    Value* output, dim4_t output_strides, dim4_t shape,
                                    MatrixOrEmpty inv_matrix, const geometry::Symmetry& symmetry,
                                    ShiftOrEmpty shift, float cutoff, bool normalize,
                                    cuda::Stream& stream) {
        // TODO Move symmetry matrices to constant memory?
        const dim_t count = symmetry.count();
        const float33_t* symmetry_matrices = symmetry.get();
        using unique_ptr_t = cuda::memory::PtrDevice<float33_t>::alloc_unique_t;
        unique_ptr_t d_matrices = cuda::memory::PtrDevice<float33_t>::alloc(count, stream);
        cuda::memory::copy(symmetry_matrices, d_matrices.get(), count, stream);
        const float scaling = normalize ? 1 / static_cast<float>(count + 1) : 1;

        const auto iwise_shape = safe_cast<int4_t>(shape).fft();
        const auto output_accessor = AccessorRestrict<Value, 4, uint32_t>(output, safe_cast<uint4_t>(output_strides));

        switch (texture_interp_mode) {
            case INTERP_NEAREST: {
                using interpolator_t = cuda::geometry::Interpolator3D<INTERP_NEAREST, Value>;
                const auto kernel = noa::geometry::fft::details::transformSymmetry3D<REMAP, int32_t>(
                        interpolator_t(texture), output_accessor, shape,
                        inv_matrix, d_matrices.get(), count, scaling, shift, cutoff);
                return cuda::utils::iwise4D("geometry::fft::transform3D", iwise_shape, kernel, stream);
            }
            case INTERP_LINEAR: {
                using interpolator_t = cuda::geometry::Interpolator3D<INTERP_LINEAR, Value>;
                const auto kernel = noa::geometry::fft::details::transformSymmetry3D<REMAP, int32_t>(
                        interpolator_t(texture), output_accessor, shape,
                        inv_matrix, d_matrices.get(), count, scaling, shift, cutoff);
                return cuda::utils::iwise4D("geometry::fft::transform3D", iwise_shape, kernel, stream);
            }
            case INTERP_COSINE: {
                using interpolator_t = cuda::geometry::Interpolator3D<INTERP_COSINE, Value>;
                const auto kernel = noa::geometry::fft::details::transformSymmetry3D<REMAP, int32_t>(
                        interpolator_t(texture), output_accessor, shape,
                        inv_matrix, d_matrices.get(), count, scaling, shift, cutoff);
                return cuda::utils::iwise4D("geometry::fft::transform3D", iwise_shape, kernel, stream);
            }
            case INTERP_LINEAR_FAST: {
                using interpolator_t = cuda::geometry::Interpolator3D<INTERP_LINEAR_FAST, Value>;
                const auto kernel = noa::geometry::fft::details::transformSymmetry3D<REMAP, int32_t>(
                        interpolator_t(texture), output_accessor, shape,
                        inv_matrix, d_matrices.get(), count, scaling, shift, cutoff);
                return cuda::utils::iwise4D("geometry::fft::transform3D", iwise_shape, kernel, stream);
            }
            case INTERP_COSINE_FAST: {
                using interpolator_t = cuda::geometry::Interpolator3D<INTERP_COSINE_FAST, Value>;
                const auto kernel = noa::geometry::fft::details::transformSymmetry3D<REMAP, int32_t>(
                        interpolator_t(texture), output_accessor, shape,
                        inv_matrix, d_matrices.get(), count, scaling, shift, cutoff);
                return cuda::utils::iwise4D("geometry::fft::transform3D", iwise_shape, kernel, stream);
            }
            default:
                NOA_THROW_FUNC("transform3D", "{} is not supported", texture_interp_mode);
        }
    }

    template<fft::Remap REMAP, typename Value>
    void launchLinearTransformSymmetry3D_(hipTextureObject_t texture, InterpMode texture_interp_mode,
                                          Value* output, dim4_t output_strides, dim4_t output_shape,
                                          float33_t inv_matrix, const geometry::Symmetry& symmetry, float3_t shift,
                                          float cutoff, bool normalize, cuda::Stream& stream) {
        const bool apply_shift = any(shift != 0.f);
        const bool apply_inv_matrix = inv_matrix != float33_t{};

        if (apply_shift && apply_inv_matrix) {
            linearTransformSymmetry3D_<REMAP>(
                    texture, texture_interp_mode, output, output_strides, output_shape,
                    inv_matrix, symmetry, shift, cutoff, normalize, stream);
        } else if (apply_shift) {
            linearTransformSymmetry3D_<REMAP>(
                    texture, texture_interp_mode, output, output_strides, output_shape,
                    empty_t{}, symmetry, shift, cutoff, normalize, stream);
        } else if (apply_inv_matrix) {
            linearTransformSymmetry3D_<REMAP>(
                    texture, texture_interp_mode, output, output_strides, output_shape,
                    inv_matrix, symmetry, empty_t{}, cutoff, normalize, stream);
        } else {
            linearTransformSymmetry3D_<REMAP>(
                    texture, texture_interp_mode, output, output_strides, output_shape,
                    empty_t{}, symmetry, empty_t{}, cutoff, normalize, stream);
        }
    }
}

namespace noa::cuda::geometry::fft {
    template<Remap REMAP, typename Value, typename Matrix, typename Shift, typename>
    void transform3D(const shared_t<Value[]>& input, dim4_t input_strides,
                     const shared_t<Value[]>& output, dim4_t output_strides, dim4_t shape,
                     const Matrix& inv_matrices, const Shift& shifts,
                     float cutoff, InterpMode interp_mode, Stream& stream) {
        NOA_ASSERT(input && all(shape > 0));
        NOA_ASSERT_DEVICE_PTR(output.get(), stream.device());

        // Ensure transformation parameters are accessible to the GPU:
        memory::PtrDevice<float33_t> inv_matrices_buffer;
        memory::PtrDevice<float3_t> shift_buffer;
        auto inv_matrices_ = matrixOrShiftOrRawConstPtrOnDevice_<false>(inv_matrices, shape[0], inv_matrices_buffer, stream);
        auto shifts_ = matrixOrShiftOrRawConstPtrOnDevice_<true>(shifts, shape[0], shift_buffer, stream);

        memory::PtrArray<Value> array({1, shape[1], shape[2], shape[3] / 2 + 1});
        memory::PtrTexture texture(array.get(), interp_mode, BORDER_ZERO);

        dim_t iterations;
        dim4_t output_shape;
        if (input_strides[0] == 0) {
            iterations = 1;
            output_shape = shape;
        } else {
            iterations = shape[0];
            output_shape = {1, shape[1], shape[2], shape[3]};
        }
        for (dim_t i = 0; i < iterations; ++i) {
            memory::copy(input.get() + i * input_strides[0], input_strides,
                         array.get(), array.shape(), stream);
            launchLinearTransform3D_<REMAP>(
                    texture.get(), interp_mode,
                    output.get() + i * output_strides[0], output_strides,
                    output_shape, inv_matrices_, shifts_, cutoff, stream);

            if constexpr (!traits::is_float33_v<Matrix>)
                ++inv_matrices_;
            if constexpr (!traits::is_float3_v<Shift>)
                ++shifts_;
        }
        stream.attach(input, output, array.share(), texture.share());
        if constexpr (!traits::is_float33_v<Matrix>)
            stream.attach(inv_matrices);
        if constexpr (!traits::is_float3_v<Shift>)
            stream.attach(shifts);
    }

    template<Remap REMAP, typename Value, typename Matrix, typename Shift, typename>
    void transform3D(const shared_t<hipArray>& array,
                     const shared_t<hipTextureObject_t>& texture, InterpMode texture_interp_mode,
                     const shared_t<Value[]>& output, dim4_t output_strides, dim4_t shape,
                     const Matrix& inv_matrices, const Shift& shifts, float cutoff, Stream& stream) {
        NOA_ASSERT(array && texture && all(shape > 0));
        NOA_ASSERT_DEVICE_PTR(output.get(), stream.device());
        NOA_ASSERT(memory::PtrTexture::array(*texture) == array.get());

        // Ensure transformation parameters are accessible to the GPU:
        memory::PtrDevice<float33_t> inv_matrices_buffer;
        memory::PtrDevice<float3_t> shift_buffer;
        auto inv_matrices_ = matrixOrShiftOrRawConstPtrOnDevice_<false>(inv_matrices, shape[0], inv_matrices_buffer, stream);
        auto shifts_ = matrixOrShiftOrRawConstPtrOnDevice_<true>(shifts, shape[0], shift_buffer, stream);

        launchLinearTransform3D_<REMAP>(
                *texture, texture_interp_mode,
                output.get(), output_strides,
                shape, inv_matrices_, shifts_, cutoff, stream);

        stream.attach(array, texture, output);
        if constexpr (!traits::is_float33_v<Matrix>)
            stream.attach(inv_matrices);
        if constexpr (!traits::is_float3_v<Shift>)
            stream.attach(shifts);
    }

    template<Remap REMAP, typename T, typename>
    void transform3D(const shared_t<T[]>& input, dim4_t input_strides,
                     const shared_t<T[]>& output, dim4_t output_strides, dim4_t shape,
                     float33_t inv_matrix, const Symmetry& symmetry, float3_t shift,
                     float cutoff, InterpMode interp_mode, bool normalize, Stream& stream) {
        if (!symmetry.count())
            return transform3D<REMAP>(input, input_strides, output, output_strides, shape,
                                      inv_matrix, shift, cutoff, interp_mode, stream);

        NOA_ASSERT(input && all(shape > 0));
        NOA_ASSERT_DEVICE_PTR(output.get(), stream.device());

        memory::PtrArray<T> array(shape.fft());
        memory::PtrTexture texture(array.get(), interp_mode, BORDER_ZERO);

        dim_t iterations;
        dim4_t output_shape;
        if (input_strides[0] == 0) {
            iterations = 1;
            output_shape = shape;
        } else {
            iterations = shape[0];
            output_shape = {1, shape[1], shape[2], shape[3]};
        }
        for (dim_t i = 0; i < iterations; ++i) {
            cuda::memory::copy(input.get() + i * input_strides[0], input_strides,
                               array.get(), array.shape(), stream);
            launchLinearTransformSymmetry3D_<REMAP>(
                    texture.get(), interp_mode,
                    output.get() + i * output_strides[0], output_strides, output_shape,
                    inv_matrix, symmetry, shift, cutoff, normalize, stream);
        }
        stream.attach(input, output, symmetry.share(), array.share(), texture.share());
    }

    template<Remap REMAP, typename Value, typename>
    void transform3D(const shared_t<hipArray>& array,
                     const shared_t<hipTextureObject_t>& texture, InterpMode texture_interp_mode,
                     const shared_t<Value[]>& output, dim4_t output_strides, dim4_t shape,
                     float33_t inv_matrix, const Symmetry& symmetry, float3_t shift,
                     float cutoff, bool normalize, Stream& stream) {
        NOA_ASSERT(array && texture && all(shape > 0));
        NOA_ASSERT_DEVICE_PTR(output.get(), stream.device());
        launchLinearTransformSymmetry3D_<REMAP>(
                *texture, texture_interp_mode,
                output.get(), output_strides,  shape,
                inv_matrix, symmetry, shift, cutoff, normalize, stream);
        stream.attach(array, texture, output, symmetry.share());
    }

    #define NOA_INSTANTIATE_TRANSFORM_3D_(T, M, S)                                                                                                                                                                  \
    template void transform3D<Remap::HC2H,  T, M, S, void>(const shared_t<T[]>&, dim4_t, const shared_t<T[]>&, dim4_t, dim4_t, const M&, const S&, float, InterpMode, Stream&);                                     \
    template void transform3D<Remap::HC2HC, T, M, S, void>(const shared_t<T[]>&, dim4_t, const shared_t<T[]>&, dim4_t, dim4_t, const M&, const S&, float, InterpMode, Stream&);                                     \
    template void transform3D<Remap::HC2H,  T, M, S, void>(const shared_t<hipArray>&, const shared_t<hipTextureObject_t>&, InterpMode, const shared_t<T[]>&, dim4_t, dim4_t, const M&, const S&, float, Stream&); \
    template void transform3D<Remap::HC2HC, T, M, S, void>(const shared_t<hipArray>&, const shared_t<hipTextureObject_t>&, InterpMode, const shared_t<T[]>&, dim4_t, dim4_t, const M&, const S&, float, Stream&)

    #define NOA_INSTANTIATE_TRANSFORM_SYMMETRY_3D_(T)                                                                                                                                                                                   \
    template void transform3D<Remap::HC2HC, T, void>(const shared_t<T[]>&, dim4_t, const shared_t<T[]>&, dim4_t, dim4_t, float33_t, const Symmetry&, float3_t, float, InterpMode, bool, Stream&);                                       \
    template void transform3D<Remap::HC2H, T, void>(const shared_t<T[]>&, dim4_t, const shared_t<T[]>&, dim4_t, dim4_t, float33_t, const Symmetry&, float3_t, float, InterpMode, bool, Stream&);                                        \
    template void transform3D<Remap::HC2HC, T, void>(const shared_t<hipArray>&, const shared_t<hipTextureObject_t>&, InterpMode, const shared_t<T[]>&, dim4_t, dim4_t, float33_t, const Symmetry&, float3_t, float, bool, Stream&);   \
    template void transform3D<Remap::HC2H, T, void>(const shared_t<hipArray>&, const shared_t<hipTextureObject_t>&, InterpMode, const shared_t<T[]>&, dim4_t, dim4_t, float33_t, const Symmetry&, float3_t, float, bool, Stream&)

    #define NOA_INSTANTIATE_TRANSFORM_3D_ALL_(T)                                     \
    NOA_INSTANTIATE_TRANSFORM_3D_(T, shared_t<float33_t[]>, shared_t<float3_t[]>);   \
    NOA_INSTANTIATE_TRANSFORM_3D_(T, shared_t<float33_t[]>, float3_t);               \
    NOA_INSTANTIATE_TRANSFORM_3D_(T, float33_t, shared_t<float3_t[]>);               \
    NOA_INSTANTIATE_TRANSFORM_3D_(T, float33_t, float3_t);                           \
    NOA_INSTANTIATE_TRANSFORM_SYMMETRY_3D_(T)

    NOA_INSTANTIATE_TRANSFORM_3D_ALL_(float);
    NOA_INSTANTIATE_TRANSFORM_3D_ALL_(cfloat_t);
}
