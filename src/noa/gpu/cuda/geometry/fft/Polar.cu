#include "noa/core/Assert.hpp"
#include "noa/core/geometry/Interpolator.hpp"
#include "noa/algorithms/geometry/PolarTransformRFFT.hpp"
#include "noa/algorithms/geometry/RotationalAverage.hpp"

#include "noa/gpu/cuda/geometry/fft/Polar.hpp"
#include "noa/gpu/cuda/geometry/Interpolator.hpp"
#include "noa/gpu/cuda/memory/Copy.hpp"
#include "noa/gpu/cuda/memory/Set.hpp"
#include "noa/gpu/cuda/memory/AllocatorArray.hpp"
#include "noa/gpu/cuda/memory/AllocatorDevice.hpp"
#include "noa/gpu/cuda/memory/AllocatorTexture.hpp"
#include "noa/gpu/cuda/utils/Iwise.cuh"
#include "noa/gpu/cuda/utils/EwiseBinary.cuh"

namespace {
    using namespace ::noa;

    template<bool LAYERED, typename Input, typename Output>
    void launch_cartesian2polar_rfft_(
            hipTextureObject_t cartesian, InterpMode cartesian_interp, const Shape4<i64>& cartesian_shape,
            Output* polar, const Strides4<i64>& polar_strides, const Shape4<i64>& polar_shape,
            const Vec2<f32>& frequency_range, bool frequency_range_endpoint,
            const Vec2<f32>& angle_range, bool angle_range_endpoint,
            noa::cuda::Stream& stream
    ) {
        NOA_ASSERT(polar_shape[1] == 1);
        NOA_ASSERT((LAYERED && polar_shape[0] == cartesian_shape[0]) ||
                   (!LAYERED && cartesian_shape[0] == 1));

        const auto i_cartesian_shape = cartesian_shape.as_safe<u32>();
        const auto i_polar_shape = polar_shape.as_safe<u32>();
        const auto iwise_shape = i_polar_shape.filter(0, 2, 3);
        const auto polar_accessor = AccessorRestrict<Output, 3, u32>(polar, polar_strides.filter(0, 2, 3).as_safe<u32>());

        switch (cartesian_interp) {
            case InterpMode::NEAREST: {
                using interpolator_t = noa::cuda::geometry::Interpolator2D<InterpMode::NEAREST, Input, false, LAYERED>;
                const auto kernel = noa::algorithm::geometry::cartesian2polar_rfft(
                        interpolator_t(cartesian), i_cartesian_shape, polar_accessor, i_polar_shape,
                        frequency_range, frequency_range_endpoint, angle_range, angle_range_endpoint);
                return noa::cuda::utils::iwise_3d(iwise_shape, kernel, stream);
            }
            case InterpMode::LINEAR: {
                using interpolator_t = noa::cuda::geometry::Interpolator2D<InterpMode::LINEAR, Input, false, LAYERED>;
                const auto kernel = noa::algorithm::geometry::cartesian2polar_rfft(
                        interpolator_t(cartesian), i_cartesian_shape, polar_accessor, i_polar_shape,
                        frequency_range, frequency_range_endpoint, angle_range, angle_range_endpoint);
                return noa::cuda::utils::iwise_3d(iwise_shape, kernel, stream);
            }
            case InterpMode::COSINE: {
                using interpolator_t = noa::cuda::geometry::Interpolator2D<InterpMode::COSINE, Input, false, LAYERED>;
                const auto kernel = noa::algorithm::geometry::cartesian2polar_rfft(
                        interpolator_t(cartesian), i_cartesian_shape, polar_accessor, i_polar_shape,
                        frequency_range, frequency_range_endpoint, angle_range, angle_range_endpoint);
                return noa::cuda::utils::iwise_3d(iwise_shape, kernel, stream);
            }
            case InterpMode::LINEAR_FAST: {
                using interpolator_t = noa::cuda::geometry::Interpolator2D<InterpMode::LINEAR_FAST, Input, false, LAYERED>;
                const auto kernel = noa::algorithm::geometry::cartesian2polar_rfft(
                        interpolator_t(cartesian), i_cartesian_shape, polar_accessor, i_polar_shape,
                        frequency_range, frequency_range_endpoint, angle_range, angle_range_endpoint);
                return noa::cuda::utils::iwise_3d(iwise_shape, kernel, stream);
            }
            case InterpMode::COSINE_FAST: {
                using interpolator_t = noa::cuda::geometry::Interpolator2D<InterpMode::COSINE_FAST, Input, false, LAYERED>;
                const auto kernel = noa::algorithm::geometry::cartesian2polar_rfft(
                        interpolator_t(cartesian), i_cartesian_shape, polar_accessor, i_polar_shape,
                        frequency_range, frequency_range_endpoint, angle_range, angle_range_endpoint);
                return noa::cuda::utils::iwise_3d(iwise_shape, kernel, stream);
            }
            case InterpMode::CUBIC:
            case InterpMode::CUBIC_BSPLINE:
            case InterpMode::CUBIC_BSPLINE_FAST:
                NOA_THROW("{} is not supported", cartesian_interp);
        }
    }
}

namespace noa::cuda::geometry::fft {
    template<Remap REMAP, typename Input, typename Output, typename>
    void cartesian2polar(
            const Input* cartesian, Strides4<i64> cartesian_strides, Shape4<i64> cartesian_shape,
            Output* polar, const Strides4<i64>& polar_strides, const Shape4<i64>& polar_shape,
            const Vec2<f32>& frequency_range, bool frequency_range_endpoint,
            const Vec2<f32>& angle_range, bool angle_range_endpoint,
            InterpMode interp_mode, Stream& stream
    ) {
        NOA_ASSERT(cartesian && noa::all(cartesian_shape > 0) && noa::all(polar_shape > 0));
        NOA_ASSERT_DEVICE_PTR(polar, stream.device());
        NOA_ASSERT(cartesian_shape[0] == 1 || cartesian_shape[0] == polar_shape[0]);

        // Broadcast the input to every output batch.
        if (cartesian_shape[0] == 1)
            cartesian_strides[0] = 0;
        else if (cartesian_strides[0] == 0)
            cartesian_shape[0] = 1;

        const auto i_cartesian_shape = cartesian_shape.as_safe<i32>();
        const auto i_polar_shape = polar_shape.as_safe<i32>();
        const auto cartesian_shape_2d = i_cartesian_shape.filter(2, 3);
        const auto polar_shape_2d = i_polar_shape.filter(0, 2, 3);
        const auto cartesian_accessor = AccessorRestrict<const Input, 3, u32>(
                cartesian, cartesian_strides.filter(0, 2, 3).as_safe<u32>());
        const auto polar_accessor = AccessorRestrict<Output, 3, u32>(
                polar, polar_strides.filter(0, 2, 3).as_safe<u32>());

        switch (interp_mode) {
            case InterpMode::NEAREST: {
                const auto interpolator = noa::geometry::interpolator_2d<BorderMode::ZERO, InterpMode::NEAREST>(
                        cartesian_accessor, cartesian_shape_2d);
                const auto kernel = noa::algorithm::geometry::cartesian2polar_rfft(
                        interpolator, i_cartesian_shape, polar_accessor, i_polar_shape,
                        frequency_range, frequency_range_endpoint, angle_range, angle_range_endpoint);
                return noa::cuda::utils::iwise_3d(polar_shape_2d, kernel, stream);
            }
            case InterpMode::LINEAR: {
                const auto interpolator = noa::geometry::interpolator_2d<BorderMode::ZERO, InterpMode::LINEAR>(
                        cartesian_accessor, cartesian_shape_2d);
                const auto kernel = noa::algorithm::geometry::cartesian2polar_rfft(
                        interpolator, i_cartesian_shape, polar_accessor, i_polar_shape,
                        frequency_range, frequency_range_endpoint, angle_range, angle_range_endpoint);
                return noa::cuda::utils::iwise_3d(polar_shape_2d, kernel, stream);
            }
            case InterpMode::COSINE: {
                const auto interpolator = noa::geometry::interpolator_2d<BorderMode::ZERO, InterpMode::COSINE>(
                        cartesian_accessor, cartesian_shape_2d);
                const auto kernel = noa::algorithm::geometry::cartesian2polar_rfft(
                        interpolator, i_cartesian_shape, polar_accessor, i_polar_shape,
                        frequency_range, frequency_range_endpoint, angle_range, angle_range_endpoint);
                return noa::cuda::utils::iwise_3d(polar_shape_2d, kernel, stream);
            }
            case InterpMode::LINEAR_FAST: {
                const auto interpolator = noa::geometry::interpolator_2d<BorderMode::ZERO, InterpMode::LINEAR_FAST>(
                        cartesian_accessor, cartesian_shape_2d);
                const auto kernel = noa::algorithm::geometry::cartesian2polar_rfft(
                        interpolator, i_cartesian_shape, polar_accessor, i_polar_shape,
                        frequency_range, frequency_range_endpoint, angle_range, angle_range_endpoint);
                return noa::cuda::utils::iwise_3d(polar_shape_2d, kernel, stream);
            }
            case InterpMode::COSINE_FAST: {
                const auto interpolator = noa::geometry::interpolator_2d<BorderMode::ZERO, InterpMode::COSINE_FAST>(
                        cartesian_accessor, cartesian_shape_2d);
                const auto kernel = noa::algorithm::geometry::cartesian2polar_rfft(
                        interpolator, i_cartesian_shape, polar_accessor, i_polar_shape,
                        frequency_range, frequency_range_endpoint, angle_range, angle_range_endpoint);
                return noa::cuda::utils::iwise_3d(polar_shape_2d, kernel, stream);
            }
            case InterpMode::CUBIC:
            case InterpMode::CUBIC_BSPLINE:
            case InterpMode::CUBIC_BSPLINE_FAST:
                NOA_THROW("{} is not supported", interp_mode);
        }
    }

    template<typename Input, typename Output, typename>
    void cartesian2polar(
            hipArray* array, hipTextureObject_t cartesian,
            InterpMode cartesian_interp, const Shape4<i64>& cartesian_shape,
            Output* polar, const Strides4<i64>& polar_strides, const Shape4<i64>& polar_shape,
            const Vec2<f32>& frequency_range, bool frequency_range_endpoint,
            const Vec2<f32>& angle_range, bool angle_range_endpoint,
            Stream& stream
    ) {
        NOA_ASSERT(array && cartesian && noa::all(cartesian_shape > 0) && noa::all(polar_shape > 0));
        NOA_ASSERT_DEVICE_PTR(polar, stream.device());

        const bool is_layered = noa::cuda::memory::AllocatorArray<Output>::is_layered(array);
        NOA_ASSERT(noa::cuda::memory::AllocatorTexture::array(cartesian) == array);

        if (is_layered) {
            launch_cartesian2polar_rfft_<true, Input>(
                    cartesian, cartesian_interp, cartesian_shape,
                    polar, polar_strides, polar_shape,
                    frequency_range, frequency_range_endpoint,
                    angle_range, angle_range_endpoint, stream);
        } else {
            launch_cartesian2polar_rfft_<false, Input>(
                    cartesian, cartesian_interp, cartesian_shape,
                    polar, polar_strides, polar_shape,
                    frequency_range, frequency_range_endpoint,
                    angle_range, angle_range_endpoint, stream);
        }
    }

    template<noa::fft::Remap REMAP, typename Input, typename Ctf, typename Output, typename Weight, typename>
    void rotational_average(
            const Input* input,
            const Strides4<i64>& input_strides, const Shape4<i64>& input_shape, const Ctf& input_ctf,
            Output* output, i64 output_batch_stride, Weight* weight, i64 weight_batch_stride, i64 n_shells,
            const Vec2<f32>& frequency_range, bool frequency_range_endpoint, bool average, Stream& stream
    ) {
        const auto n_batches = input_shape[0];
        const auto shell_shape = Shape4<i64>{n_batches, 1, 1, n_shells};
        const auto output_strides = Strides4<i64>{output_batch_stride, 0, 0, 1};
        auto weight_strides = Strides4<i64>{weight_batch_stride, 0, 0, 1};
        noa::cuda::memory::set(output, output_strides, shell_shape, Output{0}, stream);

        // When computing the average, the weights must be valid.
        using unique_t = typename noa::cuda::memory::AllocatorDevice<Weight>::unique_type;
        unique_t weight_buffer;
        Weight* weight_ptr = weight;
        if (average) {
            if (weight_ptr == nullptr) {
                weight_buffer = noa::cuda::memory::AllocatorDevice<Weight>::allocate_async(
                        n_batches * n_shells, stream);
                weight_ptr = weight_buffer.get();
                weight_strides[0] = n_shells; // contiguous batches
            }
            noa::cuda::memory::set(weight_ptr, weight_strides, shell_shape, Weight{0}, stream);
        }

        // Use 32-bits precision for indexes. // TODO benchmark
        auto input_strides_32 = input_strides.as_safe<u32>();
        auto input_shape_32 = input_shape.as_safe<i32>();
        const auto output_strides_32 = output_strides.as_safe<u32>();
        const auto weight_strides_32 = weight_strides.as_safe<u32>();
        const auto n_shells_32 = safe_cast<i32>(n_shells);

        constexpr bool IS_HALF = static_cast<u8>(REMAP) & noa::fft::Layout::SRC_HALF;
        if (input_shape.ndim() == 2) {
            // Reorder HW-dimensions to rightmost.
            const auto order = noa::indexing::order(input_strides.filter(2, 3), input_shape.filter(2, 3));
            if (noa::any(order != Vec2<i64>{0, 1})) {
                std::swap(input_strides_32[2], input_strides_32[3]);
                std::swap(input_shape_32[2], input_shape_32[3]);
            }

            const auto kernel = noa::algorithm::geometry::rotational_average_2d<REMAP>(
                    input, input_strides_32, input_shape_32, input_ctf,
                    output, output_strides_32[0], weight_ptr, weight_strides_32[0], n_shells_32,
                    frequency_range, frequency_range_endpoint);

            auto iwise_shape = input_shape_32.filter(0, 2, 3);
            if constexpr (IS_HALF)
                iwise_shape = iwise_shape.rfft();
            noa::cuda::utils::iwise_3d(iwise_shape, kernel, stream);

        } else {
            // Reorder BHW dimensions to rightmost.
            const auto order = noa::indexing::order(input_strides.pop_front(), input_shape.pop_front());
            if (noa::any(order != Vec3<i64>{0, 1, 2})) {
                const auto order_3d = (order + 1).push_front(0);
                input_strides_32 = noa::indexing::reorder(input_strides_32, order_3d);
                input_shape_32 = noa::indexing::reorder(input_shape_32, order_3d);
            }

            const auto kernel = noa::algorithm::geometry::rotational_average_3d<REMAP>(
                    input, input_strides_32, input_shape_32,
                    output, output_strides_32[0], weight_ptr, weight_strides_32[0], n_shells_32,
                    frequency_range, frequency_range_endpoint);

            noa::cuda::utils::iwise_4d(IS_HALF ? input_shape_32.rfft() : input_shape_32, kernel, stream);
        }

        if (average) {
            // Some shells can be 0, so use divide_safe_t.
            noa::cuda::utils::ewise_binary(
                    output, output_strides,
                    weight_ptr, weight_strides,
                    output, output_strides, shell_shape,
                    stream, noa::divide_safe_t{});
        }
    }

    #define INSTANTIATE_POLAR(Input, Output) \
    template void cartesian2polar<Remap::HC2FC, Input, Output, void>(   \
        const Input*, Strides4<i64>, Shape4<i64>,                       \
        Output*, const Strides4<i64>&, const Shape4<i64>&,              \
        const Vec2<f32>&, bool, const Vec2<f32>&, bool,                 \
        InterpMode, Stream&)

    #define INSTANTIATE_POLAR_TEXTURE(Input, Output)        \
    template void cartesian2polar<Input, Output, void>(     \
        hipArray*, hipTextureObject_t,                    \
        InterpMode, const Shape4<i64>&,                     \
        Output*, const Strides4<i64>&, const Shape4<i64>&,  \
        const Vec2<f32>&, bool, const Vec2<f32>&, bool,     \
        Stream&)

    INSTANTIATE_POLAR(f32, f32);
    INSTANTIATE_POLAR(f64, f64);
    INSTANTIATE_POLAR(c32, c32);
    INSTANTIATE_POLAR(c64, c64);
    INSTANTIATE_POLAR(c32, f32);
    INSTANTIATE_POLAR(c64, f64);

    INSTANTIATE_POLAR_TEXTURE(f32, f32);
    INSTANTIATE_POLAR_TEXTURE(c32, c32);
    INSTANTIATE_POLAR_TEXTURE(c32, f32);

    #define NOA_INSTANTIATE_ROTATIONAL_AVERAGE(Remap, Input, Ctf, Output, Weight)   \
    template void rotational_average<Remap, Input, Ctf, Output, Weight, void>(      \
            const Input*, const Strides4<i64>&, const Shape4<i64>&, Ctf const&,     \
            Output*, i64, Weight*, i64, i64,                                        \
            const Vec2<f32>&, bool, bool, Stream&)

    #define NOA_INSTANTIATE_ROTATIONAL_AVERAGE_CTF(Remap, Input, Output, Weight)                                \
    NOA_INSTANTIATE_ROTATIONAL_AVERAGE(Remap, Input, noa::traits::Empty, Output, Weight);                       \
    NOA_INSTANTIATE_ROTATIONAL_AVERAGE(Remap, Input, signal::fft::CTFAnisotropic<f32>, Output, Weight);         \
    NOA_INSTANTIATE_ROTATIONAL_AVERAGE(Remap, Input, signal::fft::CTFAnisotropic<f64>, Output, Weight);         \
    NOA_INSTANTIATE_ROTATIONAL_AVERAGE(Remap, Input, const signal::fft::CTFAnisotropic<f32>*, Output, Weight);  \
    NOA_INSTANTIATE_ROTATIONAL_AVERAGE(Remap, Input, const signal::fft::CTFAnisotropic<f64>*, Output, Weight)

    #define NOA_INSTANT_ROTATIONAL_AVERAGE_REMAP(Input, Output, Weight)                     \
    NOA_INSTANTIATE_ROTATIONAL_AVERAGE_CTF(noa::fft::Remap::H2H, Input, Output, Weight);    \
    NOA_INSTANTIATE_ROTATIONAL_AVERAGE_CTF(noa::fft::Remap::HC2H, Input, Output, Weight);   \
    NOA_INSTANTIATE_ROTATIONAL_AVERAGE_CTF(noa::fft::Remap::F2H, Input, Output, Weight);    \
    NOA_INSTANTIATE_ROTATIONAL_AVERAGE_CTF(noa::fft::Remap::FC2H, Input, Output, Weight)

    NOA_INSTANT_ROTATIONAL_AVERAGE_REMAP(f32, f32, f32);
    NOA_INSTANT_ROTATIONAL_AVERAGE_REMAP(f64, f64, f64);
    NOA_INSTANT_ROTATIONAL_AVERAGE_REMAP(c32, c32, f32);
    NOA_INSTANT_ROTATIONAL_AVERAGE_REMAP(c64, c64, f64);
    NOA_INSTANT_ROTATIONAL_AVERAGE_REMAP(c32, f32, f32);
    NOA_INSTANT_ROTATIONAL_AVERAGE_REMAP(c64, f64, f64);
}
