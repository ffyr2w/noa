#include "noa/core/geometry/Interpolator.hpp"
#include "noa/core/geometry/InterpolatorValue.hpp"
#include "noa/algorithms/geometry/ProjectionsFFT.hpp"

#include "noa/gpu/cuda/Exception.hpp"
#include "noa/gpu/cuda/memory/AllocatorArray.hpp"
#include "noa/gpu/cuda/memory/AllocatorTexture.hpp"
#include "noa/gpu/cuda/memory/Set.hpp"
#include "noa/gpu/cuda/utils/Iwise.cuh"
#include "noa/gpu/cuda/utils/Pointers.hpp"
#include "noa/gpu/cuda/geometry/Interpolator.hpp"
#include "noa/gpu/cuda/geometry/fft/Project.hpp"

namespace {
    using namespace ::noa;

    template<fft::Remap REMAP, typename Interpolator, typename Value, typename Scale, typename Rotate>
    void insert_interpolate_3d_(
            Interpolator slice_interpolator, const Shape4<i64>& slice_shape,
            Value* volume, const Strides4<i64>& volume_strides, const Shape4<i64>& volume_shape,
            const Scale& fwd_scaling_matrices, const Rotate& inv_rotation_matrices,
            f32 fftfreq_cutoff, f32 fftfreq_sinc, f32 fftfreq_blackman,
            const Shape4<i64>& target_shape, const Vec2<f32>& ews_radius,
            noa::cuda::Stream& stream
    ) {
        const auto i_volume_shape = volume_shape.as_safe<i32>();
        const auto volume_strides_3d = volume_strides.filter(1, 2, 3).as_safe<u32>();
        const auto volume_accessor = AccessorRestrict<Value, 3, u32>(volume, volume_strides_3d);
        const auto iwise_shape = i_volume_shape.pop_front().rfft();

        const auto apply_ews = noa::any(ews_radius != 0);
        const bool apply_scale = fwd_scaling_matrices != Scale{};

        if (apply_ews || apply_scale) {
            const auto op = na::geometry::fourier_insert_interpolate<REMAP, i32>(
                    slice_interpolator, slice_shape.as_safe<i32>(), volume_accessor, i_volume_shape,
                    fwd_scaling_matrices, inv_rotation_matrices,
                    fftfreq_cutoff, fftfreq_sinc, fftfreq_blackman,
                    target_shape.as_safe<i32>(), ews_radius);
            return noa::cuda::utils::iwise_3d(iwise_shape, op, stream);
        } else {
            const auto op = na::geometry::fourier_insert_interpolate<REMAP, i32>(
                    slice_interpolator, slice_shape.as_safe<i32>(), volume_accessor, i_volume_shape,
                    Empty{}, inv_rotation_matrices,
                    fftfreq_cutoff, fftfreq_sinc, fftfreq_blackman,
                    target_shape.as_safe<i32>(), Empty{});
            return noa::cuda::utils::iwise_3d(iwise_shape, op, stream);
        }
    }

    template<fft::Remap REMAP, typename Value, typename Interpolator, typename Scale, typename Rotate>
    void extract_3d_(
            Interpolator volume, const Shape4<i64>& volume_shape,
            Value* slice, const Strides4<i64>& slice_strides, const Shape4<i64>& slice_shape,
            const Scale& inv_scaling_matrices, const Rotate& fwd_rotation_matrices,
            f32 fftfreq_cutoff, f32 fftfreq_z_sinc, f32 fftfreq_z_blackman,
            const Shape4<i64>& target_shape, const Vec2<f32>& ews_radius,
            noa::cuda::Stream& stream
    ) {
        const auto i_slice_shape = slice_shape.as_safe<i32>();
        const auto slice_strides_3d = slice_strides.filter(0, 2, 3).as_safe<u32>();
        const auto slice_accessor = AccessorRestrict<Value, 3, u32>(slice, slice_strides_3d);

        const auto apply_ews = noa::any(ews_radius != 0);
        const bool apply_scale = inv_scaling_matrices != Scale{};

        if (apply_ews || apply_scale) {
            const auto op = na::geometry::fourier_extract<REMAP>(
                    volume, volume_shape.as_safe<i32>(), slice_accessor, i_slice_shape,
                    inv_scaling_matrices, fwd_rotation_matrices,
                    fftfreq_cutoff, fftfreq_z_sinc, fftfreq_z_blackman,
                    target_shape.as_safe<i32>(), ews_radius);
            if (op.windowed_sinc_size() > 1) {
                noa::cuda::memory::set(slice, slice_strides, slice_shape.rfft(), Value{0}, stream);
                noa::cuda::utils::iwise_4d(i_slice_shape.rfft().set<1>(op.windowed_sinc_size()), op, stream);
            } else {
                noa::cuda::utils::iwise_3d(i_slice_shape.filter(0, 2, 3).rfft(), op, stream);
            }
        } else {
            const auto op = na::geometry::fourier_extract<REMAP>(
                    volume, volume_shape.as_safe<i32>(), slice_accessor, i_slice_shape,
                    Empty{}, fwd_rotation_matrices,
                    fftfreq_cutoff, fftfreq_z_sinc, fftfreq_z_blackman,
                    target_shape.as_safe<i32>(), Empty{});
            if (op.windowed_sinc_size() > 1) {
                noa::cuda::memory::set(slice, slice_strides, slice_shape.rfft(), Value{0}, stream);
                noa::cuda::utils::iwise_4d(i_slice_shape.rfft().set<1>(op.windowed_sinc_size()), op, stream);
            } else {
                noa::cuda::utils::iwise_3d(i_slice_shape.filter(0, 2, 3).rfft(), op, stream);
            }
        }
    }

    template<fft::Remap REMAP, typename Interpolator, typename Value,
             typename Scale0, typename Scale1, typename Rotate0, typename Rotate1>
    void insert_interpolate_and_extract_3d_(
            Interpolator input_slice_interpolator, const Shape4<i64>& input_slice_shape,
            Value* output_slice, const Strides4<i64>& output_slice_strides, const Shape4<i64>& output_slice_shape,
            const Scale0& insert_fwd_scaling_matrices, const Rotate0& insert_inv_rotation_matrices,
            const Scale1& extract_inv_scaling_matrices, const Rotate1& extract_fwd_rotation_matrices,
            f32 fftfreq_cutoff, f32 fftfreq_input_sinc, f32 fftfreq_input_blackman,
            f32 fftfreq_z_sinc, f32 fftfreq_z_blackman, bool add_to_output,
            const Vec2<f32>& ews_radius, noa::cuda::Stream& stream
    ) {
        const auto output_slice_strides_2d = output_slice_strides.filter(0, 2, 3).as_safe<u32>();
        const auto output_slice_accessor = AccessorRestrict<Value, 3, u32>(output_slice, output_slice_strides_2d);
        const auto i_output_slice_shape = output_slice_shape.as_safe<i32>();

        const auto apply_ews = noa::any(ews_radius != 0);
        const bool apply_scale = insert_fwd_scaling_matrices != Scale0{};

        if (apply_ews || apply_scale) {
            const auto op = na::geometry::fourier_insert_and_extract<REMAP, i32>(
                    input_slice_interpolator, input_slice_shape.as_safe<i32>(),
                    output_slice_accessor, i_output_slice_shape,
                    insert_fwd_scaling_matrices, insert_inv_rotation_matrices,
                    extract_inv_scaling_matrices, extract_fwd_rotation_matrices,
                    fftfreq_cutoff, fftfreq_input_sinc, fftfreq_input_blackman,
                    fftfreq_z_sinc, fftfreq_z_blackman,
                    add_to_output, ews_radius);
            if (op.windowed_sinc_size() > 1) {
                if (!add_to_output)
                    noa::cuda::memory::set(output_slice, output_slice_strides, output_slice_shape.rfft(), Value{0}, stream);
                noa::cuda::utils::iwise_4d(i_output_slice_shape.rfft().set<1>(op.windowed_sinc_size()), op, stream);
            } else {
                noa::cuda::utils::iwise_3d(i_output_slice_shape.filter(0, 2, 3).rfft(), op, stream);
            }
        } else {
            const auto op = na::geometry::fourier_insert_and_extract<REMAP, i32>(
                    input_slice_interpolator, input_slice_shape.as_safe<i32>(),
                    output_slice_accessor, i_output_slice_shape,
                    Empty{}, insert_inv_rotation_matrices,
                    extract_inv_scaling_matrices, extract_fwd_rotation_matrices,
                    fftfreq_cutoff, fftfreq_input_sinc, fftfreq_input_blackman,
                    fftfreq_z_sinc, fftfreq_z_blackman, add_to_output, Empty{});
            if (op.windowed_sinc_size() > 1) {
                if (!add_to_output)
                    noa::cuda::memory::set(output_slice, output_slice_strides, output_slice_shape.rfft(), Value{0}, stream);
                noa::cuda::utils::iwise_4d(i_output_slice_shape.rfft().set<1>(op.windowed_sinc_size()), op, stream);
            } else {
                noa::cuda::utils::iwise_3d(i_output_slice_shape.filter(0, 2, 3).rfft(), op, stream);
            }
        }
    }
}

namespace noa::cuda::geometry::fft {
    template<Remap REMAP, typename Value, typename Scale, typename Rotate, typename>
    void insert_rasterize_3d(
            const Value* slice, const Strides4<i64>& slice_strides, const Shape4<i64>& slice_shape,
            Value* volume, const Strides4<i64>& volume_strides, const Shape4<i64>& volume_shape,
            const Scale& inv_scaling_matrices, const Rotate& fwd_rotation_matrices,
            f32 fftfreq_cutoff, const Shape4<i64>& target_shape, const Vec2<f32>& ews_radius, Stream& stream
    ) {
        NOA_ASSERT_DEVICE_PTR(slice, stream.device());
        NOA_ASSERT_DEVICE_PTR(volume, stream.device());

        const auto slice_strides_3d = slice_strides.filter(0, 2, 3).as_safe<u32>();
        const auto volume_strides_3d = volume_strides.pop_front().as_safe<u32>();
        const auto slice_accessor = AccessorRestrict<const Value, 3, u32>(slice, slice_strides_3d);
        const auto volume_accessor = AccessorRestrict<Value, 3, u32>(volume, volume_strides_3d);
        const auto i_slice_shape = slice_shape.as_safe<i32>();
        const auto iwise_shape = i_slice_shape.filter(0, 2, 3).rfft();

        const auto apply_ews = noa::any(ews_radius != 0);
        const bool apply_scale = inv_scaling_matrices != Scale{};

        if (apply_ews || apply_scale) {
            const auto op = na::geometry::fourier_insert_rasterize<REMAP, i32>(
                    slice_accessor, i_slice_shape, volume_accessor, volume_shape.as_safe<i32>(),
                    inv_scaling_matrices, fwd_rotation_matrices,
                    fftfreq_cutoff, target_shape.as_safe<i32>(), ews_radius);
            return noa::cuda::utils::iwise_3d(iwise_shape, op, stream);
        } else {
            const auto op = na::geometry::fourier_insert_rasterize<REMAP, i32>(
                    slice_accessor, i_slice_shape, volume_accessor, volume_shape.as_safe<i32>(),
                    Empty{}, fwd_rotation_matrices,
                    fftfreq_cutoff, target_shape.as_safe<i32>(), Empty{});
            return noa::cuda::utils::iwise_3d(iwise_shape, op, stream);
        }
    }

    template<Remap REMAP, typename Value, typename Scale, typename Rotate, typename>
    void insert_rasterize_3d(
            Value slice, const Shape4<i64>& slice_shape,
            Value* volume, const Strides4<i64>& volume_strides, const Shape4<i64>& volume_shape,
            const Scale& inv_scaling_matrices, const Rotate& fwd_rotation_matrices,
            f32 fftfreq_cutoff, const Shape4<i64>& target_shape, const Vec2<f32>& ews_radius, Stream& stream
    ) {
        NOA_ASSERT_DEVICE_PTR(volume, stream.device());

        const auto i_slice_shape = slice_shape.as_safe<i32>();
        const auto iwise_shape = i_slice_shape.filter(0, 2, 3).rfft();
        const auto volume_accessor = AccessorRestrict<Value, 3, uint32_t>(volume, volume_strides.pop_front().as_safe<u32>());

        const auto apply_ews = noa::any(ews_radius != 0);
        const bool apply_scale = inv_scaling_matrices != Scale{};

        if (apply_ews || apply_scale) {
            const auto op = na::geometry::fourier_insert_rasterize<REMAP>(
                    slice, i_slice_shape, volume_accessor, volume_shape.as_safe<i32>(),
                    inv_scaling_matrices, fwd_rotation_matrices,
                    fftfreq_cutoff, target_shape.as_safe<i32>(), ews_radius);
            return noa::cuda::utils::iwise_3d(iwise_shape, op, stream);
        } else {
            const auto op = na::geometry::fourier_insert_rasterize<REMAP>(
                    slice, i_slice_shape, volume_accessor, volume_shape.as_safe<i32>(),
                    Empty{}, fwd_rotation_matrices,
                    fftfreq_cutoff, target_shape.as_safe<i32>(), Empty{});
            return noa::cuda::utils::iwise_3d(iwise_shape, op, stream);
        }
    }

    template<Remap REMAP, typename Value, typename Scale, typename Rotate, typename>
    void insert_interpolate_3d(
            const Value* slice, const Strides4<i64>& slice_strides, const Shape4<i64>& slice_shape,
            Value* volume, const Strides4<i64>& volume_strides, const Shape4<i64>& volume_shape,
            const Scale& fwd_scaling_matrices, const Rotate& inv_rotation_matrices,
            f32 fftfreq_cutoff, f32 fftfreq_sinc, f32 fftfreq_blackman,
            const Shape4<i64>& target_shape, const Vec2<f32>& ews_radius, Stream& stream
    ) {
        NOA_ASSERT_DEVICE_PTR(volume, stream.device());
        NOA_ASSERT_DEVICE_PTR(slice, stream.device());

        const auto slice_accessor = AccessorRestrict<const Value, 3, u32>(
                slice, slice_strides.filter(0, 2, 3).as_safe<u32>());
        const auto slice_interpolator = noa::geometry::interpolator_2d<BorderMode::ZERO, InterpMode::LINEAR>(
                slice_accessor, slice_shape.filter(2, 3).as_safe<i32>().rfft(), Value{0});

        insert_interpolate_3d_<REMAP>(
                slice_interpolator, slice_shape, volume, volume_strides, volume_shape,
                fwd_scaling_matrices, inv_rotation_matrices, fftfreq_cutoff, fftfreq_sinc, fftfreq_blackman,
                target_shape, ews_radius, stream);
    }

    template<Remap REMAP, typename Value, typename Scale, typename Rotate, typename>
    void insert_interpolate_3d(
            Value slice, const Shape4<i64>& slice_shape,
            Value* volume, const Strides4<i64>& volume_strides, const Shape4<i64>& volume_shape,
            const Scale& fwd_scaling_matrices, const Rotate& inv_rotation_matrices,
            f32 fftfreq_cutoff, f32 fftfreq_sinc, f32 fftfreq_blackman,
            const Shape4<i64>& target_shape, const Vec2<f32>& ews_radius, Stream& stream
    ) {
        NOA_ASSERT_DEVICE_PTR(volume, stream.device());

        const auto slice_interpolator = noa::geometry::interpolator_value_2d<BorderMode::ZERO, InterpMode::LINEAR>(
                slice, slice_shape.filter(2, 3).as_safe<i32>().rfft(), Value{0});

        insert_interpolate_3d_<REMAP>(
                slice_interpolator, slice_shape, volume, volume_strides, volume_shape,
                fwd_scaling_matrices, inv_rotation_matrices, fftfreq_cutoff, fftfreq_sinc, fftfreq_blackman,
                target_shape, ews_radius, stream);
    }

    template<Remap REMAP, typename Value, typename Scale, typename Rotate, typename>
    void insert_interpolate_3d(
            hipArray* array, hipTextureObject_t slice,
            InterpMode slice_interpolation_mode, const Shape4<i64>& slice_shape,
            Value* volume, const Strides4<i64>& volume_strides, const Shape4<i64>& volume_shape,
            const Scale& fwd_scaling_matrices, const Rotate& inv_rotation_matrices,
            f32 fftfreq_cutoff, f32 fftfreq_sinc, f32 fftfreq_blackman,
            const Shape4<i64>& target_shape, const Vec2<f32>& ews_radius, Stream& stream
    ) {
        NOA_ASSERT(noa::cuda::memory::AllocatorTexture::array(slice) == array);
        NOA_ASSERT_DEVICE_PTR(volume, stream.device());
        NOA_CHECK(slice_interpolation_mode == InterpMode::LINEAR || slice_interpolation_mode == InterpMode::LINEAR_FAST,
                  "The interpolation mode should be {} or {}, got {}",
                  InterpMode::LINEAR, InterpMode::LINEAR_FAST, slice_interpolation_mode);

        const bool is_layered = noa::cuda::memory::AllocatorArray<Value>::is_layered(array);
        if (is_layered) {
            if (slice_interpolation_mode == InterpMode::LINEAR) {
                using interpolator_t = noa::cuda::geometry::Interpolator2D<InterpMode::LINEAR, Value, false, true>;
                insert_interpolate_3d_<REMAP>(
                        interpolator_t(slice), slice_shape, volume, volume_strides, volume_shape,
                        fwd_scaling_matrices, inv_rotation_matrices, fftfreq_cutoff, fftfreq_sinc, fftfreq_blackman,
                        target_shape, ews_radius, stream);
            } else {
                using interpolator_t = noa::cuda::geometry::Interpolator2D<InterpMode::LINEAR_FAST, Value, false, true>;
                insert_interpolate_3d_<REMAP>(
                        interpolator_t(slice), slice_shape, volume, volume_strides, volume_shape,
                        fwd_scaling_matrices, inv_rotation_matrices, fftfreq_cutoff, fftfreq_sinc, fftfreq_blackman,
                        target_shape, ews_radius, stream);
            }
        } else {
            if (slice_interpolation_mode == InterpMode::LINEAR) {
                using interpolator_t = noa::cuda::geometry::Interpolator2D<InterpMode::LINEAR, Value>;
                insert_interpolate_3d_<REMAP>(
                        interpolator_t(slice), slice_shape, volume, volume_strides, volume_shape,
                        fwd_scaling_matrices, inv_rotation_matrices, fftfreq_cutoff, fftfreq_sinc, fftfreq_blackman,
                        target_shape, ews_radius, stream);
            } else {
                using interpolator_t = noa::cuda::geometry::Interpolator2D<InterpMode::LINEAR_FAST, Value>;
                insert_interpolate_3d_<REMAP>(
                        interpolator_t(slice), slice_shape, volume, volume_strides, volume_shape,
                        fwd_scaling_matrices, inv_rotation_matrices, fftfreq_cutoff, fftfreq_sinc, fftfreq_blackman,
                        target_shape, ews_radius, stream);
            }
        }
    }

    template<Remap REMAP, typename Value, typename Scale, typename Rotate, typename>
    void extract_3d(
            const Value* volume, const Strides4<i64>& volume_strides, const Shape4<i64>& volume_shape,
            Value* slice, const Strides4<i64>& slice_strides, const Shape4<i64>& slice_shape,
            const Scale& inv_scaling_matrices, const Rotate& fwd_rotation_matrices,
            f32 fftfreq_cutoff, f32 fftfreq_z_sinc, f32 fftfreq_z_blackman,
            const Shape4<i64>& target_shape, const Vec2<f32>& ews_radius, Stream& stream
    ) {
        NOA_ASSERT_DEVICE_PTR(slice, stream.device());
        NOA_ASSERT_DEVICE_PTR(volume, stream.device());

        const auto volume_accessor = AccessorRestrict<const Value, 3, u32>(volume, volume_strides.pop_front().as_safe<u32>());
        const auto volume_interpolator = noa::geometry::interpolator_3d<BorderMode::ZERO, InterpMode::LINEAR>(
                volume_accessor, volume_shape.pop_front().as_safe<i32>().rfft(), Value{0});

        extract_3d_<REMAP>(volume_interpolator, volume_shape, slice, slice_strides, slice_shape,
                           inv_scaling_matrices, fwd_rotation_matrices,
                           fftfreq_cutoff, fftfreq_z_sinc, fftfreq_z_blackman,
                           target_shape, ews_radius, stream);
    }

    template<Remap REMAP, typename Value, typename Scale, typename Rotate, typename>
    void extract_3d(
            hipArray* array, hipTextureObject_t volume,
            InterpMode volume_interpolation_mode, const Shape4<i64>& volume_shape,
            Value* slice, const Strides4<i64>& slice_strides, const Shape4<i64>& slice_shape,
            const Scale& inv_scaling_matrices, const Rotate& fwd_rotation_matrices,
            f32 fftfreq_cutoff, f32 fftfreq_z_sinc, f32 fftfreq_z_blackman,
            const Shape4<i64>& target_shape, const Vec2<f32>& ews_radius, Stream& stream
    ) {
        NOA_ASSERT(noa::cuda::memory::AllocatorTexture::array(volume) == array);
        NOA_ASSERT_DEVICE_PTR(slice, stream.device());
        NOA_CHECK(volume_interpolation_mode == InterpMode::LINEAR || volume_interpolation_mode == InterpMode::LINEAR_FAST,
                  "The interpolation mode should be {} or {}, got {}",
                  InterpMode::LINEAR, InterpMode::LINEAR_FAST, volume_interpolation_mode);

        if (volume_interpolation_mode == InterpMode::LINEAR) {
            using interpolator_t = noa::cuda::geometry::Interpolator3D<InterpMode::LINEAR, Value>;
            extract_3d_<REMAP>(interpolator_t(volume), volume_shape, slice, slice_strides, slice_shape,
                               inv_scaling_matrices, fwd_rotation_matrices,
                               fftfreq_cutoff, fftfreq_z_sinc, fftfreq_z_blackman,
                               target_shape, ews_radius, stream);
        } else if (volume_interpolation_mode == InterpMode::LINEAR_FAST) {
            using interpolator_t = noa::cuda::geometry::Interpolator3D<InterpMode::LINEAR_FAST, Value>;
            extract_3d_<REMAP>(interpolator_t(volume), volume_shape, slice, slice_strides, slice_shape,
                               inv_scaling_matrices, fwd_rotation_matrices,
                               fftfreq_cutoff, fftfreq_z_sinc, fftfreq_z_blackman,
                               target_shape, ews_radius, stream);
        } else {
            NOA_THROW("The interpolation mode should be {} or {}, got {}",
                      InterpMode::LINEAR, InterpMode::LINEAR_FAST, volume_interpolation_mode);
        }
    }

    template<Remap REMAP, typename Value,
             typename Scale0, typename Scale1,
             typename Rotate0, typename Rotate1, typename>
    void insert_interpolate_and_extract_3d(
            const Value* input_slice, const Strides4<i64>& input_slice_strides, const Shape4<i64>& input_slice_shape,
            Value* output_slice, const Strides4<i64>& output_slice_strides, const Shape4<i64>& output_slice_shape,
            const Scale0& insert_fwd_scaling_matrices, const Rotate0& insert_inv_rotation_matrices,
            const Scale1& extract_inv_scaling_matrices, const Rotate1& extract_fwd_rotation_matrices,
            f32 fftfreq_cutoff, f32 fftfreq_input_sinc, f32 fftfreq_input_blackman,
            f32 fftfreq_z_sinc, f32 fftfreq_z_blackman, bool add_to_output,
            const Vec2<f32>& ews_radius, Stream& stream
    ) {
        NOA_ASSERT_DEVICE_PTR(output_slice, stream.device());
        NOA_ASSERT_DEVICE_PTR(input_slice, stream.device());

        const auto input_slice_accessor = AccessorRestrict<const Value, 3, u32>(
                input_slice, input_slice_strides.filter(0, 2, 3).as_safe<u32>());
        const auto input_slice_interpolator = noa::geometry::interpolator_2d<BorderMode::ZERO, InterpMode::LINEAR>(
                input_slice_accessor, input_slice_shape.filter(2, 3).as_safe<i32>().rfft(), Value{0});

        insert_interpolate_and_extract_3d_<REMAP>(
                input_slice_interpolator, input_slice_shape,
                output_slice, output_slice_strides, output_slice_shape,
                insert_fwd_scaling_matrices, insert_inv_rotation_matrices,
                extract_inv_scaling_matrices, extract_fwd_rotation_matrices,
                fftfreq_cutoff, fftfreq_input_sinc, fftfreq_input_blackman,
                fftfreq_z_sinc, fftfreq_z_blackman, add_to_output,
                ews_radius, stream);
    }

    template<Remap REMAP, typename Value,
             typename Scale0, typename Scale1,
             typename Rotate0, typename Rotate1, typename>
    void insert_interpolate_and_extract_3d(
            Value input_slice, const Shape4<i64>& input_slice_shape,
            Value* output_slice, const Strides4<i64>& output_slice_strides, const Shape4<i64>& output_slice_shape,
            const Scale0& insert_fwd_scaling_matrices, const Rotate0& insert_inv_rotation_matrices,
            const Scale1& extract_inv_scaling_matrices, const Rotate1& extract_fwd_rotation_matrices,
            f32 fftfreq_cutoff, f32 fftfreq_input_sinc, f32 fftfreq_input_blackman,
            f32 fftfreq_z_sinc, f32 fftfreq_z_blackman, bool add_to_output,
            const Vec2<f32>& ews_radius, Stream& stream
    ) {
        NOA_ASSERT_DEVICE_PTR(output_slice, stream.device());

        const auto input_slice_interpolator = noa::geometry::interpolator_value_2d<BorderMode::ZERO, InterpMode::LINEAR>(
                input_slice, input_slice_shape.filter(2, 3).as_safe<i32>().rfft(), Value{0});

        insert_interpolate_and_extract_3d_<REMAP>(
                input_slice_interpolator, input_slice_shape,
                output_slice, output_slice_strides, output_slice_shape,
                insert_fwd_scaling_matrices, insert_inv_rotation_matrices,
                extract_inv_scaling_matrices, extract_fwd_rotation_matrices,
                fftfreq_cutoff, fftfreq_input_sinc, fftfreq_input_blackman,
                fftfreq_z_sinc, fftfreq_z_blackman, add_to_output,
                ews_radius, stream);
    }

    template<Remap REMAP, typename Value,
             typename Scale0, typename Scale1,
             typename Rotate0, typename Rotate1, typename>
    void insert_interpolate_and_extract_3d(
            hipArray* input_slice_array, hipTextureObject_t input_slice_texture,
            InterpMode input_slice_interpolation_mode, const Shape4<i64>& input_slice_shape,
            Value* output_slice, const Strides4<i64>& output_slice_strides, const Shape4<i64>& output_slice_shape,
            const Scale0& insert_fwd_scaling_matrices, const Rotate0& insert_inv_rotation_matrices,
            const Scale1& extract_inv_scaling_matrices, const Rotate1& extract_fwd_rotation_matrices,
            f32 fftfreq_cutoff, f32 fftfreq_input_sinc, f32 fftfreq_input_blackman,
            f32 fftfreq_z_sinc, f32 fftfreq_z_blackman, bool add_to_output,
            const Vec2<f32>& ews_radius, Stream& stream
    ) {
        // Input texture requirements:
        NOA_ASSERT(noa::cuda::memory::AllocatorTexture::array(input_slice_texture) == input_slice_array);
        NOA_ASSERT_DEVICE_PTR(output_slice, stream.device());
        NOA_CHECK(input_slice_interpolation_mode == InterpMode::LINEAR ||
                  input_slice_interpolation_mode == InterpMode::LINEAR_FAST,
                  "The interpolation mode should be {} or {}, got {}",
                  InterpMode::LINEAR, InterpMode::LINEAR_FAST, input_slice_interpolation_mode);

        const bool is_layered = noa::cuda::memory::AllocatorArray<Value>::is_layered(input_slice_array);
        if (is_layered) {
            if (input_slice_interpolation_mode == InterpMode::LINEAR) {
                using interpolator_t = noa::cuda::geometry::Interpolator2D<InterpMode::LINEAR, Value, false, true>;
                insert_interpolate_and_extract_3d_<REMAP>(
                        interpolator_t(input_slice_texture), input_slice_shape,
                        output_slice, output_slice_strides, output_slice_shape,
                        insert_fwd_scaling_matrices, insert_inv_rotation_matrices,
                        extract_inv_scaling_matrices, extract_fwd_rotation_matrices,
                        fftfreq_cutoff, fftfreq_input_sinc, fftfreq_input_blackman,
                        fftfreq_z_sinc, fftfreq_z_blackman, add_to_output,
                        ews_radius, stream);
            } else if (input_slice_interpolation_mode == InterpMode::LINEAR_FAST) {
                using interpolator_t = noa::cuda::geometry::Interpolator2D<InterpMode::LINEAR_FAST, Value, false, true>;
                insert_interpolate_and_extract_3d_<REMAP>(
                        interpolator_t(input_slice_texture), input_slice_shape,
                        output_slice, output_slice_strides, output_slice_shape,
                        insert_fwd_scaling_matrices, insert_inv_rotation_matrices,
                        extract_inv_scaling_matrices, extract_fwd_rotation_matrices,
                        fftfreq_cutoff, fftfreq_input_sinc, fftfreq_input_blackman,
                        fftfreq_z_sinc, fftfreq_z_blackman, add_to_output,
                        ews_radius, stream);
            }
        } else {
            if (input_slice_interpolation_mode == InterpMode::LINEAR) {
                using interpolator_t = cuda::geometry::Interpolator2D<InterpMode::LINEAR, Value>;
                insert_interpolate_and_extract_3d_<REMAP>(
                        interpolator_t(input_slice_texture), input_slice_shape,
                        output_slice, output_slice_strides, output_slice_shape,
                        insert_fwd_scaling_matrices, insert_inv_rotation_matrices,
                        extract_inv_scaling_matrices, extract_fwd_rotation_matrices,
                        fftfreq_cutoff, fftfreq_input_sinc, fftfreq_input_blackman,
                        fftfreq_z_sinc, fftfreq_z_blackman, add_to_output,
                        ews_radius, stream);
            } else if (input_slice_interpolation_mode == InterpMode::LINEAR_FAST) {
                using interpolator_t = cuda::geometry::Interpolator2D<InterpMode::LINEAR_FAST, Value>;
                insert_interpolate_and_extract_3d_<REMAP>(
                        interpolator_t(input_slice_texture), input_slice_shape,
                        output_slice, output_slice_strides, output_slice_shape,
                        insert_fwd_scaling_matrices, insert_inv_rotation_matrices,
                        extract_inv_scaling_matrices, extract_fwd_rotation_matrices,
                        fftfreq_cutoff, fftfreq_input_sinc, fftfreq_input_blackman,
                        fftfreq_z_sinc, fftfreq_z_blackman, add_to_output,
                        ews_radius, stream);
            }
        }
    }

    template<typename Value, typename>
    void gridding_correction(
            const Value* input, const Strides4<i64>& input_strides,
            Value* output, const Strides4<i64>& output_strides,
            const Shape4<i64>& shape, bool post_correction, Stream& stream
    ) {
        NOA_ASSERT(noa::all(shape > 0));
        NOA_ASSERT_DEVICE_PTR(input, stream.device());
        NOA_ASSERT_DEVICE_PTR(output, stream.device());

        const auto i_shape = shape.as_safe<u32>();
        const auto input_accessor = Accessor<const Value, 4, u32>(input, input_strides.as_safe<u32>());
        const auto output_accessor = Accessor<Value, 4, u32>(output, output_strides.as_safe<u32>());

        if (post_correction) {
            const auto op = na::geometry::gridding_correction<true>(
                    input_accessor, output_accessor, i_shape);
            noa::cuda::utils::iwise_4d(i_shape, op, stream);
        } else {
            const auto op = na::geometry::gridding_correction<false>(
                    input_accessor, output_accessor, i_shape);
            noa::cuda::utils::iwise_4d(i_shape, op, stream);
        }
    }
    template void gridding_correction<f32, void>(const f32*, const Strides4<i64>&, f32*, const Strides4<i64>&, const Shape4<i64>&, bool, Stream&);
    template void gridding_correction<f64, void>(const f64*, const Strides4<i64>&, f64*, const Strides4<i64>&, const Shape4<i64>&, bool, Stream&);

    #define NOA_INSTANTIATE_INSERT_RASTERIZE_(T, REMAP, S, R)                       \
    template void insert_rasterize_3d<REMAP, T, S, R, void>(                        \
        const T*, const Strides4<i64>&, const Shape4<i64>&,                         \
        T*, const Strides4<i64>&, const Shape4<i64>&,                               \
        S const&, R const&, f32, const Shape4<i64>&, const Vec2<f32>&, Stream&);    \
    template void insert_rasterize_3d<REMAP, T, S, R, void>(                        \
        T, const Shape4<i64>&,                                                      \
        T*, const Strides4<i64>&, const Shape4<i64>&,                               \
        S const&, R const&, f32, const Shape4<i64>&, const Vec2<f32>&, Stream&)

    #define NOA_INSTANTIATE_INSERT_INTERPOLATE_(T, REMAP, S, R)     \
    template void insert_interpolate_3d<REMAP, T, S, R, void>(      \
        const T*, const Strides4<i64>&, const Shape4<i64>&,         \
        T*, const Strides4<i64>&, const Shape4<i64>&,               \
        S const&, R const&, f32, f32, f32,                          \
        const Shape4<i64>&, const Vec2<f32>&, Stream&);             \
    template void insert_interpolate_3d<REMAP, T, S, R, void>(      \
        T, const Shape4<i64>&,                                      \
        T*, const Strides4<i64>&, const Shape4<i64>&,               \
        S const&, R const&, f32, f32, f32,                          \
        const Shape4<i64>&, const Vec2<f32>&, Stream&)

    #define NOA_INSTANTIATE_EXTRACT_(T, REMAP, S, R)        \
    template void extract_3d<REMAP, T, S, R, void>(         \
        const T*, const Strides4<i64>&, const Shape4<i64>&, \
        T*, const Strides4<i64>&, const Shape4<i64>&,       \
        S const&, R const&, f32, f32, f32,                  \
        const Shape4<i64>&, const Vec2<f32>&, Stream&)

    #define NOA_INSTANTIATE_INSERT_EXTRACT_(T, REMAP, S0, S1, R0, R1)                   \
    template void insert_interpolate_and_extract_3d<REMAP, T, S0, S1, R0, R1, void>(    \
        const T*, const Strides4<i64>&, const Shape4<i64>&,                             \
        T*, const Strides4<i64>&, const Shape4<i64>&,                                   \
        S0 const&, R0 const&, S1 const&, R1 const&,                                     \
        f32, f32, f32, f32, f32, bool, const Vec2<f32>&, Stream&);                      \
    template void insert_interpolate_and_extract_3d<REMAP, T, S0, S1, R0, R1, void>(    \
        T, const Shape4<i64>&,                                                          \
        T*, const Strides4<i64>&, const Shape4<i64>&,                                   \
        S0 const&, R0 const&, S1 const&, R1 const&,                                     \
        f32, f32, f32, f32, f32, bool, const Vec2<f32>&, Stream&)

    #define NOA_INSTANTIATE_PROJECT_ALL_REMAP(T, S, R)          \
    NOA_INSTANTIATE_INSERT_RASTERIZE_(T, Remap::H2H, S, R);     \
    NOA_INSTANTIATE_INSERT_RASTERIZE_(T, Remap::H2HC, S, R);    \
    NOA_INSTANTIATE_INSERT_RASTERIZE_(T, Remap::HC2H, S, R);    \
    NOA_INSTANTIATE_INSERT_RASTERIZE_(T, Remap::HC2HC, S, R);   \
    NOA_INSTANTIATE_INSERT_INTERPOLATE_(T, Remap::HC2H, S, R);  \
    NOA_INSTANTIATE_INSERT_INTERPOLATE_(T, Remap::HC2HC, S, R); \
    NOA_INSTANTIATE_EXTRACT_(T, Remap::HC2H, S, R);             \
    NOA_INSTANTIATE_EXTRACT_(T, Remap::HC2HC, S, R)

    #define NOA_INSTANTIATE_PROJECT_MERGE_ALL_REMAP(T, S0, S1, R0, R1)  \
    NOA_INSTANTIATE_INSERT_EXTRACT_(T, Remap::HC2H, S0, S1, R0, R1);    \
    NOA_INSTANTIATE_INSERT_EXTRACT_(T, Remap::HC2HC, S0, S1, R0, R1)

    #define NOA_INSTANTIATE_PROJECT_MERGE_ALL_SCALE(T, R0, R1)                      \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_REMAP(T, Float22, Float22, R0, R1);           \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_REMAP(T, const Float22*, Float22, R0, R1);    \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_REMAP(T, Float22, const Float22*, R0, R1);    \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_REMAP(T, const Float22*, const Float22*, R0, R1)

    #define NOA_INSTANTIATE_PROJECT_MERGE_ALL_ROTATE(T)                     \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_SCALE(T, Float33, Float33);           \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_SCALE(T, const Float33*, Float33);    \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_SCALE(T, Float33, const Float33*);    \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_SCALE(T, const Float33*, const Float33*)

    #define NOA_INSTANTIATE_PROJECT_ALL_(T)                                 \
    NOA_INSTANTIATE_PROJECT_ALL_REMAP(T, Float22, Float33);                 \
    NOA_INSTANTIATE_PROJECT_ALL_REMAP(T, const Float22*, Float33);          \
    NOA_INSTANTIATE_PROJECT_ALL_REMAP(T, Float22, const Float33*);          \
    NOA_INSTANTIATE_PROJECT_ALL_REMAP(T, const Float22*, const Float33*);   \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_ROTATE(T)

    NOA_INSTANTIATE_PROJECT_ALL_(f32);
    NOA_INSTANTIATE_PROJECT_ALL_(f64);
    NOA_INSTANTIATE_PROJECT_ALL_(c32);
    NOA_INSTANTIATE_PROJECT_ALL_(c64);

    #define NOA_INSTANTIATE_INSERT_THICK_TEXTURE(T, REMAP, S, R)            \
    template void insert_interpolate_3d<REMAP, T, S, R, void>(              \
        hipArray*, hipTextureObject_t, InterpMode, const Shape4<i64>&,    \
        T*, const Strides4<i64>&, const Shape4<i64>&,                       \
        S const&, R const&, f32, f32, f32,                                  \
        const Shape4<i64>&, const Vec2<f32>&, Stream&)

    #define NOA_INSTANTIATE_EXTRACT_TEXTURE(T, REMAP, S, R)                 \
    template void extract_3d<REMAP, T, S, R, void>(                         \
        hipArray*, hipTextureObject_t, InterpMode, const Shape4<i64>&,    \
        T*, const Strides4<i64>&, const Shape4<i64>&,                       \
        S const&, R const&, f32, f32, f32,                                  \
        const Shape4<i64>&, const Vec2<f32>&, Stream&)

    #define NOA_INSTANTIATE_INSERT_EXTRACT_TEXTURE_(T, REMAP, S0, S1, R0, R1)       \
    template void insert_interpolate_and_extract_3d<REMAP, T, S0, S1, R0, R1, void>(\
        hipArray*, hipTextureObject_t, InterpMode, const Shape4<i64>&,            \
        T*, const Strides4<i64>&, const Shape4<i64>&,                               \
        S0 const&, R0 const&, S1 const&, R1 const&,                                 \
        f32, f32, f32, f32, f32, bool, const Vec2<f32>&, Stream&)

    #define NOA_INSTANTIATE_PROJECT_TEXTURE_ALL_REMAP(T, S, R)  \
    NOA_INSTANTIATE_INSERT_THICK_TEXTURE(T, Remap::HC2H, S, R); \
    NOA_INSTANTIATE_INSERT_THICK_TEXTURE(T, Remap::HC2HC, S, R);\
    NOA_INSTANTIATE_EXTRACT_TEXTURE(T, Remap::HC2H, S, R);      \
    NOA_INSTANTIATE_EXTRACT_TEXTURE(T, Remap::HC2HC, S, R)

    #define NOA_INSTANTIATE_PROJECT_MERGE_ALL_REMAP_TEXTURE(T, S0, S1, R0, R1)  \
    NOA_INSTANTIATE_INSERT_EXTRACT_TEXTURE_(T, Remap::HC2H, S0, S1, R0, R1);    \
    NOA_INSTANTIATE_INSERT_EXTRACT_TEXTURE_(T, Remap::HC2HC, S0, S1, R0, R1)

    #define NOA_INSTANTIATE_PROJECT_MERGE_ALL_SCALE_TEXTURE(T, R0, R1)                      \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_REMAP_TEXTURE(T, Float22, Float22, R0, R1);           \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_REMAP_TEXTURE(T, const Float22*, Float22, R0, R1);    \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_REMAP_TEXTURE(T, Float22, const Float22*, R0, R1);    \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_REMAP_TEXTURE(T, const Float22*, const Float22*, R0, R1)

    #define NOA_INSTANTIATE_PROJECT_MERGE_ALL_ROTATE_TEXTURE(T)                     \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_SCALE_TEXTURE(T, Float33, Float33);           \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_SCALE_TEXTURE(T, const Float33*, Float33);    \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_SCALE_TEXTURE(T, Float33, const Float33*);    \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_SCALE_TEXTURE(T, const Float33*, const Float33*)

    #define NOA_INSTANTIATE_PROJECT_TEXTURE_ALL(T)                                  \
    NOA_INSTANTIATE_PROJECT_TEXTURE_ALL_REMAP(T, Float22, Float33);                 \
    NOA_INSTANTIATE_PROJECT_TEXTURE_ALL_REMAP(T, const Float22*, Float33);          \
    NOA_INSTANTIATE_PROJECT_TEXTURE_ALL_REMAP(T, Float22, const Float33*);          \
    NOA_INSTANTIATE_PROJECT_TEXTURE_ALL_REMAP(T, const Float22*, const Float33*);   \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_ROTATE_TEXTURE(T)

    NOA_INSTANTIATE_PROJECT_TEXTURE_ALL(f32);
    NOA_INSTANTIATE_PROJECT_TEXTURE_ALL(c32);
}
