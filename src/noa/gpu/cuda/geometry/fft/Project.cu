#include "hip/hip_runtime.h"
#include "noa/gpu/cuda/Exception.h"
#include "noa/gpu/cuda/memory/Copy.h"
#include "noa/gpu/cuda/memory/PtrArray.h"
#include "noa/gpu/cuda/memory/PtrDevice.h"
#include "noa/gpu/cuda/memory/PtrTexture.h"
#include "noa/gpu/cuda/util/Atomic.cuh"
#include "noa/gpu/cuda/util/Pointers.h"
#include "noa/gpu/cuda/geometry/Interpolate.h"
#include "noa/gpu/cuda/geometry/fft/Project.h"

// This implementation is almost identical to the CPU backend's.
// See implementation details/comments there.
namespace {
    using namespace ::noa;
    constexpr dim3 THREADS(32, 8);

    template<typename T>
    struct GridNoTexture {
        const T* __restrict__ ptr;
        uint3_t strides;
        int3_t shape;
    };

    template<bool IS_CENTERED>
    [[nodiscard]] __device__ __forceinline__ int32_t getIndex_(int32_t frequency, int32_t volume_dim) {
        if constexpr (IS_CENTERED) {
            return frequency + volume_dim / 2;
        } else {
            return frequency < 0 ? frequency + volume_dim : frequency;
        }
        return 0; // unreachable - remove false warning
    }

    template<bool IS_CENTERED>
    [[nodiscard]] __device__ __forceinline__ int32_t getFrequency_(int32_t index, int32_t shape) {
        if constexpr (IS_CENTERED)
            return index - shape / 2;
        else
            return index < (shape + 1) / 2 ? index : index - shape;
        return 0; // unreachable
    }

    __device__ inline void setGriddingWeights_(int3_t base0, float3_t freq, float o_weights[2][2][2]) {
        float3_t fraction[2];
        fraction[1] = freq - float3_t(base0);
        fraction[0] = 1.f - fraction[1];
        for (int64_t w = 0; w < 2; ++w)
            for (int64_t v = 0; v < 2; ++v)
                for (int64_t u = 0; u < 2; ++u)
                    o_weights[w][v][u] = fraction[w][0] * fraction[v][1] * fraction[u][2];
    }

    __device__ inline void setBoundary_(int3_t base0, int3_t shape, bool2_t o_bound[3]) {
        const int3_t base1(base0 + 1);
        const int3_t idx_max = (shape - 1) / 2;

        o_bound[0][0] = base0[0] >= -idx_max[0] && base0[0] <= idx_max[0];
        o_bound[0][1] = base1[0] >= -idx_max[0] && base1[0] <= idx_max[0];

        o_bound[1][0] = base0[1] >= -idx_max[1] && base0[1] <= idx_max[1];
        o_bound[1][1] = base1[1] >= -idx_max[1] && base1[1] <= idx_max[1];

        o_bound[2][0] = base0[2] <= idx_max[2];
        o_bound[2][1] = base1[2] <= idx_max[2];
    }

    template<bool IS_CENTERED, typename T>
    __device__ void addByGridding_(T* grid, uint3_t grid_strides, int3_t grid_shape, T data, float3_t frequency) {
        using real_t = traits::value_type_t<T>;
        namespace atomic = noa::cuda::util::atomic;

        const int3_t base0(math::floor(frequency));

        float kernel[2][2][2];
        setGriddingWeights_(base0, frequency, kernel);

        bool2_t is_valid[3];
        setBoundary_(base0, grid_shape, is_valid);

        for (int32_t w = 0; w < 2; ++w) {
            for (int32_t v = 0; v < 2; ++v) {
                for (int32_t u = 0; u < 2; ++u) {
                    if (is_valid[0][w] && is_valid[1][v] && is_valid[2][u]) {
                        const int32_t idx_w = getIndex_<IS_CENTERED>(base0[0] + w, grid_shape[0]);
                        const int32_t idx_v = getIndex_<IS_CENTERED>(base0[1] + v, grid_shape[1]);
                        const int32_t idx_u = base0[2] + u;
                        const auto fraction = static_cast<real_t>(kernel[w][v][u]);
                        atomic::add(grid + indexing::at(idx_w, idx_v, idx_u, grid_strides), data * fraction);
                    }
                }
            }
        }

        if (base0[2] == 0 && (base0[1] != 0 || base0[0] != 0)) {
            if constexpr (traits::is_complex_v<T>)
                data.imag = -data.imag;
            for (int32_t w = 0; w < 2; ++w) {
                for (int32_t v = 0; v < 2; ++v) {
                    if (is_valid[0][w] && is_valid[1][v]) {
                        const int32_t idx_w = getIndex_<IS_CENTERED>(-(base0[0] + w), grid_shape[0]);
                        const int32_t idx_v = getIndex_<IS_CENTERED>(-(base0[1] + v), grid_shape[1]);
                        const auto fraction = static_cast<real_t>(kernel[w][v][0]);
                        atomic::add(grid + indexing::at(idx_w, idx_v, grid_strides), data * fraction);
                    }
                }
            }
        }
    }

    template<typename T>
    [[nodiscard]] __device__ T linear3D_(const T* __restrict__ grid, uint3_t strides, int3_t shape, float3_t frequency) {
        int3_t idx[2];
        idx[0] = int3_t(noa::math::floor(frequency));
        idx[1] = idx[0] + 1;

        const bool cond_z[2] = {idx[0][0] >= 0 && idx[0][0] < shape[0], idx[1][0] >= 0 && idx[1][0] < shape[0]};
        const bool cond_y[2] = {idx[0][1] >= 0 && idx[0][1] < shape[1], idx[1][1] >= 0 && idx[1][1] < shape[1]};
        const bool cond_x[2] = {idx[0][2] >= 0 && idx[0][2] < shape[2], idx[1][2] >= 0 && idx[1][2] < shape[2]};

        const uint32_t off_z[2] = {idx[0][0] * strides[0], idx[1][0] * strides[0]};
        const uint32_t off_y[2] = {idx[0][1] * strides[1], idx[1][1] * strides[1]};
        const uint32_t off_x[2] = {idx[0][2] * strides[2], idx[1][2] * strides[2]};

        const float rx = frequency[2] - static_cast<float>(idx[0][2]);
        const float ry = frequency[1] - static_cast<float>(idx[0][1]);

        const T v000 = cond_z[0] && cond_y[0] && cond_x[0] ? grid[off_z[0] + off_y[0] + off_x[0]] : T{0};
        const T v001 = cond_z[0] && cond_y[0] && cond_x[1] ? grid[off_z[0] + off_y[0] + off_x[1]] : T{0};
        const T v010 = cond_z[0] && cond_y[1] && cond_x[0] ? grid[off_z[0] + off_y[1] + off_x[0]] : T{0};
        const T v011 = cond_z[0] && cond_y[1] && cond_x[1] ? grid[off_z[0] + off_y[1] + off_x[1]] : T{0};
        T tmp1 = cuda::geometry::details::linear2D(v000, v001, v010, v011, rx, ry);

        const T v100 = cond_z[1] && cond_y[0] && cond_x[0] ? grid[off_z[1] + off_y[0] + off_x[0]] : T{0};
        const T v101 = cond_z[1] && cond_y[0] && cond_x[1] ? grid[off_z[1] + off_y[0] + off_x[1]] : T{0};
        const T v110 = cond_z[1] && cond_y[1] && cond_x[0] ? grid[off_z[1] + off_y[1] + off_x[0]] : T{0};
        const T v111 = cond_z[1] && cond_y[1] && cond_x[1] ? grid[off_z[1] + off_y[1] + off_x[1]] : T{0};
        T tmp2 = cuda::geometry::details::linear2D(v100, v101, v110, v111, rx, ry);

        const float rz = frequency[0] - static_cast<float>(idx[0][0]);
        return cuda::geometry::details::linear1D(tmp1, tmp2, rz);
    }

    template<bool IS_SRC_CENTERED, bool IS_DST_CENTERED, typename T>
    __global__ void __launch_bounds__(THREADS.x * THREADS.y)
    fourierInsert_(const T* __restrict__ slice, uint3_t slice_strides, int2_t slice_shape, float2_t f_slice_shape,
                   T* __restrict__ grid, uint3_t grid_strides, int3_t grid_shape, float3_t f_grid_shape,
                   const float22_t* inv_scaling_factors, const float33_t* rotations,
                   float cutoff_sqd, float2_t ews_diam_inv) {
        using real_t = traits::value_type_t<T>;
        const int3_t gid{blockIdx.z,
                         blockIdx.y * THREADS.y + threadIdx.y,
                         blockIdx.x * THREADS.x + threadIdx.x};
        if (gid[1] >= slice_shape[0] || gid[2] >= slice_shape[1])
            return;

        const int32_t v = getFrequency_<IS_SRC_CENTERED>(gid[1], slice_shape[0]);
        const float2_t orig_freq{v, gid[2]};
        float2_t freq_2d = orig_freq / f_slice_shape;

        if (inv_scaling_factors)
            freq_2d = inv_scaling_factors[gid[0]] * freq_2d;

        const float z = math::sum(ews_diam_inv * freq_2d * freq_2d);
        float3_t freq_3d{z, freq_2d[0], freq_2d[1]};
        freq_3d = rotations[gid[0]] * freq_3d;

        if (math::dot(freq_3d, freq_3d) > cutoff_sqd)
            return;

        real_t conj = 1;
        if (freq_3d[2] < 0) {
            freq_3d = -freq_3d;
            if constexpr(traits::is_complex_v<T>)
                conj = -1;
        }
        freq_3d *= f_grid_shape;

        T value = slice[indexing::at(gid, slice_strides)];
        if constexpr(traits::is_complex_v<T>)
            value.imag *= conj;
        else
            (void) conj;

        addByGridding_<IS_DST_CENTERED>(grid, grid_strides, grid_shape, value, freq_3d);
    }

    template<bool IS_DST_CENTERED, typename T, typename U>
    __global__ void __launch_bounds__(THREADS.x * THREADS.y)
    fourierExtract_(T grid, float3_t f_grid_shape,
                    U* __restrict__ slice, uint3_t slice_strides, int2_t slice_shape, float2_t f_slice_shape,
                    const float22_t* inv_scaling_factors, const float33_t* rotations,
                    float cutoff_sqd, float2_t ews_diam_inv) {
        using real_t = traits::value_type_t<U>;
        const int3_t gid{blockIdx.z,
                         blockIdx.y * THREADS.y + threadIdx.y,
                         blockIdx.x * THREADS.x + threadIdx.x};
        if (gid[1] >= slice_shape[0] || gid[2] >= slice_shape[1])
            return;

        // -------------------------------- //
        const int32_t v = getFrequency_<IS_DST_CENTERED>(gid[1], slice_shape[0]);
        const float2_t orig_freq{v, gid[2]};
        float2_t freq_2d = orig_freq / f_slice_shape;

        if (inv_scaling_factors)
            freq_2d = inv_scaling_factors[gid[0]] * freq_2d;

        const float z = math::sum(ews_diam_inv * freq_2d * freq_2d);
        float3_t freq_3d{z, freq_2d[0], freq_2d[1]};
        freq_3d = rotations[gid[0]] * freq_3d;

        if (math::dot(freq_3d, freq_3d) > cutoff_sqd) {
            slice[indexing::at(gid, slice_strides)] = U{0};
            return;
        }

        real_t conj = 1;
        if (freq_3d[2] < 0) {
            freq_3d = -freq_3d;
            if constexpr(traits::is_complex_v<U>)
                conj = -1;
        }
        freq_3d[0] += 0.5f;
        freq_3d[1] += 0.5f;
        freq_3d *= f_grid_shape;
        // -------------------------------- //

        U value;
        if constexpr (std::is_same_v<T, hipTextureObject_t>)
            value = cuda::geometry::tex3D<U, INTERP_LINEAR>(grid, freq_3d + 0.5f);
        else
            value = linear3D_(grid.ptr, grid.strides, grid.shape, freq_3d);

        if constexpr(traits::is_complex_v<U>)
            value.imag *= conj;
        else
            (void) conj;

        slice[indexing::at(gid, slice_strides)] = value;
    }

    template<bool POST_CORRECTION, typename T>
    __global__ void __launch_bounds__(THREADS.x * THREADS.y)
    correctGriddingSinc2_(Accessor<const T, 4, uint32_t> input,
                          Accessor<T, 4, uint32_t> output,
                          uint2_t shape, float3_t f_shape, float3_t half, uint32_t blocks_x) {
        constexpr float PI = math::Constants<float>::PI;

        const uint2_t indexes = indexing::indexes(blockIdx.x, blocks_x);
        const uint4_t gid{blockIdx.z,
                          blockIdx.y,
                          indexes[0] * THREADS.y + threadIdx.y,
                          indexes[1] * THREADS.x + threadIdx.x};
        if (gid[2] >= shape[0] || gid[3] >= shape[1])
            return;

        float3_t dist{gid[1], gid[2], gid[3]};
        dist -= half;
        dist /= f_shape;

        const float radius = math::sqrt(math::dot(dist, dist));
        const float sinc = math::sinc(PI * radius);
        const T sinc2 = static_cast<T>(sinc * sinc); // > 0.05
        output(gid) = POST_CORRECTION ? input(gid) / sinc2 : input(gid) * sinc2;
    }

    template<fft::Remap REMAP, typename T, typename U>
    void launchExtract3D_(const T* grid, dim4_t grid_strides, int3_t grid_shape,
                          U* slice, dim4_t slice_strides, dim4_t slice_shape,
                          const float22_t* scaling_factors, const float33_t* rotations,
                          float cutoff, float3_t sampling_factor, float2_t ews_radius, cuda::Stream& stream) {
        using Layout = ::noa::fft::Layout;
        constexpr auto REMAP_ = static_cast<uint8_t>(REMAP);
        constexpr bool IS_DST_CENTERED = REMAP_ & Layout::DST_CENTERED;
        if constexpr (REMAP_ & Layout::SRC_NON_CENTERED ||
                      REMAP_ & Layout::SRC_FULL ||
                      REMAP_ & Layout::DST_FULL)
            static_assert(traits::always_false_v<T>);

        // Dimensions:
        const dim_t count = slice_shape[0];
        const auto slice_shape_ = safe_cast<int2_t>(dim2_t{slice_shape[2], slice_shape[3]});
        const auto slice_strides_ = safe_cast<uint3_t>(dim3_t{slice_strides[0], slice_strides[2], slice_strides[3]});
        const float2_t f_slice_shape(slice_shape_ / 2 * 2 + int2_t(slice_shape_ == 1));

        float3_t f_grid_shape(grid_shape / 2 * 2 + int3_t(grid_shape == 1));
        f_grid_shape *= sampling_factor;

        // Launch config:
        const uint2_t tmp(slice_shape.get(2));
        const dim3 blocks(math::divideUp(tmp[1] / 2 + 1, THREADS.x),
                          math::divideUp(tmp[0], THREADS.y),
                          count);
        const cuda::LaunchConfig config{blocks, THREADS};

        // Some preprocessing:
        const float2_t ews_diam_inv = any(ews_radius != 0) ? 1 / (2 * ews_radius) : float2_t{};
        cutoff = math::clamp(cutoff, 0.f, 0.5f);
        cutoff *= cutoff;

        // Ensure transformation parameters are accessible to the GPU:
        cuda::memory::PtrDevice<float22_t> b0;
        cuda::memory::PtrDevice<float33_t> b1;
        using namespace cuda::util;
        const auto* ptr0 = scaling_factors ? ensureDeviceAccess(scaling_factors, stream, b0, count) : nullptr;
        const auto* ptr1 = ensureDeviceAccess(rotations, stream, b1, count);

        if constexpr (noa::traits::is_almost_same_v<T, hipTextureObject_t>) {
            (void) grid_strides;
            stream.enqueue("geometry::fft::extract3D", fourierExtract_<IS_DST_CENTERED, hipTextureObject_t, U>, config,
                           *grid, f_grid_shape,
                           slice, slice_strides_, slice_shape_, f_slice_shape,
                           ptr0, ptr1, cutoff, ews_diam_inv);
        } else {
            const auto grid_strides_ = safe_cast<uint3_t>(dim3_t(grid_strides.get(1)));
            GridNoTexture<T> grid_{grid, grid_strides_, grid_shape};
            stream.enqueue("geometry::fft::extract3D", fourierExtract_<IS_DST_CENTERED, GridNoTexture<T>, U>, config,
                           grid_, f_grid_shape,
                           slice, slice_strides_, slice_shape_, f_slice_shape,
                           ptr0, ptr1, cutoff, ews_diam_inv);
        }
    }
}

namespace noa::cuda::geometry::fft {
    template<Remap REMAP, typename T, typename>
    void insert3D(const shared_t<T[]>& slice, dim4_t slice_strides, dim4_t slice_shape,
                  const shared_t<T[]>& grid, dim4_t grid_strides, dim4_t grid_shape,
                  const shared_t<float22_t[]>& scaling_factors,
                  const shared_t<float33_t[]>& rotations,
                  float cutoff, float3_t sampling_factor, float2_t ews_radius, Stream& stream) {
        using Layout = ::noa::fft::Layout;
        constexpr auto REMAP_ = static_cast<uint8_t>(REMAP);
        constexpr bool IS_SRC_CENTERED = REMAP_ & Layout::SRC_CENTERED;
        constexpr bool IS_DST_CENTERED = REMAP_ & Layout::DST_CENTERED;
        if constexpr (REMAP_ & Layout::SRC_FULL || REMAP_ & Layout::DST_FULL)
            static_assert(traits::always_false_v<T>);

        NOA_ASSERT(rotations && slice.get() != grid.get() && all(slice_shape > 0) && all(grid_shape > 0));
        NOA_ASSERT_DEVICE_PTR(slice.get(), stream.device());
        NOA_ASSERT_DEVICE_PTR(grid.get(), stream.device());
        NOA_ASSERT(slice_shape[1] == 1);
        NOA_ASSERT(grid_shape[0] == 1);

        // Dimensions:
        const dim_t count = slice_shape[0];
        const auto slice_shape_ = safe_cast<int2_t>(dim2_t{slice_shape[2], slice_shape[3]});
        const auto grid_shape_ = safe_cast<int3_t>(dim3_t(grid_shape.get(1)));
        const auto slice_strides_ = safe_cast<uint3_t>(dim3_t{slice_strides[0], slice_strides[2], slice_strides[3]});
        const auto grid_strides_ = safe_cast<uint3_t>(dim3_t(grid_strides.get(1)));
        const float2_t f_slice_shape(slice_shape_ / 2 * 2 + int2_t(slice_shape_ == 1));
        float3_t f_grid_shape(grid_shape_ / 2 * 2 + int3_t(grid_shape_ == 1));
        f_grid_shape *= sampling_factor;

        // Launch config:
        const uint2_t tmp(slice_shape_);
        const dim3 blocks(math::divideUp(tmp[1] / 2 + 1, THREADS.x),
                          math::divideUp(tmp[0], THREADS.y),
                          count);
        const LaunchConfig config{blocks, THREADS};

        // Some preprocessing:
        const float2_t ews_diam_inv = any(ews_radius != 0) ? 1 / (2 * ews_radius) : float2_t{};
        cutoff = math::clamp(cutoff, 0.f, 0.5f);
        cutoff *= cutoff;

        // Ensure transformation parameters are accessible to the GPU:
        memory::PtrDevice<float22_t> b0;
        memory::PtrDevice<float33_t> b1;
        using namespace util;
        const auto* ptr0 = scaling_factors ? ensureDeviceAccess(scaling_factors.get(), stream, b0, count) : nullptr;
        const auto* ptr1 = ensureDeviceAccess(rotations.get(), stream, b1, count);

        stream.enqueue("geometry::fft::insert3D", fourierInsert_<IS_SRC_CENTERED, IS_DST_CENTERED, T>, config,
                       slice.get(), slice_strides_, slice_shape_, f_slice_shape,
                       grid.get(), grid_strides_, grid_shape_, f_grid_shape,
                       ptr0, ptr1, cutoff, ews_diam_inv);
        stream.attach(slice, grid, scaling_factors, rotations);
    }

    template<Remap REMAP, typename T, typename>
    void extract3D(const shared_t<T[]>& grid, dim4_t grid_strides, dim4_t grid_shape,
                   const shared_t<T[]>& slice, dim4_t slice_strides, dim4_t slice_shape,
                   const shared_t<float22_t[]>& scaling_factors,
                   const shared_t<float33_t[]>& rotations,
                   float cutoff, float3_t sampling_factor, float2_t ews_radius,
                   bool no_texture, Stream& stream) {
        NOA_ASSERT(rotations && all(slice_shape > 0) && all(grid_shape > 0));
        NOA_ASSERT_DEVICE_PTR(slice.get(), stream.device());
        NOA_ASSERT(slice_shape[1] == 1);
        NOA_ASSERT(grid_shape[0] == 1);
        const auto int3_grid_shape = safe_cast<int3_t>(dim3_t(grid_shape.get(1)));

        if (no_texture) {
            NOA_ASSERT(slice.get() != grid.get());
            NOA_ASSERT_DEVICE_PTR(grid.get(), stream.device());
            launchExtract3D_<REMAP>(grid.get(), grid_strides, int3_grid_shape,
                                    slice.get(), slice_strides, slice_shape,
                                    scaling_factors.get(), rotations.get(),
                                    cutoff, sampling_factor, ews_radius, stream);
            stream.attach(grid, slice, scaling_factors, rotations);
        } else {
            if constexpr (traits::is_any_v<T, double, cdouble_t>) {
                NOA_THROW("Double precision is not supported in this mode. Use no_texture=true instead");
            } else {
                NOA_ASSERT(grid_strides[1] == 1 && indexing::isContiguous(grid_strides, grid_shape)[1]);
                memory::PtrArray<T> array(dim3_t(int3_grid_shape.fft()));
                memory::PtrTexture texture(array.get(), INTERP_LINEAR_FAST, BORDER_ZERO);
                memory::copy(grid, grid_strides[2], array.share(), array.shape(), stream);

                const auto tex = texture.get();
                launchExtract3D_<REMAP>(&tex, {}, int3_grid_shape,
                                        slice.get(), slice_strides, slice_shape,
                                        scaling_factors.get(), rotations.get(),
                                        cutoff, sampling_factor, ews_radius, stream);
                stream.attach(array.share(), texture.share(), slice, scaling_factors, rotations);
            }
        }
    }

    template<Remap REMAP, typename T, typename>
    void extract3D(const shared_t<hipArray>& array,
                   const shared_t<hipTextureObject_t>& grid, int3_t grid_shape,
                   const shared_t<T[]>& slice, dim4_t slice_strides, dim4_t slice_shape,
                   const shared_t<float22_t[]>& scaling_factors,
                   const shared_t<float33_t[]>& rotations,
                   float cutoff, float3_t sampling_factor, float2_t ews_radius, Stream& stream) {
        NOA_ASSERT(array && grid && rotations && all(slice_shape > 0) && all(grid_shape > 0));
        NOA_ASSERT_DEVICE_PTR(slice.get(), stream.device());
        launchExtract3D_<REMAP>(grid.get(), {}, grid_shape,
                                slice.get(), slice_strides, slice_shape,
                                scaling_factors.get(), rotations.get(),
                                cutoff, sampling_factor, ews_radius, stream);
        stream.attach(array, grid, slice, scaling_factors, rotations);
    }

    template<typename T, typename>
    void griddingCorrection(const shared_t<T[]>& input, dim4_t input_strides,
                            const shared_t<T[]>& output, dim4_t output_strides,
                            dim4_t shape, bool post_correction, Stream& stream) {
        NOA_ASSERT(all(shape > 0));
        NOA_ASSERT_DEVICE_PTR(input.get(), stream.device());
        NOA_ASSERT_DEVICE_PTR(output.get(), stream.device());

        const uint2_t shape_(shape.get(2));
        const uint32_t blocks_x = math::divideUp(shape_[1], THREADS.x);
        const uint32_t blocks_y = math::divideUp(shape_[0], THREADS.y);
        const dim3 blocks(blocks_x * blocks_y,
                          shape[1],
                          shape[0]);
        const LaunchConfig config{blocks, THREADS};

        const auto i_shape = safe_cast<int3_t>(dim3_t(shape.get(1)));
        const float3_t f_shape(i_shape);
        const float3_t half(f_shape / 2 * float3_t(i_shape != 1)); // if size == 1, half should be 0
        const Accessor<const T, 4, uint32_t> input_accessor(input.get(), safe_cast<uint4_t>(input_strides));
        const Accessor<T, 4, uint32_t> output_accessor(output.get(), safe_cast<uint4_t>(output_strides));

        stream.enqueue("geometry::fft::griddingCorrection",
                       post_correction ? correctGriddingSinc2_<true, T> : correctGriddingSinc2_<false, T>, config,
                       input_accessor, output_accessor, shape_, f_shape, half, blocks_x);
        stream.attach(input, output);
    }

    #define NOA_INSTANTIATE_INSERT_(T, R)                                                                           \
    template void insert3D<R, T, void>(const shared_t<T[]>&, dim4_t, dim4_t, const shared_t<T[]>&, dim4_t, dim4_t,  \
                                       const shared_t<float22_t[]>&, const shared_t<float33_t[]>&, float, float3_t, float2_t, Stream&)

    #define NOA_INSTANTIATE_PROJECT_(T)         \
    NOA_INSTANTIATE_INSERT_(T, Remap::H2H);     \
    NOA_INSTANTIATE_INSERT_(T, Remap::H2HC);    \
    NOA_INSTANTIATE_INSERT_(T, Remap::HC2H);    \
    NOA_INSTANTIATE_INSERT_(T, Remap::HC2HC);   \
    template void griddingCorrection<T, void>(const shared_t<T[]>&, dim4_t, const shared_t<T[]>&, dim4_t, dim4_t, bool, Stream&)

    NOA_INSTANTIATE_PROJECT_(float);
    NOA_INSTANTIATE_PROJECT_(double);
    NOA_INSTANTIATE_PROJECT_(cfloat_t);
    NOA_INSTANTIATE_PROJECT_(cdouble_t);

    #define NOA_INSTANTIATE_EXTRACT_NO_TEXTURE_(T, R)                                                               \
    template void extract3D<R, T, void>(const shared_t<T[]>&, dim4_t, dim4_t, const shared_t<T[]>&, dim4_t, dim4_t, \
                                        const shared_t<float22_t[]>&, const shared_t<float33_t[]>&, float, float3_t, float2_t, bool, Stream&)

    #define NOA_INSTANTIATE_EXTRACT_TEXTURE_(T, R)                                                                          \
    template void extract3D<R, T, void>(const shared_t<hipArray>&,                                                         \
                                        const shared_t<hipTextureObject_t>&, int3_t, const shared_t<T[]>&, dim4_t, dim4_t, \
                                        const shared_t<float22_t[]>&, const shared_t<float33_t[]>&, float, float3_t, float2_t, Stream&)

    #define NOA_INSTANTIATE_EXTRACT_(T, R)      \
    NOA_INSTANTIATE_EXTRACT_NO_TEXTURE_(T, R);  \
    NOA_INSTANTIATE_EXTRACT_TEXTURE_(T, R)

    NOA_INSTANTIATE_EXTRACT_(float, Remap::HC2HC);
    NOA_INSTANTIATE_EXTRACT_(float, Remap::HC2H);
    NOA_INSTANTIATE_EXTRACT_(cfloat_t, Remap::HC2HC);
    NOA_INSTANTIATE_EXTRACT_(cfloat_t, Remap::HC2H);
    NOA_INSTANTIATE_EXTRACT_NO_TEXTURE_(double, Remap::HC2HC);
    NOA_INSTANTIATE_EXTRACT_NO_TEXTURE_(double, Remap::HC2H);
    NOA_INSTANTIATE_EXTRACT_NO_TEXTURE_(cdouble_t, Remap::HC2HC);
    NOA_INSTANTIATE_EXTRACT_NO_TEXTURE_(cdouble_t, Remap::HC2H);
}
