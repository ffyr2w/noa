#include "hip/hip_runtime.h"
#include "noa/common/geometry/Interpolator.h"
#include "noa/gpu/cuda/Exception.h"
#include "noa/gpu/cuda/memory/Copy.h"
#include "noa/gpu/cuda/memory/PtrArray.h"
#include "noa/gpu/cuda/memory/PtrDevice.h"
#include "noa/gpu/cuda/memory/PtrTexture.h"
#include "noa/gpu/cuda/memory/PtrPinned.h"
#include "noa/gpu/cuda/utils/Atomic.cuh"
#include "noa/gpu/cuda/utils/Iwise.cuh"
#include "noa/gpu/cuda/utils/Pointers.h"
#include "noa/gpu/cuda/geometry/Interpolator.h"
#include "noa/gpu/cuda/geometry/fft/Project.h"

// The Project.h header contains the implementation, which includes
// a call to cuda's atomicAdd. For now, leave it like this, but make
// sure the Atomic.cub header is included before this point.
#include "noa/common/geometry/details/FourierProjections.h"

namespace {
    using namespace ::noa;

    template<typename T>
    using matrixOrRawConstPtr_t = std::conditional_t<
            traits::is_floatXX_v<T>,
            traits::remove_ref_cv_t<T>,
            const traits::element_type_t<T>*>;

    template<bool ASSERT_VALID_PTR, typename T>
    auto matrixOrRawConstPtr_(T v) {
        using output_t = matrixOrRawConstPtr_t<T>;
        if constexpr (traits::is_floatXX_v<T>) {
            return output_t(v);
        } else {
            NOA_ASSERT(!ASSERT_VALID_PTR || v.get() != nullptr);
            return static_cast<output_t>(v.get());
        }
    }

    template<bool ASSERT_VALID_PTR, typename matrix_wrapper_t, typename matrix_value_t>
    auto matrixOrRawConstPtrOnDevice_(matrix_wrapper_t matrices, size_t count,
                                      cuda::memory::PtrDevice<matrix_value_t>& buffer, cuda::Stream& stream) {
        using output_t = matrixOrRawConstPtr_t<matrix_wrapper_t>;
        if constexpr (traits::is_floatXX_v<matrix_wrapper_t>) {
            return output_t(matrices);
        } else {
            NOA_ASSERT(!ASSERT_VALID_PTR || matrices.get() != nullptr);
            return output_t(cuda::utils::ensureDeviceAccess(matrices.get(), stream, buffer, count));
        }
    }

    template<typename matrix_wrapper_t, typename matrix_value_t>
    auto inverseMatrices_(matrix_wrapper_t matrices, size_t count,
                          cuda::memory::PtrPinned<matrix_value_t>& buffer) {
        if constexpr (traits::is_floatXX_v<matrix_wrapper_t>) {
            return math::inverse(matrices);
        } else {
            NOA_ASSERT(count == 0 || cuda::utils::hostPointer(matrices) != nullptr);
            buffer = cuda::memory::PtrPinned<matrix_value_t>(count);
            for (size_t i = 0; i < count; ++i)
                buffer[i] = math::inverse(matrices[i]);
            using output_t = const matrix_value_t*;
            return output_t(buffer.get());
        }
    }

    template<fft::Remap REMAP, typename interpolator_t, typename data_t, typename scale_t, typename rotate_t>
    void insert3D_(interpolator_t slice_interpolator, dim4_t slice_shape,
                   data_t* grid, dim4_t grid_strides, dim4_t grid_shape,
                   const scale_t& inv_scaling_matrices,
                   const rotate_t& fwd_rotation_matrices,
                   float cutoff, dim4_t target_shape, float2_t ews_radius,
                   float slice_z_radius, cuda::Stream& stream) {

        const auto slice_count = static_cast<size_t>(slice_shape[0]);
        const auto grid_strides_3d = safe_cast<uint3_t>(dim3_t{grid_strides[1], grid_strides[2], grid_strides[3]});
        const auto grid_accessor = AccessorRestrict<data_t, 3, uint32_t>(grid, grid_strides_3d);
        const auto iwise_shape = safe_cast<int3_t>(dim3_t{grid_shape.get(1)}).fft();

        const auto apply_ews = any(ews_radius != 0);
        const bool apply_scale = inv_scaling_matrices != scale_t{};

        cuda::memory::PtrPinned<float22_t> fwd_scaling_matrices_buffer;
        cuda::memory::PtrPinned<float33_t> inv_rotation_matrices_buffer;
        const auto fwd_scaling_matrices = inverseMatrices_(
                matrixOrRawConstPtr_<true>(inv_scaling_matrices), slice_count, fwd_scaling_matrices_buffer);
        const auto inv_rotation_matrices = inverseMatrices_(
                matrixOrRawConstPtr_<true>(fwd_rotation_matrices), slice_count, inv_rotation_matrices_buffer);

        using namespace noa::geometry::fft::details;
        if (apply_ews && apply_scale) {
            const auto functor = fourierInsertionExplicitThickness<REMAP, int32_t>(
                    slice_interpolator, slice_shape, grid_accessor, grid_shape,
                    fwd_scaling_matrices, inv_rotation_matrices,
                    cutoff, target_shape, ews_radius, slice_z_radius);
            cuda::utils::iwise3D("geometry::fft::insert3D", iwise_shape, functor, stream);
        } else if (apply_ews) {
            const auto functor = fourierInsertionExplicitThickness<REMAP, int32_t>(
                    slice_interpolator, slice_shape, grid_accessor, grid_shape,
                    empty_t{}, inv_rotation_matrices,
                    cutoff, target_shape, ews_radius, slice_z_radius);
            cuda::utils::iwise3D("geometry::fft::insert3D", iwise_shape, functor, stream);
        } else if (apply_scale) {
            const auto functor = fourierInsertionExplicitThickness<REMAP, int32_t>(
                    slice_interpolator, slice_shape, grid_accessor, grid_shape,
                    fwd_scaling_matrices, inv_rotation_matrices,
                    cutoff, target_shape, empty_t{}, slice_z_radius);
            cuda::utils::iwise3D("geometry::fft::insert3D", iwise_shape, functor, stream);
        } else {
            const auto functor = fourierInsertionExplicitThickness<REMAP, int32_t>(
                    slice_interpolator, slice_shape, grid_accessor, grid_shape,
                    empty_t{}, inv_rotation_matrices,
                    cutoff, target_shape, empty_t{}, slice_z_radius);
            cuda::utils::iwise3D("geometry::fft::insert3D", iwise_shape, functor, stream);
        }

        if constexpr (!traits::is_floatXX_v<scale_t>)
            stream.attach(inv_scaling_matrices, fwd_scaling_matrices_buffer.share());
        if constexpr (!traits::is_floatXX_v<rotate_t>)
            stream.attach(fwd_rotation_matrices, inv_rotation_matrices_buffer.share());
    }


    template<fft::Remap REMAP, typename data_t, typename interpolator_t, typename scale_t, typename rotate_t>
    void extract3D_(interpolator_t grid, dim4_t grid_shape,
                    data_t* slice, dim4_t slice_strides, dim4_t slice_shape,
                    const scale_t& inv_scaling_matrices,
                    const rotate_t& fwd_rotation_matrices,
                    float cutoff, dim4_t target_shape, float2_t ews_radius, cuda::Stream& stream) {

        const auto slice_count = static_cast<size_t>(slice_shape[0]);
        const auto iwise_shape = safe_cast<int3_t>(dim3_t{slice_shape[0], slice_shape[2], slice_shape[3]}).fft();
        const auto slice_strides_3d = safe_cast<uint3_t>(dim3_t{slice_strides[0], slice_strides[2], slice_strides[3]});
        const auto slice_accessor = AccessorRestrict<data_t, 3, uint32_t>(slice, slice_strides_3d);

        const auto apply_ews = any(ews_radius != 0);
        const bool apply_scale = inv_scaling_matrices != scale_t{};

        // Ensure transformation parameters are accessible to the GPU:
        cuda::memory::PtrDevice<float22_t> inv_scaling_matrices_buffer;
        cuda::memory::PtrDevice<float33_t> fwd_rotation_matrices_buffer;
        const auto inv_scaling_matrices_ = matrixOrRawConstPtrOnDevice_<false>(
                inv_scaling_matrices, slice_count, inv_scaling_matrices_buffer, stream);
        const auto fwd_rotation_matrices_ = matrixOrRawConstPtrOnDevice_<true>(
                fwd_rotation_matrices, slice_count, fwd_rotation_matrices_buffer, stream);

        using namespace noa::geometry::fft::details;
        if (apply_ews && apply_scale) {
            const auto functor = fourierExtraction<REMAP, int32_t>(
                    grid, grid_shape, slice_accessor, slice_shape,
                    inv_scaling_matrices_, fwd_rotation_matrices_,
                    cutoff, target_shape, ews_radius);
            cuda::utils::iwise3D("geometry::fft::extract3D", iwise_shape, functor, stream);
        } else if (apply_ews) {
            const auto functor = fourierExtraction<REMAP, int32_t>(
                    grid, grid_shape, slice_accessor, slice_shape,
                    empty_t{}, fwd_rotation_matrices_,
                    cutoff, target_shape, ews_radius);
            cuda::utils::iwise3D("geometry::fft::extract3D", iwise_shape, functor, stream);
        } else if (apply_scale) {
            const auto functor = fourierExtraction<REMAP, int32_t>(
                    grid, grid_shape, slice_accessor, slice_shape,
                    inv_scaling_matrices_, fwd_rotation_matrices_,
                    cutoff, target_shape, empty_t{});
            cuda::utils::iwise3D("geometry::fft::extract3D", iwise_shape, functor, stream);
        } else {
            const auto functor = fourierExtraction<REMAP, int32_t>(
                    grid, grid_shape, slice_accessor, slice_shape,
                    empty_t{}, fwd_rotation_matrices_,
                    cutoff, target_shape, empty_t{});
            cuda::utils::iwise3D("geometry::fft::extract3D", iwise_shape, functor, stream);
        }

        if constexpr (!traits::is_floatXX_v<scale_t>)
            stream.attach(inv_scaling_matrices);
        if constexpr (!traits::is_floatXX_v<rotate_t>)
            stream.attach(fwd_rotation_matrices);
    }


    template<fft::Remap REMAP, typename interpolator_t, typename data_t,
             typename scale0_t, typename scale1_t, typename rotate0_t, typename rotate1_t>
    void extract3D_(interpolator_t input_slice_interpolator, dim4_t input_slice_shape,
                    data_t* output_slice, dim4_t output_slice_strides, dim4_t output_slice_shape,
                    const scale0_t& insert_inv_scaling_matrices, const rotate0_t& insert_fwd_rotation_matrices,
                    const scale1_t& extract_inv_scaling_matrices, const rotate1_t& extract_fwd_rotation_matrices,
                    float cutoff, float2_t ews_radius, float slice_z_radius, cuda::Stream& stream) {

        const auto output_slice_strides_2d = safe_cast<uint3_t>(
                dim3_t{output_slice_strides[0], output_slice_strides[2], output_slice_strides[3]});
        const auto output_slice_accessor = AccessorRestrict<data_t, 3, uint32_t>(
                output_slice, output_slice_strides_2d);
        const auto iwise_shape = safe_cast<int3_t>(
                dim3_t{output_slice_shape[0], output_slice_shape[2], output_slice_shape[3]}).fft();

        const auto apply_ews = any(ews_radius != 0);
        const bool apply_scale = insert_inv_scaling_matrices != scale0_t{};

        // The transformation for the insertion needs to be inverted.
        cuda::memory::PtrPinned<float22_t> insert_fwd_scaling_matrices_buffer;
        cuda::memory::PtrPinned<float33_t> insert_inv_rotation_matrices_buffer;
        const auto insert_fwd_scaling_matrices = inverseMatrices_(
                matrixOrRawConstPtr_<false>(insert_inv_scaling_matrices),
                input_slice_shape[0], insert_fwd_scaling_matrices_buffer);
        const auto insert_inv_rotation_matrices = inverseMatrices_(
                matrixOrRawConstPtr_<true>(insert_fwd_rotation_matrices),
                input_slice_shape[0], insert_inv_rotation_matrices_buffer);

        const auto extract_inv_scaling_matrices_ = matrixOrRawConstPtr_<false>(extract_inv_scaling_matrices);
        const auto extract_fwd_rotation_matrices_ = matrixOrRawConstPtr_<true>(extract_fwd_rotation_matrices);

        using namespace noa::geometry::fft::details;
        if (apply_ews && apply_scale) {
            const auto functor = fourierInsertExtraction<REMAP, int32_t>(
                    input_slice_interpolator, input_slice_shape,
                    output_slice_accessor, output_slice_shape,
                    insert_fwd_scaling_matrices, insert_inv_rotation_matrices,
                    extract_inv_scaling_matrices_, extract_fwd_rotation_matrices_,
                    cutoff, ews_radius, slice_z_radius);
            cuda::utils::iwise3D("geometry::fft::extract3D", iwise_shape, functor, stream);
        } else if (apply_ews) {
            const auto functor = fourierInsertExtraction<REMAP, int32_t>(
                    input_slice_interpolator, input_slice_shape,
                    output_slice_accessor, output_slice_shape,
                    empty_t{}, insert_inv_rotation_matrices,
                    extract_inv_scaling_matrices_, extract_fwd_rotation_matrices_,
                    cutoff, ews_radius, slice_z_radius);
            cuda::utils::iwise3D("geometry::fft::extract3D", iwise_shape, functor, stream);
        } else if (apply_scale) {
            const auto functor = fourierInsertExtraction<REMAP, int32_t>(
                    input_slice_interpolator, input_slice_shape,
                    output_slice_accessor, output_slice_shape,
                    insert_fwd_scaling_matrices, insert_inv_rotation_matrices,
                    extract_inv_scaling_matrices_, extract_fwd_rotation_matrices_,
                    cutoff, empty_t{}, slice_z_radius);
            cuda::utils::iwise3D("geometry::fft::extract3D", iwise_shape, functor, stream);
        } else {
            const auto functor = fourierInsertExtraction<REMAP, int32_t>(
                    input_slice_interpolator, input_slice_shape,
                    output_slice_accessor, output_slice_shape,
                    empty_t{}, insert_inv_rotation_matrices,
                    extract_inv_scaling_matrices_, extract_fwd_rotation_matrices_,
                    cutoff, empty_t{}, slice_z_radius);
            cuda::utils::iwise3D("geometry::fft::extract3D", iwise_shape, functor, stream);
        }
    }
}

namespace noa::cuda::geometry::fft {
    template<Remap REMAP, typename data_t, typename scale_t, typename rotate_t, typename>
    void insert3D(const shared_t<data_t[]>& slice, dim4_t slice_strides, dim4_t slice_shape,
                  const shared_t<data_t[]>& grid, dim4_t grid_strides, dim4_t grid_shape,
                  const scale_t& inv_scaling_matrices, const rotate_t& fwd_rotation_matrices,
                  float cutoff, dim4_t target_shape, float2_t ews_radius, Stream& stream) {

        NOA_ASSERT_DEVICE_PTR(slice.get(), stream.device());
        NOA_ASSERT_DEVICE_PTR(grid.get(), stream.device());

        const auto slice_strides_3d = safe_cast<uint3_t>(dim3_t{slice_strides[0], slice_strides[2], slice_strides[3]});
        const auto grid_strides_3d = safe_cast<uint3_t>(dim3_t(grid_strides.get(1)));
        const auto slice_accessor = AccessorRestrict<const data_t, 3, uint32_t>(slice.get(), slice_strides_3d);
        const auto grid_accessor = AccessorRestrict<data_t, 3, uint32_t>(grid.get(), grid_strides_3d);
        const auto iwise_shape = safe_cast<int3_t>(dim3_t{slice_shape[0], slice_shape[2], slice_shape[3]}).fft();

        const auto apply_ews = any(ews_radius != 0);
        const bool apply_scale = inv_scaling_matrices != scale_t{};

        // Ensure transformation parameters are accessible to the GPU:
        memory::PtrDevice<float22_t> inv_scaling_matrices_buffer;
        memory::PtrDevice<float33_t> fwd_rotation_matrices_buffer;
        const auto inv_scaling_matrices_ = matrixOrRawConstPtrOnDevice_<false>(
                inv_scaling_matrices, iwise_shape[0], inv_scaling_matrices_buffer, stream);
        const auto fwd_rotation_matrices_ = matrixOrRawConstPtrOnDevice_<true>(
                fwd_rotation_matrices, iwise_shape[0], fwd_rotation_matrices_buffer, stream);

        using namespace noa::geometry::fft::details;
        if (apply_ews && apply_scale) {
            const auto functor = fourierInsertionByGridding<REMAP, int32_t>(
                    slice_accessor, slice_shape, grid_accessor, grid_shape,
                    inv_scaling_matrices_, fwd_rotation_matrices_,
                    cutoff, target_shape, ews_radius);
            utils::iwise3D("geometry::fft::insert3D", iwise_shape, functor, stream);
        } else if (apply_ews) {
            const auto functor = fourierInsertionByGridding<REMAP, int32_t>(
                    slice_accessor, slice_shape, grid_accessor, grid_shape,
                    empty_t{}, fwd_rotation_matrices_,
                    cutoff, target_shape, ews_radius);
            utils::iwise3D("geometry::fft::insert3D", iwise_shape, functor, stream);
        } else if (apply_scale) {
            const auto functor = fourierInsertionByGridding<REMAP, int32_t>(
                    slice_accessor, slice_shape, grid_accessor, grid_shape,
                    inv_scaling_matrices_, fwd_rotation_matrices_,
                    cutoff, target_shape, empty_t{});
            utils::iwise3D("geometry::fft::insert3D", iwise_shape, functor, stream);
        } else {
            const auto functor = fourierInsertionByGridding<REMAP, int32_t>(
                    slice_accessor, slice_shape, grid_accessor, grid_shape,
                    empty_t{}, fwd_rotation_matrices_,
                    cutoff, target_shape, empty_t{});
            utils::iwise3D("geometry::fft::insert3D", iwise_shape, functor, stream);
        }

        stream.attach(slice, grid);
        if constexpr (!traits::is_floatXX_v<scale_t>)
            if (inv_scaling_matrices)
                stream.attach(inv_scaling_matrices);
        if constexpr (!traits::is_floatXX_v<rotate_t>)
            stream.attach(fwd_rotation_matrices);
    }

    template<Remap REMAP, typename data_t, typename scale_t, typename rotate_t, typename>
    void insert3D(const shared_t<data_t[]>& slice, dim4_t slice_strides, dim4_t slice_shape,
                  const shared_t<data_t[]>& grid, dim4_t grid_strides, dim4_t grid_shape,
                  const scale_t& inv_scaling_matrices, const rotate_t& fwd_rotation_matrices,
                  float cutoff, dim4_t target_shape, float2_t ews_radius,
                  float slice_z_radius, bool use_texture, Stream& stream) {
        NOA_ASSERT_DEVICE_PTR(grid.get(), stream.device());

        if (use_texture) {
            if constexpr (traits::is_any_v<data_t, double, cdouble_t>) {
                NOA_THROW("Double precision is not supported in this mode. Use use_texture=false instead");
            } else {
                // Be conservative on the memory that is allocated, do one slice at a time.
                // Users can use the overload with the texture anyway, so they still have
                // the choice to batch everything.
                dim_t slice_count = slice_shape[0];
                if (slice_strides[0] == 0)
                    slice_count = 1;
                memory::PtrArray<data_t> array({1, 1, slice_shape[2], slice_shape[3]}, hipArrayLayered);
                memory::PtrTexture texture(array.get(), INTERP_LINEAR, BORDER_ZERO);
                using interpolator_t = cuda::geometry::Interpolator2D<INTERP_LINEAR, data_t, false, true>;
                const auto slice_interpolator = interpolator_t(texture.get());

                for (dim_t i = 0; i < slice_count; ++i) {
                    memory::copy(slice.get() + slice_strides[0] * i, slice_strides,
                                 array.get(), array.shape(), stream);
                    insert3D_<REMAP>(slice_interpolator, slice_shape, grid.get(), grid_strides, grid_shape,
                                     inv_scaling_matrices, fwd_rotation_matrices, cutoff, target_shape, ews_radius,
                                     slice_z_radius, stream);
                }
                stream.attach(slice, grid, array.share(), texture.share());
            }
        } else {
            NOA_ASSERT_DEVICE_PTR(slice.get(), stream.device());
            const auto slice_shape_2d = safe_cast<int2_t>(dim2_t(slice_shape.get(2)));
            const auto slice_strides_3d = safe_cast<uint3_t>(dim3_t{slice_strides[0], slice_strides[2], slice_strides[3]});
            const auto slice_accessor = AccessorRestrict<const data_t, 3, uint32_t>(slice.get(), slice_strides_3d);
            const auto slice_interpolator = noa::geometry::interpolator2D<BORDER_ZERO, INTERP_LINEAR>(
                    slice_accessor, slice_shape_2d.fft(), data_t{0});

            insert3D_<REMAP>(slice_interpolator, slice_shape, grid.get(), grid_strides, grid_shape,
                             inv_scaling_matrices, fwd_rotation_matrices, cutoff, target_shape, ews_radius,
                             slice_z_radius, stream);
            stream.attach(slice, grid);
        }
    }

    template<Remap REMAP, typename data_t, typename scale_t, typename rotate_t, typename>
    void insert3D(const shared_t<hipArray>& array,
                  const shared_t<hipTextureObject_t>& slice, InterpMode slice_interpolation_mode, dim4_t slice_shape,
                  const shared_t<data_t[]>& grid, dim4_t grid_strides, dim4_t grid_shape,
                  const scale_t& inv_scaling_matrices, const rotate_t& fwd_rotation_matrices,
                  float cutoff, dim4_t target_shape, float2_t ews_radius,
                  float slice_z_radius, Stream& stream) {

        // Input texture requirements:
        constexpr bool NORMALIZED = false;
        constexpr bool LAYERED = true;
        NOA_ASSERT(memory::PtrTexture::array(*slice) == array.get());

        if (slice_interpolation_mode == INTERP_LINEAR) {
            using interpolator_t = cuda::geometry::Interpolator2D<INTERP_LINEAR, data_t, NORMALIZED, LAYERED>;
            const auto slice_interpolator = interpolator_t(*slice);
            insert3D_<REMAP>(slice_interpolator, slice_shape, grid.get(), grid_strides, grid_shape,
                             inv_scaling_matrices, fwd_rotation_matrices, cutoff, target_shape, ews_radius,
                             slice_z_radius, stream);
        } else if (slice_interpolation_mode == INTERP_LINEAR_FAST) {
            using interpolator_t = cuda::geometry::Interpolator2D<INTERP_LINEAR_FAST, data_t, NORMALIZED, LAYERED>;
            const auto slice_interpolator = interpolator_t(*slice);
            insert3D_<REMAP>(slice_interpolator, slice_shape, grid.get(), grid_strides, grid_shape,
                             inv_scaling_matrices, fwd_rotation_matrices, cutoff, target_shape, ews_radius,
                             slice_z_radius, stream);
        } else {
            NOA_THROW("The interpolation mode should be {} or {}, got {}",
                      INTERP_LINEAR, INTERP_LINEAR_FAST, slice_interpolation_mode);
        }
        stream.attach(array, grid, slice);
    }

    template<Remap REMAP, typename data_t, typename scale_t, typename rotate_t, typename>
    void extract3D(const shared_t<data_t[]>& grid, dim4_t grid_strides, dim4_t grid_shape,
                   const shared_t<data_t[]>& slice, dim4_t slice_strides, dim4_t slice_shape,
                   const scale_t& inv_scaling_matrices, const rotate_t& fwd_rotation_matrices,
                   float cutoff, dim4_t target_shape, float2_t ews_radius,
                   bool use_texture, Stream& stream) {
        NOA_ASSERT_DEVICE_PTR(slice.get(), stream.device());
        NOA_ASSERT_DEVICE_PTR(grid.get(), stream.device());

        if (use_texture) {
            if constexpr (traits::is_any_v<data_t, double, cdouble_t>) {
                NOA_THROW("Double precision is not supported in this mode. Use use_texture=false instead");
            } else {
                memory::PtrArray<data_t> array(grid_shape);
                memory::PtrTexture texture(array.get(), INTERP_LINEAR, BORDER_ZERO);
                using interpolator_t = cuda::geometry::Interpolator3D<INTERP_LINEAR, data_t>;
                const auto grid_interpolator = interpolator_t(texture.get());

                memory::copy(grid.get(), grid_strides, array.get(), array.shape(), stream);
                extract3D_<REMAP>(grid_interpolator, grid_shape, slice.get(), slice_strides, slice_shape,
                                  inv_scaling_matrices, fwd_rotation_matrices, cutoff, target_shape,
                                  ews_radius, stream);
                stream.attach(slice, grid, array.share(), texture.share());
            }
        } else {
            const auto grid_shape_3d = safe_cast<int3_t>(dim3_t(grid_shape.get(1)));
            const auto grid_strides_3d = safe_cast<uint3_t>(dim3_t(grid_strides.get(1)));
            const auto grid_accessor = AccessorRestrict<const data_t, 3, uint32_t>(grid.get(), grid_strides_3d);
            const auto grid_interpolator = noa::geometry::interpolator3D<BORDER_ZERO, INTERP_LINEAR>(
                    grid_accessor, grid_shape_3d.fft(), data_t{0});

            extract3D_<REMAP>(grid_interpolator, grid_shape, slice.get(), slice_strides, slice_shape,
                              inv_scaling_matrices, fwd_rotation_matrices, cutoff, target_shape,
                              ews_radius, stream);
            stream.attach(slice, grid);
        }
    }

    template<Remap REMAP, typename data_t, typename scale_t, typename rotate_t, typename>
    void extract3D(const shared_t<hipArray>& array,
                   const shared_t<hipTextureObject_t>& grid, InterpMode grid_interpolation_mode, dim4_t grid_shape,
                   const shared_t<data_t[]>& slice, dim4_t slice_strides, dim4_t slice_shape,
                   const scale_t& inv_scaling_matrices, const rotate_t& fwd_rotation_matrices,
                   float cutoff, dim4_t target_shape, float2_t ews_radius, Stream& stream) {
        NOA_ASSERT(memory::PtrTexture::array(*grid) == array.get());

        if (grid_interpolation_mode == INTERP_LINEAR) {
            using interpolator_t = cuda::geometry::Interpolator3D<INTERP_LINEAR, data_t>;
            const auto grid_interpolator = interpolator_t(*grid);
            extract3D_<REMAP>(grid_interpolator, grid_shape, slice.get(), slice_strides, slice_shape,
                              inv_scaling_matrices, fwd_rotation_matrices, cutoff, target_shape,
                              ews_radius, stream);
        } else if (grid_interpolation_mode == INTERP_LINEAR_FAST) {
            using interpolator_t = cuda::geometry::Interpolator3D<INTERP_LINEAR_FAST, data_t>;
            const auto grid_interpolator = interpolator_t(*grid);
            extract3D_<REMAP>(grid_interpolator, grid_shape, slice.get(), slice_strides, slice_shape,
                              inv_scaling_matrices, fwd_rotation_matrices, cutoff, target_shape,
                              ews_radius, stream);
        } else {
            NOA_THROW("The interpolation mode should be {} or {}, got {}",
                      INTERP_LINEAR, INTERP_LINEAR_FAST, grid_interpolation_mode);
        }
        stream.attach(array, grid, slice);
    }

    template<Remap REMAP, typename data_t,
             typename scale0_t, typename scale1_t,
             typename rotate0_t, typename rotate1_t, typename>
    void extract3D(const shared_t<data_t[]>& input_slice, dim4_t input_slice_strides, dim4_t input_slice_shape,
                   const shared_t<data_t[]>& output_slice, dim4_t output_slice_strides, dim4_t output_slice_shape,
                   const scale0_t& insert_inv_scaling_matrices, const rotate0_t& insert_fwd_rotation_matrices,
                   const scale1_t& extract_inv_scaling_matrices, const rotate1_t& extract_fwd_rotation_matrices,
                   float cutoff, float2_t ews_radius, float slice_z_radius, bool use_texture, Stream& stream) {
        NOA_ASSERT_DEVICE_PTR(output_slice.get(), stream.device());

        if (use_texture) {
            if constexpr (traits::is_any_v<data_t, double, cdouble_t>) {
                NOA_THROW("Double precision is not supported in this mode. Use use_texture=false instead");
            } else {
                // Be conservative on the memory that is allocated, do one slice at a time.
                // Users can use the overload with the texture anyway, so they still
                // have the choice to batch everything.
                dim_t input_slice_count = input_slice_shape[0];
                if (input_slice_strides[0] == 0)
                    input_slice_count = 1;
                memory::PtrArray<data_t> array({1, 1, input_slice_shape[2], input_slice_shape[3]}, hipArrayLayered);
                memory::PtrTexture texture(array.get(), INTERP_LINEAR, BORDER_ZERO);
                using interpolator_t = cuda::geometry::Interpolator2D<INTERP_LINEAR, data_t, false, true>;
                const auto input_slice_interpolator = interpolator_t(texture.get());

                for (dim_t i = 0; i < input_slice_count; ++i) {
                    memory::copy(input_slice.get() + input_slice_strides[0] * i, input_slice_strides,
                                 array.get(), array.shape(), stream);
                    extract3D_<REMAP>(input_slice_interpolator, input_slice_shape,
                                      output_slice.get(), output_slice_strides, output_slice_shape,
                                      insert_inv_scaling_matrices, insert_fwd_rotation_matrices,
                                      extract_inv_scaling_matrices, extract_fwd_rotation_matrices,
                                      cutoff, ews_radius, slice_z_radius, stream);
                }
                stream.attach(input_slice, output_slice, array.share(), texture.share());
            }
        } else {
            NOA_ASSERT_DEVICE_PTR(input_slice.get(), stream.device());
            const dim3_t input_slice_strides_2d{input_slice_strides[0], input_slice_strides[2], input_slice_strides[3]};
            const auto input_slice_accessor = AccessorRestrict<const data_t, 3, uint32_t>(
                    input_slice.get(), input_slice_strides_2d);
            const auto input_slice_interpolator = noa::geometry::interpolator2D<BORDER_ZERO, INTERP_LINEAR>(
                    input_slice_accessor, safe_cast<int2_t>(dim2_t(input_slice_shape.get(2))).fft(), data_t{0});

            extract3D_<REMAP>(input_slice_interpolator, input_slice_shape,
                              output_slice.get(), output_slice_strides, output_slice_shape,
                              insert_inv_scaling_matrices, insert_fwd_rotation_matrices,
                              extract_inv_scaling_matrices, extract_fwd_rotation_matrices,
                              cutoff, ews_radius, slice_z_radius, stream);
            stream.attach(input_slice, output_slice);
        }
    }

    template<Remap REMAP, typename data_t,
             typename scale0_t, typename scale1_t,
             typename rotate0_t, typename rotate1_t, typename>
    void extract3D(const shared_t<hipArray>& input_slice_array,
                   const shared_t<hipTextureObject_t>& input_slice_texture,
                   InterpMode input_slice_interpolation_mode, dim4_t input_slice_shape,
                   const shared_t<data_t[]>& output_slice, dim4_t output_slice_strides, dim4_t output_slice_shape,
                   const scale0_t& insert_inv_scaling_matrices, const rotate0_t& insert_fwd_rotation_matrices,
                   const scale1_t& extract_inv_scaling_matrices, const rotate1_t& extract_fwd_rotation_matrices,
                   float cutoff, float2_t ews_radius, float slice_z_radius, Stream& stream) {

        // Input texture requirements:
        constexpr bool NORMALIZED = false;
        constexpr bool LAYERED = true;
        NOA_ASSERT(memory::PtrTexture::array(*input_slice_texture) == input_slice_array.get());

        if (input_slice_interpolation_mode == INTERP_LINEAR) {
            using interpolator_t = cuda::geometry::Interpolator2D<INTERP_LINEAR, data_t, NORMALIZED, LAYERED>;
            const auto input_slice_interpolator = interpolator_t(*input_slice_texture);
            extract3D_<REMAP>(input_slice_interpolator, input_slice_shape,
                              output_slice.get(), output_slice_strides, output_slice_shape,
                              insert_inv_scaling_matrices, insert_fwd_rotation_matrices,
                              extract_inv_scaling_matrices, extract_fwd_rotation_matrices,
                              cutoff, ews_radius, slice_z_radius, stream);
        } else if (input_slice_interpolation_mode == INTERP_LINEAR_FAST) {
            using interpolator_t = cuda::geometry::Interpolator2D<INTERP_LINEAR_FAST, data_t, NORMALIZED, LAYERED>;
            const auto input_slice_interpolator = interpolator_t(*input_slice_texture);
            extract3D_<REMAP>(input_slice_interpolator, input_slice_shape,
                              output_slice.get(), output_slice_strides, output_slice_shape,
                              insert_inv_scaling_matrices, insert_fwd_rotation_matrices,
                              extract_inv_scaling_matrices, extract_fwd_rotation_matrices,
                              cutoff, ews_radius, slice_z_radius, stream);
        } else {
            NOA_THROW("The interpolation mode should be {} or {}, got {}",
                      INTERP_LINEAR, INTERP_LINEAR_FAST, input_slice_interpolation_mode);
        }
        stream.attach(input_slice_array, input_slice_texture, output_slice);
    }

    template<typename data_t, typename>
    void griddingCorrection(const shared_t<data_t[]>& input, dim4_t input_strides,
                            const shared_t<data_t[]>& output, dim4_t output_strides,
                            dim4_t shape, bool post_correction, Stream& stream) {
        NOA_ASSERT(all(shape > 0));
        NOA_ASSERT_DEVICE_PTR(input.get(), stream.device());
        NOA_ASSERT_DEVICE_PTR(output.get(), stream.device());

        const auto iwise_shape = safe_cast<uint4_t>(shape);
        const auto input_accessor = Accessor<const data_t, 4, uint32_t>(input.get(), safe_cast<uint4_t>(input_strides));
        const auto output_accessor = Accessor<data_t, 4, uint32_t>(output.get(), safe_cast<uint4_t>(output_strides));

        if (post_correction) {
            const auto kernel = noa::geometry::fft::details::griddingCorrection<true>(
                    input_accessor, output_accessor, shape);
            utils::iwise4D("geometry::fft::griddingCorrection", iwise_shape, kernel, stream);
        } else {
            const auto kernel = noa::geometry::fft::details::griddingCorrection<false>(
                    input_accessor, output_accessor, shape);
            utils::iwise4D("geometry::fft::griddingCorrection", iwise_shape, kernel, stream);
        }
        stream.attach(input, output);
    }

    #define NOA_INSTANTIATE_INSERT_(T, REMAP, S, R) \
    template void insert3D<REMAP, T, S, R, void>(   \
        const shared_t<T[]>&, dim4_t, dim4_t,       \
        const shared_t<T[]>&, dim4_t, dim4_t,       \
        const S&, const R&, float, dim4_t, float2_t, Stream&)

    #define NOA_INSTANTIATE_INSERT_THICK_(T, REMAP, S, R)                   \
    template void insert3D<REMAP, T, S, R, void>(                           \
        const shared_t<T[]>&, dim4_t, dim4_t,                               \
        const shared_t<T[]>&, dim4_t, dim4_t,                               \
        const S&, const R&, float, dim4_t, float2_t, float, bool, Stream&)

    #define NOA_INSTANTIATE_EXTRACT_(T, REMAP, S, R)    \
    template void extract3D<REMAP, T, S, R, void>(      \
        const shared_t<T[]>&, dim4_t, dim4_t,           \
        const shared_t<T[]>&, dim4_t, dim4_t,           \
        const S&, const R&, float, dim4_t, float2_t, bool, Stream&)

    #define NOA_INSTANTIATE_INSERT_EXTRACT_(T, REMAP, S0, S1, R0, R1)   \
    template void extract3D<REMAP, T, S0, S1, R0, R1, void>(            \
        const shared_t<T[]>&, dim4_t, dim4_t,                           \
        const shared_t<T[]>&, dim4_t, dim4_t,                           \
        const S0&, const R0&, const S1&, const R1&, float,              \
        float2_t, float, bool, Stream&)

    #define NOA_INSTANTIATE_PROJECT_ALL_REMAP(T, S, R)      \
    NOA_INSTANTIATE_INSERT_(T, Remap::H2H, S, R);           \
    NOA_INSTANTIATE_INSERT_(T, Remap::H2HC, S, R);          \
    NOA_INSTANTIATE_INSERT_(T, Remap::HC2H, S, R);          \
    NOA_INSTANTIATE_INSERT_(T, Remap::HC2HC, S, R);         \
    NOA_INSTANTIATE_INSERT_THICK_(T, Remap::HC2H, S, R);    \
    NOA_INSTANTIATE_INSERT_THICK_(T, Remap::HC2HC, S, R);   \
    NOA_INSTANTIATE_EXTRACT_(T, Remap::HC2H, S, R);         \
    NOA_INSTANTIATE_EXTRACT_(T, Remap::HC2HC, S, R)

    #define NOA_INSTANTIATE_PROJECT_MERGE_ALL_REMAP(T, S0, S1, R0, R1)  \
    NOA_INSTANTIATE_INSERT_EXTRACT_(T, Remap::HC2H, S0, S1, R0, R1);    \
    NOA_INSTANTIATE_INSERT_EXTRACT_(T, Remap::HC2HC, S0, S1, R0, R1)

    #define NOA_INSTANTIATE_PROJECT_MERGE_ALL_SCALE(T, R0, R1)                              \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_REMAP(T, float22_t, float22_t, R0, R1);               \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_REMAP(T, shared_t<float22_t[]>, float22_t, R0, R1);   \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_REMAP(T, float22_t, shared_t<float22_t[]>, R0, R1);   \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_REMAP(T, shared_t<float22_t[]>, shared_t<float22_t[]>, R0, R1)

    #define NOA_INSTANTIATE_PROJECT_MERGE_ALL_ROTATE(T)                             \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_SCALE(T, float33_t, float33_t);               \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_SCALE(T, shared_t<float33_t[]>, float33_t);   \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_SCALE(T, float33_t, shared_t<float33_t[]>);   \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_SCALE(T, shared_t<float33_t[]>, shared_t<float33_t[]>)

    #define NOA_INSTANTIATE_PROJECT_ALL(T)                                  \
    NOA_INSTANTIATE_PROJECT_ALL_REMAP(T, float22_t, float33_t);             \
    NOA_INSTANTIATE_PROJECT_ALL_REMAP(T, shared_t<float22_t[]>, float33_t); \
    NOA_INSTANTIATE_PROJECT_ALL_REMAP(T, float22_t, shared_t<float33_t[]>); \
    NOA_INSTANTIATE_PROJECT_ALL_REMAP(T, shared_t<float22_t[]>, shared_t<float33_t[]>);\
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_ROTATE(T)

    NOA_INSTANTIATE_PROJECT_ALL(float);
    NOA_INSTANTIATE_PROJECT_ALL(cfloat_t);
    NOA_INSTANTIATE_PROJECT_ALL(double);
    NOA_INSTANTIATE_PROJECT_ALL(cdouble_t);

    #define NOA_INSTANTIATE_INSERT_THICK_TEXTURE(T, REMAP, S, R)            \
    template void insert3D<REMAP, T, S, R, void>(                           \
        const shared_t<hipArray>&,                                         \
        const shared_t<hipTextureObject_t>& slice, InterpMode, dim4_t,     \
        const shared_t<T[]>&, dim4_t, dim4_t,                               \
        const S&, const R&, float, dim4_t, float2_t, float, Stream&)

    #define NOA_INSTANTIATE_EXTRACT_TEXTURE(T, REMAP, S, R)         \
    template void extract3D<REMAP, T, S, R, void>(                  \
        const shared_t<hipArray>&,                                 \
        const shared_t<hipTextureObject_t>&, InterpMode, dim4_t,   \
        const shared_t<T[]>&, dim4_t, dim4_t,                       \
        const S&, const R&, float, dim4_t, float2_t, Stream&)

    #define NOA_INSTANTIATE_INSERT_EXTRACT_TEXTURE_(T, REMAP, S0, S1, R0, R1)   \
    template void extract3D<REMAP, T, S0, S1, R0, R1, void>(                    \
        const shared_t<hipArray>&,                                             \
        const shared_t<hipTextureObject_t>&, InterpMode, dim4_t,               \
        const shared_t<T[]>&, dim4_t, dim4_t,                                   \
        const S0&, const R0&, const S1&, const R1&, float,                      \
        float2_t, float, Stream&)

    #define NOA_INSTANTIATE_PROJECT_TEXTURE_ALL_REMAP(T, S, R)  \
    NOA_INSTANTIATE_INSERT_THICK_TEXTURE(T, Remap::HC2H, S, R); \
    NOA_INSTANTIATE_INSERT_THICK_TEXTURE(T, Remap::HC2HC, S, R);\
    NOA_INSTANTIATE_EXTRACT_TEXTURE(T, Remap::HC2H, S, R);      \
    NOA_INSTANTIATE_EXTRACT_TEXTURE(T, Remap::HC2HC, S, R)

    #define NOA_INSTANTIATE_PROJECT_MERGE_ALL_REMAP_TEXTURE(T, S0, S1, R0, R1)  \
    NOA_INSTANTIATE_INSERT_EXTRACT_TEXTURE_(T, Remap::HC2H, S0, S1, R0, R1);    \
    NOA_INSTANTIATE_INSERT_EXTRACT_TEXTURE_(T, Remap::HC2HC, S0, S1, R0, R1)

    #define NOA_INSTANTIATE_PROJECT_MERGE_ALL_SCALE_TEXTURE(T, R0, R1)                              \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_REMAP_TEXTURE(T, float22_t, float22_t, R0, R1);               \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_REMAP_TEXTURE(T, shared_t<float22_t[]>, float22_t, R0, R1);   \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_REMAP_TEXTURE(T, float22_t, shared_t<float22_t[]>, R0, R1);   \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_REMAP_TEXTURE(T, shared_t<float22_t[]>, shared_t<float22_t[]>, R0, R1)

    #define NOA_INSTANTIATE_PROJECT_MERGE_ALL_ROTATE_TEXTURE(T)                             \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_SCALE_TEXTURE(T, float33_t, float33_t);               \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_SCALE_TEXTURE(T, shared_t<float33_t[]>, float33_t);   \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_SCALE_TEXTURE(T, float33_t, shared_t<float33_t[]>);   \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_SCALE_TEXTURE(T, shared_t<float33_t[]>, shared_t<float33_t[]>)

    #define NOA_INSTANTIATE_PROJECT_TEXTURE_ALL(T)                                              \
    NOA_INSTANTIATE_PROJECT_TEXTURE_ALL_REMAP(T, float22_t, float33_t);                         \
    NOA_INSTANTIATE_PROJECT_TEXTURE_ALL_REMAP(T, shared_t<float22_t[]>, float33_t);             \
    NOA_INSTANTIATE_PROJECT_TEXTURE_ALL_REMAP(T, float22_t, shared_t<float33_t[]>);             \
    NOA_INSTANTIATE_PROJECT_TEXTURE_ALL_REMAP(T, shared_t<float22_t[]>, shared_t<float33_t[]>); \
    NOA_INSTANTIATE_PROJECT_MERGE_ALL_ROTATE_TEXTURE(T)

    NOA_INSTANTIATE_PROJECT_TEXTURE_ALL(float);
    NOA_INSTANTIATE_PROJECT_TEXTURE_ALL(cfloat_t);

    template void griddingCorrection<float, void>(const shared_t<float[]>&, dim4_t, const shared_t<float[]>&, dim4_t, dim4_t, bool, Stream&);
    template void griddingCorrection<double, void>(const shared_t<double[]>&, dim4_t, const shared_t<double[]>&, dim4_t, dim4_t, bool, Stream&);
}
