#include "hip/hip_runtime.h"
#include "noa/common/Assert.h"
#include "noa/common/Math.h"

#include "noa/gpu/cuda/Exception.h"
#include "noa/gpu/cuda/util/Pointers.h"
#include "noa/gpu/cuda/memory/PtrArray.h"
#include "noa/gpu/cuda/memory/PtrDevice.h"
#include "noa/gpu/cuda/memory/PtrTexture.h"
#include "noa/gpu/cuda/memory/Copy.h"

#include "noa/gpu/cuda/geometry/Interpolate.h"
#include "noa/gpu/cuda/geometry/Prefilter.h"
#include "noa/gpu/cuda/geometry/Shift.h"

namespace {
    using namespace ::noa;
    constexpr dim3 THREADS(16, 16);

    template<InterpMode MODE, bool NORMALIZED, typename T>
    __global__ void __launch_bounds__(THREADS.x * THREADS.y)
    shift2D_(hipTextureObject_t texture, float2_t texture_shape,
             Accessor<T, 3, uint32_t> output, uint2_t output_shape,
             const float2_t* shifts) {
        const uint3_t gid{blockIdx.z,
                          blockIdx.y * blockDim.y + threadIdx.y,
                          blockIdx.x * blockDim.x + threadIdx.x};
        if (gid[1] >= output_shape[0] || gid[2] >= output_shape[1])
            return;

        float2_t pos{gid[1], gid[2]};
        pos -= shifts[gid[0]];
        pos += 0.5f;
        if constexpr (NORMALIZED)
            pos /= texture_shape;
        else
            (void) texture_shape;

        output(gid) = cuda::geometry::tex2D<T, MODE>(texture, pos);
    }

    template<InterpMode MODE, bool NORMALIZED, typename T>
    __global__ void __launch_bounds__(THREADS.x * THREADS.y)
    shift2D_single_(hipTextureObject_t texture, float2_t texture_shape,
                    Accessor<T, 3, uint32_t> output, uint2_t output_shape,
                    float2_t shift) {
        const uint3_t gid{blockIdx.z,
                          blockIdx.y * blockDim.y + threadIdx.y,
                          blockIdx.x * blockDim.x + threadIdx.x};
        if (gid[1] >= output_shape[0] || gid[2] >= output_shape[1])
            return;

        float2_t pos{gid[1], gid[2]};
        pos -= shift;
        pos += 0.5f;
        if constexpr (NORMALIZED)
            pos /= texture_shape;
        else
            (void) texture_shape;

        output(gid) = cuda::geometry::tex2D<T, MODE>(texture, pos);
    }

    template<typename T>
    void launchShiftTexture2D_(hipTextureObject_t texture, dim2_t texture_shape,
                               InterpMode texture_interp_mode, BorderMode texture_border_mode,
                               T* output, dim4_t output_strides, dim4_t output_shape,
                               const float2_t* shifts, cuda::Stream& stream) {
        NOA_ASSERT(output_shape[1] == 1);
        const float2_t i_shape(texture_shape);
        const auto o_shape = safe_cast<uint2_t>(dim2_t(output_shape.get(2)));
        const auto o_strides = safe_cast<uint3_t>(dim3_t{output_strides[0], output_strides[2], output_strides[3]});
        const dim3 blocks(math::divideUp(o_shape[1], THREADS.x),
                          math::divideUp(o_shape[0], THREADS.y),
                          output_shape[0]);
        const cuda::LaunchConfig config{blocks, THREADS};
        const Accessor<T, 3, uint32_t> output_accessor(output, o_strides);

        cuda::memory::PtrDevice<float2_t> buffer;
        shifts = cuda::util::ensureDeviceAccess(shifts, stream, buffer, output_shape[0]);

        if (texture_border_mode == BORDER_PERIODIC || texture_border_mode == BORDER_MIRROR) {
            NOA_ASSERT(cuda::memory::PtrTexture::hasNormalizedCoordinates(texture));
            if (texture_interp_mode == INTERP_NEAREST) {
                stream.enqueue("geometry::shift2D",
                               shift2D_<INTERP_NEAREST, true, T>,
                               config, texture, i_shape, output_accessor, o_shape, shifts);
            } else if (texture_interp_mode == INTERP_LINEAR_FAST) {
                stream.enqueue("geometry::shift2D",
                               shift2D_<INTERP_LINEAR_FAST, true, T>,
                               config, texture, i_shape, output_accessor, o_shape, shifts);
            } else {
                NOA_THROW("{} is not supported with {}", texture_interp_mode, texture_border_mode);
            }
        } else {
            NOA_ASSERT(!cuda::memory::PtrTexture::hasNormalizedCoordinates(texture));
            switch (texture_interp_mode) {
                case INTERP_NEAREST:
                    return stream.enqueue("geometry::shift2D",
                                          shift2D_<INTERP_NEAREST, false, T>,
                                          config, texture, i_shape, output_accessor, o_shape, shifts);
                case INTERP_LINEAR:
                    return stream.enqueue("geometry::shift2D",
                                          shift2D_<INTERP_LINEAR, false, T>,
                                          config, texture, i_shape, output_accessor, o_shape, shifts);
                case INTERP_COSINE:
                    return stream.enqueue("geometry::shift2D",
                                          shift2D_<INTERP_COSINE, false, T>,
                                          config, texture, i_shape, output_accessor, o_shape, shifts);
                case INTERP_CUBIC:
                    return stream.enqueue("geometry::shift2D",
                                          shift2D_<INTERP_CUBIC, false, T>,
                                          config, texture, i_shape, output_accessor, o_shape, shifts);
                case INTERP_CUBIC_BSPLINE:
                    return stream.enqueue("geometry::shift2D",
                                          shift2D_<INTERP_CUBIC_BSPLINE, false, T>,
                                          config, texture, i_shape, output_accessor, o_shape, shifts);
                case INTERP_LINEAR_FAST:
                    return stream.enqueue("geometry::shift2D",
                                          shift2D_<INTERP_LINEAR_FAST, false, T>,
                                          config, texture, i_shape, output_accessor, o_shape, shifts);
                case INTERP_COSINE_FAST:
                    return stream.enqueue("geometry::shift2D",
                                          shift2D_<INTERP_COSINE_FAST, false, T>,
                                          config, texture, i_shape, output_accessor, o_shape, shifts);
                case INTERP_CUBIC_BSPLINE_FAST:
                    return stream.enqueue("geometry::shift2D",
                                          shift2D_<INTERP_CUBIC_BSPLINE_FAST, false, T>,
                                          config, texture, i_shape, output_accessor, o_shape, shifts);
                default:
                    NOA_THROW("{} is not supported", texture_interp_mode);
            }
        }
    }

    template<typename T>
    void launchShiftTextureSingle2D_(hipTextureObject_t texture, dim2_t texture_shape,
                                     InterpMode texture_interp_mode, BorderMode texture_border_mode,
                                     T* output, dim4_t output_strides, dim4_t output_shape,
                                     float2_t shift, cuda::Stream& stream) {
        NOA_ASSERT(output_shape[1] == 1);
        const float2_t i_shape(texture_shape);
        const auto o_shape = safe_cast<uint2_t>(dim2_t(output_shape.get(2)));
        const auto o_strides = safe_cast<uint3_t>(dim3_t{output_strides[0], output_strides[2], output_strides[3]});
        const dim3 blocks(math::divideUp(o_shape[1], THREADS.x),
                          math::divideUp(o_shape[0], THREADS.y),
                          output_shape[0]);
        const cuda::LaunchConfig config{blocks, THREADS};
        const Accessor<T, 3, uint32_t> output_accessor(output, o_strides);

        if (texture_border_mode == BORDER_PERIODIC || texture_border_mode == BORDER_MIRROR) {
            NOA_ASSERT(cuda::memory::PtrTexture::hasNormalizedCoordinates(texture));
            if (texture_interp_mode == INTERP_NEAREST) {
                stream.enqueue("geometry::shift2D",
                               shift2D_single_<INTERP_NEAREST, true, T>,
                               config, texture, i_shape, output_accessor, o_shape, shift);
            } else if (texture_interp_mode == INTERP_LINEAR_FAST) {
                stream.enqueue("geometry::shift2D",
                               shift2D_single_<INTERP_LINEAR_FAST, true, T>,
                               config, texture, i_shape, output_accessor, o_shape, shift);
            } else {
                NOA_THROW("{} is not supported with {}", texture_interp_mode, texture_border_mode);
            }
        } else {
            NOA_ASSERT(!cuda::memory::PtrTexture::hasNormalizedCoordinates(texture));
            switch (texture_interp_mode) {
                case INTERP_NEAREST:
                    return stream.enqueue("geometry::shift2D",
                                          shift2D_single_<INTERP_NEAREST, false, T>,
                                          config, texture, i_shape, output_accessor, o_shape, shift);
                case INTERP_LINEAR:
                    return stream.enqueue("geometry::shift2D",
                                          shift2D_single_<INTERP_LINEAR, false, T>,
                                          config, texture, i_shape, output_accessor, o_shape, shift);
                case INTERP_COSINE:
                    return stream.enqueue("geometry::shift2D",
                                          shift2D_single_<INTERP_COSINE, false, T>,
                                          config, texture, i_shape, output_accessor, o_shape, shift);
                case INTERP_CUBIC:
                    return stream.enqueue("geometry::shift2D",
                                          shift2D_single_<INTERP_CUBIC, false, T>,
                                          config, texture, i_shape, output_accessor, o_shape, shift);
                case INTERP_CUBIC_BSPLINE:
                    return stream.enqueue("geometry::shift2D",
                                          shift2D_single_<INTERP_CUBIC_BSPLINE, false, T>,
                                          config, texture, i_shape, output_accessor, o_shape, shift);
                case INTERP_LINEAR_FAST:
                    return stream.enqueue("geometry::shift2D",
                                          shift2D_single_<INTERP_LINEAR_FAST, false, T>,
                                          config, texture, i_shape, output_accessor, o_shape, shift);
                case INTERP_COSINE_FAST:
                    return stream.enqueue("geometry::shift2D",
                                          shift2D_single_<INTERP_COSINE_FAST, false, T>,
                                          config, texture, i_shape, output_accessor, o_shape, shift);
                case INTERP_CUBIC_BSPLINE_FAST:
                    return stream.enqueue("geometry::shift2D",
                                          shift2D_single_<INTERP_CUBIC_BSPLINE_FAST, false, T>,
                                          config, texture, i_shape, output_accessor, o_shape, shift);
                default:
                    NOA_THROW("{} is not supported", texture_interp_mode);
            }
        }
    }

    // NOTE: almost identical to launchTransform2D_
    template<typename T, typename U>
    void launchShift2D_(const shared_t<T[]>& input, dim4_t input_strides, dim4_t input_shape,
                        const shared_t<T[]>& output, dim4_t output_strides, dim4_t output_shape,
                        U shifts, InterpMode interp_mode, BorderMode border_mode, bool prefilter,
                        cuda::Stream& stream) {
        NOA_ASSERT(input_shape[0] == 1 || input_shape[0] == output_shape[0]);
        NOA_ASSERT(input_shape[1] == 1 && output_shape[1] == 1);

        if (input_strides[0] == 0)
            input_shape[0] = 1;

        // Prepare the input array:
        cuda::memory::PtrDevice<T> buffer;
        const T* buffer_ptr;
        dim_t buffer_pitch;
        dim_t buffer_offset;
        if (prefilter && (interp_mode == INTERP_CUBIC_BSPLINE || interp_mode == INTERP_CUBIC_BSPLINE_FAST)) {
            if (input_shape[2] != output_shape[2] || input_shape[3] != output_shape[3]) {
                buffer = cuda::memory::PtrDevice<T>(input_shape.elements(), stream);
                const dim4_t contiguous_strides = input_shape.strides();
                cuda::geometry::bspline::prefilter(input, input_strides,
                                                   buffer.share(), contiguous_strides, input_shape, stream);
                buffer_ptr = buffer.get();
                buffer_pitch = contiguous_strides[2];
                buffer_offset = contiguous_strides[0];
            } else {
                NOA_ASSERT(indexing::isContiguous(output_strides, output_shape)[3]);
                // Whether input is batched or not, since we copy to the CUDA array, we can use the output as buffer.
                cuda::geometry::bspline::prefilter(input, input_strides, output, output_strides, input_shape, stream);
                buffer_ptr = output.get();
                buffer_pitch = output_strides[2];
                buffer_offset = output_strides[0];
            }
        } else {
            NOA_ASSERT(indexing::isContiguous(input_strides, input_shape)[3]);
            buffer_ptr = input.get();
            buffer_pitch = input_strides[2];
            buffer_offset = input_strides[0];
        }

        // Broadcast input if it is not batched:
        const dim4_t o_shape{input_shape[0] > 1 ? 1 : output_shape[0],
                             output_shape[1], output_shape[2], output_shape[3]};

        // Copy to texture and launch (per input batch):
        const dim3_t shape_3d{1, input_shape[2], input_shape[3]};
        cuda::memory::PtrArray<T> array(shape_3d);
        cuda::memory::PtrTexture texture(array.get(), interp_mode, border_mode);
        for (dim_t i = 0; i < input_shape[0]; ++i) {
            cuda::memory::copy(buffer_ptr + i * buffer_offset, buffer_pitch, array.get(), shape_3d, stream);
            if constexpr (traits::is_floatX_v<U>) {
                launchShiftTextureSingle2D_(
                        texture.get(), dim2_t(input_shape.get(2)), interp_mode, border_mode,
                        output.get() + i * output_strides[0], output_strides, o_shape, shifts, stream);
            } else {
                NOA_ASSERT(shifts);
                launchShiftTexture2D_(
                        texture.get(), dim2_t(input_shape.get(2)), interp_mode, border_mode,
                        output.get() + i * output_strides[0], output_strides, o_shape, shifts.get() + i, stream);
            }
        }
        stream.attach(input, output, array.share(), texture.share());
        if constexpr (!traits::is_floatX_v<U>)
            stream.attach(shifts);
    }
}

namespace noa::cuda::geometry {
    template<typename T, typename S, typename>
    void shift2D(const shared_t<T[]>& input, dim4_t input_strides, dim4_t input_shape,
                 const shared_t<T[]>& output, dim4_t output_strides, dim4_t output_shape,
                 const S& shifts, InterpMode interp_mode, BorderMode border_mode, bool prefilter,
                 Stream& stream) {
        NOA_ASSERT(all(input_shape > 0) && all(output_shape > 0) && input);
        NOA_ASSERT_DEVICE_PTR(output.get(), stream.device());
        launchShift2D_(input, input_strides, input_shape, output, output_strides, output_shape,
                       shifts, interp_mode, border_mode, prefilter, stream);
    }

    template<typename T, typename S, typename>
    void shift2D(const shared_t<hipArray>& array,
                 const shared_t<hipTextureObject_t>& texture, dim2_t texture_shape,
                 InterpMode texture_interp_mode, BorderMode texture_border_mode,
                 const shared_t<T[]>& output, dim4_t output_strides, dim4_t output_shape,
                 const S& shifts, Stream& stream) {
        NOA_ASSERT(all(texture_shape > 0) && all(output_shape > 0) && array && texture);
        NOA_ASSERT_DEVICE_PTR(output.get(), stream.device());
        if constexpr (traits::is_floatX_v<S>) {
            launchShiftTextureSingle2D_(
                    *texture, texture_shape, texture_interp_mode, texture_border_mode,
                    output.get(), output_strides, output_shape, shifts, stream);
            stream.attach(array, texture, output);
        } else {
            NOA_ASSERT(shifts);
            launchShiftTexture2D_(
                    *texture, texture_shape, texture_interp_mode, texture_border_mode,
                    output.get(), output_strides, output_shape, shifts.get(), stream);
            stream.attach(array, texture, output, shifts);
        }
    }

    #define NOA_INSTANTIATE_SHIFT_3D_VECTOR_(T, S)                                                                                                                                                                          \
    template void shift2D<T, shared_t<S[]>, void>(const shared_t<T[]>&, dim4_t, dim4_t, const shared_t<T[]>&, dim4_t, dim4_t, const shared_t<S[]>&, InterpMode, BorderMode, bool, Stream&);                                 \
    template void shift2D<T, S, void>(const shared_t<T[]>&, dim4_t, dim4_t, const shared_t<T[]>&, dim4_t, dim4_t, const S&, InterpMode, BorderMode, bool, Stream&);                                                         \
    template void shift2D<T, shared_t<S[]>, void>(const shared_t<hipArray>&, const shared_t<hipTextureObject_t>&, dim2_t, InterpMode, BorderMode, const shared_t<T[]>&, dim4_t, dim4_t, const shared_t<S[]>&, Stream&);   \
    template void shift2D<T, S, void>(const shared_t<hipArray>&, const shared_t<hipTextureObject_t>&, dim2_t, InterpMode, BorderMode, const shared_t<T[]>&, dim4_t, dim4_t, const S&, Stream&)

    #define NOA_INSTANTIATE_SHIFT_3D_(T)           \
    NOA_INSTANTIATE_SHIFT_3D_VECTOR_(T, float2_t); \

    NOA_INSTANTIATE_SHIFT_3D_(float);
    NOA_INSTANTIATE_SHIFT_3D_(cfloat_t);
}
