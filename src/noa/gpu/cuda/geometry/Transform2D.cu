#include "noa/common/Assert.h"
#include "noa/common/Math.h"
#include "noa/common/geometry/details/LinearTransformations2D.h"

#include "noa/gpu/cuda/util/Iwise.cuh"
#include "noa/gpu/cuda/util/Pointers.h"
#include "noa/gpu/cuda/memory/Copy.h"
#include "noa/gpu/cuda/memory/PtrArray.h"
#include "noa/gpu/cuda/memory/PtrDevice.h"
#include "noa/gpu/cuda/memory/PtrTexture.h"

#include "noa/gpu/cuda/geometry/Interpolator.h"
#include "noa/gpu/cuda/geometry/Prefilter.h"
#include "noa/gpu/cuda/geometry/Transform.h"

namespace {
    using namespace ::noa;

    template<bool LAYERED, typename data_t, typename matrix_t>
    void launchTransformTexture2D_(hipTextureObject_t texture, dim4_t texture_shape,
                                   InterpMode texture_interp_mode, BorderMode texture_border_mode,
                                   data_t* output, dim4_t output_strides, dim4_t output_shape,
                                   matrix_t matrices, cuda::Stream& stream) {
        NOA_ASSERT(output_shape[1] == 1);
        NOA_ASSERT(texture_shape[1] == 1);
        NOA_ASSERT((LAYERED && texture_shape[0] == output_shape[0]) ||
                   (!LAYERED && texture_shape[0] == 1));

        const auto iwise_shape = safe_cast<uint3_t>(dim3_t{output_shape[0], output_shape[2], output_shape[3]});
        const auto output_accessor = AccessorRestrict<data_t, 3, uint32_t>(
                output, safe_cast<uint3_t>(dim3_t{output_strides[0], output_strides[2], output_strides[3]}));

        // Copy matrices to device if not available yet.
        using value_t = std::remove_cv_t<std::remove_pointer_t<matrix_t>>;
        cuda::memory::PtrDevice<value_t> buffer;
        if constexpr (std::is_pointer_v<matrix_t>)
            matrices = cuda::util::ensureDeviceAccess(matrices, stream, buffer, output_shape[0]);

        if (texture_border_mode == BORDER_PERIODIC || texture_border_mode == BORDER_MIRROR) {
            const float2_t i_shape(texture_shape.get(2));

            if (texture_interp_mode == INTERP_NEAREST) {
                using interpolator_t = cuda::geometry::Interpolator2D<INTERP_NEAREST, data_t, true, LAYERED>;
                const auto kernel = noa::geometry::details::transform2D<LAYERED, uint32_t>(
                        interpolator_t(texture, i_shape), output_accessor, matrices);
                cuda::utils::iwise3D("geometry::transform2D", iwise_shape, kernel, stream);
            } else if (texture_interp_mode == INTERP_LINEAR_FAST) {
                using interpolator_t = cuda::geometry::Interpolator2D<INTERP_LINEAR_FAST, data_t, true, LAYERED>;
                const auto kernel = noa::geometry::details::transform2D<LAYERED, uint32_t>(
                        interpolator_t(texture, i_shape), output_accessor, matrices);
                cuda::utils::iwise3D("geometry::transform2D", iwise_shape, kernel, stream);
            } else {
                NOA_THROW("{} is not supported with {}", texture_interp_mode, texture_border_mode);
            }

        } else {
            switch (texture_interp_mode) {
                case INTERP_NEAREST: {
                    using interpolator_t = cuda::geometry::Interpolator2D<INTERP_NEAREST, data_t, false, LAYERED>;
                    const auto kernel = noa::geometry::details::transform2D<LAYERED, uint32_t>(
                            interpolator_t(texture), output_accessor, matrices);
                    return cuda::utils::iwise3D("geometry::transform2D", iwise_shape, kernel, stream);
                }
                case INTERP_LINEAR: {
                    using interpolator_t = cuda::geometry::Interpolator2D<INTERP_LINEAR, data_t, false, LAYERED>;
                    const auto kernel = noa::geometry::details::transform2D<LAYERED, uint32_t>(
                            interpolator_t(texture), output_accessor, matrices);
                    return cuda::utils::iwise3D("geometry::transform2D", iwise_shape, kernel, stream);
                }
                case INTERP_COSINE: {
                    using interpolator_t = cuda::geometry::Interpolator2D<INTERP_COSINE, data_t, false, LAYERED>;
                    const auto kernel = noa::geometry::details::transform2D<LAYERED, uint32_t>(
                            interpolator_t(texture), output_accessor, matrices);
                    return cuda::utils::iwise3D("geometry::transform2D", iwise_shape, kernel, stream);
                }
                case INTERP_CUBIC: {
                    using interpolator_t = cuda::geometry::Interpolator2D<INTERP_CUBIC, data_t, false, LAYERED>;
                    const auto kernel = noa::geometry::details::transform2D<LAYERED, uint32_t>(
                            interpolator_t(texture), output_accessor, matrices);
                    return cuda::utils::iwise3D("geometry::transform2D", iwise_shape, kernel, stream);
                }
                case INTERP_CUBIC_BSPLINE: {
                    using interpolator_t = cuda::geometry::Interpolator2D<INTERP_CUBIC_BSPLINE, data_t, false, LAYERED>;
                    const auto kernel = noa::geometry::details::transform2D<LAYERED, uint32_t>(
                            interpolator_t(texture), output_accessor, matrices);
                    return cuda::utils::iwise3D("geometry::transform2D", iwise_shape, kernel, stream);
                }
                case INTERP_LINEAR_FAST: {
                    using interpolator_t = cuda::geometry::Interpolator2D<INTERP_LINEAR_FAST, data_t, false, LAYERED>;
                    const auto kernel = noa::geometry::details::transform2D<LAYERED, uint32_t>(
                            interpolator_t(texture), output_accessor, matrices);
                    return cuda::utils::iwise3D("geometry::transform2D", iwise_shape, kernel, stream);
                }
                case INTERP_COSINE_FAST: {
                    using interpolator_t = cuda::geometry::Interpolator2D<INTERP_COSINE_FAST, data_t, false, LAYERED>;
                    const auto kernel = noa::geometry::details::transform2D<LAYERED, uint32_t>(
                            interpolator_t(texture), output_accessor, matrices);
                    return cuda::utils::iwise3D("geometry::transform2D", iwise_shape, kernel, stream);
                }
                case INTERP_CUBIC_BSPLINE_FAST: {
                    using interpolator_t = cuda::geometry::Interpolator2D<INTERP_CUBIC_BSPLINE_FAST, data_t, false, LAYERED>;
                    const auto kernel = noa::geometry::details::transform2D<LAYERED, uint32_t>(
                            interpolator_t(texture), output_accessor, matrices);
                    return cuda::utils::iwise3D("geometry::transform2D", iwise_shape, kernel, stream);
                }
            }
        }
    }

    template<bool LAYERED, typename data_t>
    void launchTransformSymmetryTexture2D_(hipTextureObject_t texture, InterpMode texture_interp_mode,
                                           data_t* output, dim4_t output_strides, dim4_t output_shape,
                                           float2_t shift, float22_t matrix, const geometry::Symmetry& symmetry,
                                           float2_t center, bool normalize, cuda::Stream& stream) {
        // TODO Move symmetry matrices to constant memory?
        const dim_t count = symmetry.count();
        const float33_t* symmetry_matrices = symmetry.get();
        using unique_ptr = cuda::memory::PtrDevice<float33_t>::alloc_unique_t;
        unique_ptr d_matrices = cuda::memory::PtrDevice<float33_t>::alloc(count, stream);
        cuda::memory::copy(symmetry_matrices, d_matrices.get(), count, stream);
        const float scaling = normalize ? 1 / static_cast<float>(count + 1) : 1;

        const auto iwise_shape = safe_cast<uint3_t>(dim3_t{output_shape[0], output_shape[2], output_shape[3]});
        const auto output_accessor = AccessorRestrict<data_t, 3, uint32_t>(
                output, safe_cast<uint3_t>(dim3_t{output_strides[0], output_strides[2], output_strides[3]}));

        switch (texture_interp_mode) {
            case INTERP_NEAREST: {
                using interpolator_t = cuda::geometry::Interpolator2D<INTERP_NEAREST, data_t, false, LAYERED>;
                const auto kernel = noa::geometry::details::transformSymmetry2D<LAYERED, int32_t>(
                        interpolator_t(texture), output_accessor, shift, matrix, center,
                        d_matrices.get(), count, scaling);
                return cuda::utils::iwise3D("geometry::transform2D", iwise_shape, kernel, stream);
            }
            case INTERP_LINEAR: {
                using interpolator_t = cuda::geometry::Interpolator2D<INTERP_LINEAR, data_t, false, LAYERED>;
                const auto kernel = noa::geometry::details::transformSymmetry2D<LAYERED, int32_t>(
                        interpolator_t(texture), output_accessor, shift, matrix, center,
                        d_matrices.get(), count, scaling);
                return cuda::utils::iwise3D("geometry::transform2D", iwise_shape, kernel, stream);
            }
            case INTERP_COSINE: {
                using interpolator_t = cuda::geometry::Interpolator2D<INTERP_COSINE, data_t, false, LAYERED>;
                const auto kernel = noa::geometry::details::transformSymmetry2D<LAYERED, int32_t>(
                        interpolator_t(texture), output_accessor, shift, matrix, center,
                        d_matrices.get(), count, scaling);
                return cuda::utils::iwise3D("geometry::transform2D", iwise_shape, kernel, stream);
            }
            case INTERP_CUBIC: {
                using interpolator_t = cuda::geometry::Interpolator2D<INTERP_CUBIC, data_t, false, LAYERED>;
                const auto kernel = noa::geometry::details::transformSymmetry2D<LAYERED, int32_t>(
                        interpolator_t(texture), output_accessor, shift, matrix, center,
                        d_matrices.get(), count, scaling);
                return cuda::utils::iwise3D("geometry::transform2D", iwise_shape, kernel, stream);
            }
            case INTERP_CUBIC_BSPLINE: {
                using interpolator_t = cuda::geometry::Interpolator2D<INTERP_CUBIC_BSPLINE, data_t, false, LAYERED>;
                const auto kernel = noa::geometry::details::transformSymmetry2D<LAYERED, int32_t>(
                        interpolator_t(texture), output_accessor, shift, matrix, center,
                        d_matrices.get(), count, scaling);
                return cuda::utils::iwise3D("geometry::transform2D", iwise_shape, kernel, stream);
            }
            case INTERP_LINEAR_FAST: {
                using interpolator_t = cuda::geometry::Interpolator2D<INTERP_LINEAR_FAST, data_t, false, LAYERED>;
                const auto kernel = noa::geometry::details::transformSymmetry2D<LAYERED, int32_t>(
                        interpolator_t(texture), output_accessor, shift, matrix, center,
                        d_matrices.get(), count, scaling);
                return cuda::utils::iwise3D("geometry::transform2D", iwise_shape, kernel, stream);
            }
            case INTERP_COSINE_FAST: {
                using interpolator_t = cuda::geometry::Interpolator2D<INTERP_COSINE_FAST, data_t, false, LAYERED>;
                const auto kernel = noa::geometry::details::transformSymmetry2D<LAYERED, int32_t>(
                        interpolator_t(texture), output_accessor, shift, matrix, center,
                        d_matrices.get(), count, scaling);
                return cuda::utils::iwise3D("geometry::transform2D", iwise_shape, kernel, stream);
            }
            case INTERP_CUBIC_BSPLINE_FAST: {
                using interpolator_t = cuda::geometry::Interpolator2D<INTERP_CUBIC_BSPLINE_FAST, data_t, false, LAYERED>;
                const auto kernel = noa::geometry::details::transformSymmetry2D<LAYERED, int32_t>(
                        interpolator_t(texture), output_accessor, shift, matrix, center,
                        d_matrices.get(), count, scaling);
                return cuda::utils::iwise3D("geometry::transform2D", iwise_shape, kernel, stream);
            }
        }
    }

    template<bool LAYERED, typename data_t>
    void launchSymmetrize2D_(hipTextureObject_t texture, InterpMode texture_interp_mode,
                             data_t* output, dim4_t output_strides, dim4_t output_shape,
                             const geometry::Symmetry& symmetry, float2_t center, bool normalize,
                             cuda::Stream& stream) {
        // TODO Move symmetry matrices to constant memory?
        const dim_t count = symmetry.count();
        const float33_t* symmetry_matrices = symmetry.get();
        using unique_ptr = cuda::memory::PtrDevice<float33_t>::alloc_unique_t;
        unique_ptr d_matrices = cuda::memory::PtrDevice<float33_t>::alloc(count, stream);
        cuda::memory::copy(symmetry_matrices, d_matrices.get(), count, stream);
        const float scaling = normalize ? 1 / static_cast<float>(count + 1) : 1;

        const auto iwise_shape = safe_cast<uint3_t>(dim3_t{output_shape[0], output_shape[2], output_shape[3]});
        const auto output_accessor = AccessorRestrict<data_t, 3, uint32_t>(
                output, safe_cast<uint3_t>(dim3_t{output_strides[0], output_strides[2], output_strides[3]}));

        switch (texture_interp_mode) {
            case INTERP_NEAREST: {
                using interpolator_t = cuda::geometry::Interpolator2D<INTERP_NEAREST, data_t, false, LAYERED>;
                const auto kernel = noa::geometry::details::symmetry2D<LAYERED, int32_t>(
                        interpolator_t(texture), output_accessor, center,
                        d_matrices.get(), count, scaling);
                return cuda::utils::iwise3D("geometry::symmetry2D", iwise_shape, kernel, stream);
            }
            case INTERP_LINEAR: {
                using interpolator_t = cuda::geometry::Interpolator2D<INTERP_LINEAR, data_t, false, LAYERED>;
                const auto kernel = noa::geometry::details::symmetry2D<LAYERED, int32_t>(
                        interpolator_t(texture), output_accessor, center,
                        d_matrices.get(), count, scaling);
                return cuda::utils::iwise3D("geometry::symmetry2D", iwise_shape, kernel, stream);
            }
            case INTERP_COSINE: {
                using interpolator_t = cuda::geometry::Interpolator2D<INTERP_COSINE, data_t, false, LAYERED>;
                const auto kernel = noa::geometry::details::symmetry2D<LAYERED, int32_t>(
                        interpolator_t(texture), output_accessor, center,
                        d_matrices.get(), count, scaling);
                return cuda::utils::iwise3D("geometry::symmetry2D", iwise_shape, kernel, stream);
            }
            case INTERP_CUBIC: {
                using interpolator_t = cuda::geometry::Interpolator2D<INTERP_CUBIC, data_t, false, LAYERED>;
                const auto kernel = noa::geometry::details::symmetry2D<LAYERED, int32_t>(
                        interpolator_t(texture), output_accessor, center,
                        d_matrices.get(), count, scaling);
                return cuda::utils::iwise3D("geometry::symmetry2D", iwise_shape, kernel, stream);
            }
            case INTERP_CUBIC_BSPLINE: {
                using interpolator_t = cuda::geometry::Interpolator2D<INTERP_CUBIC_BSPLINE, data_t, false, LAYERED>;
                const auto kernel = noa::geometry::details::symmetry2D<LAYERED, int32_t>(
                        interpolator_t(texture), output_accessor, center,
                        d_matrices.get(), count, scaling);
                return cuda::utils::iwise3D("geometry::symmetry2D", iwise_shape, kernel, stream);
            }
            case INTERP_LINEAR_FAST: {
                using interpolator_t = cuda::geometry::Interpolator2D<INTERP_LINEAR_FAST, data_t, false, LAYERED>;
                const auto kernel = noa::geometry::details::symmetry2D<LAYERED, int32_t>(
                        interpolator_t(texture), output_accessor, center,
                        d_matrices.get(), count, scaling);
                return cuda::utils::iwise3D("geometry::symmetry2D", iwise_shape, kernel, stream);
            }
            case INTERP_COSINE_FAST: {
                using interpolator_t = cuda::geometry::Interpolator2D<INTERP_COSINE_FAST, data_t, false, LAYERED>;
                const auto kernel = noa::geometry::details::symmetry2D<LAYERED, int32_t>(
                        interpolator_t(texture), output_accessor, center,
                        d_matrices.get(), count, scaling);
                return cuda::utils::iwise3D("geometry::symmetry2D", iwise_shape, kernel, stream);
            }
            case INTERP_CUBIC_BSPLINE_FAST: {
                using interpolator_t = cuda::geometry::Interpolator2D<INTERP_CUBIC_BSPLINE_FAST, data_t, false, LAYERED>;
                const auto kernel = noa::geometry::details::symmetry2D<LAYERED, int32_t>(
                        interpolator_t(texture), output_accessor, center,
                        d_matrices.get(), count, scaling);
                return cuda::utils::iwise3D("geometry::symmetry2D", iwise_shape, kernel, stream);
            }
        }
    }

    // Updates the input and output shape to correctly broadcast the input.
    // Prefilter the input if needed.
    template<typename T>
    auto preprocess2D_(const shared_t<T[]>& input, dim4_t& input_strides, dim4_t& input_shape,
                      const shared_t<T[]>& output, dim4_t output_strides, dim4_t& output_shape,
                      InterpMode interp_mode, bool prefilter, cuda::Stream& stream) {

        // Be careful about the symmetry case, where the input and output shape are the same objects.
        // In this case, there's no need to update the shapes.
        if (&input_shape != &output_shape) {
            // If the output is batched, the input is allowed to either have the same number of batches
            // or have a single batch. In the later case, the single batch is used to compute all output
            // batches. The following makes sure to correctly identify whether the input is batched.
            // We update this before doing the prefilter, since it can simplify the processing.
            if (input_strides[0] == 0)
                input_shape[0] = 1;
            else if (input_shape[0] == 1)
                input_strides[0] = 0;

            // If the input is not batched, then we need to ensure that the processing loop will compute
            // one batch at a time, for both the input and the output. Otherwise, the processing loop
            // should run once, processing all output batches at the same time using the unique input batch.
            if (input_shape[0] > 1)
                output_shape[0] = 1;
        }

        shared_t<T[]> buffer;
        const T* buffer_ptr;
        dim4_t buffer_strides;
        if (prefilter && (interp_mode == INTERP_CUBIC_BSPLINE || interp_mode == INTERP_CUBIC_BSPLINE_FAST)) {
            if (input_shape[2] != output_shape[2] || input_shape[3] != output_shape[3]) {
                buffer = cuda::memory::PtrDevice<T>::alloc(input_shape.elements(), stream);
                const dim4_t contiguous_strides = input_shape.strides();
                cuda::geometry::bspline::prefilter(
                        input, input_strides, buffer, contiguous_strides, input_shape, stream);
                buffer_ptr = buffer.get();
                buffer_strides = contiguous_strides;
            } else {
                // Whether input is batched or not, since we copy to the CUDA array, we can use the output as buffer.
                cuda::geometry::bspline::prefilter(input, input_strides, output, output_strides, input_shape, stream);
                buffer_ptr = output.get();
                buffer_strides = output_strides;
            }
        } else {
            buffer_ptr = input.get();
            buffer_strides = input_strides;
        }

        return std::tuple<shared_t<T[]>, const T*, dim4_t>(buffer, buffer_ptr, buffer_strides);
    }

    template<typename T>
    auto matrixOrRawConstPtr(const T& v, size_t index = 0) {
        if constexpr (traits::is_float23_v<T> || traits::is_float33_v<T>) {
            return float23_t(v);
        } else {
            NOA_ASSERT(v != nullptr);
            using clean_t = traits::remove_ref_cv_t<T>;
            using raw_const_ptr_t = const typename clean_t::element_type*;
            return static_cast<raw_const_ptr_t>(v.get() + index);
        }
    }
}

namespace noa::cuda::geometry {
    template<typename T, typename MAT, typename>
    void transform2D(const shared_t<T[]>& input, dim4_t input_strides, dim4_t input_shape,
                     const shared_t<T[]>& output, dim4_t output_strides, dim4_t output_shape,
                     const MAT& matrices, InterpMode interp_mode, BorderMode border_mode,
                     bool prefilter, Stream& stream) {
        NOA_ASSERT(input && all(input_shape > 0) && all(output_shape > 0));
        NOA_ASSERT_DEVICE_PTR(output.get(), stream.device());
        NOA_ASSERT(input_shape[0] == 1 || input_shape[0] == output_shape[0]);
        NOA_ASSERT(input_shape[1] == 1 && output_shape[1] == 1);

        // Prepare the input array:
        auto [buffer, buffer_ptr, buffer_strides] = preprocess2D_(
                input, input_strides, input_shape,
                output, output_strides, output_shape,
                interp_mode, prefilter, stream);

        // Copy to texture and launch (per input batch):
        cuda::memory::PtrArray<T> array({1, 1, input_shape[2], input_shape[3]});
        cuda::memory::PtrTexture texture(array.get(), interp_mode, border_mode);
        for (dim_t i = 0; i < input_shape[0]; ++i) {
            cuda::memory::copy(buffer_ptr + i * buffer_strides[0],
                               buffer_strides, array.get(), array.shape(), stream);
            launchTransformTexture2D_<false>(
                    texture.get(), array.shape(), interp_mode, border_mode,
                    output.get() + i * output_strides[0], output_strides, output_shape,
                    matrixOrRawConstPtr(matrices, i), stream);
        }
        stream.attach(input, output, array.share(), texture.share());
        if constexpr (!traits::is_floatXX_v<MAT>)
            stream.attach(matrices);
    }

    template<typename T, typename MAT, typename>
    void transform2D(const shared_t<hipArray>& array,
                     const shared_t<hipTextureObject_t>& texture, dim4_t texture_shape,
                     InterpMode texture_interp_mode, BorderMode texture_border_mode,
                     const shared_t<T[]>& output, dim4_t output_strides, dim4_t output_shape,
                     const MAT& matrices, Stream& stream) {
        NOA_ASSERT(array && texture && all(texture_shape > 0) && all(output_shape > 0));
        NOA_ASSERT_DEVICE_PTR(output.get(), stream.device());
        const bool is_layered = memory::PtrArray<T>::isLayered(array.get());
        NOA_ASSERT(memory::PtrTexture::array(*texture) == array.get());

        if (is_layered) {
            launchTransformTexture2D_<true>(
                    *texture, texture_shape, texture_interp_mode, texture_border_mode,
                    output.get(), output_strides, output_shape,
                    matrixOrRawConstPtr(matrices, 0), stream);
        } else {
            launchTransformTexture2D_<false>(
                    *texture, texture_shape, texture_interp_mode, texture_border_mode,
                    output.get(), output_strides, output_shape,
                    matrixOrRawConstPtr(matrices, 0), stream);
        }

        if constexpr (traits::is_floatXX_v<MAT>)
            stream.attach(array, texture, output);
        else
            stream.attach(array, texture, output, matrices);
    }

    template<typename T, typename>
    void transform2D(const shared_t<T[]>& input, dim4_t input_strides, dim4_t input_shape,
                     const shared_t<T[]>& output, dim4_t output_strides, dim4_t output_shape,
                     float2_t shift, float22_t matrix, const Symmetry& symmetry, float2_t center,
                     InterpMode interp_mode, bool prefilter, bool normalize, Stream& stream) {
        NOA_ASSERT(input && all(input_shape > 0) && all(output_shape > 0));
        NOA_ASSERT_DEVICE_PTR(output.get(), stream.device());
        NOA_ASSERT(input_shape[0] == 1 || input_shape[0] == output_shape[0]);
        NOA_ASSERT(input_shape[1] == 1 && output_shape[1] == 1);

        // Prepare the input array:
        auto [buffer, buffer_ptr, buffer_strides] = preprocess2D_(
                input, input_strides, input_shape,
                output, output_strides, output_shape,
                interp_mode, prefilter, stream);

        // Copy to texture and launch (per input batch):
        cuda::memory::PtrArray<T> array({1, 1, input_shape[2], input_shape[3]});
        cuda::memory::PtrTexture texture(array.get(), interp_mode, BORDER_ZERO);
        for (dim_t i = 0; i < input_shape[0]; ++i) {
            cuda::memory::copy(buffer_ptr + i * buffer_strides[0],
                               buffer_strides, array.get(), array.shape(), stream);
            launchTransformSymmetryTexture2D_<false>(
                    texture.get(), interp_mode,
                    output.get() + i * output_strides[0], output_strides, output_shape,
                    shift, matrix, symmetry, center, normalize, stream);
        }
        stream.attach(input, output, symmetry.share(), array.share(), texture.share());
        if (buffer)
            stream.attach(buffer);
    }

    template<typename T, typename>
    void transform2D(const shared_t<hipArray>& array,
                     const shared_t<hipTextureObject_t>& texture,
                     InterpMode texture_interp_mode, dim4_t texture_shape,
                     const shared_t<T[]>& output, dim4_t output_strides, dim4_t output_shape,
                     float2_t shift, float22_t matrix, const Symmetry& symmetry, float2_t center,
                     bool normalize, Stream& stream) {
        NOA_ASSERT(array && texture && all(output_shape > 0));
        NOA_ASSERT_DEVICE_PTR(output.get(), stream.device());
        const bool is_layered = memory::PtrArray<T>::isLayered(array.get());
        NOA_ASSERT(memory::PtrTexture::array(*texture) == array.get());

        NOA_ASSERT(texture_shape[1] == 1);
        NOA_ASSERT((is_layered && texture_shape[0] == output_shape[0]) ||
                   (!is_layered && texture_shape[0] == 1));
        (void) texture_shape;

        if (is_layered) {
            launchTransformSymmetryTexture2D_<true>(
                    *texture, texture_interp_mode,
                    output.get(), output_strides, output_shape,
                    shift, matrix, symmetry, center, normalize, stream);
        } else {
            launchTransformSymmetryTexture2D_<false>(
                    *texture, texture_interp_mode,
                    output.get(), output_strides, output_shape,
                    shift, matrix, symmetry, center, normalize, stream);
        }
        stream.attach(array, texture, output, symmetry.share());
    }

    template<typename T, typename>
    void symmetrize2D(const shared_t<T[]>& input, dim4_t input_strides,
                      const shared_t<T[]>& output, dim4_t output_strides,
                      dim4_t shape, const Symmetry& symmetry, float2_t center,
                      InterpMode interp_mode, bool prefilter, bool normalize, Stream& stream) {
        NOA_ASSERT(all(shape > 0) && input);
        NOA_ASSERT_DEVICE_PTR(output.get(), stream.device());
        NOA_ASSERT(shape[1] == 1);

        if (!symmetry.count()) {
            if (input != output)
                memory::copy(input, input_strides, output, output_strides, shape, stream);
            return;
        }

        auto [buffer, buffer_ptr, buffer_strides] = preprocess2D_(
                input, input_strides, shape,
                output, output_strides, shape,
                interp_mode, prefilter, stream);

        // Copy to texture and launch (per input batch):
        cuda::memory::PtrArray<T> array({1, 1, shape[2], shape[3]});
        cuda::memory::PtrTexture texture(array.get(), interp_mode, BORDER_ZERO);
        for (dim_t i = 0; i < shape[0]; ++i) {
            cuda::memory::copy(buffer_ptr + i * buffer_strides[0], buffer_strides, array.get(), shape, stream);
            launchSymmetrize2D_<false>(
                    texture.get(), interp_mode,
                    output.get() + i * output_strides[0], output_strides, shape,
                    symmetry, center, normalize, stream);
        }
        stream.attach(input, output, symmetry.share(), array.share(), texture.share());
    }

    template<typename T, typename>
    void symmetrize2D(const shared_t<hipArray>& array,
                      const shared_t<hipTextureObject_t>& texture,
                      InterpMode texture_interp_mode, dim4_t texture_shape,
                      const shared_t<T[]>& output, dim4_t output_strides, dim4_t output_shape,
                      const Symmetry& symmetry, float2_t center, bool normalize, Stream& stream) {
        NOA_ASSERT(array && texture && all(output_shape > 0));
        NOA_ASSERT_DEVICE_PTR(output.get(), stream.device());
        const bool is_layered = memory::PtrArray<T>::isLayered(array.get());
        NOA_ASSERT(memory::PtrTexture::array(*texture) == array.get());

        NOA_ASSERT(texture_shape[1] == 1);
        NOA_ASSERT((is_layered && texture_shape[0] == output_shape[0]) ||
                   (!is_layered && texture_shape[0] == 1));
        (void) texture_shape;

        if (is_layered) {
            launchSymmetrize2D_<true>(
                    *texture, texture_interp_mode,
                    output.get(), output_strides, output_shape,
                    symmetry, center, normalize, stream);
        } else {
            launchSymmetrize2D_<false>(
                    *texture, texture_interp_mode,
                    output.get(), output_strides, output_shape,
                    symmetry, center, normalize, stream);
        }
        stream.attach(array, texture, output, symmetry.share());
    }

    #define NOA_INSTANTIATE_TRANSFORM_2D_MATRIX(T, M)                                                                                                                                                                           \
    template void transform2D<T, shared_t<M[]>, void>(const shared_t<T[]>&, dim4_t, dim4_t, const shared_t<T[]>&, dim4_t, dim4_t, const shared_t<M[]>&, InterpMode, BorderMode, bool, Stream&);                                 \
    template void transform2D<T, M, void>(const shared_t<T[]>&, dim4_t, dim4_t, const shared_t<T[]>&, dim4_t, dim4_t, const M&, InterpMode, BorderMode, bool, Stream&);                                                         \
    template void transform2D<T, shared_t<M[]>, void>(const shared_t<hipArray>&, const shared_t<hipTextureObject_t>&, dim4_t, InterpMode, BorderMode, const shared_t<T[]>&, dim4_t, dim4_t, const shared_t<M[]>&, Stream&);   \
    template void transform2D<T, M, void>(const shared_t<hipArray>&, const shared_t<hipTextureObject_t>&, dim4_t, InterpMode, BorderMode, const shared_t<T[]>&, dim4_t, dim4_t, const M&, Stream&)

    #define NOA_INSTANTIATE_TRANSFORM_SYM_(T)                                                                                                                                                           \
    template void transform2D<T, void>(const shared_t<T[]>&, dim4_t, dim4_t, const shared_t<T[]>&, dim4_t, dim4_t, float2_t, float22_t, const Symmetry&, float2_t, InterpMode, bool, bool, Stream&);    \
    template void transform2D<T, void>(const shared_t<hipArray>&, const shared_t<hipTextureObject_t>&, InterpMode, dim4_t, const shared_t<T[]>&, dim4_t, dim4_t, float2_t, float22_t, const Symmetry&, float2_t, bool, Stream&)

    #define NOA_INSTANTIATE_SYM_(T)                                                                                                                                         \
    template void symmetrize2D<T, void>(const shared_t<T[]>&, dim4_t, const shared_t<T[]>&, dim4_t, dim4_t, const Symmetry&, float2_t, InterpMode, bool, bool, Stream&);    \
    template void symmetrize2D<T, void>(const shared_t<hipArray>&, const shared_t<hipTextureObject_t>&, InterpMode, dim4_t, const shared_t<T[]>&, dim4_t, dim4_t, const Symmetry&, float2_t, bool, Stream&)

    #define NOA_INSTANTIATE_TRANSFORM_2D_(T)            \
    NOA_INSTANTIATE_TRANSFORM_2D_MATRIX(T, float23_t);  \
    NOA_INSTANTIATE_TRANSFORM_2D_MATRIX(T, float33_t);  \
    NOA_INSTANTIATE_TRANSFORM_SYM_(T);                  \
    NOA_INSTANTIATE_SYM_(T)

    NOA_INSTANTIATE_TRANSFORM_2D_(float);
    NOA_INSTANTIATE_TRANSFORM_2D_(cfloat_t);
}
