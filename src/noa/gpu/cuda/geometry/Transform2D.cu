#include "hip/hip_runtime.h"
#include "noa/common/Assert.h"
#include "noa/common/Math.h"

#include "noa/gpu/cuda/Exception.h"
#include "noa/gpu/cuda/Types.h"
#include "noa/gpu/cuda/util/Pointers.h"

#include "noa/gpu/cuda/memory/PtrArray.h"
#include "noa/gpu/cuda/memory/PtrDevice.h"
#include "noa/gpu/cuda/memory/PtrTexture.h"
#include "noa/gpu/cuda/memory/Copy.h"

#include "noa/gpu/cuda/geometry/Transform.h"
#include "noa/gpu/cuda/geometry/Interpolate.h"
#include "noa/gpu/cuda/geometry/Prefilter.h"

namespace {
    using namespace ::noa;
    constexpr dim3 THREADS(16, 16);

    // 2D, batched
    template<InterpMode MODE, bool NORMALIZED, typename T, typename MATRIX>
    __global__ void __launch_bounds__(THREADS.x * THREADS.y)
    transform2D_(hipTextureObject_t texture, float2_t texture_shape,
                 Accessor<T, 3, uint32_t> output, uint2_t output_shape,
                 const MATRIX* matrices) {
        const uint3_t gid{blockIdx.z,
                          blockIdx.y * blockDim.y + threadIdx.y,
                          blockIdx.x * blockDim.x + threadIdx.x};
        if (gid[1] >= output_shape[0] || gid[2] >= output_shape[1])
            return;

        const float3_t pos{gid[1], gid[2], 1.f};
        const float23_t matrix(matrices[gid[0]]);
        float2_t coordinates = matrix * pos;
        coordinates += 0.5f;
        if constexpr (NORMALIZED)
            coordinates /= texture_shape;
        else
            (void) texture_shape;

        output(gid) = cuda::geometry::tex2D<T, MODE>(texture, coordinates);
    }

    // 2D, single
    template<InterpMode MODE, bool NORMALIZED, typename T>
    __global__ void __launch_bounds__(THREADS.x * THREADS.y)
    transform2D_single_(hipTextureObject_t texture, float2_t texture_shape,
                        Accessor<T, 3, uint32_t> output, uint2_t output_shape,
                        float23_t matrix) {
        const uint3_t gid{blockIdx.z,
                          blockIdx.y * blockDim.y + threadIdx.y,
                          blockIdx.x * blockDim.x + threadIdx.x};
        if (gid[1] >= output_shape[0] || gid[2] >= output_shape[1])
            return;

        const float3_t pos{gid[1], gid[2], 1.f};
        float2_t coordinates = matrix * pos;
        coordinates += 0.5f;
        if constexpr (NORMALIZED)
            coordinates /= texture_shape;
        else
            (void) texture_shape;

        output(gid) = cuda::geometry::tex2D<T, MODE>(texture, coordinates);
    }


    template<typename T, typename MAT>
    void launchTransformTexture2D_(hipTextureObject_t texture, dim2_t texture_shape,
                                   InterpMode texture_interp_mode, BorderMode texture_border_mode,
                                   T* output, dim4_t output_strides, dim4_t output_shape,
                                   const MAT* matrices, cuda::Stream& stream) {
        NOA_ASSERT(output_shape[1] == 1);
        const float2_t i_shape(texture_shape);
        const auto o_shape = safe_cast<uint2_t>(dim2_t(output_shape.get(2)));
        const auto o_strides = safe_cast<uint3_t>(dim3_t{output_strides[0], output_strides[2], output_strides[3]});
        const dim3 blocks(math::divideUp(o_shape[1], THREADS.x),
                          math::divideUp(o_shape[0], THREADS.y),
                          output_shape[0]);
        const cuda::LaunchConfig config{blocks, THREADS};
        const Accessor<T, 3, uint32_t> output_accessor(output, o_strides);

        cuda::memory::PtrDevice<MAT> buffer;
        matrices = cuda::util::ensureDeviceAccess(matrices, stream, buffer, output_shape[0]);

        if (texture_border_mode == BORDER_PERIODIC || texture_border_mode == BORDER_MIRROR) {
            NOA_ASSERT(cuda::memory::PtrTexture::hasNormalizedCoordinates(texture));
            if (texture_interp_mode == INTERP_NEAREST) {
                stream.enqueue("geometry::transform2D",
                               transform2D_<INTERP_NEAREST, true, T, MAT>,
                               config, texture, i_shape, output_accessor, o_shape, matrices);
            } else if (texture_interp_mode == INTERP_LINEAR_FAST) {
                stream.enqueue("geometry::transform2D",
                               transform2D_<INTERP_LINEAR_FAST, true, T, MAT>,
                               config, texture, i_shape, output_accessor, o_shape, matrices);
            } else {
                NOA_THROW("{} is not supported with {}", texture_interp_mode, texture_border_mode);
            }
        } else {
            NOA_ASSERT(!cuda::memory::PtrTexture::hasNormalizedCoordinates(texture));
            switch (texture_interp_mode) {
                case INTERP_NEAREST:
                    return stream.enqueue("geometry::transform2D",
                                          transform2D_<INTERP_NEAREST, false, T, MAT>,
                                          config, texture, i_shape, output_accessor, o_shape, matrices);
                case INTERP_LINEAR:
                    return stream.enqueue("geometry::transform2D",
                                          transform2D_<INTERP_LINEAR, false, T, MAT>,
                                          config, texture, i_shape, output_accessor, o_shape, matrices);
                case INTERP_COSINE:
                    return stream.enqueue("geometry::transform2D",
                                          transform2D_<INTERP_COSINE, false, T, MAT>,
                                          config, texture, i_shape, output_accessor, o_shape, matrices);
                case INTERP_CUBIC:
                    return stream.enqueue("geometry::transform2D",
                                          transform2D_<INTERP_CUBIC, false, T, MAT>,
                                          config, texture, i_shape, output_accessor, o_shape, matrices);
                case INTERP_CUBIC_BSPLINE:
                    return stream.enqueue("geometry::transform2D",
                                          transform2D_<INTERP_CUBIC_BSPLINE, false, T, MAT>,
                                          config, texture, i_shape, output_accessor, o_shape, matrices);
                case INTERP_LINEAR_FAST:
                    return stream.enqueue("geometry::transform2D",
                                          transform2D_<INTERP_LINEAR_FAST, false, T, MAT>,
                                          config, texture, i_shape, output_accessor, o_shape, matrices);
                case INTERP_COSINE_FAST:
                    return stream.enqueue("geometry::transform2D",
                                          transform2D_<INTERP_COSINE_FAST, false, T, MAT>,
                                          config, texture, i_shape, output_accessor, o_shape, matrices);
                case INTERP_CUBIC_BSPLINE_FAST:
                    return stream.enqueue("geometry::transform2D",
                                          transform2D_<INTERP_CUBIC_BSPLINE_FAST, false, T, MAT>,
                                          config, texture, i_shape, output_accessor, o_shape, matrices);
                default:
                    NOA_THROW("{} is not supported", texture_interp_mode);
            }
        }
    }

    template<typename T, typename MAT>
    void launchTransformTextureSingle2D_(hipTextureObject_t texture, dim2_t texture_shape,
                                         InterpMode texture_interp_mode, BorderMode texture_border_mode,
                                         T* output, dim4_t output_strides, dim4_t output_shape,
                                         MAT matrix, cuda::Stream& stream) {
        NOA_ASSERT(output_shape[1] == 1);
        const float2_t i_shape(texture_shape);
        const auto o_shape = safe_cast<uint2_t>(dim2_t(output_shape.get(2)));
        const auto o_strides = safe_cast<uint3_t>(dim3_t{output_strides[0], output_strides[2], output_strides[3]});
        const dim3 blocks(math::divideUp(o_shape[1], THREADS.x),
                          math::divideUp(o_shape[0], THREADS.y),
                          output_shape[0]);
        const cuda::LaunchConfig config{blocks, THREADS};
        const Accessor<T, 3, uint32_t> output_accessor(output, o_strides);

        const float23_t matrix_(matrix);

        if (texture_border_mode == BORDER_PERIODIC || texture_border_mode == BORDER_MIRROR) {
            NOA_ASSERT(cuda::memory::PtrTexture::hasNormalizedCoordinates(texture));
            if (texture_interp_mode == INTERP_NEAREST) {
                stream.enqueue("geometry::transform2D",
                               transform2D_single_<INTERP_NEAREST, true, T>,
                               config, texture, i_shape, output_accessor, o_shape, matrix_);
            } else if (texture_interp_mode == INTERP_LINEAR_FAST) {
                stream.enqueue("geometry::transform2D",
                               transform2D_single_<INTERP_LINEAR_FAST, true, T>,
                               config, texture, i_shape, output_accessor, o_shape, matrix_);
            } else {
                NOA_THROW("{} is not supported with {}", texture_interp_mode, texture_border_mode);
            }
        } else {
            NOA_ASSERT(!cuda::memory::PtrTexture::hasNormalizedCoordinates(texture));
            switch (texture_interp_mode) {
                case INTERP_NEAREST:
                    return stream.enqueue("geometry::transform2D",
                                          transform2D_single_<INTERP_NEAREST, false, T>,
                                          config, texture, i_shape, output_accessor, o_shape, matrix_);
                case INTERP_LINEAR:
                    return stream.enqueue("geometry::transform2D",
                                          transform2D_single_<INTERP_LINEAR, false, T>,
                                          config, texture, i_shape, output_accessor, o_shape, matrix_);
                case INTERP_COSINE:
                    return stream.enqueue("geometry::transform2D",
                                          transform2D_single_<INTERP_COSINE, false, T>,
                                          config, texture, i_shape, output_accessor, o_shape, matrix_);
                case INTERP_CUBIC:
                    return stream.enqueue("geometry::transform2D",
                                          transform2D_single_<INTERP_CUBIC, false, T>,
                                          config, texture, i_shape, output_accessor, o_shape, matrix_);
                case INTERP_CUBIC_BSPLINE:
                    return stream.enqueue("geometry::transform2D",
                                          transform2D_single_<INTERP_CUBIC_BSPLINE, false, T>,
                                          config, texture, i_shape, output_accessor, o_shape, matrix_);
                case INTERP_LINEAR_FAST:
                    return stream.enqueue("geometry::transform2D",
                                          transform2D_single_<INTERP_LINEAR_FAST, false, T>,
                                          config, texture, i_shape, output_accessor, o_shape, matrix_);
                case INTERP_COSINE_FAST:
                    return stream.enqueue("geometry::transform2D",
                                          transform2D_single_<INTERP_COSINE_FAST, false, T>,
                                          config, texture, i_shape, output_accessor, o_shape, matrix_);
                case INTERP_CUBIC_BSPLINE_FAST:
                    return stream.enqueue("geometry::transform2D",
                                          transform2D_single_<INTERP_CUBIC_BSPLINE_FAST, false, T>,
                                          config, texture, i_shape, output_accessor, o_shape, matrix_);
                default:
                    NOA_THROW("{} is not supported", texture_interp_mode);
            }
        }
    }

    template<typename T, typename U>
    void launchTransform2D_(const shared_t<T[]>& input, dim4_t input_strides, dim4_t input_shape,
                            const shared_t<T[]>& output, dim4_t output_strides, dim4_t output_shape,
                            U matrices, InterpMode interp_mode, BorderMode border_mode, bool prefilter,
                            cuda::Stream& stream) {
        NOA_ASSERT(input_shape[0] == 1 || input_shape[0] == output_shape[0]);
        NOA_ASSERT(input_shape[1] == 1 && output_shape[1] == 1);

        if (input_strides[0] == 0)
            input_shape[0] = 1;

        // Prepare the input array:
        cuda::memory::PtrDevice<T> buffer;
        const T* buffer_ptr;
        dim_t buffer_pitch;
        dim_t buffer_offset;
        if (prefilter && (interp_mode == INTERP_CUBIC_BSPLINE || interp_mode == INTERP_CUBIC_BSPLINE_FAST)) {
            if (input_shape[2] != output_shape[2] || input_shape[3] != output_shape[3]) {
                buffer = cuda::memory::PtrDevice<T>(input_shape.elements(), stream);
                const dim4_t contiguous_strides = input_shape.strides();
                cuda::geometry::bspline::prefilter(input, input_strides,
                                                   buffer.share(), contiguous_strides, input_shape, stream);
                buffer_ptr = buffer.get();
                buffer_pitch = contiguous_strides[2];
                buffer_offset = contiguous_strides[0];
            } else {
                NOA_ASSERT(indexing::isContiguous(output_strides, output_shape)[3]);
                // Whether input is batched or not, since we copy to the CUDA array, we can use the output as buffer.
                cuda::geometry::bspline::prefilter(input, input_strides, output, output_strides, input_shape, stream);
                buffer_ptr = output.get();
                buffer_pitch = output_strides[2];
                buffer_offset = output_strides[0];
            }
        } else {
            NOA_ASSERT(indexing::isContiguous(input_strides, input_shape)[3]);
            buffer_ptr = input.get();
            buffer_pitch = input_strides[2];
            buffer_offset = input_strides[0];
        }

        // Broadcast input if it is not batched:
        const dim4_t o_shape{input_shape[0] > 1 ? 1 : output_shape[0],
                             output_shape[1], output_shape[2], output_shape[3]};

        // Copy to texture and launch (per input batch):
        const dim3_t shape_3d{1, input_shape[2], input_shape[3]};
        cuda::memory::PtrArray<T> array(shape_3d);
        cuda::memory::PtrTexture texture(array.get(), interp_mode, border_mode);
        for (dim_t i = 0; i < input_shape[0]; ++i) {
            cuda::memory::copy(buffer_ptr + i * buffer_offset, buffer_pitch, array.get(), shape_3d, stream);
            if constexpr (traits::is_floatXX_v<U>) {
                launchTransformTextureSingle2D_(
                        texture.get(), dim2_t(input_shape.get(2)), interp_mode, border_mode,
                        output.get() + i * output_strides[0], output_strides, o_shape,
                        matrices, stream);
            } else {
                launchTransformTexture2D_(
                        texture.get(), dim2_t(input_shape.get(2)), interp_mode, border_mode,
                        output.get() + i * output_strides[0], output_strides, o_shape,
                        matrices.get() + i, stream);
            }
        }
        stream.attach(input, output, array.share(), texture.share());
        if (!buffer.empty())
            stream.attach(buffer.share());
        if constexpr (!traits::is_floatXX_v<U>)
            stream.attach(matrices);
    }
}

namespace noa::cuda::geometry {
    template<typename T, typename MAT, typename>
    void transform2D(const shared_t<T[]>& input, dim4_t input_strides, dim4_t input_shape,
                     const shared_t<T[]>& output, dim4_t output_strides, dim4_t output_shape,
                     const MAT& matrices, InterpMode interp_mode, BorderMode border_mode,
                     bool prefilter, Stream& stream) {
        return launchTransform2D_(input, input_strides, input_shape, output, output_strides, output_shape,
                                  matrices, interp_mode, border_mode, prefilter, stream);
    }

    template<typename T, typename MAT, typename>
    void transform2D(const shared_t<hipArray>& array,
                     const shared_t<hipTextureObject_t>& texture, dim2_t texture_shape,
                     InterpMode texture_interp_mode, BorderMode texture_border_mode,
                     const shared_t<T[]>& output, dim4_t output_strides, dim4_t output_shape,
                     const MAT& matrices, Stream& stream) {
        if constexpr (traits::is_floatXX_v<MAT>) {
            launchTransformTextureSingle2D_(*texture, texture_shape, texture_interp_mode, texture_border_mode,
                                            output.get(), output_strides, output_shape,
                                            matrices, stream);
            stream.attach(array, texture, output);
        } else {
            launchTransformTexture2D_(*texture, texture_shape, texture_interp_mode, texture_border_mode,
                                      output.get(), output_strides, output_shape,
                                      matrices.get(), stream);
            stream.attach(array, texture, output, matrices);
        }
    }

    #define NOA_INSTANTIATE_TRANSFORM_2D_MATRIX(T, M)                                                                                                                                                                           \
    template void transform2D<T, shared_t<M[]>, void>(const shared_t<T[]>&, dim4_t, dim4_t, const shared_t<T[]>&, dim4_t, dim4_t, const shared_t<M[]>&, InterpMode, BorderMode, bool, Stream&);                                 \
    template void transform2D<T, M, void>(const shared_t<T[]>&, dim4_t, dim4_t, const shared_t<T[]>&, dim4_t, dim4_t, const M&, InterpMode, BorderMode, bool, Stream&);                                                         \
    template void transform2D<T, shared_t<M[]>, void>(const shared_t<hipArray>&, const shared_t<hipTextureObject_t>&, dim2_t, InterpMode, BorderMode, const shared_t<T[]>&, dim4_t, dim4_t, const shared_t<M[]>&, Stream&);   \
    template void transform2D<T, M, void>(const shared_t<hipArray>&, const shared_t<hipTextureObject_t>&, dim2_t, InterpMode, BorderMode, const shared_t<T[]>&, dim4_t, dim4_t, const M&, Stream&)

    #define NOA_INSTANTIATE_TRANSFORM_2D_(T)            \
    NOA_INSTANTIATE_TRANSFORM_2D_MATRIX(T, float23_t);  \
    NOA_INSTANTIATE_TRANSFORM_2D_MATRIX(T, float33_t)

    NOA_INSTANTIATE_TRANSFORM_2D_(float);
    NOA_INSTANTIATE_TRANSFORM_2D_(cfloat_t);
}
