#include "hip/hip_runtime.h"
#include "noa/common/Assert.h"
#include "noa/common/Profiler.h"
#include "noa/gpu/cuda/Exception.h"

#include "noa/gpu/cuda/memory/Copy.h"
#include "noa/gpu/cuda/memory/PtrArray.h"
#include "noa/gpu/cuda/memory/PtrDevice.h"
#include "noa/gpu/cuda/memory/PtrTexture.h"

#include "noa/gpu/cuda/geometry/Interpolate.h"
#include "noa/gpu/cuda/geometry/Prefilter.h"
#include "noa/gpu/cuda/geometry/Symmetry.h"

namespace {
    using namespace ::noa;
    constexpr dim3 THREADS(16, 16);

    template<typename T, InterpMode INTERP>
    __global__ void __launch_bounds__(THREADS.x * THREADS.y)
    symmetrize3D_(hipTextureObject_t texture, T* output, uint4_t output_stride, uint2_t shape,
                  const float33_t* symmetry_matrices, uint symmetry_count, float3_t center,
                  float scaling, uint blocks_x) {
        const uint2_t index = indexes(blockIdx.x, blocks_x);
        const uint4_t gid{blockIdx.z,
                          blockIdx.y,
                          index[0] * THREADS.y + threadIdx.y,
                          index[1] * THREADS.x + threadIdx.x};
        if (gid[2] >= shape[0] || gid[3] >= shape[1])
            return;

        float3_t coordinates{gid[1], gid[2], gid[3]};
        T value = cuda::geometry::tex3D<T, INTERP>(texture, coordinates + 0.5f);
        coordinates -= center;
        for (uint i = 0; i < symmetry_count; ++i) {
            float3_t i_coordinates(symmetry_matrices[i] * coordinates);
            value += cuda::geometry::tex3D<T, INTERP>(texture, i_coordinates + center + 0.5f);
        }

        output[at(gid, output_stride)] = value * scaling;
    }
}

namespace noa::cuda::geometry {
    template<bool PREFILTER, typename T>
    void symmetrize3D(const T* input, size4_t input_stride, T* output, size4_t output_stride,
                      size4_t shape, const Symmetry& symmetry, float3_t center,
                      InterpMode interp_mode, bool normalize, Stream& stream) {
        NOA_PROFILE_FUNCTION();
        NOA_ASSERT(shape[1] > 1);

        if (!symmetry.count())
            return memory::copy(input, input_stride, output, output_stride, shape, stream);

        size4_t input_shape{shape};
        if (input_stride[0] == 0)
            input_shape[0] = 1;

        // Prepare the input array:
        const T* buffer_ptr;
        size_t buffer_pitch;
        size_t buffer_offset;
        if (PREFILTER && (interp_mode == INTERP_CUBIC_BSPLINE || interp_mode == INTERP_CUBIC_BSPLINE_FAST)) {
            NOA_ASSERT(isContiguous(output_stride, shape)[3]);
            NOA_ASSERT(isContiguous(output_stride, shape)[1]);
            // Whether input is batched or not, since we copy to the CUDA array, we can use the output as buffer.
            cuda::geometry::bspline::prefilter(input, input_stride, output, output_stride, input_shape, stream);
            buffer_ptr = output;
            buffer_pitch = output_stride[2];
            buffer_offset = output_stride[0];
        } else {
            NOA_ASSERT(isContiguous(input_stride, input_shape)[3]);
            NOA_ASSERT(isContiguous(input_stride, input_shape)[1]);
            buffer_ptr = input;
            buffer_pitch = input_stride[2];
            buffer_offset = input_stride[0];
        }

        // Broadcast input if it is not batched:
        size4_t o_shape{input_shape[0] > 1 ? 1 : shape[0],
                        shape[1], shape[2], shape[3]};

        // Copy to texture and launch (per input batch):
        const size3_t shape_3d{input_shape.get() + 1};
        cuda::memory::PtrArray<T> array(shape_3d);
        cuda::memory::PtrTexture<T> texture(array.get(), interp_mode, BORDER_ZERO);
        for (size_t i = 0; i < input_shape[0]; ++i) {
            cuda::memory::copy(buffer_ptr + i * buffer_offset, buffer_pitch, array.get(), shape_3d, stream);
            cuda::geometry::symmetrize3D(
                    texture.get(), interp_mode, output + i * output_stride[0], output_stride, o_shape,
                    symmetry, center, normalize, stream);
        }
        stream.synchronize();
    }

    template<typename T>
    void symmetrize3D(hipTextureObject_t texture, InterpMode texture_interp_mode,
                      T* output, size4_t output_stride, size4_t output_shape,
                      const Symmetry& symmetry, float3_t center, bool normalize, Stream& stream) {
        NOA_PROFILE_FUNCTION();
        NOA_ASSERT(!memory::PtrTexture<T>::hasNormalizedCoordinates(texture));

        // TODO Move symmetry matrices to constant memory?
        const size_t count = symmetry.count();
        const float33_t* symmetry_matrices = symmetry.matrices();
        memory::PtrDevice<float33_t> d_matrices(count, stream);
        memory::copy(symmetry_matrices, d_matrices.get(), count, stream);
        const float scaling = normalize ? 1 / static_cast<float>(count + 1) : 1;

        const uint2_t o_shape{output_shape.get() + 2};
        const uint4_t o_stride{output_stride};
        const uint blocks_x = math::divideUp(o_shape[1], THREADS.x);
        const dim3 blocks(blocks_x * math::divideUp(o_shape[0], THREADS.y),
                          output_shape[1],
                          output_shape[0]);
        const LaunchConfig config{blocks, THREADS};

        switch (texture_interp_mode) {
            case INTERP_NEAREST:
                return stream.enqueue("geometry::symmetrize3D", symmetrize3D_<T, INTERP_NEAREST>,
                                      config, texture, output, o_stride, o_shape,
                                      d_matrices.get(), count, center, scaling, blocks_x);
            case INTERP_LINEAR:
                return stream.enqueue("geometry::symmetrize3D", symmetrize3D_<T, INTERP_LINEAR>,
                                      config, texture, output, o_stride, o_shape,
                                      d_matrices.get(), count, center, scaling, blocks_x);
            case INTERP_COSINE:
                return stream.enqueue("geometry::symmetrize3D", symmetrize3D_<T, INTERP_COSINE>,
                                      config, texture, output, o_stride, o_shape,
                                      d_matrices.get(), count, center, scaling, blocks_x);
            case INTERP_CUBIC:
                return stream.enqueue("geometry::symmetrize3D", symmetrize3D_<T, INTERP_CUBIC>,
                                      config, texture, output, o_stride, o_shape,
                                      d_matrices.get(), count, center, scaling, blocks_x);
            case INTERP_CUBIC_BSPLINE:
                return stream.enqueue("geometry::symmetrize3D", symmetrize3D_<T, INTERP_CUBIC_BSPLINE>,
                                      config, texture, output, o_stride, o_shape,
                                      d_matrices.get(), count, center, scaling, blocks_x);
            case INTERP_LINEAR_FAST:
                return stream.enqueue("geometry::symmetrize3D", symmetrize3D_<T, INTERP_LINEAR_FAST>,
                                      config, texture, output, o_stride, o_shape,
                                      d_matrices.get(), count, center, scaling, blocks_x);
            case INTERP_COSINE_FAST:
                return stream.enqueue("geometry::symmetrize3D", symmetrize3D_<T, INTERP_COSINE_FAST>,
                                      config, texture, output, o_stride, o_shape,
                                      d_matrices.get(), count, center, scaling, blocks_x);
            case INTERP_CUBIC_BSPLINE_FAST:
                return stream.enqueue("geometry::symmetrize3D", symmetrize3D_<T, INTERP_CUBIC_BSPLINE_FAST>,
                                      config, texture, output, o_stride, o_shape,
                                      d_matrices.get(), count, center, scaling, blocks_x);
            default:
                NOA_THROW("{} is not supported", texture_interp_mode);
        }
    }

    #define NOA_INSTANTIATE_TRANSFORM_SYM_(T)                                                                                           \
    template void symmetrize3D<true, T>(const T*, size4_t, T*, size4_t, size4_t, const Symmetry&, float3_t, InterpMode, bool, Stream&); \
    template void symmetrize3D<false, T>(const T*, size4_t, T*, size4_t, size4_t, const Symmetry&, float3_t, InterpMode, bool, Stream&);

    NOA_INSTANTIATE_TRANSFORM_SYM_(float);
    NOA_INSTANTIATE_TRANSFORM_SYM_(cfloat_t);
}
