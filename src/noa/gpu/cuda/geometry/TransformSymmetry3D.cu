#include "hip/hip_runtime.h"
#include "noa/common/Assert.h"
#include "noa/gpu/cuda/Exception.h"

#include "noa/gpu/cuda/memory/Copy.h"
#include "noa/gpu/cuda/memory/PtrArray.h"
#include "noa/gpu/cuda/memory/PtrDevice.h"
#include "noa/gpu/cuda/memory/PtrTexture.h"

#include "noa/gpu/cuda/geometry/Transform.h"
#include "noa/gpu/cuda/geometry/Interpolate.h"
#include "noa/gpu/cuda/geometry/Prefilter.h"

namespace {
    using namespace ::noa;
    constexpr dim3 THREADS(16, 16);

    template<typename T, InterpMode INTERP>
    __global__ void __launch_bounds__(THREADS.x * THREADS.y)
    transformWithSymmetry3D_(hipTextureObject_t texture, T* output, uint4_t output_strides, uint2_t shape,
                             float3_t shift, float33_t matrix, float3_t center,
                             const float33_t* symmetry_matrices, uint symmetry_count, float scaling, uint blocks_x) {
        const uint2_t index = indexing::indexes(blockIdx.x, blocks_x);
        const uint4_t gid{blockIdx.z,
                          blockIdx.y,
                          index[0] * THREADS.y + threadIdx.y,
                          index[1] * THREADS.x + threadIdx.x};
        if (gid[2] >= shape[0] || gid[3] >= shape[1])
            return;

        float3_t coordinates{gid[1], gid[2], gid[3]};
        coordinates -= center;
        coordinates = matrix * coordinates;
        T value = cuda::geometry::tex3D<T, INTERP>(texture, coordinates + center + shift + 0.5f);
        for (uint i = 0; i < symmetry_count; ++i) {
            const float3_t i_coordinates(symmetry_matrices[i] * coordinates);
            value += cuda::geometry::tex3D<T, INTERP>(texture, i_coordinates + center + shift + 0.5f);
        }

        output[indexing::at(gid, output_strides)] = value * scaling;
    }

    template<typename T>
    void launchTransformSymmetryTexture3D_(hipTextureObject_t texture, InterpMode texture_interp_mode,
                                           T* output, size4_t output_strides, size4_t output_shape,
                                           float3_t shift, float33_t matrix, const geometry::Symmetry& symmetry,
                                           float3_t center, bool normalize, cuda::Stream& stream) {
        NOA_ASSERT(!cuda::memory::PtrTexture::hasNormalizedCoordinates(texture));

        // TODO Move symmetry matrices to constant memory?
        const size_t count = symmetry.count();
        const float33_t* symmetry_matrices = symmetry.matrices();
        cuda::memory::PtrDevice<float33_t> d_matrices(count, stream);
        cuda::memory::copy(symmetry_matrices, d_matrices.get(), count, stream);
        const float scaling = normalize ? 1 / static_cast<float>(count + 1) : 1;

        const uint2_t o_shape(output_shape.get(2));
        const uint4_t o_strides(output_strides);
        const uint blocks_x = math::divideUp(o_shape[1], THREADS.x);
        const dim3 blocks(blocks_x * math::divideUp(o_shape[0], THREADS.y),
                          output_shape[1],
                          output_shape[0]);
        const cuda::LaunchConfig config{blocks, THREADS};

        switch (texture_interp_mode) {
            case INTERP_NEAREST:
                return stream.enqueue("geometry::transform3D", transformWithSymmetry3D_<T, INTERP_NEAREST>,
                                      config, texture, output, o_strides, o_shape, shift, matrix, center,
                                      d_matrices.get(), count, scaling, blocks_x);
            case INTERP_LINEAR:
                return stream.enqueue("geometry::transform3D", transformWithSymmetry3D_<T, INTERP_LINEAR>,
                                      config, texture, output, o_strides, o_shape, shift, matrix, center,
                                      d_matrices.get(), count, scaling, blocks_x);
            case INTERP_COSINE:
                return stream.enqueue("geometry::transform3D", transformWithSymmetry3D_<T, INTERP_COSINE>,
                                      config, texture, output, o_strides, o_shape, shift, matrix, center,
                                      d_matrices.get(), count, scaling, blocks_x);
            case INTERP_CUBIC:
                return stream.enqueue("geometry::transform3D", transformWithSymmetry3D_<T, INTERP_CUBIC>,
                                      config, texture, output, o_strides, o_shape, shift, matrix, center,
                                      d_matrices.get(), count, scaling, blocks_x);
            case INTERP_CUBIC_BSPLINE:
                return stream.enqueue("geometry::transform3D", transformWithSymmetry3D_<T, INTERP_CUBIC_BSPLINE>,
                                      config, texture, output, o_strides, o_shape, shift, matrix, center,
                                      d_matrices.get(), count, scaling, blocks_x);
            case INTERP_LINEAR_FAST:
                return stream.enqueue("geometry::transform3D", transformWithSymmetry3D_<T, INTERP_LINEAR_FAST>,
                                      config, texture, output, o_strides, o_shape, shift, matrix, center,
                                      d_matrices.get(), count, scaling, blocks_x);
            case INTERP_COSINE_FAST:
                return stream.enqueue("geometry::transform3D", transformWithSymmetry3D_<T, INTERP_COSINE_FAST>,
                                      config, texture, output, o_strides, o_shape, shift, matrix, center,
                                      d_matrices.get(), count, scaling, blocks_x);
            case INTERP_CUBIC_BSPLINE_FAST:
                return stream.enqueue("geometry::transform3D", transformWithSymmetry3D_<T, INTERP_CUBIC_BSPLINE_FAST>,
                                      config, texture, output, o_strides, o_shape, shift, matrix, center,
                                      d_matrices.get(), count, scaling, blocks_x);
            default:
                NOA_THROW_FUNC("transform3D", "{} is not supported", texture_interp_mode);
        }
    }
}

namespace noa::cuda::geometry {
    template<typename T, typename>
    void transform3D(const shared_t<T[]>& input, size4_t input_strides, size4_t input_shape,
                     const shared_t<T[]>& output, size4_t output_strides, size4_t output_shape,
                     float3_t shift, float33_t matrix, const Symmetry& symmetry, float3_t center,
                     InterpMode interp_mode, bool prefilter, bool normalize, Stream& stream) {
        NOA_ASSERT(input_shape[0] == 1 || input_shape[0] == output_shape[0]);

        if (input_strides[0] == 0)
            input_shape[0] = 1;

        // Prepare the input array:
        cuda::memory::PtrDevice<T> buffer;
        const T* buffer_ptr;
        size_t buffer_pitch;
        size_t buffer_offset;
        if (prefilter && (interp_mode == INTERP_CUBIC_BSPLINE || interp_mode == INTERP_CUBIC_BSPLINE_FAST)) {
            if (input_shape[1] != output_shape[1] ||
                input_shape[2] != output_shape[2] ||
                input_shape[3] != output_shape[3]) {
                buffer = cuda::memory::PtrDevice<T>(input_shape.elements(), stream);
                const size4_t contiguous_strides = input_shape.strides();
                cuda::geometry::bspline::prefilter(input, input_strides,
                                                   buffer.share(), contiguous_strides, input_shape, stream);
                buffer_ptr = buffer.get();
                buffer_pitch = contiguous_strides[2];
                buffer_offset = contiguous_strides[0];
            } else {
                NOA_ASSERT(indexing::isContiguous(output_strides, output_shape)[3]);
                NOA_ASSERT(indexing::isContiguous(output_strides, output_shape)[1]);
                // Whether input is batched or not, since we copy to the CUDA array, we can use the output as buffer.
                cuda::geometry::bspline::prefilter(input, input_strides, output, output_strides, input_shape, stream);
                buffer_ptr = output.get();
                buffer_pitch = output_strides[2];
                buffer_offset = output_strides[0];
            }
        } else {
            NOA_ASSERT(indexing::isContiguous(input_strides, input_shape)[3]);
            NOA_ASSERT(indexing::isContiguous(input_strides, input_shape)[1]);
            buffer_ptr = input.get();
            buffer_pitch = input_strides[2];
            buffer_offset = input_strides[0];
        }

        // Broadcast input if it is not batched:
        const size4_t o_shape{input_shape[0] > 1 ? 1 : output_shape[0],
                              output_shape[1], output_shape[2], output_shape[3]};

        // Copy to texture and launch (per input batch):
        const size3_t shape_3d(input_shape.get(1));
        cuda::memory::PtrArray<T> array(shape_3d);
        cuda::memory::PtrTexture texture(array.get(), interp_mode, BORDER_ZERO);
        for (size_t i = 0; i < input_shape[0]; ++i) {
            cuda::memory::copy(buffer_ptr + i * buffer_offset, buffer_pitch, array.get(), shape_3d, stream);
            launchTransformSymmetryTexture3D_(
                    texture.get(), interp_mode, output.get() + i * output_strides[0], output_strides, o_shape,
                    shift, matrix, symmetry, center, normalize, stream);
        }
        stream.attach(input, output, array.share(), texture.share());
        if (!buffer.empty())
            stream.attach(buffer.share());
    }

    template<typename T, typename>
    void transform3D(const shared_t<hipTextureObject_t>& texture, InterpMode texture_interp_mode,
                     const shared_t<T[]>& output, size4_t output_strides, size4_t output_shape,
                     float3_t shift, float33_t matrix, const Symmetry& symmetry, float3_t center,
                     bool normalize, Stream& stream) {
        launchTransformSymmetryTexture3D_(*texture, texture_interp_mode,
                                          output.get(), output_strides, output_shape,
                                          shift, matrix, symmetry, center, normalize, stream);
        stream.attach(texture, output);
    }

    #define NOA_INSTANTIATE_TRANSFORM_SYM_(T)                                                                                                                                                               \
    template void transform3D<T, void>(const shared_t<T[]>&, size4_t, size4_t, const shared_t<T[]>&, size4_t, size4_t, float3_t, float33_t, const Symmetry&, float3_t, InterpMode, bool, bool, Stream&);    \
    template void transform3D<T, void>(const shared_t<hipTextureObject_t>&, InterpMode, const shared_t<T[]>&, size4_t, size4_t, float3_t, float33_t, const Symmetry&, float3_t, bool, Stream&)

    NOA_INSTANTIATE_TRANSFORM_SYM_(float);
    NOA_INSTANTIATE_TRANSFORM_SYM_(cfloat_t);
}
