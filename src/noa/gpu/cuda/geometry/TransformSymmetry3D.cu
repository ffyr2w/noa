#include "hip/hip_runtime.h"
#include "noa/common/Assert.h"
#include "noa/common/Profiler.h"
#include "noa/gpu/cuda/Exception.h"

#include "noa/gpu/cuda/memory/Copy.h"
#include "noa/gpu/cuda/memory/PtrArray.h"
#include "noa/gpu/cuda/memory/PtrDevice.h"
#include "noa/gpu/cuda/memory/PtrTexture.h"

#include "noa/gpu/cuda/geometry/Transform.h"
#include "noa/gpu/cuda/geometry/Interpolate.h"
#include "noa/gpu/cuda/geometry/Prefilter.h"

namespace {
    using namespace ::noa;
    constexpr dim3 THREADS(16, 16);

    template<typename T, InterpMode INTERP>
    __global__ void __launch_bounds__(THREADS.x * THREADS.y)
    transformWithSymmetry3D_(hipTextureObject_t texture, T* output, uint4_t output_stride, uint2_t shape,
                             float3_t shift, float33_t matrix, float3_t center,
                             const float33_t* symmetry_matrices, uint symmetry_count, float scaling, uint blocks_x) {
        const uint2_t index = indexing::indexes(blockIdx.x, blocks_x);
        const uint4_t gid{blockIdx.z,
                          blockIdx.y,
                          index[0] * THREADS.y + threadIdx.y,
                          index[1] * THREADS.x + threadIdx.x};
        if (gid[2] >= shape[0] || gid[3] >= shape[1])
            return;

        float3_t coordinates{gid[1], gid[2], gid[3]};
        coordinates -= center;
        coordinates = matrix * coordinates;
        T value = cuda::geometry::tex3D<T, INTERP>(texture, coordinates + center + shift + 0.5f);
        for (uint i = 0; i < symmetry_count; ++i) {
            const float3_t i_coordinates{symmetry_matrices[i] * coordinates};
            value += cuda::geometry::tex3D<T, INTERP>(texture, i_coordinates + center + shift + 0.5f);
        }

        output[indexing::at(gid, output_stride)] = value * scaling;
    }
}

namespace noa::cuda::geometry {
    template<bool PREFILTER, typename T>
    void transform3D(const shared_t<T[]>& input, size4_t input_stride, size4_t input_shape,
                     const shared_t<T[]>& output, size4_t output_stride, size4_t output_shape,
                     float3_t shift, float33_t matrix, const Symmetry& symmetry, float3_t center,
                     InterpMode interp_mode, bool normalize, Stream& stream) {
        NOA_PROFILE_FUNCTION();
        NOA_ASSERT(input_shape[0] == 1 || input_shape[0] == output_shape[0]);

        if (input_stride[0] == 0)
            input_shape[0] = 1;

        // Prepare the input array:
        cuda::memory::PtrDevice<T> buffer;
        const T* buffer_ptr;
        size_t buffer_pitch;
        size_t buffer_offset;
        if (PREFILTER && (interp_mode == INTERP_CUBIC_BSPLINE || interp_mode == INTERP_CUBIC_BSPLINE_FAST)) {
            if (input_shape[1] != output_shape[1] ||
                input_shape[2] != output_shape[2] ||
                input_shape[3] != output_shape[3]) {
                buffer = cuda::memory::PtrDevice<T>{input_shape.elements(), stream};
                const size4_t contiguous_stride = input_shape.stride();
                cuda::geometry::bspline::prefilter(input, input_stride,
                                                   buffer.share(), contiguous_stride, input_shape, stream);
                buffer_ptr = buffer.get();
                buffer_pitch = contiguous_stride[2];
                buffer_offset = contiguous_stride[0];
            } else {
                NOA_ASSERT(indexing::isContiguous(output_stride, output_shape)[3]);
                NOA_ASSERT(indexing::isContiguous(output_stride, output_shape)[1]);
                // Whether input is batched or not, since we copy to the CUDA array, we can use the output as buffer.
                cuda::geometry::bspline::prefilter(input, input_stride, output, output_stride, input_shape, stream);
                buffer_ptr = output.get();
                buffer_pitch = output_stride[2];
                buffer_offset = output_stride[0];
            }
        } else {
            NOA_ASSERT(indexing::isContiguous(input_stride, input_shape)[3]);
            NOA_ASSERT(indexing::isContiguous(input_stride, input_shape)[1]);
            buffer_ptr = input.get();
            buffer_pitch = input_stride[2];
            buffer_offset = input_stride[0];
        }

        // Broadcast input if it is not batched:
        const size4_t o_shape{input_shape[0] > 1 ? 1 : output_shape[0],
                              output_shape[1], output_shape[2], output_shape[3]};

        // Copy to texture and launch (per input batch):
        const size3_t shape_3d{input_shape.get() + 1};
        cuda::memory::PtrArray<T> array{shape_3d};
        cuda::memory::PtrTexture texture{array.get(), interp_mode, BORDER_ZERO};
        for (size_t i = 0; i < input_shape[0]; ++i) {
            cuda::memory::copy(buffer_ptr + i * buffer_offset, buffer_pitch, array.get(), shape_3d, stream);
            cuda::geometry::transform3D(
                    texture.get(), interp_mode, output.get() + i * output_stride[0], output_stride, o_shape,
                    shift, matrix, symmetry, center, normalize, stream);
        }
        stream.attach(input, output, array.share(), texture.share());
        if (!buffer.empty())
            stream.attach(buffer.share());
    }

    template<typename T>
    void transform3D(hipTextureObject_t texture, InterpMode texture_interp_mode,
                     T* output, size4_t output_stride, size4_t output_shape,
                     float3_t shift, float33_t matrix, const Symmetry& symmetry, float3_t center,
                     bool normalize, Stream& stream) {
        NOA_PROFILE_FUNCTION();
        NOA_ASSERT(!memory::PtrTexture::hasNormalizedCoordinates(texture));

        // TODO Move symmetry matrices to constant memory?
        const size_t count = symmetry.count();
        const float33_t* symmetry_matrices = symmetry.matrices();
        memory::PtrDevice<float33_t> d_matrices{count, stream};
        memory::copy(symmetry_matrices, d_matrices.get(), count, stream);
        const float scaling = normalize ? 1 / static_cast<float>(count + 1) : 1;

        const uint2_t o_shape{output_shape.get() + 2};
        const uint4_t o_stride{output_stride};
        const uint blocks_x = math::divideUp(o_shape[1], THREADS.x);
        const dim3 blocks(blocks_x * math::divideUp(o_shape[0], THREADS.y),
                          output_shape[1],
                          output_shape[0]);
        const LaunchConfig config{blocks, THREADS};

        switch (texture_interp_mode) {
            case INTERP_NEAREST:
                return stream.enqueue("geometry::transform3D", transformWithSymmetry3D_<T, INTERP_NEAREST>,
                                      config, texture, output, o_stride, o_shape, shift, matrix, center,
                                      d_matrices.get(), count, scaling, blocks_x);
            case INTERP_LINEAR:
                return stream.enqueue("geometry::transform3D", transformWithSymmetry3D_<T, INTERP_LINEAR>,
                                      config, texture, output, o_stride, o_shape, shift, matrix, center,
                                      d_matrices.get(), count, scaling, blocks_x);
            case INTERP_COSINE:
                return stream.enqueue("geometry::transform3D", transformWithSymmetry3D_<T, INTERP_COSINE>,
                                      config, texture, output, o_stride, o_shape, shift, matrix, center,
                                      d_matrices.get(), count, scaling, blocks_x);
            case INTERP_CUBIC:
                return stream.enqueue("geometry::transform3D", transformWithSymmetry3D_<T, INTERP_CUBIC>,
                                      config, texture, output, o_stride, o_shape, shift, matrix, center,
                                      d_matrices.get(), count, scaling, blocks_x);
            case INTERP_CUBIC_BSPLINE:
                return stream.enqueue("geometry::transform3D", transformWithSymmetry3D_<T, INTERP_CUBIC_BSPLINE>,
                                      config, texture, output, o_stride, o_shape, shift, matrix, center,
                                      d_matrices.get(), count, scaling, blocks_x);
            case INTERP_LINEAR_FAST:
                return stream.enqueue("geometry::transform3D", transformWithSymmetry3D_<T, INTERP_LINEAR_FAST>,
                                      config, texture, output, o_stride, o_shape, shift, matrix, center,
                                      d_matrices.get(), count, scaling, blocks_x);
            case INTERP_COSINE_FAST:
                return stream.enqueue("geometry::transform3D", transformWithSymmetry3D_<T, INTERP_COSINE_FAST>,
                                      config, texture, output, o_stride, o_shape, shift, matrix, center,
                                      d_matrices.get(), count, scaling, blocks_x);
            case INTERP_CUBIC_BSPLINE_FAST:
                return stream.enqueue("geometry::transform3D", transformWithSymmetry3D_<T, INTERP_CUBIC_BSPLINE_FAST>,
                                      config, texture, output, o_stride, o_shape, shift, matrix, center,
                                      d_matrices.get(), count, scaling, blocks_x);
            default:
                NOA_THROW_FUNC("transform3D", "{} is not supported", texture_interp_mode);
        }
    }

    #define NOA_INSTANTIATE_TRANSFORM_SYM_(T)                                                                                                                                                       \
    template void transform3D<true, T>(const shared_t<T[]>&, size4_t, size4_t, const shared_t<T[]>&, size4_t, size4_t, float3_t, float33_t, const Symmetry&, float3_t, InterpMode, bool, Stream&);  \
    template void transform3D<false, T>(const shared_t<T[]>&, size4_t, size4_t, const shared_t<T[]>&, size4_t, size4_t, float3_t, float33_t, const Symmetry&, float3_t, InterpMode, bool, Stream&);

    NOA_INSTANTIATE_TRANSFORM_SYM_(float);
    NOA_INSTANTIATE_TRANSFORM_SYM_(cfloat_t);
}
