#include "hip/hip_runtime.h"
#include "noa/common/Assert.h"
#include "noa/common/Profiler.h"
#include "noa/gpu/cuda/Exception.h"

#include "noa/gpu/cuda/memory/Copy.h"
#include "noa/gpu/cuda/memory/PtrArray.h"
#include "noa/gpu/cuda/memory/PtrDevice.h"
#include "noa/gpu/cuda/memory/PtrTexture.h"

#include "noa/gpu/cuda/geometry/Transform.h"
#include "noa/gpu/cuda/geometry/Interpolate.h"
#include "noa/gpu/cuda/geometry/Prefilter.h"

namespace {
    using namespace ::noa;
    constexpr dim3 THREADS(16, 16);

    template<typename T, InterpMode INTERP>
    __global__ void __launch_bounds__(THREADS.x * THREADS.y)
    transformWithSymmetry3D_(hipTextureObject_t texture, T* output, uint4_t output_stride, uint2_t shape,
                             float3_t shift, float33_t matrix, float3_t center,
                             const float33_t* symmetry_matrices, uint symmetry_count, float scaling, uint blocks_x) {
        const uint2_t index = indexes(blockIdx.x, blocks_x);
        const uint4_t gid{blockIdx.z,
                          blockIdx.y,
                          index[0] * THREADS.y + threadIdx.y,
                          index[1] * THREADS.x + threadIdx.x};
        if (gid[2] >= shape[0] || gid[3] >= shape[1])
            return;

        float3_t coordinates{gid[1], gid[2], gid[3]};
        coordinates -= center;
        coordinates = matrix * coordinates;
        T value = cuda::geometry::tex3D<T, INTERP>(texture, coordinates + center + shift + 0.5f);
        for (uint i = 0; i < symmetry_count; ++i) {
            float3_t i_coordinates(symmetry_matrices[i] * coordinates);
            value += cuda::geometry::tex3D<T, INTERP>(texture, i_coordinates + center + shift + 0.5f);
        }

        output[at(gid, output_stride)] = value * scaling;
    }
}

namespace noa::cuda::geometry {
    template<bool PREFILTER, typename T>
    void transform3D(const T* input, size4_t input_stride, size4_t input_shape,
                     T* output, size4_t output_stride, size4_t output_shape,
                     float3_t shift, float33_t matrix, const Symmetry& symmetry, float3_t center,
                     InterpMode interp_mode, bool normalize, Stream& stream) {
        NOA_PROFILE_FUNCTION();
        NOA_ASSERT(input_shape[0] == 1 || input_shape[0] == output_shape[0]);
        NOA_ASSERT(input_shape[1] > 1);

        if (input_stride[0] == 0)
            input_shape[0] = 1;

        // Prepare the input array:
        cuda::memory::PtrDevice<T> buffer;
        const T* buffer_ptr;
        size_t buffer_pitch;
        size_t buffer_offset;
        if (PREFILTER && (interp_mode == INTERP_CUBIC_BSPLINE || interp_mode == INTERP_CUBIC_BSPLINE_FAST)) {
            if (input_shape[1] != output_shape[1] ||
                input_shape[2] != output_shape[2] ||
                input_shape[3] != output_shape[3]) {
                buffer.reset(input_shape.elements(), stream);
                const size4_t contiguous_stride = input_shape.strides();
                cuda::geometry::bspline::prefilter(input, input_stride,
                                                   buffer.get(), contiguous_stride, input_shape, stream);
                buffer_ptr = buffer.get();
                buffer_pitch = contiguous_stride[2];
                buffer_offset = contiguous_stride[0];
            } else {
                NOA_ASSERT(isContiguous(output_stride, output_shape)[3]);
                NOA_ASSERT(isContiguous(output_stride, output_shape)[1]);
                // Whether input is batched or not, since we copy to the CUDA array, we can use the output as buffer.
                cuda::geometry::bspline::prefilter(input, input_stride, output, output_stride, input_shape, stream);
                buffer_ptr = output;
                buffer_pitch = output_stride[2];
                buffer_offset = output_stride[0];
            }
        } else {
            NOA_ASSERT(isContiguous(input_stride, input_shape)[3]);
            NOA_ASSERT(isContiguous(input_stride, input_shape)[1]);
            buffer_ptr = input;
            buffer_pitch = input_stride[2];
            buffer_offset = input_stride[0];
        }

        // Broadcast input if it is not batched:
        size4_t o_shape{input_shape[0] > 1 ? 1 : output_shape[0],
                        output_shape[1], output_shape[2],output_shape[3]};

        // Copy to texture and launch (per input batch):
        const size3_t shape_3d{input_shape.get() + 1};
        cuda::memory::PtrArray<T> i_array(shape_3d);
        cuda::memory::PtrTexture<T> i_texture;
        for (size_t i = 0; i < input_shape[0]; ++i) {
            cuda::memory::copy(buffer_ptr + i * buffer_offset, buffer_pitch, i_array.get(), shape_3d, stream);
            i_texture.reset(i_array.get(), interp_mode, BORDER_ZERO); // no need to wait here
            cuda::geometry::transform3D(
                    i_texture.get(), interp_mode, output + i * output_stride[0], output_stride, o_shape,
                    shift, matrix, symmetry, center, normalize, stream);
            stream.synchronize();
        }
    }

    template<typename T>
    void transform3D(hipTextureObject_t texture, InterpMode texture_interp_mode,
                     T* output, size4_t output_stride, size4_t output_shape,
                     float3_t shift, float33_t matrix, const Symmetry& symmetry, float3_t center,
                     bool normalize, Stream& stream) {
        NOA_PROFILE_FUNCTION();
        NOA_ASSERT(!memory::PtrTexture<T>::hasNormalizedCoordinates(texture));

        // TODO Move symmetry matrices to constant memory?
        const size_t count = symmetry.count();
        const float33_t* symmetry_matrices = symmetry.matrices();
        memory::PtrDevice<float33_t> d_matrices(count, stream);
        memory::copy(symmetry_matrices, d_matrices.get(), count, stream);
        const float scaling = 1 / static_cast<float>(count + 1);

        const uint2_t o_shape{output_shape.get() + 2};
        const uint4_t o_stride{output_stride};
        const uint blocks_x = math::divideUp(o_shape[1], THREADS.x);
        const dim3 blocks(blocks_x * math::divideUp(o_shape[0], THREADS.y),
                          output_shape[1],
                          output_shape[0]);
        const LaunchConfig config{blocks, THREADS};

        switch (texture_interp_mode) {
            case INTERP_NEAREST:
                return stream.enqueue("geometry::transform3D", transformWithSymmetry3D_<T, INTERP_NEAREST>,
                                      config, texture, output, o_stride, o_shape, shift, matrix, center,
                                      d_matrices.get(), count, scaling, blocks_x);
            case INTERP_LINEAR:
                return stream.enqueue("geometry::transform3D", transformWithSymmetry3D_<T, INTERP_LINEAR>,
                                      config, texture, output, o_stride, o_shape, shift, matrix, center,
                                      d_matrices.get(), count, scaling, blocks_x);
            case INTERP_COSINE:
                return stream.enqueue("geometry::transform3D", transformWithSymmetry3D_<T, INTERP_COSINE>,
                                      config, texture, output, o_stride, o_shape, shift, matrix, center,
                                      d_matrices.get(), count, scaling, blocks_x);
            case INTERP_CUBIC:
                return stream.enqueue("geometry::transform3D", transformWithSymmetry3D_<T, INTERP_CUBIC>,
                                      config, texture, output, o_stride, o_shape, shift, matrix, center,
                                      d_matrices.get(), count, scaling, blocks_x);
            case INTERP_CUBIC_BSPLINE:
                return stream.enqueue("geometry::transform3D", transformWithSymmetry3D_<T, INTERP_CUBIC_BSPLINE>,
                                      config, texture, output, o_stride, o_shape, shift, matrix, center,
                                      d_matrices.get(), count, scaling, blocks_x);
            case INTERP_LINEAR_FAST:
                return stream.enqueue("geometry::transform3D", transformWithSymmetry3D_<T, INTERP_LINEAR_FAST>,
                                      config, texture, output, o_stride, o_shape, shift, matrix, center,
                                      d_matrices.get(), count, scaling, blocks_x);
            case INTERP_COSINE_FAST:
                return stream.enqueue("geometry::transform3D", transformWithSymmetry3D_<T, INTERP_COSINE_FAST>,
                                      config, texture, output, o_stride, o_shape, shift, matrix, center,
                                      d_matrices.get(), count, scaling, blocks_x);
            case INTERP_CUBIC_BSPLINE_FAST:
                return stream.enqueue("geometry::transform3D", transformWithSymmetry3D_<T, INTERP_CUBIC_BSPLINE_FAST>,
                                      config, texture, output, o_stride, o_shape, shift, matrix, center,
                                      d_matrices.get(), count, scaling, blocks_x);
            default:
                NOA_THROW_FUNC("transform3D", "{} is not supported", texture_interp_mode);
        }
    }

    #define NOA_INSTANTIATE_TRANSFORM_SYM_(T)                                                                                                                        \
    template void transform3D<true, T>(const T*, size4_t, size4_t, T*, size4_t, size4_t, float3_t, float33_t, const Symmetry&, float3_t, InterpMode, bool, Stream&); \
    template void transform3D<false, T>(const T*, size4_t, size4_t, T*, size4_t, size4_t, float3_t, float33_t, const Symmetry&, float3_t, InterpMode, bool, Stream&);

    NOA_INSTANTIATE_TRANSFORM_SYM_(float);
    NOA_INSTANTIATE_TRANSFORM_SYM_(cfloat_t);
}
